#include "hip/hip_runtime.h"
#include "updates.cuh"
#ifdef TRI
__global__ void flip1_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  extern __shared__ unsigned rngShmem[];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  int i, j, ib, jb;
  //0..
  //...
  //...
  i = tx;
  j = ty;
  ib = (i + SpinSize - 1) % SpinSize;
  if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, i+1)] + BYPxx * confx[coo2D(j+1, i)] + BWPxx * confx[coo2D(j+1, i+1)] + BXMxx * confx[coo2D(j, ib)] + BYMxx * confx[coo2D(jb, i)] + BWMxx * confx[coo2D(jb, ib)]\
     + BXPxy * confy[coo2D(j, i+1)] + BYPxy * confy[coo2D(j+1, i)] + BWPxy * confy[coo2D(j+1, i+1)] + BXMxy * confy[coo2D(j, ib)] + BYMxy * confy[coo2D(jb, i)] + BWMxy * confy[coo2D(jb, ib)]\
     + BXPxz * confz[coo2D(j, i+1)] + BYPxz * confz[coo2D(j+1, i)] + BWPxz * confz[coo2D(j+1, i+1)] + BXMxz * confz[coo2D(j, ib)] + BYMxz * confz[coo2D(jb, i)] + BWMxz * confz[coo2D(jb, ib)];
  hy = BXPyx * confx[coo2D(j, i+1)] + BYPyx * confx[coo2D(j+1, i)] + BWPyx * confx[coo2D(j+1, i+1)] + BXMyx * confx[coo2D(j, ib)] + BYMyx * confx[coo2D(jb, i)] + BWMyx * confx[coo2D(jb, ib)]\
     + BXPyy * confy[coo2D(j, i+1)] + BYPyy * confy[coo2D(j+1, i)] + BWPyy * confy[coo2D(j+1, i+1)] + BXMyy * confy[coo2D(j, ib)] + BYMyy * confy[coo2D(jb, i)] + BWMyy * confy[coo2D(jb, ib)]\
     + BXPyz * confz[coo2D(j, i+1)] + BYPyz * confz[coo2D(j+1, i)] + BWPyz * confz[coo2D(j+1, i+1)] + BXMyz * confz[coo2D(j, ib)] + BYMyz * confz[coo2D(jb, i)] + BWMyz * confz[coo2D(jb, ib)];
  hz = BXPzx * confx[coo2D(j, i+1)] + BYPzx * confx[coo2D(j+1, i)] + BWPzx * confx[coo2D(j+1, i+1)] + BXMzx * confx[coo2D(j, ib)] + BYMzx * confx[coo2D(jb, i)] + BWMzx * confx[coo2D(jb, ib)]\
     + BXPzy * confy[coo2D(j, i+1)] + BYPzy * confy[coo2D(j+1, i)] + BWPzy * confy[coo2D(j+1, i+1)] + BXMzy * confy[coo2D(j, ib)] + BYMzy * confy[coo2D(jb, i)] + BWMzy * confy[coo2D(jb, ib)]\
     + BXPzz * confz[coo2D(j, i+1)] + BYPzz * confz[coo2D(j+1, i)] + BWPzz * confz[coo2D(j+1, i+1)] + BXMzz * confz[coo2D(j, ib)] + BYMzz * confz[coo2D(jb, i)] + BWMzz * confz[coo2D(jb, ib)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //..0
  //...
  i = tx + 2;
  j = ty + 1;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(j-1, i)] + BWMxx * confx[coo2D(j-1, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(j-1, i)] + BWMxy * confy[coo2D(j-1, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(j-1, i)] + BWMxz * confz[coo2D(j-1, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(j-1, i)] + BWMyx * confx[coo2D(j-1, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(j-1, i)] + BWMyy * confy[coo2D(j-1, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(j-1, i)] + BWMyz * confz[coo2D(j-1, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(j-1, i)] + BWMzx * confx[coo2D(j-1, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(j-1, i)] + BWMzy * confy[coo2D(j-1, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(j-1, i)] + BWMzz * confz[coo2D(j-1, i-1)] + H;
  du = -confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //...
  //.0.
  i = tx + 1;
  j = ty + 2;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(j-1, i)] + BWMxx * confx[coo2D(j-1, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(j-1, i)] + BWMxy * confy[coo2D(j-1, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(j-1, i)] + BWMxz * confz[coo2D(j-1, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(j-1, i)] + BWMyx * confx[coo2D(j-1, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(j-1, i)] + BWMyy * confy[coo2D(j-1, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(j-1, i)] + BWMyz * confz[coo2D(j-1, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(j-1, i)] + BWMzx * confx[coo2D(j-1, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(j-1, i)] + BWMzy * confy[coo2D(j-1, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(j-1, i)] + BWMzz * confz[coo2D(j-1, i-1)] + H;
  du = -confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}



__global__ void flip2_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  extern __shared__ unsigned rngShmem[];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);// % BlockSize_y;
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty = (blockIdx.x / BN) * SpinSize + 3 * (((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  i = tx;
  j = ty + 1;
  ib = (i + SpinSize - 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //...
  //0..
  //...
  hx = BXPxx * confx[coo2D(j, i+1)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, i+1)] + BXMxx * confx[coo2D(j, ib)] + BYMxx * confx[coo2D((j-1), i)] + BWMxx * confx[coo2D((j-1), ib)]\
     + BXPxy * confy[coo2D(j, i+1)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, i+1)] + BXMxy * confy[coo2D(j, ib)] + BYMxy * confy[coo2D((j-1), i)] + BWMxy * confy[coo2D((j-1), ib)]\
     + BXPxz * confz[coo2D(j, i+1)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, i+1)] + BXMxz * confz[coo2D(j, ib)] + BYMxz * confz[coo2D((j-1), i)] + BWMxz * confz[coo2D((j-1), ib)];
  hy = BXPyx * confx[coo2D(j, i+1)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, i+1)] + BXMyx * confx[coo2D(j, ib)] + BYMyx * confx[coo2D((j-1), i)] + BWMyx * confx[coo2D((j-1), ib)]\
     + BXPyy * confy[coo2D(j, i+1)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, i+1)] + BXMyy * confy[coo2D(j, ib)] + BYMyy * confy[coo2D((j-1), i)] + BWMyy * confy[coo2D((j-1), ib)]\
     + BXPyz * confz[coo2D(j, i+1)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, i+1)] + BXMyz * confz[coo2D(j, ib)] + BYMyz * confz[coo2D((j-1), i)] + BWMyz * confz[coo2D((j-1), ib)];
  hz = BXPzx * confx[coo2D(j, i+1)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, i+1)] + BXMzx * confx[coo2D(j, ib)] + BYMzx * confx[coo2D((j-1), i)] + BWMzx * confx[coo2D((j-1), ib)]\
     + BXPzy * confy[coo2D(j, i+1)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, i+1)] + BXMzy * confy[coo2D(j, ib)] + BYMzy * confy[coo2D((j-1), i)] + BWMzy * confy[coo2D((j-1), ib)]\
     + BXPzz * confz[coo2D(j, i+1)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, i+1)] + BWMzz * confz[coo2D(j, ib)] + BYMzz * confz[coo2D((j-1), i)] + BWMzz * confz[coo2D((j-1), ib)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the top and right corner of each thread sqare----------
  i = tx + 1;
  j = ty;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //.0.
  //...
  //...
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D((j+1), i)] + BWPxx * confx[coo2D((j+1), ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(jb, i)] + BWMxx * confx[coo2D(jb, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D((j+1), i)] + BWPxy * confy[coo2D((j+1), ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(jb, i)] + BWMxy * confy[coo2D(jb, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D((j+1), i)] + BWPxz * confz[coo2D((j+1), ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(jb, i)] + BWMxz * confz[coo2D(jb, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D((j+1), i)] + BWPyx * confx[coo2D((j+1), ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(jb, i)] + BWMyx * confx[coo2D(jb, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D((j+1), i)] + BWPyy * confy[coo2D((j+1), ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(jb, i)] + BWMyy * confy[coo2D(jb, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D((j+1), i)] + BWPyz * confz[coo2D((j+1), ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(jb, i)] + BWMyz * confz[coo2D(jb, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D((j+1), i)] + BWPzx * confx[coo2D((j+1), ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(jb, i)] + BWMzx * confx[coo2D(jb, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D((j+1), i)] + BWPzy * confy[coo2D((j+1), ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(jb, i)] + BWMzy * confy[coo2D(jb, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D((j+1), i)] + BWPzz * confz[coo2D((j+1), ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(jb, i)] + BWMzz * confz[coo2D(jb, i-1)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //...
  //..0
  i = tx + 2;
  j = ty + 2;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(j-1, i)] + BWMxx * confx[coo2D(j-1, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(j-1, i)] + BWMxy * confy[coo2D(j-1, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(j-1, i)] + BWMxz * confz[coo2D(j-1, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(j-1, i)] + BWMyx * confx[coo2D(j-1, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(j-1, i)] + BWMyy * confy[coo2D(j-1, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(j-1, i)] + BWMyz * confz[coo2D(j-1, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(j-1, i)] + BWMzx * confx[coo2D(j-1, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(j-1, i)] + BWMzy * confy[coo2D(j-1, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(j-1, i)] + BWMzz * confz[coo2D(j-1, i-1)] + H;
  du = -confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}


__global__ void flip3_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  extern __shared__ unsigned rngShmem[];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);// % BlockSize_y;
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty = (blockIdx.x / BN) * SpinSize + 3 * (((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  i = tx;
  j = ty + 2;
  ib = (i + SpinSize - 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //...
  //...
  //0..
  hx = BXPxx * confx[coo2D(j, i+1)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, i+1)] + BXMxx * confx[coo2D(j, ib)] + BYMxx * confx[coo2D((j-1), i)] + BWMxx * confx[coo2D((j-1), ib)]\
     + BXPxy * confy[coo2D(j, i+1)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, i+1)] + BXMxy * confy[coo2D(j, ib)] + BYMxy * confy[coo2D((j-1), i)] + BWMxy * confy[coo2D((j-1), ib)]\
     + BXPxz * confz[coo2D(j, i+1)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, i+1)] + BXMxz * confz[coo2D(j, ib)] + BYMxz * confz[coo2D((j-1), i)] + BWMxz * confz[coo2D((j-1), ib)];
  hy = BXPyx * confx[coo2D(j, i+1)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, i+1)] + BXMyx * confx[coo2D(j, ib)] + BYMyx * confx[coo2D((j-1), i)] + BWMyx * confx[coo2D((j-1), ib)]\
     + BXPyy * confy[coo2D(j, i+1)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, i+1)] + BXMyy * confy[coo2D(j, ib)] + BYMyy * confy[coo2D((j-1), i)] + BWMyy * confy[coo2D((j-1), ib)]\
     + BXPyz * confz[coo2D(j, i+1)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, i+1)] + BXMyz * confz[coo2D(j, ib)] + BYMyz * confz[coo2D((j-1), i)] + BWMyz * confz[coo2D((j-1), ib)];
  hz = BXPzx * confx[coo2D(j, i+1)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, i+1)] + BXMzx * confx[coo2D(j, ib)] + BYMzx * confx[coo2D((j-1), i)] + BWMzx * confx[coo2D((j-1), ib)]\
     + BXPzy * confy[coo2D(j, i+1)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, i+1)] + BXMzy * confy[coo2D(j, ib)] + BYMzy * confy[coo2D((j-1), i)] + BWMzy * confy[coo2D((j-1), ib)]\
     + BXPzz * confz[coo2D(j, i+1)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, i+1)] + BWMzz * confz[coo2D(j, ib)] + BYMzz * confz[coo2D((j-1), i)] + BWMzz * confz[coo2D((j-1), ib)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the top and right corner of each thread sqare----------
  i = tx + 2;
  j = ty;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //..0
  //...
  //...
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D((j+1), i)] + BWPxx * confx[coo2D((j+1), ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(jb, i)] + BWMxx * confx[coo2D(jb, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D((j+1), i)] + BWPxy * confy[coo2D((j+1), ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(jb, i)] + BWMxy * confy[coo2D(jb, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D((j+1), i)] + BWPxz * confz[coo2D((j+1), ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(jb, i)] + BWMxz * confz[coo2D(jb, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D((j+1), i)] + BWPyx * confx[coo2D((j+1), ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(jb, i)] + BWMyx * confx[coo2D(jb, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D((j+1), i)] + BWPyy * confy[coo2D((j+1), ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(jb, i)] + BWMyy * confy[coo2D(jb, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D((j+1), i)] + BWPyz * confz[coo2D((j+1), ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(jb, i)] + BWMyz * confz[coo2D(jb, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D((j+1), i)] + BWPzx * confx[coo2D((j+1), ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(jb, i)] + BWMzx * confx[coo2D(jb, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D((j+1), i)] + BWPzy * confy[coo2D((j+1), ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(jb, i)] + BWMzy * confy[coo2D(jb, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D((j+1), i)] + BWPzz * confz[coo2D((j+1), ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(jb, i)] + BWMzz * confz[coo2D(jb, i-1)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //.0.
  //...
  i = tx + 1;
  j = ty + 1;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D(jb, i)] + BWPxx * confx[coo2D(jb, ib)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(j-1, i)] + BWMxx * confx[coo2D(j-1, i-1)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D(jb, i)] + BWPxy * confy[coo2D(jb, ib)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(j-1, i)] + BWMxy * confy[coo2D(j-1, i-1)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D(jb, i)] + BWPxz * confz[coo2D(jb, ib)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(j-1, i)] + BWMxz * confz[coo2D(j-1, i-1)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D(jb, i)] + BWPyx * confx[coo2D(jb, ib)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(j-1, i)] + BWMyx * confx[coo2D(j-1, i-1)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D(jb, i)] + BWPyy * confy[coo2D(jb, ib)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(j-1, i)] + BWMyy * confy[coo2D(j-1, i-1)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D(jb, i)] + BWPyz * confz[coo2D(jb, ib)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(j-1, i)] + BWMyz * confz[coo2D(j-1, i-1)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D(jb, i)] + BWPzx * confx[coo2D(jb, ib)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(j-1, i)] + BWMzx * confx[coo2D(j-1, i-1)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D(jb, i)] + BWPzy * confy[coo2D(jb, ib)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(j-1, i)] + BWMzy * confy[coo2D(j-1, i-1)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D(jb, i)] + BWPzz * confz[coo2D(jb, ib)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(j-1, i)] + BWMzz * confz[coo2D(j-1, i-1)] + H;
  du = -confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}
#endif
