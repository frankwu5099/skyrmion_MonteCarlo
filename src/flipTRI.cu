#include "hip/hip_runtime.h"
#include "updates.cuh"
#ifdef TRI
__constant__ unsigned int flip_SpinSize;
__constant__ unsigned int flip_SpinSize_z;
__constant__ unsigned int flip_BlockSize_x;
__constant__ unsigned int flip_BlockSize_y;
__constant__ unsigned int flip_GridSize_x;
__constant__ unsigned int flip_GridSize_y;
__constant__ unsigned int flip_BN;
__constant__ float flip_A; //(0.0)
void move_params_device_flip(){
  hipMemcpyToSymbol(HIP_SYMBOL( flip_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BN), &H_BN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_A ), &H_A , sizeof(float));
}
__global__ void flip1_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  //extern __shared__ unsigned rngShmem[];
  __shared__ unsigned rngShmem[1024];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / flip_BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty =(blockIdx.x / flip_BN) * flip_SpinSize +  3 * ((((blockIdx.x % flip_BN) / flip_GridSize_x) % flip_GridSize_y) * flip_BlockSize_y + y);
  int i, j, ib, jb;
  //0..
  //...
  //...
  i = tx;
  j = ty;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  hx = BXPxx * confx[flip_coo2D(j, i+1)] + BYPxx * confx[flip_coo2D(j+1, i)] + BWPxx * confx[flip_coo2D(j+1, i+1)] + BXMxx * confx[flip_coo2D(j, ib)] + BYMxx * confx[flip_coo2D(jb, i)] + BWMxx * confx[flip_coo2D(jb, ib)]\
     + BXPxy * confy[flip_coo2D(j, i+1)] + BYPxy * confy[flip_coo2D(j+1, i)] + BWPxy * confy[flip_coo2D(j+1, i+1)] + BXMxy * confy[flip_coo2D(j, ib)] + BYMxy * confy[flip_coo2D(jb, i)] + BWMxy * confy[flip_coo2D(jb, ib)]\
     + BXPxz * confz[flip_coo2D(j, i+1)] + BYPxz * confz[flip_coo2D(j+1, i)] + BWPxz * confz[flip_coo2D(j+1, i+1)] + BXMxz * confz[flip_coo2D(j, ib)] + BYMxz * confz[flip_coo2D(jb, i)] + BWMxz * confz[flip_coo2D(jb, ib)];
  hy = BXPyx * confx[flip_coo2D(j, i+1)] + BYPyx * confx[flip_coo2D(j+1, i)] + BWPyx * confx[flip_coo2D(j+1, i+1)] + BXMyx * confx[flip_coo2D(j, ib)] + BYMyx * confx[flip_coo2D(jb, i)] + BWMyx * confx[flip_coo2D(jb, ib)]\
     + BXPyy * confy[flip_coo2D(j, i+1)] + BYPyy * confy[flip_coo2D(j+1, i)] + BWPyy * confy[flip_coo2D(j+1, i+1)] + BXMyy * confy[flip_coo2D(j, ib)] + BYMyy * confy[flip_coo2D(jb, i)] + BWMyy * confy[flip_coo2D(jb, ib)]\
     + BXPyz * confz[flip_coo2D(j, i+1)] + BYPyz * confz[flip_coo2D(j+1, i)] + BWPyz * confz[flip_coo2D(j+1, i+1)] + BXMyz * confz[flip_coo2D(j, ib)] + BYMyz * confz[flip_coo2D(jb, i)] + BWMyz * confz[flip_coo2D(jb, ib)];
  hz = BXPzx * confx[flip_coo2D(j, i+1)] + BYPzx * confx[flip_coo2D(j+1, i)] + BWPzx * confx[flip_coo2D(j+1, i+1)] + BXMzx * confx[flip_coo2D(j, ib)] + BYMzx * confx[flip_coo2D(jb, i)] + BWMzx * confx[flip_coo2D(jb, ib)]\
     + BXPzy * confy[flip_coo2D(j, i+1)] + BYPzy * confy[flip_coo2D(j+1, i)] + BWPzy * confy[flip_coo2D(j+1, i+1)] + BXMzy * confy[flip_coo2D(j, ib)] + BYMzy * confy[flip_coo2D(jb, i)] + BWMzy * confy[flip_coo2D(jb, ib)]\
     + BXPzz * confz[flip_coo2D(j, i+1)] + BYPzz * confz[flip_coo2D(j+1, i)] + BWPzz * confz[flip_coo2D(j+1, i+1)] + BXMzz * confz[flip_coo2D(j, ib)] + BYMzz * confz[flip_coo2D(jb, i)] + BWMzz * confz[flip_coo2D(jb, ib)] + H;
  du = - confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //..0
  //...
  i = tx + 2;
  j = ty + 1;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(j-1, i)] + BWMxx * confx[flip_coo2D(j-1, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(j-1, i)] + BWMxy * confy[flip_coo2D(j-1, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(j-1, i)] + BWMxz * confz[flip_coo2D(j-1, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(j-1, i)] + BWMyx * confx[flip_coo2D(j-1, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(j-1, i)] + BWMyy * confy[flip_coo2D(j-1, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(j-1, i)] + BWMyz * confz[flip_coo2D(j-1, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(j-1, i)] + BWMzx * confx[flip_coo2D(j-1, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(j-1, i)] + BWMzy * confy[flip_coo2D(j-1, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(j-1, i)] + BWMzz * confz[flip_coo2D(j-1, i-1)] + H;
  du = -confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //...
  //.0.
  i = tx + 1;
  j = ty + 2;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(j-1, i)] + BWMxx * confx[flip_coo2D(j-1, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(j-1, i)] + BWMxy * confy[flip_coo2D(j-1, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(j-1, i)] + BWMxz * confz[flip_coo2D(j-1, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(j-1, i)] + BWMyx * confx[flip_coo2D(j-1, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(j-1, i)] + BWMyy * confy[flip_coo2D(j-1, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(j-1, i)] + BWMyz * confz[flip_coo2D(j-1, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(j-1, i)] + BWMzx * confx[flip_coo2D(j-1, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(j-1, i)] + BWMzy * confy[flip_coo2D(j-1, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(j-1, i)] + BWMzz * confz[flip_coo2D(j-1, i-1)] + H;
  du = -confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}



__global__ void flip2_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  __shared__ unsigned rngShmem[1024];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / flip_BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);// % flip_BlockSize_y;
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty = (blockIdx.x / flip_BN) * flip_SpinSize + 3 * (((blockIdx.x % flip_BN) / flip_GridSize_x) * flip_BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  i = tx;
  j = ty + 1;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //...
  //0..
  //...
  hx = BXPxx * confx[flip_coo2D(j, i+1)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, i+1)] + BXMxx * confx[flip_coo2D(j, ib)] + BYMxx * confx[flip_coo2D((j-1), i)] + BWMxx * confx[flip_coo2D((j-1), ib)]\
     + BXPxy * confy[flip_coo2D(j, i+1)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, i+1)] + BXMxy * confy[flip_coo2D(j, ib)] + BYMxy * confy[flip_coo2D((j-1), i)] + BWMxy * confy[flip_coo2D((j-1), ib)]\
     + BXPxz * confz[flip_coo2D(j, i+1)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, i+1)] + BXMxz * confz[flip_coo2D(j, ib)] + BYMxz * confz[flip_coo2D((j-1), i)] + BWMxz * confz[flip_coo2D((j-1), ib)];
  hy = BXPyx * confx[flip_coo2D(j, i+1)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, i+1)] + BXMyx * confx[flip_coo2D(j, ib)] + BYMyx * confx[flip_coo2D((j-1), i)] + BWMyx * confx[flip_coo2D((j-1), ib)]\
     + BXPyy * confy[flip_coo2D(j, i+1)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, i+1)] + BXMyy * confy[flip_coo2D(j, ib)] + BYMyy * confy[flip_coo2D((j-1), i)] + BWMyy * confy[flip_coo2D((j-1), ib)]\
     + BXPyz * confz[flip_coo2D(j, i+1)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, i+1)] + BXMyz * confz[flip_coo2D(j, ib)] + BYMyz * confz[flip_coo2D((j-1), i)] + BWMyz * confz[flip_coo2D((j-1), ib)];
  hz = BXPzx * confx[flip_coo2D(j, i+1)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, i+1)] + BXMzx * confx[flip_coo2D(j, ib)] + BYMzx * confx[flip_coo2D((j-1), i)] + BWMzx * confx[flip_coo2D((j-1), ib)]\
     + BXPzy * confy[flip_coo2D(j, i+1)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, i+1)] + BXMzy * confy[flip_coo2D(j, ib)] + BYMzy * confy[flip_coo2D((j-1), i)] + BWMzy * confy[flip_coo2D((j-1), ib)]\
     + BXPzz * confz[flip_coo2D(j, i+1)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, i+1)] + BWMzz * confz[flip_coo2D(j, ib)] + BYMzz * confz[flip_coo2D((j-1), i)] + BWMzz * confz[flip_coo2D((j-1), ib)] + H;
  du = - confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the top and right corner of each thread sqare----------
  i = tx + 1;
  j = ty;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //.0.
  //...
  //...
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D((j+1), i)] + BWPxx * confx[flip_coo2D((j+1), ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(jb, i)] + BWMxx * confx[flip_coo2D(jb, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D((j+1), i)] + BWPxy * confy[flip_coo2D((j+1), ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(jb, i)] + BWMxy * confy[flip_coo2D(jb, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D((j+1), i)] + BWPxz * confz[flip_coo2D((j+1), ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(jb, i)] + BWMxz * confz[flip_coo2D(jb, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D((j+1), i)] + BWPyx * confx[flip_coo2D((j+1), ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(jb, i)] + BWMyx * confx[flip_coo2D(jb, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D((j+1), i)] + BWPyy * confy[flip_coo2D((j+1), ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(jb, i)] + BWMyy * confy[flip_coo2D(jb, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D((j+1), i)] + BWPyz * confz[flip_coo2D((j+1), ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(jb, i)] + BWMyz * confz[flip_coo2D(jb, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D((j+1), i)] + BWPzx * confx[flip_coo2D((j+1), ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(jb, i)] + BWMzx * confx[flip_coo2D(jb, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D((j+1), i)] + BWPzy * confy[flip_coo2D((j+1), ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(jb, i)] + BWMzy * confy[flip_coo2D(jb, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D((j+1), i)] + BWPzz * confz[flip_coo2D((j+1), ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(jb, i)] + BWMzz * confz[flip_coo2D(jb, i-1)] + H;
  du = - confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //...
  //..0
  i = tx + 2;
  j = ty + 2;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(j-1, i)] + BWMxx * confx[flip_coo2D(j-1, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(j-1, i)] + BWMxy * confy[flip_coo2D(j-1, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(j-1, i)] + BWMxz * confz[flip_coo2D(j-1, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(j-1, i)] + BWMyx * confx[flip_coo2D(j-1, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(j-1, i)] + BWMyy * confy[flip_coo2D(j-1, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(j-1, i)] + BWMyz * confz[flip_coo2D(j-1, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(j-1, i)] + BWMzx * confx[flip_coo2D(j-1, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(j-1, i)] + BWMzy * confy[flip_coo2D(j-1, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(j-1, i)] + BWMzz * confz[flip_coo2D(j-1, i-1)] + H;
  du = -confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}


__global__ void flip3_TRI(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  __shared__ unsigned rngShmem[1024];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / flip_BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);// % flip_BlockSize_y;
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty = (blockIdx.x / flip_BN) * flip_SpinSize + 3 * (((blockIdx.x % flip_BN) / flip_GridSize_x) * flip_BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  i = tx;
  j = ty + 2;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //...
  //...
  //0..
  hx = BXPxx * confx[flip_coo2D(j, i+1)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, i+1)] + BXMxx * confx[flip_coo2D(j, ib)] + BYMxx * confx[flip_coo2D((j-1), i)] + BWMxx * confx[flip_coo2D((j-1), ib)]\
     + BXPxy * confy[flip_coo2D(j, i+1)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, i+1)] + BXMxy * confy[flip_coo2D(j, ib)] + BYMxy * confy[flip_coo2D((j-1), i)] + BWMxy * confy[flip_coo2D((j-1), ib)]\
     + BXPxz * confz[flip_coo2D(j, i+1)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, i+1)] + BXMxz * confz[flip_coo2D(j, ib)] + BYMxz * confz[flip_coo2D((j-1), i)] + BWMxz * confz[flip_coo2D((j-1), ib)];
  hy = BXPyx * confx[flip_coo2D(j, i+1)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, i+1)] + BXMyx * confx[flip_coo2D(j, ib)] + BYMyx * confx[flip_coo2D((j-1), i)] + BWMyx * confx[flip_coo2D((j-1), ib)]\
     + BXPyy * confy[flip_coo2D(j, i+1)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, i+1)] + BXMyy * confy[flip_coo2D(j, ib)] + BYMyy * confy[flip_coo2D((j-1), i)] + BWMyy * confy[flip_coo2D((j-1), ib)]\
     + BXPyz * confz[flip_coo2D(j, i+1)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, i+1)] + BXMyz * confz[flip_coo2D(j, ib)] + BYMyz * confz[flip_coo2D((j-1), i)] + BWMyz * confz[flip_coo2D((j-1), ib)];
  hz = BXPzx * confx[flip_coo2D(j, i+1)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, i+1)] + BXMzx * confx[flip_coo2D(j, ib)] + BYMzx * confx[flip_coo2D((j-1), i)] + BWMzx * confx[flip_coo2D((j-1), ib)]\
     + BXPzy * confy[flip_coo2D(j, i+1)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, i+1)] + BXMzy * confy[flip_coo2D(j, ib)] + BYMzy * confy[flip_coo2D((j-1), i)] + BWMzy * confy[flip_coo2D((j-1), ib)]\
     + BXPzz * confz[flip_coo2D(j, i+1)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, i+1)] + BWMzz * confz[flip_coo2D(j, ib)] + BYMzz * confz[flip_coo2D((j-1), i)] + BWMzz * confz[flip_coo2D((j-1), ib)] + H;
  du = - confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the top and right corner of each thread sqare----------
  i = tx + 2;
  j = ty;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //..0
  //...
  //...
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D((j+1), i)] + BWPxx * confx[flip_coo2D((j+1), ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(jb, i)] + BWMxx * confx[flip_coo2D(jb, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D((j+1), i)] + BWPxy * confy[flip_coo2D((j+1), ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(jb, i)] + BWMxy * confy[flip_coo2D(jb, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D((j+1), i)] + BWPxz * confz[flip_coo2D((j+1), ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(jb, i)] + BWMxz * confz[flip_coo2D(jb, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D((j+1), i)] + BWPyx * confx[flip_coo2D((j+1), ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(jb, i)] + BWMyx * confx[flip_coo2D(jb, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D((j+1), i)] + BWPyy * confy[flip_coo2D((j+1), ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(jb, i)] + BWMyy * confy[flip_coo2D(jb, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D((j+1), i)] + BWPyz * confz[flip_coo2D((j+1), ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(jb, i)] + BWMyz * confz[flip_coo2D(jb, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D((j+1), i)] + BWPzx * confx[flip_coo2D((j+1), ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(jb, i)] + BWMzx * confx[flip_coo2D(jb, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D((j+1), i)] + BWPzy * confy[flip_coo2D((j+1), ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(jb, i)] + BWMzy * confy[flip_coo2D(jb, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D((j+1), i)] + BWPzz * confz[flip_coo2D((j+1), ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(jb, i)] + BWMzz * confz[flip_coo2D(jb, i-1)] + H;
  du = - confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //...
  //.0.
  //...
  i = tx + 1;
  j = ty + 1;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[flip_coo2D(j, ib)] + BYPxx * confx[flip_coo2D(jb, i)] + BWPxx * confx[flip_coo2D(jb, ib)] + BXMxx * confx[flip_coo2D(j, i-1)] + BYMxx * confx[flip_coo2D(j-1, i)] + BWMxx * confx[flip_coo2D(j-1, i-1)]\
     + BXPxy * confy[flip_coo2D(j, ib)] + BYPxy * confy[flip_coo2D(jb, i)] + BWPxy * confy[flip_coo2D(jb, ib)] + BXMxy * confy[flip_coo2D(j, i-1)] + BYMxy * confy[flip_coo2D(j-1, i)] + BWMxy * confy[flip_coo2D(j-1, i-1)]\
     + BXPxz * confz[flip_coo2D(j, ib)] + BYPxz * confz[flip_coo2D(jb, i)] + BWPxz * confz[flip_coo2D(jb, ib)] + BXMxz * confz[flip_coo2D(j, i-1)] + BYMxz * confz[flip_coo2D(j-1, i)] + BWMxz * confz[flip_coo2D(j-1, i-1)];
  hy = BXPyx * confx[flip_coo2D(j, ib)] + BYPyx * confx[flip_coo2D(jb, i)] + BWPyx * confx[flip_coo2D(jb, ib)] + BXMyx * confx[flip_coo2D(j, i-1)] + BYMyx * confx[flip_coo2D(j-1, i)] + BWMyx * confx[flip_coo2D(j-1, i-1)]\
     + BXPyy * confy[flip_coo2D(j, ib)] + BYPyy * confy[flip_coo2D(jb, i)] + BWPyy * confy[flip_coo2D(jb, ib)] + BXMyy * confy[flip_coo2D(j, i-1)] + BYMyy * confy[flip_coo2D(j-1, i)] + BWMyy * confy[flip_coo2D(j-1, i-1)]\
     + BXPyz * confz[flip_coo2D(j, ib)] + BYPyz * confz[flip_coo2D(jb, i)] + BWPyz * confz[flip_coo2D(jb, ib)] + BXMyz * confz[flip_coo2D(j, i-1)] + BYMyz * confz[flip_coo2D(j-1, i)] + BWMyz * confz[flip_coo2D(j-1, i-1)];
  hz = BXPzx * confx[flip_coo2D(j, ib)] + BYPzx * confx[flip_coo2D(jb, i)] + BWPzx * confx[flip_coo2D(jb, ib)] + BXMzx * confx[flip_coo2D(j, i-1)] + BYMzx * confx[flip_coo2D(j-1, i)] + BWMzx * confx[flip_coo2D(j-1, i-1)]\
     + BXPzy * confy[flip_coo2D(j, ib)] + BYPzy * confy[flip_coo2D(jb, i)] + BWPzy * confy[flip_coo2D(jb, ib)] + BXMzy * confy[flip_coo2D(j, i-1)] + BYMzy * confy[flip_coo2D(j-1, i)] + BWMzy * confy[flip_coo2D(j-1, i-1)]\
     + BXPzz * confz[flip_coo2D(j, ib)] + BYPzz * confz[flip_coo2D(jb, i)] + BWPzz * confz[flip_coo2D(jb, ib)] + BXMzz * confz[flip_coo2D(j, i-1)] + BYMzz * confz[flip_coo2D(j-1, i)] + BWMzz * confz[flip_coo2D(j-1, i-1)] + H;
  du = -confx[flip_coo2D(j, i)] * hx - confy[flip_coo2D(j, i)] * hy - confz[flip_coo2D(j, i)] * hz + flip_A * confz[flip_coo2D(j, i)] * confz[flip_coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - flip_A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[flip_coo2D(j, i)] = sx;
    confy[flip_coo2D(j, i)] = sy;
    confz[flip_coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}
#endif
