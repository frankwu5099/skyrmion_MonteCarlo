#include "hip/hip_runtime.h"
#include "updates.cuh"
#ifdef TRI
__constant__ unsigned int flip_SpinSize;
__constant__ unsigned int flip_SpinSize_z;
__constant__ unsigned int flip_BlockSize_x;
__constant__ unsigned int flip_BlockSize_y;
__constant__ unsigned int flip_GridSize_x;
__constant__ unsigned int flip_GridSize_y;
__constant__ unsigned int flip_Nplane;
__constant__ unsigned int flip_BN;
__constant__ float flip_A; //(0.0)
__constant__ float BXPyz;
__constant__ float BYPyz;
__constant__ float BWPyz;
__constant__ float BXMyz;
__constant__ float BYMyz;
__constant__ float BWMyz;
__constant__ float BXPzy;
__constant__ float BYPzy;
__constant__ float BWPzy;
__constant__ float BXMzy;
__constant__ float BYMzy;
__constant__ float BWMzy;
__constant__ float BXPxz;
__constant__ float BYPxz;
__constant__ float BWPxz;
__constant__ float BXMxz;
__constant__ float BYMxz;
__constant__ float BWMxz;
__constant__ float BXPzx;
__constant__ float BYPzx;
__constant__ float BWPzx;
__constant__ float BXMzx;
__constant__ float BYMzx;
__constant__ float BWMzx;
__constant__ float BZMxy;
__constant__ float BZMyx;
__constant__ float BZPxy;
__constant__ float BZPyx;

void move_params_device_flip(){
  float tmpp;
  hipMemcpyToSymbol(HIP_SYMBOL( flip_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_Nplane), &H_Nplane, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_BN), &H_BN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( flip_A ), &H_A , sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BXPyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYPyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWPyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BXMyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYMyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWMyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BXPzy), &tmpp, sizeof(float));
  tmpp =  (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYPzy), &tmpp, sizeof(float));
  tmpp =  (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWPzy), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BXMzy), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYMzy), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWMzy), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BXPxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYPxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWPxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BXMxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYMxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWMxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BXPzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYPzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWPzx), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BXMzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BYMzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( BWMzx), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BZPxy), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BZPyx), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BZMxy), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( BZMyx), &tmpp, sizeof(float));
}

__device__ inline void single_update(const float &invT, float &hx, float &hy, float &hz, float &x, float &y, float &z, hiprandState &state){
  unsigned int r;
  float du;	//-dE
  float sx, sy, Heff, nx, ny, nz;
  float costh, sinth, phi;
  r = hiprand(&state);//WarpStandard_Generate(rngRegs, rngShmem);
  Heff = sqrt(hx*hx + hy*hy + hz*hz);
    hx /= Heff;
    hy /= Heff;
    hz /= Heff;
  sx = r * NORM / 2.0;
  if (sx<1.0)
  costh =  log(exp(Heff*invT)*(1.00-sx) + exp(-Heff*invT)*sx) / invT /Heff;
  else
    costh = -1.0;
  if (1>costh*costh)
    sinth = sqrt(1 - costh*costh);
  else
    sinth = 0.0;
  r = hiprand(&state);//WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  du = sqrt(hx*hx + hy*hy);
    sx = cos( phi );
    sy = sin( phi );
  if (du>0.0000001){
    nx = sinth * (hy*sx/du + hx*hz*sy/du);
    ny = sinth * (-hx*sx/du + hy*hz*sy/du);
    nz = -sinth * (hx*hx + hy*hy)/du*sy;
    x = hx * costh + nx;
    y = hy * costh + ny;
    z = hz * costh + nz;
  }
  else{
    x = sinth * sx;
    y = sinth * sy;
    z = hz * costh;
  }
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * flip_BlockSize_x* flip_BlockSize_y;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void flip1_TRI(float *confx, float *confy, float *confz, float* Hs, float* invTs, hiprandState *state){
  //Energy variables
  //__shared__ unsigned rngShmem[1024];
  float H = Hs[blockIdx.x / flip_BN];
  float invT = invTs[blockIdx.x / flip_BN];
  float hx, hy, hz;
  //float norm;
  hiprandState localState = state[threadIdx.x + blockIdx.x * flip_BlockSize_x* flip_BlockSize_y];
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty =(blockIdx.x / flip_BN) * flip_SpinSize * flip_SpinSize_z +  3 * (((blockIdx.x % flip_BN) / flip_GridSize_x) * flip_BlockSize_y + y);
  int i, j, ib, jb, k;
  //0..
  //...
  //...
  i = tx;
  j = ty;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, j+1, i)] + BWPxx * confx[flip_coo(k, j+1, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, ib)]\
     + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, j+1, i)] + BWPxy * confy[flip_coo(k, j+1, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, ib)]\
     + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, j+1, i)] + BWPxz * confz[flip_coo(k, j+1, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, ib)];
  hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, j+1, i)] + BWPyx * confx[flip_coo(k, j+1, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, ib)]\
     + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, j+1, i)] + BWPyy * confy[flip_coo(k, j+1, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, ib)]\
     + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, j+1, i)] + BWPyz * confz[flip_coo(k, j+1, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, ib)];
  hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, j+1, i)] + BWPzx * confx[flip_coo(k, j+1, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, ib)]\
     + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, j+1, i)] + BWPzy * confy[flip_coo(k, j+1, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, ib)]\
     + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, j+1, i)] + BWPzz * confz[flip_coo(k, j+1, i+1)] + BXMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, ib)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();
  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, j+1, i)] + BWPxx * confx[flip_coo(k, j+1, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, ib)]\
			 + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, j+1, i)] + BWPxy * confy[flip_coo(k, j+1, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, ib)]\
			 + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, j+1, i)] + BWPxz * confz[flip_coo(k, j+1, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, ib)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, j+1, i)] + BWPyx * confx[flip_coo(k, j+1, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, ib)]\
			 + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, j+1, i)] + BWPyy * confy[flip_coo(k, j+1, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, ib)]\
			 + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, j+1, i)] + BWPyz * confz[flip_coo(k, j+1, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, ib)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, j+1, i)] + BWPzx * confx[flip_coo(k, j+1, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, ib)]\
			 + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, j+1, i)] + BWPzy * confy[flip_coo(k, j+1, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, ib)]\
			 + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, j+1, i)] + BWPzz * confz[flip_coo(k, j+1, i+1)] + BXMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, ib)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, j+1, i)] + BWPxx * confx[flip_coo(k, j+1, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, ib)]\
       + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, j+1, i)] + BWPxy * confy[flip_coo(k, j+1, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, ib)]\
       + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, j+1, i)] + BWPxz * confz[flip_coo(k, j+1, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, ib)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, j+1, i)] + BWPyx * confx[flip_coo(k, j+1, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, ib)]\
       + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, j+1, i)] + BWPyy * confy[flip_coo(k, j+1, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, ib)]\
       + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, j+1, i)] + BWPyz * confz[flip_coo(k, j+1, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, ib)]\
        + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, j+1, i)] + BWPzx * confx[flip_coo(k, j+1, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, ib)]\
       + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, j+1, i)] + BWPzy * confy[flip_coo(k, j+1, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, ib)]\
       + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, j+1, i)] + BWPzz * confz[flip_coo(k, j+1, i+1)] + BXMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, ib)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //...
  //..0
  //...
  i = tx + 2;
  j = ty + 1;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();
  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //...
  //...
  //.0.
  i = tx + 1;
  j = ty + 2;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //Load random number back to global memory
  state[threadIdx.x + blockIdx.x * flip_BlockSize_x * flip_BlockSize_y] = localState;
}



__global__ void flip2_TRI(float *confx, float *confy, float *confz, float* Hs, float* invTs, hiprandState *state){
  //Energy variables
  //__shared__ unsigned rngShmem[1024];
  hiprandState localState = state[threadIdx.x + blockIdx.x * flip_BlockSize_x * flip_BlockSize_y];
  float H = Hs[blockIdx.x / flip_BN];
  float invT = invTs[blockIdx.x / flip_BN];
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);// % flip_BlockSize_y;
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty = (blockIdx.x / flip_BN) * flip_SpinSize * flip_SpinSize_z + 3 * (((blockIdx.x % flip_BN) / flip_GridSize_x) * flip_BlockSize_y + y);
  int i, j, ib, jb, k;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  //...
  //0..
  //...
  i = tx;
  j = ty + 1;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
     + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
     + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)];
  hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
     + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
     + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)];
  hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
     + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
     + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)]\
				 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
       + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
       + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
       + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
       + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
       + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
       + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //----------Spin flip at the top and right corner of each thread sqare----------
  //.0.
  //...
  //...
  i = tx + 1;
  j = ty;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)]\
				 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //...
  //...
  //..0
  i = tx + 2;
  j = ty + 2;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
				 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //Load random number back to global memory
  state[threadIdx.x + blockIdx.x * flip_BlockSize_x * flip_BlockSize_y] = localState;
}


__global__ void flip3_TRI(float *confx, float *confy, float *confz, float* Hs, float* invTs, hiprandState *state){
  //Energy variables
  //__shared__ unsigned rngShmem[1024];
  hiprandState localState = state[threadIdx.x + blockIdx.x * flip_BlockSize_x * flip_BlockSize_y];
  float H = Hs[blockIdx.x / flip_BN];
  float invT = invTs[blockIdx.x / flip_BN];
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (flip_BlockSize_x);
  const int y = (threadIdx.x / flip_BlockSize_x);// % flip_BlockSize_y;
  const int tx = 3 * (((blockIdx.x % flip_BN) % flip_GridSize_x) * flip_BlockSize_x + x);
  const int ty = (blockIdx.x / flip_BN) * flip_SpinSize * flip_SpinSize_z + 3 * (((blockIdx.x % flip_BN) / flip_GridSize_x) * flip_BlockSize_y + y);
  int i, j, ib, jb, k;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  //...
  //...
  //0..
  i = tx;
  j = ty + 2;
  ib = (i + flip_SpinSize - 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
     + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
     + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)];
  hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
     + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
     + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)];
  hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
     + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
     + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
			 + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
			 + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, i+1)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, i+1)] + BXMxx * confx[flip_coo(k, j, ib)] + BYMxx * confx[flip_coo(k, (j-1), i)] + BWMxx * confx[flip_coo(k, (j-1), ib)]\
       + BXPxy * confy[flip_coo(k, j, i+1)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, i+1)] + BXMxy * confy[flip_coo(k, j, ib)] + BYMxy * confy[flip_coo(k, (j-1), i)] + BWMxy * confy[flip_coo(k, (j-1), ib)]\
       + BXPxz * confz[flip_coo(k, j, i+1)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, i+1)] + BXMxz * confz[flip_coo(k, j, ib)] + BYMxz * confz[flip_coo(k, (j-1), i)] + BWMxz * confz[flip_coo(k, (j-1), ib)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, i+1)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, i+1)] + BXMyx * confx[flip_coo(k, j, ib)] + BYMyx * confx[flip_coo(k, (j-1), i)] + BWMyx * confx[flip_coo(k, (j-1), ib)]\
       + BXPyy * confy[flip_coo(k, j, i+1)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, i+1)] + BXMyy * confy[flip_coo(k, j, ib)] + BYMyy * confy[flip_coo(k, (j-1), i)] + BWMyy * confy[flip_coo(k, (j-1), ib)]\
       + BXPyz * confz[flip_coo(k, j, i+1)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, i+1)] + BXMyz * confz[flip_coo(k, j, ib)] + BYMyz * confz[flip_coo(k, (j-1), i)] + BWMyz * confz[flip_coo(k, (j-1), ib)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, i+1)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, i+1)] + BXMzx * confx[flip_coo(k, j, ib)] + BYMzx * confx[flip_coo(k, (j-1), i)] + BWMzx * confx[flip_coo(k, (j-1), ib)]\
       + BXPzy * confy[flip_coo(k, j, i+1)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, i+1)] + BXMzy * confy[flip_coo(k, j, ib)] + BYMzy * confy[flip_coo(k, (j-1), i)] + BWMzy * confy[flip_coo(k, (j-1), ib)]\
       + BXPzz * confz[flip_coo(k, j, i+1)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, i+1)] + BWMzz * confz[flip_coo(k, j, ib)] + BYMzz * confz[flip_coo(k, (j-1), i)] + BWMzz * confz[flip_coo(k, (j-1), ib)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //----------Spin flip at the top and right corner of each thread sqare----------
  //..0
  //...
  //...
  i = tx + 2;
  j = ty;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == 0)	jb = j + flip_SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, (j+1), i)] + BWPxx * confx[flip_coo(k, (j+1), ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, jb, i)] + BWMxx * confx[flip_coo(k, jb, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, (j+1), i)] + BWPxy * confy[flip_coo(k, (j+1), ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, jb, i)] + BWMxy * confy[flip_coo(k, jb, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, (j+1), i)] + BWPxz * confz[flip_coo(k, (j+1), ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, jb, i)] + BWMxz * confz[flip_coo(k, jb, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, (j+1), i)] + BWPyx * confx[flip_coo(k, (j+1), ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, jb, i)] + BWMyx * confx[flip_coo(k, jb, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, (j+1), i)] + BWPyy * confy[flip_coo(k, (j+1), ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, jb, i)] + BWMyy * confy[flip_coo(k, jb, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, (j+1), i)] + BWPyz * confz[flip_coo(k, (j+1), ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, jb, i)] + BWMyz * confz[flip_coo(k, jb, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, (j+1), i)] + BWPzx * confx[flip_coo(k, (j+1), ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, jb, i)] + BWMzx * confx[flip_coo(k, jb, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, (j+1), i)] + BWPzy * confy[flip_coo(k, (j+1), ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, jb, i)] + BWMzy * confy[flip_coo(k, jb, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, (j+1), i)] + BWPzz * confz[flip_coo(k, (j+1), ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, jb, i)] + BWMzz * confz[flip_coo(k, jb, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //...
  //.0.
  //...
  i = tx + 1;
  j = ty + 1;
  ib = (i + 1) % flip_SpinSize;
  if((j % flip_SpinSize) == flip_SpinSize - 1)	jb = j - flip_SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  //first layer
	k = 0;
  hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
     + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
     + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)];
  hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
     + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
     + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)];
  hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
     + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
     + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H;
  if (flip_SpinSize_z>1){
    hx += BZPxy * confy[flip_coo(k+1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)];
    hy += BZPyx * confx[flip_coo(k+1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)];
    hz += BZPzz * confz[flip_coo(k+1, j, i)];
  }
  single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
  __syncthreads();

  for (k = 1;k < flip_SpinSize_z - 1; k++){//middle layers
		hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPxy * confy[flip_coo(k+1, j, i)] + BZMxy * confy[flip_coo(k-1, j, i)] + BZPxx * confx[flip_coo(k+1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
		hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
			 + BZPyx * confx[flip_coo(k+1, j, i)] + BZMyx * confx[flip_coo(k-1, j, i)] + BZPyy * confy[flip_coo(k+1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
		hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
			 + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
			 + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
			 + BZPzz * confz[flip_coo(k+1, j, i)] + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();

	}
  //last layer
  if (flip_SpinSize_z>1){
    k = flip_SpinSize_z - 1;
    hx = BXPxx * confx[flip_coo(k, j, ib)] + BYPxx * confx[flip_coo(k, jb, i)] + BWPxx * confx[flip_coo(k, jb, ib)] + BXMxx * confx[flip_coo(k, j, i-1)] + BYMxx * confx[flip_coo(k, j-1, i)] + BWMxx * confx[flip_coo(k, j-1, i-1)]\
       + BXPxy * confy[flip_coo(k, j, ib)] + BYPxy * confy[flip_coo(k, jb, i)] + BWPxy * confy[flip_coo(k, jb, ib)] + BXMxy * confy[flip_coo(k, j, i-1)] + BYMxy * confy[flip_coo(k, j-1, i)] + BWMxy * confy[flip_coo(k, j-1, i-1)]\
       + BXPxz * confz[flip_coo(k, j, ib)] + BYPxz * confz[flip_coo(k, jb, i)] + BWPxz * confz[flip_coo(k, jb, ib)] + BXMxz * confz[flip_coo(k, j, i-1)] + BYMxz * confz[flip_coo(k, j-1, i)] + BWMxz * confz[flip_coo(k, j-1, i-1)]\
       + BZMxy * confy[flip_coo(k-1, j, i)] + BZMxx * confx[flip_coo(k-1, j, i)];
    hy = BXPyx * confx[flip_coo(k, j, ib)] + BYPyx * confx[flip_coo(k, jb, i)] + BWPyx * confx[flip_coo(k, jb, ib)] + BXMyx * confx[flip_coo(k, j, i-1)] + BYMyx * confx[flip_coo(k, j-1, i)] + BWMyx * confx[flip_coo(k, j-1, i-1)]\
       + BXPyy * confy[flip_coo(k, j, ib)] + BYPyy * confy[flip_coo(k, jb, i)] + BWPyy * confy[flip_coo(k, jb, ib)] + BXMyy * confy[flip_coo(k, j, i-1)] + BYMyy * confy[flip_coo(k, j-1, i)] + BWMyy * confy[flip_coo(k, j-1, i-1)]\
       + BXPyz * confz[flip_coo(k, j, ib)] + BYPyz * confz[flip_coo(k, jb, i)] + BWPyz * confz[flip_coo(k, jb, ib)] + BXMyz * confz[flip_coo(k, j, i-1)] + BYMyz * confz[flip_coo(k, j-1, i)] + BWMyz * confz[flip_coo(k, j-1, i-1)]\
       + BZMyx * confx[flip_coo(k-1, j, i)] + BZMyy * confy[flip_coo(k-1, j, i)];
    hz = BXPzx * confx[flip_coo(k, j, ib)] + BYPzx * confx[flip_coo(k, jb, i)] + BWPzx * confx[flip_coo(k, jb, ib)] + BXMzx * confx[flip_coo(k, j, i-1)] + BYMzx * confx[flip_coo(k, j-1, i)] + BWMzx * confx[flip_coo(k, j-1, i-1)]\
       + BXPzy * confy[flip_coo(k, j, ib)] + BYPzy * confy[flip_coo(k, jb, i)] + BWPzy * confy[flip_coo(k, jb, ib)] + BXMzy * confy[flip_coo(k, j, i-1)] + BYMzy * confy[flip_coo(k, j-1, i)] + BWMzy * confy[flip_coo(k, j-1, i-1)]\
       + BXPzz * confz[flip_coo(k, j, ib)] + BYPzz * confz[flip_coo(k, jb, i)] + BWPzz * confz[flip_coo(k, jb, ib)] + BXMzz * confz[flip_coo(k, j, i-1)] + BYMzz * confz[flip_coo(k, j-1, i)] + BWMzz * confz[flip_coo(k, j-1, i-1)] + H\
       + BZMzz * confz[flip_coo(k-1, j, i)];
    single_update(invT, hx, hy, hz, confx[flip_coo(k, j, i)], confy[flip_coo(k, j, i)], confz[flip_coo(k, j, i)], localState);
    __syncthreads();
  }

  //Load random number back to global memory
  state[threadIdx.x + blockIdx.x * flip_BlockSize_x * flip_BlockSize_y] = localState;
}
#endif
