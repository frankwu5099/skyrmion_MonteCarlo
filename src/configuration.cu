#include "hip/hip_runtime.h"
#ifndef CONFIGURATION_H
#define CONFIGURATION_H
#include "configuration.cuh"
#endif


configuration.configuration(int Pnum, char* conf_dir){
	Spin_mem_size = Pnum * N * sizeof(float);
	spins_num = Pnum * N;
	configurations_num = Pnum;
	sprintf(Confxfn, "%s/Confx", conf_dir);
	sprintf(Confyfn, "%s/Confy", conf_dir);
	sprintf(Confzfn, "%s/Confz", conf_dir);
	Hx = (float*)malloc(Spin_mem_size);
	Hy = (float*)malloc(Spin_mem_size);
	Hz = (float*)malloc(Spin_mem_size);
  Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
  Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
  Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
  if(hipMalloc((void**)&Dx, Spin_mem_size)){
    fprintf(stderr, "Error couldn't allocate Device Memory (x)!!\n");
    exit(1);
  }
  if(hipMalloc((void**)&Dy, Spin_mem_size)){
    fprintf(stderr, "Error couldn't allocate Device Memory (y)!!\n");
    exit(1);
  }
  if(hipMalloc((void**)&Dz, Spin_mem_size)){
    fprintf(stderr, "Error couldn't allocate Device Memory (z)!!\n");
    exit(1);
  }
}
void initialize (bool order){
	if (order == 0){
		double pi = 3.141592653589793;
		double th, phi;
		for(int i = 0; i < spins_num; i++){
			th = uni01_sampler() * pi;
			phi = uni01_sampler() * 2 * pi;
			Hconfx[i] = cos(th);
			th = sin(th);
			Hconfy[i] = th * cos(phi);
			Hconfz[i] = th * sin(phi);
		}
	}
	else {
		for(int i = 0; i < spins_num; i++){
			Hconfx[i] = 0;
			Hconfy[i] = 0;
			Hconfz[i] = 1;
		}
	}
  hipMemcpy(Dx, Hx, Spin_mem_size, hipMemcpyHostToDevice);
  hipMemcpy(Dy, Hy, Spin_mem_size, hipMemcpyHostToDevice);
  hipMemcpy(Dz, Hz, Spin_mem_size, hipMemcpyHostToDevice);
}
void configuration.backtoHost(){
	hipMemcpy(Hx, Dx, Spin_mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(Hy, Dy, Spin_mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(Hz, Dz, Spin_mem_size, hipMemcpyDeviceToHost);
  free(Hx);
  free(Hy);
  free(Hz);
  hipFree(Dx);
  hipFree(Dy);
  hipFree(Dz);
  //hipFree(Dcorr);
}
void configuration.write(){
	write(Confxfd, Hconfx, Spin_mem_size);
	write(Confyfd, Hconfy, Spin_mem_size);
	write(Confzfd, Hconfz, Spin_mem_size);
}

configuration.~configuration(){
  close(Confxfd);
  close(Confyfd);
  close(Confzfd);
}
