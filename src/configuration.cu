#include "hip/hip_runtime.h"
#include "configuration.cuh"


configuration::configuration(int Pnum, char* conf_dir){
  Spin_mem_size = Pnum * N * sizeof(float);
  spins_num = Pnum * N;
  configurations_num = Pnum;
  sprintf(Confxfn, "%s/Confx", conf_dir);
  sprintf(Confyfn, "%s/Confy", conf_dir);
  sprintf(Confzfn, "%s/Confz", conf_dir);
  Hx = (float*)malloc(Spin_mem_size);
  Hy = (float*)malloc(Spin_mem_size);
  Hz = (float*)malloc(Spin_mem_size);
  Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
  Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
  Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
  CudaSafeCall(hipMalloc((void**)&Dx, Spin_mem_size));
  CudaSafeCall(hipMalloc((void**)&Dy, Spin_mem_size));
  CudaSafeCall(hipMalloc((void**)&Dz, Spin_mem_size));
}



void configuration::initialize (bool order){
  if (order == 0){
    double pi = 3.141592653589793;
    double th, phi;
    for(int i = 0; i < spins_num; i++){
      th = uni01_sampler() * pi;
      phi = uni01_sampler() * 2 * pi;
      Hx[i] = cos(th);
      th = sin(th);
      Hy[i] = th * cos(phi);
      Hz[i] = th * sin(phi);
    }
  }
  else {
    for(int i = 0; i < spins_num; i++){
      Hx[i] = 0;
      Hy[i] = 0;
      Hz[i] = 1;
    }
  }
  CudaSafeCall(hipMemcpy(Dx, Hx, Spin_mem_size, hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(Dy, Hy, Spin_mem_size, hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(Dz, Hz, Spin_mem_size, hipMemcpyHostToDevice));
}
void configuration::backtoHost(){
  CudaSafeCall(hipMemcpy(Hx, Dx, Spin_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(Hy, Dy, Spin_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(Hz, Dz, Spin_mem_size, hipMemcpyDeviceToHost));
  free(Hx);
  free(Hy);
  free(Hz);
  CudaSafeCall(hipFree(Dx));
  CudaSafeCall(hipFree(Dy));
  CudaSafeCall(hipFree(Dz));
  //hipFree(Dcorr);
}
void configuration::writedata(){
  write(Confxfd, Hx, Spin_mem_size);
  write(Confyfd, Hy, Spin_mem_size);
  write(Confzfd, Hz, Spin_mem_size);
}

configuration::~configuration(){
  close(Confxfd);
  close(Confyfd);
  close(Confzfd);
}
