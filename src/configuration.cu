#include "hip/hip_runtime.h"
#include "configuration.cuh"


configuration::configuration(int Pnum, char* conf_dir){
  configurations_num = Pnum;
  configurations_num_s = Pnum/StreamN;
  Spin_mem_size = configurations_num * H_N * sizeof(float);
  Single_mem_size = H_N * sizeof(float);
  spins_num = configurations_num * H_N;
  Spin_mem_size_s = configurations_num_s * H_N * sizeof(float);
  spins_num_s = configurations_num_s * H_N;
  sprintf(Confxfn, "%s/Confx", conf_dir);
  sprintf(Confyfn, "%s/Confy", conf_dir);
  sprintf(Confzfn, "%s/Confz", conf_dir);
  Hx = (float*)malloc(Spin_mem_size);
  Hy = (float*)malloc(Spin_mem_size);
  Hz = (float*)malloc(Spin_mem_size);
  Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
  Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
  Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
  Dx = (float**)calloc(StreamN, sizeof(float*));
  Dy = (float**)calloc(StreamN, sizeof(float*));
  Dz = (float**)calloc(StreamN, sizeof(float*));
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&Dx[gpu_i], Spin_mem_size_s));
    CudaSafeCall(hipMalloc((void**)&Dy[gpu_i], Spin_mem_size_s));
    CudaSafeCall(hipMalloc((void**)&Dz[gpu_i], Spin_mem_size_s));
  }
}



void configuration::initialize (bool order){
  if (order == 0){
    double pi = 3.141592653589793;
    double th, phi;
    for(int i = 0; i < spins_num; i++){
      th = uni01_sampler() * pi;
      phi = uni01_sampler() * 2 * pi;
      Hx[i] = cos(th);
      th = sin(th);
      Hy[i] = th * cos(phi);
      Hz[i] = th * sin(phi);
    }
  }
  else {
    for(int i = 0; i < spins_num; i++){
      Hx[i] = 0;
      Hy[i] = 0;
      Hz[i] = 1;
    }
  }
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(cudaMemcpyasync(Dx[gpu_i], Hx + gpu_i * spins_nums, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(cudaMemcpyasync(Dy[gpu_i], Hy + gpu_i * spins_nums, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(cudaMemcpyasync(Dz[gpu_i], Hz + gpu_i * spins_nums, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }
}
void configuration::backtoHost(){
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(cudaMemcpyasync(Hx + gpu_i * spins_nums, Dx[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
    CudaSafeCall(cudaMemcpyasync(Hy + gpu_i * spins_nums, Dy[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
    CudaSafeCall(cudaMemcpyasync(Hz + gpu_i * spins_nums, Dz[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }
}

void configuration::Dominatestateback(int hostid, int deviceid){
  hipSetDevice(device_0 + deviceid/configurations_num_s);
  CudaSafeCall(hipMemcpy(((float*)Hx) + hostid * H_N, ((float*)Dx[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(((float*)Hy) + hostid * H_N, ((float*)Dy[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(((float*)Hz) + hostid * H_N, ((float*)Dz[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  //hipFree(Dcorr);
}
void configuration::writedata(){
  write(Confxfd, Hx, Spin_mem_size);
  write(Confyfd, Hy, Spin_mem_size);
  write(Confzfd, Hz, Spin_mem_size);
}

configuration::~configuration(){
  printf("conf free begin!\n");
  fflush(stdout);
  free(Hx);
  free(Hy);
  free(Hz);
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipFree(Dx[gpu_i]));
    CudaSafeCall(hipFree(Dy[gpu_i]));
    CudaSafeCall(hipFree(Dz[gpu_i]));
  }
  close(Confxfd);
  close(Confyfd);
  close(Confzfd);
  printf("conf free succeed!\n");
  fflush(stdout);
}
