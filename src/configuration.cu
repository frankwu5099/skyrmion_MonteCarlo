#include "hip/hip_runtime.h"
#include "configuration.cuh"


configuration::configuration(int Pnum, char* conf_dir){
	f_index = 0;
  configurations_num = Pnum;
  configurations_num_s = Pnum/StreamN;
  Spin_mem_size = configurations_num * H_N * sizeof(float);
  Single_mem_size = H_N * sizeof(float);
  spins_num = configurations_num * H_N;
  Spin_mem_size_s = configurations_num_s * H_N * sizeof(float);
  spins_num_s = configurations_num_s * H_N;
  sprintf(dirfn, "%s", conf_dir);
  sprintf(Confxfn, "%s/Confx_%d", dirfn, f_index);
  sprintf(Confyfn, "%s/Confy_%d", dirfn, f_index);
  sprintf(Confzfn, "%s/Confz_%d", dirfn, f_index);
  Hx = (float*)malloc(Spin_mem_size);
  Hy = (float*)malloc(Spin_mem_size);
  Hz = (float*)malloc(Spin_mem_size);
  Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
  Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
  Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
  Dx = (float**)calloc(StreamN, sizeof(float*));
  Dy = (float**)calloc(StreamN, sizeof(float*));
  Dz = (float**)calloc(StreamN, sizeof(float*));
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&Dx[gpu_i], Spin_mem_size_s));
    CudaSafeCall(hipMalloc((void**)&Dy[gpu_i], Spin_mem_size_s));
    CudaSafeCall(hipMalloc((void**)&Dz[gpu_i], Spin_mem_size_s));
  }
}



void configuration::initialize (bool order){
  if (order == 0){
    double pi = 3.141592653589793;
    double th, phi;
    for(int i = 0; i < spins_num; i++){
      th = uni01_sampler() * pi;
      phi = uni01_sampler() * 2 * pi;
      Hx[i] = cos(th);
      th = sin(th);
      Hy[i] = th * cos(phi);
      Hz[i] = th * sin(phi);
    }
  }
  else {
    /*
    for(int i = 0; i < spins_num; i++){
      Hx[i] = 0;
      Hy[i] = 0;
      Hz[i] = 1;
    }*/
    double pi = 3.141592653589793;
    double th, phi;
    for(int i_P = 0; i_P < configurations_num; i_P++){
      for(int iz = 0; iz < H_SpinSize_z; iz++){
        for(int ix = 0; ix < H_SpinSize; ix++){
          for(int iy = 0; iy < H_SpinSize; iy++){
            phi = (ix-0.5*iy) * pi/3.0;
            Hx[H_N * i_P + H_Nplane * iz + H_SpinSize * iy + ix] = 0;
            Hy[H_N * i_P + H_Nplane * iz + H_SpinSize * iy + ix] = cos(phi);
            Hz[H_N * i_P + H_Nplane * iz + H_SpinSize * iy + ix] = sin(phi);
          }
        }
      }
    }
  }
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(Dx[gpu_i], Hx + gpu_i * spins_num_s, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(Dy[gpu_i], Hy + gpu_i * spins_num_s, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(Dz[gpu_i], Hz + gpu_i * spins_num_s, Spin_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
  }
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }
}
void configuration::backtoHost(){
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(Hx + gpu_i * spins_num_s, Dx[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(Hy + gpu_i * spins_num_s, Dy[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(Hz + gpu_i * spins_num_s, Dz[gpu_i], Spin_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));
  }
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }
}

void configuration::Dominatestateback(int hostid, int deviceid){
  hipSetDevice(device_0 + deviceid/configurations_num_s);
  CudaSafeCall(hipMemcpy(((float*)Hx) + hostid * H_N, ((float*)Dx[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(((float*)Hy) + hostid * H_N, ((float*)Dy[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(((float*)Hz) + hostid * H_N, ((float*)Dz[deviceid/configurations_num_s]) + (deviceid%configurations_num_s) * H_N, Single_mem_size, hipMemcpyDeviceToHost));
  //hipFree(Dcorr);
}
void configuration::writedata(){
  if (f_index > 30){
    for (int i = 0; i < configurations_num; i+=2){
            write(Confzfd, Hz+(i*H_N), H_N * sizeof(float));
            write(Confyfd, Hy+(i*H_N), H_N * sizeof(float));
            write(Confxfd, Hx+(i*H_N), H_N * sizeof(float));
    }
  }
  close(Confxfd);
  close(Confyfd);
  close(Confzfd);
  f_index += 1;
  sprintf(Confxfn, "%s/Confx_%d", dirfn, f_index);
  sprintf(Confyfn, "%s/Confy_%d", dirfn, f_index);
  sprintf(Confzfn, "%s/Confz_%d", dirfn, f_index);
  Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
  Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
  Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
}

configuration::~configuration(){
  printf("conf free begin!\n");
  fflush(stdout);
  free(Hx);
  free(Hy);
  free(Hz);
  for (int gpu_i = 0 ; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipFree(Dx[gpu_i]));
    CudaSafeCall(hipFree(Dy[gpu_i]));
    CudaSafeCall(hipFree(Dz[gpu_i]));
  }
  close(Confxfd);
  close(Confyfd);
  close(Confzfd);
  printf("conf free succeed!\n");
  fflush(stdout);
}
