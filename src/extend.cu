#include "hip/hip_runtime.h"
#ifndef PARAMS_H
#define PARAMS_H
#include "params.cuh"
#endif
int setDev(){
  int num_devices,flag=0;
  hipError_t error = cudaErrorDevicesUnavailable;
  hipGetDeviceCount(&num_devices);
  bool *Dtest;
  for (int device = 0; device < num_devices; device++) {//
    hipSetDevice(device);
    error = hipMalloc((void**)&Dtest, 100*sizeof(bool));
    if (error == hipSuccess){
      printf("using device %d !\n",device);
      hipFree(Dtest);
      break;

    }else{
      /* if GPU busy*/
      if (error == cudaErrorDevicesUnavailable){
	printf("device %d is busy >> try another!\n",device);
      }else{
	printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
	printf("CUDA error: %s\n", hipGetErrorString(error));
      }
      if(device==num_devices-1)
	{printf("%s\n","ERROR! no avalible device now!");flag=1;}
    }

  }
  return flag ;
}


/*
void debugg(float *x,float *y,float *ang){

  for(int i=25;i<40;i++){

    printf("%d-Hang: %f\tHcos: %f\tHsin: %f\ngetA: %f\n",i,ang[i],x[i],y[i],getangf(x[i],y[i],sqrt(x[i]*x[i]+y[i]*y[i])));


  }


}
*/
