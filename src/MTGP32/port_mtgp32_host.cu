#include "port_mtgp32_host.h"


void mtgp32_init_state(unsigned int state[],
              const mtgp32_params_fast_t *para, unsigned int seed) {
    int i;
    int size = para->mexp / 32 + 1;
    unsigned int hidden_seed;
    unsigned int tmp;
    hidden_seed = para->tbl[4] ^ (para->tbl[8] << 16);
    tmp = hidden_seed;
    tmp += tmp >> 16;
    tmp += tmp >> 8;
    memset(state, tmp & 0xff, sizeof(unsigned int) * size);
    state[0] = seed;
    state[1] = hidden_seed;
    for (i = 1; i < size; i++) {
    state[i] ^= (1812433253) * (state[i - 1]
                        ^ (state[i - 1] >> 30))
        + i;
    }
}


/*
 * This function initializes the internal state array
 * with a 32-bit integer array. \b para should be one of the elements in
 * the parameter table (mtgp32-param-ref.c).
 *
 * @param[out] mtgp32 MTGP structure.
 * @param[in] para parameter structure
 * @param[in] array a 32-bit integer array used as a seed.
 * @param[in] length length of the array.
 * @return HIPRAND_STATUS_SUCCESS 
 */

int mtgp32_init_by_array(unsigned int state[],
             const mtgp32_params_fast_t *para,
             unsigned int *array, int length) {
    int i, j, count;
    unsigned int r;
    int lag;
    int mid;
    int size = para->mexp / 32 + 1;
    unsigned int hidden_seed;
    unsigned int tmp;

    if (size >= 623) {
    lag = 11;
    } else if (size >= 68) {
    lag = 7;
    } else if (size >= 39) {
    lag = 5;
    } else {
    lag = 3;
    }
    mid = (size - lag) / 2;

    hidden_seed = para->tbl[4] ^ (para->tbl[8] << 16);
    tmp = hidden_seed;
    tmp += tmp >> 16;
    tmp += tmp >> 8;
    memset(state, tmp & 0xff, sizeof(unsigned int) * size);
    state[0] = hidden_seed;

    if (length + 1 > size) {
    count = length + 1;
    } else {
    count = size;
    }
    r = ini_func1(state[0] ^ state[mid] ^ state[size - 1]);
    state[mid] += r;
    r += length;
    state[(mid + lag) % size] += r;
    state[0] = r;
    i = 1;
    count--;
    for (i = 1, j = 0; (j < count) && (j < length); j++) {
    r = ini_func1(state[i] ^ state[(i + mid) % size]
              ^ state[(i + size - 1) % size]);
    state[(i + mid) % size] += r;
    r += array[j] + i;
    state[(i + mid + lag) % size] += r;
    state[i] = r;
    i = (i + 1) % size;
    }
    for (; j < count; j++) {
    r = ini_func1(state[i] ^ state[(i + mid) % size]
              ^ state[(i + size - 1) % size]);
    state[(i + mid) % size] += r;
    r += i;
    state[(i + mid + lag) % size] += r;
    state[i] = r;
    i = (i + 1) % size;
    }
    for (j = 0; j < size; j++) {
    r = ini_func2(state[i] + state[(i + mid) % size]
              + state[(i + size - 1) % size]);
    state[(i + mid) % size] ^= r;
    r -= i;
    state[(i + mid + lag) % size] ^= r;
    state[i] = r;
    i = (i + 1) % size;
    }
    if (state[size - 1] == 0) {
    state[size - 1] = non_zero;
    }
    return 0;
}

/*
 * This function initializes the internal state array
 * with a character array. \b para should be one of the elements in
 * the parameter table (mtgp32-param-ref.c).
 * This is the same algorithm with mtgp32_init_by_array(), but hope to
 * be more useful.
 *
 * @param[out] mtgp32 MTGP structure.
 * @param[in] para parameter structure
 * @param[in] array a character array used as a seed. (terminated by zero.)
 * @return memory allocation result. if 0 then O.K.
 */
int mtgp32_init_by_str(unsigned int state[],
               const mtgp32_params_fast_t *para, unsigned char *array) {
    int i, j, count;
    unsigned int r;
    int lag;
    int mid;
    int size = para->mexp / 32 + 1;
    int length = (unsigned int)strlen((char *)array);
    unsigned int hidden_seed;
    unsigned int tmp;

    if (size >= 623) {
    lag = 11;
    } else if (size >= 68) {
    lag = 7;
    } else if (size >= 39) {
    lag = 5;
    } else {
    lag = 3;
    }
    mid = (size - lag) / 2;

    hidden_seed = para->tbl[4] ^ (para->tbl[8] << 16);
    tmp = hidden_seed;
    tmp += tmp >> 16;
    tmp += tmp >> 8;
    memset(state, tmp & 0xff, sizeof(unsigned int) * size);
    state[0] = hidden_seed;

    if (length + 1 > size) {
    count = length + 1;
    } else {
    count = size;
    }
    r = ini_func1(state[0] ^ state[mid] ^ state[size - 1]);
    state[mid] += r;
    r += length;
    state[(mid + lag) % size] += r;
    state[0] = r;
    i = 1;
    count--;
    for (i = 1, j = 0; (j < count) && (j < length); j++) {
    r = ini_func1(state[i] ^ state[(i + mid) % size]
              ^ state[(i + size - 1) % size]);
    state[(i + mid) % size] += r;
    r += array[j] + i;
    state[(i + mid + lag) % size] += r;
    state[i] = r;
    i = (i + 1) % size;
    }
    for (; j < count; j++) {
    r = ini_func1(state[i] ^ state[(i + mid) % size]
              ^ state[(i + size - 1) % size]);
    state[(i + mid) % size] += r;
    r += i;
    state[(i + mid + lag) % size] += r;
    state[i] = r;
    i = (i + 1) % size;
    }
    for (j = 0; j < size; j++) {
    r = ini_func2(state[i] + state[(i + mid) % size]
              + state[(i + size - 1) % size]);
    state[(i + mid) % size] ^= r;
    r -= i;
    state[(i + mid + lag) % size] ^= r;
    state[i] = r;
    i = (i + 1) % size;
    }
    if (state[size - 1] == 0) {
    state[size - 1] = non_zero;
    }
    return 0;
}



/**
 * \brief Set up constant parameters for the mtgp32 generator
 *
 * This host-side helper function re-organizes CURAND_NUM_MTGP32_PARAMS sets of 
 * generator parameters for use by kernel functions and copies the 
 * result to the specified location in device memory.
 *
 * \param params - Pointer to an array of type mtgp32_params_fast_t in host memory
 * \param p - pointer to a structure of type mtgp32_kernel_params_t in device memory.
 *
 * \return 
 * - HIPRAND_STATUS_ALLOCATION_FAILED if host memory could not be allocated
 * - HIPRAND_STATUS_INITIALIZATION_FAILED if the copy to device memory failed
 * - HIPRAND_STATUS_SUCCESS otherwise
 */
__host__ hiprandStatus_t hiprandMakeMTGP32Constants(const mtgp32_params_fast_t params[], mtgp32_kernel_params_t *& p) {
    const int block_num = CURAND_NUM_MTGP32_PARAMS;
    const int size1 = sizeof(unsigned int) * block_num;
    const int size2 = sizeof(unsigned int) * block_num * TBL_SIZE;
    unsigned int *h_pos_tbl;
    unsigned int *h_sh1_tbl;
    unsigned int *h_sh2_tbl;
    unsigned int *h_param_tbl;
    unsigned int *h_temper_tbl;
    unsigned int *h_single_temper_tbl;
    unsigned int *h_mask;
    hiprandStatus_t status = HIPRAND_STATUS_SUCCESS;
    
    h_pos_tbl = (unsigned int *)malloc(size1);
    h_sh1_tbl = (unsigned int *)malloc(size1);
    h_sh2_tbl = (unsigned int *)malloc(size1);
    h_param_tbl = (unsigned int *)malloc(size2);
    h_temper_tbl = (unsigned int *)malloc(size2);
    h_single_temper_tbl = (unsigned int *)malloc(size2);
    h_mask = (unsigned int *)malloc(sizeof(unsigned int));
    if (h_pos_tbl == NULL
	    || h_sh1_tbl == NULL
	    || h_sh2_tbl == NULL
	    || h_param_tbl == NULL
	    || h_temper_tbl == NULL
	    || h_single_temper_tbl == NULL
	    || h_mask == NULL) {
        if (h_pos_tbl != NULL) free(h_pos_tbl);
        if (h_sh1_tbl != NULL) free(h_sh1_tbl);
        if (h_sh2_tbl != NULL) free(h_sh2_tbl);
        if (h_param_tbl != NULL) free(h_param_tbl);
        if (h_temper_tbl != NULL) free(h_temper_tbl);
        if (h_single_temper_tbl != NULL) free(h_single_temper_tbl);
        if (h_mask != NULL) free(h_mask);
        status = HIPRAND_STATUS_ALLOCATION_FAILED;
    } else {       

        h_mask[0] = params[0].mask;
        for (int i = 0; i < block_num; i++) {
	        h_pos_tbl[i] = params[i].pos;
	        h_sh1_tbl[i] = params[i].sh1;
	        h_sh2_tbl[i] = params[i].sh2;
	        for (int j = 0; j < TBL_SIZE; j++) {
	            h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j];
	            h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j];
	            h_single_temper_tbl[i * TBL_SIZE + j] = params[i].flt_tmp_tbl[j];
	        }
        }
        if (hipMemcpy( p->pos_tbl, 
                        h_pos_tbl, size1, hipMemcpyHostToDevice) != hipSuccess)
        { 
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->sh1_tbl, 
                        h_sh1_tbl, size1, hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->sh2_tbl, 
                        h_sh2_tbl, size1, hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->param_tbl, 
                        h_param_tbl, size2, hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->temper_tbl, 
                        h_temper_tbl, size2, hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->single_temper_tbl, 
                        h_single_temper_tbl, size2, hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } else
        if (hipMemcpy( p->mask, 
                        h_mask, sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess)
        {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        } 
    }
    if (h_pos_tbl != NULL) free(h_pos_tbl);
    if (h_sh1_tbl != NULL) free(h_sh1_tbl);
    if (h_sh2_tbl != NULL) free(h_sh2_tbl);
    if (h_param_tbl != NULL) free(h_param_tbl);
    if (h_temper_tbl != NULL) free(h_temper_tbl);
    if (h_single_temper_tbl != NULL)free(h_single_temper_tbl);
    if (h_mask != NULL) free(h_mask);
    return status;
}

/**
 * \brief Set up initial states for the mtgp32 generator
 *
 * This host-side helper function initializes a number of states (one parameter set per state) for 
 * an mtgp32 generator. To accomplish this it allocates a state array in host memory,
 * initializes that array, and copies the result to device memory.
 *
 V* \param s - pointer to an array of states in device memory
 * \param params - Pointer to an array of type mtgp32_params_fast_t in host memory
 * \param k - pointer to a structure of type mtgp32_kernel_params_t in device memory
 * \param n - number of parameter sets/states to initialize
 * \param seed - seed value
 *
 * \return 
 * - HIPRAND_STATUS_ALLOCATION_FAILED if host memory state could not be allocated 
 * - HIPRAND_STATUS_INITIALIZATION_FAILED if the copy to device memory failed
 * - HIPRAND_STATUS_SUCCESS otherwise
 */
__host__ hiprandStatus_t CURANDAPI hiprandMakeMTGP32KernelState(hiprandStateMtgp32_t *s,
                                            mtgp32_params_fast_t params[],
                                            mtgp32_kernel_params_t *k,
                                            int n,
                                            unsigned long long seed)
{
    int i;
    hiprandStatus_t status = HIPRAND_STATUS_SUCCESS;
    hiprandStateMtgp32_t *h_status =(hiprandStateMtgp32_t *) malloc(sizeof(hiprandStateMtgp32_t) * n);
    if (h_status == NULL) {
        status = HIPRAND_STATUS_ALLOCATION_FAILED;
    } else {
        seed = seed ^ (seed >> 32);
        for (i = 0; i < n; i++) {
            mtgp32_init_state(&(h_status[i].s[0]), &params[i],(unsigned int)seed + i + 1);
            h_status[i].offset = 0;
            h_status[i].pIdx = i;
            h_status[i].k = k;
            h_status[i].precise_double_flag = 0;
        }
        if (hipMemcpy(s, h_status,
                       sizeof(hiprandStateMtgp32_t) * n,
                       hipMemcpyHostToDevice) != hipSuccess) {
            status = HIPRAND_STATUS_INITIALIZATION_FAILED;
        }
     }
    free(h_status);
    return status;
}



