#include "hip/hip_runtime.h"
#include "params.cuh"

using namespace std;
unsigned int block;
unsigned int grid;
unsigned int rngShmemsize;
unsigned int caloutputsize;
unsigned int H_SpinSize;
unsigned int H_SpinSize_z;
unsigned int H_BlockSize_x;
unsigned int H_BlockSize_y;
unsigned int H_GridSize_x;
unsigned int H_GridSize_y;
unsigned int H_N;
unsigned int H_Nplane;
unsigned int H_TN;
unsigned int H_BN;
//------ system size setting end --------

//------ gpu setting -------
int StreamN;
int device_0;
hipStream_t stream[10];
//---- gpu setting end -----

//------ system variable setting --------
//!!!!!!!!!!!!notice that the value of DD and DR are set while compile for the efficiency of triangular lattic.
float H_A; //(0.0)
float DD; //(0.0)
float DR; //(0.0)
float H_Q1x; //(0.0)
float H_Q1y; //(0.0)
float H_Q2x; //(0.0)
float H_Q2y; //(0.0)
//----- system variable setting end ------

//----- simulation setting ------
unsigned int BIN_SZ;
unsigned int BIN_NUM;
unsigned int EQUI_N;
unsigned int EQUI_Ni;
unsigned int relax_N;

float PTF = 0.1;             //Frequency of parallel tempering
string Output;  //set the output directory


void read_params(char* param_file){
  FILE* paramfp = fopen(param_file, "r");
  char tmp[128], readidx;
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize);
  if ((readidx == -1)||(strcmp(tmp,"Size")!=0)){
    printf("read size error");
    exit(0);
  }
  if (H_SpinSize % 3 != 0){
    fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize_z);
  if ((readidx == -1)||(strcmp(tmp,"Size_z")!=0)){
    printf("read size error");
    exit(0);
  }
  //if (H_SpinSize_z % 16 != 0){
  //  fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
  //  exit(0);
  //}
#ifdef THIN
#endif
#ifndef THIN
  //H_SpinSize_z = 1;
#endif
#ifdef TRI
  H_BlockSize_x = H_SpinSize / 3;
  H_BlockSize_y = H_SpinSize / 3;
  for (int tmpi = 0 ; tmpi < 10;tmpi++){
    H_BlockSize_x = (H_BlockSize_x > 16)?(H_BlockSize_x/2):H_BlockSize_x;
    H_BlockSize_y = (H_BlockSize_y > 16)?(H_BlockSize_y/2):H_BlockSize_y;
  }
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 3;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 3;
#endif
#ifndef TRI
  H_BlockSize_x = H_SpinSize / 2;
  H_BlockSize_y = H_SpinSize / 2;
  H_BlockSize_x = (H_BlockSize_x > 32)?32:H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?16:H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 2;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 2;
#endif
  H_N = H_SpinSize * H_SpinSize * H_SpinSize_z;
  H_Nplane = H_SpinSize * H_SpinSize;
  H_TN = H_BlockSize_x * H_BlockSize_y;
  H_BN = H_GridSize_x * H_GridSize_y;
  block = H_BlockSize_x * H_BlockSize_y;
  printf("%d\n", block);
  fflush(stdout);
  caloutputsize = block * sizeof(double);
  rngShmemsize = block * 8 * sizeof(unsigned);

  readidx = fscanf(paramfp, "%s %f", tmp, &H_A);
  if ((readidx == -1)||(strcmp(tmp,"A")!=0)){
    printf("read A error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &DR);
  if ((readidx == -1)||(strcmp(tmp,"DR")!=0)){
    printf("read DR error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &DD);
  if ((readidx == -1)||(strcmp(tmp,"DD")!=0)){
    printf("read DD error");
    exit(0);
  }
  H_Q1x = atan(sqrt((DD*DD+DR*DR)));//atan(sqrt((DD*DD+DR*DR)/2.0));
  H_Q1y = -0.5*atan(sqrt((DD*DD+DR*DR)));
  H_Q2x = 2*H_Q1x;
  H_Q2y = 2*H_Q1y;

  //----- system variable setting end ------

  //----- simulation setting ------
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_SZ);
  if ((readidx == -1)||(strcmp(tmp,"BIN_SIZE")!=0)){
    printf("read bin size error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_NUM);
  if ((readidx == -1)||(strcmp(tmp,"BIN_NUM")!=0)){
    printf("read bin number error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_N);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_N")!=0)){
    printf("read EQUI_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_Ni);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_Ni")!=0)){
    printf("read EQUI_Ni error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &relax_N);
  if ((readidx == -1)||(strcmp(tmp,"relax_N")!=0)){
    printf("read relax_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &PTF);
  if ((readidx == -1)||(strcmp(tmp,"PTF")!=0)){
    printf("read parallel tempering frequency error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &f_CORR);
  if ((readidx == -1)||(strcmp(tmp,"f_CORR")!=0)){
    printf("read f_CORR error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &CORR_N);
  if ((readidx == -1)||(strcmp(tmp,"CORR_N")!=0)){
    printf("read CORR_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %s", tmp, &Output);
  if ((readidx == -1)||(strcmp(tmp,"Output_dir")!=0)){
    printf("read output dir error");
    exit(0);
  }
  fclose(paramfp);
  //-- simulation setting end --
}

json read_json(){
  printf("Loading configuration...");
  fflush(stdout);
    std::ifstream jsoni("config.json");
	json allj;
	jsoni >> allj;
  json configj = allj["parameters"];
  json ensemblej = allj["ensemble"];

  H_SpinSize = configj["Size"];
  H_SpinSize_z = configj["Size_z"];
  if (H_SpinSize % 3 != 0){
    fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
    exit(0);
  }
  // cuda spin configuration setup
#ifdef THIN
#endif
#ifndef THIN
  //H_SpinSize_z = 1;
#endif
#ifdef TRI
  H_BlockSize_x = H_SpinSize / 3;
  H_BlockSize_y = H_SpinSize / 3;
  for (int tmpi = 0 ; tmpi < 10;tmpi++){
    H_BlockSize_x = (H_BlockSize_x > 16)?(H_BlockSize_x/2):H_BlockSize_x;
    H_BlockSize_y = (H_BlockSize_y > 16)?(H_BlockSize_y/2):H_BlockSize_y;
  }
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 3;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 3;
#endif
#ifndef TRI
  H_BlockSize_x = H_SpinSize / 2;
  H_BlockSize_y = H_SpinSize / 2;
  H_BlockSize_x = (H_BlockSize_x > 32)?32:H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?16:H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 2;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 2;
#endif
  H_N = H_SpinSize * H_SpinSize * H_SpinSize_z;
  H_Nplane = H_SpinSize * H_SpinSize;
  H_TN = H_BlockSize_x * H_BlockSize_y;
  H_BN = H_GridSize_x * H_GridSize_y;
  block = H_BlockSize_x * H_BlockSize_y;
  printf("%d\n", block);
  fflush(stdout);
  caloutputsize = block * sizeof(double);
  rngShmemsize = block * 8 * sizeof(unsigned);
  // end spin configuration setup

  H_A = configj["A"];
  DR = configj["DR"];
  DD = configj["DD"];

  H_Q1x = atan(sqrt((DD*DD+DR*DR)));//atan(sqrt((DD*DD+DR*DR)/2.0));
  H_Q1y = -0.5*atan(sqrt((DD*DD+DR*DR)));
  H_Q2x = 2*H_Q1x;
  H_Q2y = 2*H_Q1y;

  //----- system variable setting end ------

  //----- simulation setting ------
  BIN_SZ = configj["BIN_SIZE"];
  BIN_NUM = configj["BIN_NUM"];
  EQUI_N = configj["EQUI_N"];
  EQUI_Ni = configj["EQUI_Ni"];
  relax_N = configj["relax_N"];
  PTF = configj["PTF"];
  f_CORR = configj["f_CORR"];
  CORR_N = configj["CORR_N"];
  Output = configj["Output_dir"];
  //-- simulation setting end --
  

  //-- ensemble setting -- 
  Tnum = ensemblej["NumTaxis"];
  Hnum = ensemblej["NumHaxis"];

  //-- ensemble setting end --
    vector<float> tmpTls = ensemblej["Ts"];
    vector<float> tmpHls = ensemblej["Hs"];
    transform(tmpHls.begin(), tmpHls.end(), tmpHls.begin(),
               bind(multiplies<float>(), std::placeholders::_1, (DR*DR + DD*DD)));// bind1st(multiplies<T>(), (DR*DR + DD*DD)));//std::
	Tls.push_back(tmpTls);
	Hls.push_back(tmpHls);
	tmpTls.clear();
	tmpHls.clear();
    Pnum = Tls[0].size();
    Cnum = Tls.size();
    if (Tnum * Hnum != Pnum){
      fprintf(stderr, "wrong temperatures and fields!!!\n");
      exit(0);
    }
  // back up the config
  return allj;
}
