#include "hip/hip_runtime.h"
#include "params.cuh"
unsigned int block;
unsigned int grid;
unsigned int rngShmemsize;
unsigned int caloutputsize;
unsigned int H_SpinSize;
unsigned int H_SpinSize_z;
unsigned int H_BlockSize_x;
unsigned int H_BlockSize_y;
unsigned int H_GridSize_x;
unsigned int H_GridSize_y;
unsigned int H_N;
unsigned int H_Nplane;
unsigned int H_TN;
unsigned int H_BN;
//------ system size setting end --------

//------ gpu setting -------
int StreamN;
int device_0;
hipStream_t stream[10];
//---- gpu setting end -----

//------ system variable setting --------
//!!!!!!!!!!!!notice that the value of DD and DR are set while compile for the efficiency of triangular lattic.
float H_A; //(0.0)
float DD; //(0.0)
float DR; //(0.0)
float H_Q1x; //(0.0)
float H_Q1y; //(0.0)
float H_Q2x; //(0.0)
float H_Q2y; //(0.0)
//----- system variable setting end ------

//----- simulation setting ------
unsigned int BIN_SZ;
unsigned int BIN_NUM;
unsigned int EQUI_N;
unsigned int EQUI_Ni;
unsigned int relax_N;

float PTF = 0.1;             //Frequency of parallel tempering
char Output[128];  //set the output directory


void read_params(char* param_file){
  FILE* paramfp = fopen(param_file, "r");
  char tmp[128], readidx;
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize);
  if ((readidx == -1)||(strcmp(tmp,"Size")!=0)){
    printf("read size error");
    exit(0);
  }
  if (H_SpinSize % 24 != 0){
    fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize_z);
  if ((readidx == -1)||(strcmp(tmp,"Size_z")!=0)){
    printf("read size error");
    exit(0);
  }
  //if (H_SpinSize_z % 16 != 0){
  //  fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
  //  exit(0);
  //}
#ifdef THIN
#endif
#ifndef THIN
  //H_SpinSize_z = 1;
#endif
#ifdef TRI
  H_BlockSize_x = H_SpinSize / 3;
  H_BlockSize_y = H_SpinSize / 3;
  H_BlockSize_x = (H_BlockSize_x > 32)?(H_BlockSize_x/2):H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?(H_BlockSize_y/2):H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 3;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 3;
#endif
#ifndef TRI
  H_BlockSize_x = H_SpinSize / 2;
  H_BlockSize_y = H_SpinSize / 2;
  H_BlockSize_x = (H_BlockSize_x > 32)?32:H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?16:H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 2;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 2;
#endif
  H_N = H_SpinSize * H_SpinSize * H_SpinSize_z;
  H_Nplane = H_SpinSize * H_SpinSize;
  H_TN = H_BlockSize_x * H_BlockSize_y;
  H_BN = H_GridSize_x * H_GridSize_y;
  block = H_BlockSize_x * H_BlockSize_y;
  printf("%d\n", block);
  fflush(stdout);
  caloutputsize = block * sizeof(double);
  rngShmemsize = block * 4 * sizeof(unsigned);

  readidx = fscanf(paramfp, "%s %f", tmp, &H_A);
  if ((readidx == -1)||(strcmp(tmp,"A")!=0)){
    printf("read A error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &DR);
  if ((readidx == -1)||(strcmp(tmp,"DR")!=0)){
    printf("read DR error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &DD);
  if ((readidx == -1)||(strcmp(tmp,"DD")!=0)){
    printf("read DD error");
    exit(0);
  }
  H_Q1x = atan(sqrt((DD*DD+DR*DR)));//atan(sqrt((DD*DD+DR*DR)/2.0));
  H_Q1y = -0.5*atan(sqrt((DD*DD+DR*DR)));
  H_Q2x = 2*H_Q1x;
  H_Q2y = 2*H_Q1y;

  //----- system variable setting end ------

  //----- simulation setting ------
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_SZ);
  if ((readidx == -1)||(strcmp(tmp,"BIN_SIZE")!=0)){
    printf("read bin size error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_NUM);
  if ((readidx == -1)||(strcmp(tmp,"BIN_NUM")!=0)){
    printf("read bin number error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_N);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_N")!=0)){
    printf("read EQUI_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_Ni);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_Ni")!=0)){
    printf("read EQUI_Ni error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &relax_N);
  if ((readidx == -1)||(strcmp(tmp,"relax_N")!=0)){
    printf("read relax_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &PTF);
  if ((readidx == -1)||(strcmp(tmp,"PTF")!=0)){
    printf("read parallel tempering frequency error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &f_CORR);
  if ((readidx == -1)||(strcmp(tmp,"f_CORR")!=0)){
    printf("read f_CORR error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &CORR_N);
  if ((readidx == -1)||(strcmp(tmp,"CORR_N")!=0)){
    printf("read CORR_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %s", tmp, &Output);
  if ((readidx == -1)||(strcmp(tmp,"Output_dir")!=0)){
    printf("read output dir error");
    exit(0);
  }
  fclose(paramfp);
  //-- simulation setting end --
}
