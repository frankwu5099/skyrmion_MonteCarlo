#include "params.cuh"
unsigned int block;
unsigned int grid;
unsigned int rngShmemsize;
unsigned int caloutputsize;
__constant__ unsigned int SpinSize;
__constant__ unsigned int SpinSize_z;
__constant__ unsigned int BlockSize_x;
__constant__ unsigned int BlockSize_y;
__constant__ unsigned int GridSize_x;
__constant__ unsigned int GridSize_y;
__constant__ unsigned int N;
__constant__ unsigned int Nplane;
__constant__ unsigned int TN;
__constant__ unsigned int BN;
unsigned int H_SpinSize;
unsigned int H_SpinSize_z;
unsigned int H_BlockSize_x;
unsigned int H_BlockSize_y;
unsigned int H_GridSize_x;
unsigned int H_GridSize_y;
unsigned int H_N;
unsigned int H_Nplane;
unsigned int H_TN;
unsigned int H_BN;
//------ system size setting end --------

//------ system variable setting --------
//!!!!!!!!!!!!notice that the value of DD and DR are set while compile for the efficiency of triangular lattic.
__constant__ float A; //(0.0)
float H_A; //(0.0)
//----- system variable setting end ------

//----- simulation setting ------
unsigned int BIN_SZ;
unsigned int BIN_NUM;
unsigned int EQUI_N;
unsigned int EQUI_Ni;
unsigned int relax_N;

float PTF = 0.1;             //Frequency of parallel tempering
char Output[128];  //set the output directory


void read_params(char* param_file){
  FILE* paramfp = fopen(param_file, "r");
  char tmp[128], readidx;
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize);
  if ((readidx == -1)||(strcmp(tmp,"Size")!=0)){
    printf("read size error");
    exit(0);
  }
  if (H_SpinSize % 16 != 0){
    fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
    exit(0);
  }
#ifdef THIN
  readidx = fscanf(paramfp, "%s %d", tmp, &H_SpinSize_z);
  if ((readidx == -1)||(strcmp(tmp,"Size")!=0)){
    printf("read size error");
    exit(0);
  }
  if (H_SpinSize_z % 16 != 0){
    fprintf(stderr, "Please give a legal Size or revise cals.cu.\n");
    exit(0);
  }
#endif
#ifndef THIN
  H_SpinSize_z = 1;
#endif
#ifdef TRI
  H_BlockSize_x = H_SpinSize / 3;
  H_BlockSize_y = H_SpinSize / 3;
  H_BlockSize_x = (H_BlockSize_x > 32)?32:H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?16:H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 3;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 3;
  H_N = H_SpinSize * H_SpinSize * H_SpinSize_z;
  H_Nplane = H_SpinSize * H_SpinSize;
  H_TN = H_Nplane / 9;
#endif
#ifndef TRI
  H_BlockSize_x = H_SpinSize / 2;
  H_BlockSize_y = H_SpinSize / 2;
  H_BlockSize_x = (H_BlockSize_x > 32)?32:H_BlockSize_x;
  H_BlockSize_y = (H_BlockSize_y > 16)?16:H_BlockSize_y;
  H_GridSize_x = H_SpinSize / H_BlockSize_x / 2;
  H_GridSize_y = H_SpinSize / H_BlockSize_y / 2;
  H_N = H_SpinSize * H_SpinSize * H_SpinSize_z;
  H_Nplane = H_SpinSize * H_SpinSize;
  H_TN = H_Nplane / 4;
#endif
  H_BN = H_GridSize_x * H_GridSize_y;
  block = H_BlockSize_x * H_BlockSize_y;
  caloutputsize = block * sizeof(double);
  rngShmemsize = block * 4 * sizeof(float);
  hipMemcpyToSymbol(HIP_SYMBOL( SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( N ), &H_N , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( Nplane ), &H_Nplane , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( BN), &H_BN, sizeof(unsigned int));

  readidx = fscanf(paramfp, "%s %f", tmp, &H_A);
  if ((readidx == -1)||(strcmp(tmp,"A")!=0)){
    printf("read A error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &H_DR);
  hipMemcpyToSymbol(HIP_SYMBOL( A ), &H_A , sizeof(float));

  //----- system variable setting end ------

  //----- simulation setting ------
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_SZ);
  if ((readidx == -1)||(strcmp(tmp,"BIN_SIZE")!=0)){
    printf("read bin size error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &BIN_NUM);
  if ((readidx == -1)||(strcmp(tmp,"BIN_NUM")!=0)){
    printf("read bin number error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_N);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_N")!=0)){
    printf("read EQUI_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &EQUI_Ni);
  if ((readidx == -1)||(strcmp(tmp,"EQUI_Ni")!=0)){
    printf("read EQUI_Ni error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %d", tmp, &relax_N);
  if ((readidx == -1)||(strcmp(tmp,"relax_N")!=0)){
    printf("read relax_N error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %f", tmp, &PTF);
  if ((readidx == -1)||(strcmp(tmp,"PTF")!=0)){
    printf("read parallel tempering frequency error");
    exit(0);
  }
  readidx = fscanf(paramfp, "%s %s", tmp, &Output);
  if ((readidx == -1)||(strcmp(tmp,"Output_dir")!=0)){
    printf("read output dir error");
    exit(0);
  }
  fclose(paramfp);
  //-- simulation setting end --
}
