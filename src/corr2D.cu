#include "hip/hip_runtime.h"
__global__ void getcorr(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
	/*****************************************************************
	Set ( original_i, original_j) as our original point.
	for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
	use the periodic condition to keep the index positive.
	We need to sum over different (original_i, original_j) to get the correlation.
	*****************************************************************/
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int ox = original_i;
	const int oy =(blockIdx.x / BN) * SpinSize + original_j;
	//const int txp = tx +1 ;
	//const int typ = ty +1 ;
	//const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	float sx00, sy00, sz00, sx01, sy01, sz01, sx02, sy02, sz02, sx03, sy03, sz03,
        sx10, sy10, sz10, sx11, sy11, sz11, sx12, sy12, sz12, sx13, sy13, sz13,
        sx20, sy20, sz20, sx21, sy21, sz21, sx22, sy22, sz22, sx23, sy23, sz23,
        sx30, sy30, sz30, sx31, sy31, sz31, sx32, sy32, sz32, sx33, sy33, sz33;
	int fx0, fy0,
	    fx1, fy1,
	    fx2, fy2,
	    fx3, fy3,
	    fx4, fy4; //from o to f
	//calculate all the final position first

	fx0 = (tx + original_i) % SpinSize;
	fx1 = (tx + original_i + 1) % SpinSize;
	fx2 = (tx + original_i + 2) % SpinSize;
	fx3 = (tx + original_i + 3) % SpinSize;
	fx4 = (tx + original_i + 4) % SpinSize;

	if((ty % SpinSize + original_j) >= SpinSize)	fy0 = ty + original_j - SpinSize;
	else  fy0 = ty + original_j;
	if((ty % SpinSize + original_j + 1) >= SpinSize)	fy1 = ty + original_j + 1 - SpinSize;
	else  fy1 = ty + original_j + 1;
	if((ty % SpinSize + original_j + 2) >= SpinSize)	fy2 = ty + original_j + 2 - SpinSize;
	else  fy2 = ty + original_j + 2;
	if((ty % SpinSize + original_j + 3) >= SpinSize)	fy3 = ty + original_j + 3 - SpinSize;
	else  fy3 = ty + original_j + 3;
	if((ty % SpinSize + original_j + 4) >= SpinSize)	fy4 = ty + original_j + 4 - SpinSize;
	else  fy4 = ty + original_j + 4;

	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[(oy) * SpinSize + ox];
  sy00 = confy[(oy) * SpinSize + ox];
  sz00 = confz[(oy) * SpinSize + ox];
  sx01 = confx[(oy) * SpinSize + ox+1];
  sy01 = confy[(oy) * SpinSize + ox+1];
  sz01 = confz[(oy) * SpinSize + ox+1];
  sx02 = confx[(oy) * SpinSize + ox+2];
  sy02 = confy[(oy) * SpinSize + ox+2];
  sz02 = confz[(oy) * SpinSize + ox+2];
  sx03 = confx[(oy) * SpinSize + ox+3];
  sy03 = confy[(oy) * SpinSize + ox+3];
  sz03 = confz[(oy) * SpinSize + ox+3];
  sx10 = confx[(oy+1) * SpinSize + ox];
  sy10 = confy[(oy+1) * SpinSize + ox];
  sz10 = confz[(oy+1) * SpinSize + ox];
  sx11 = confx[(oy+1) * SpinSize + ox+1];
  sy11 = confy[(oy+1) * SpinSize + ox+1];
  sz11 = confz[(oy+1) * SpinSize + ox+1];
  sx12 = confx[(oy+1) * SpinSize + ox+2];
  sy12 = confy[(oy+1) * SpinSize + ox+2];
  sz12 = confz[(oy+1) * SpinSize + ox+2];
  sx13 = confx[(oy+1) * SpinSize + ox+3];
  sy13 = confy[(oy+1) * SpinSize + ox+3];
  sz13 = confz[(oy+1) * SpinSize + ox+3];
  sx20 = confx[(oy+2) * SpinSize + ox];
  sy20 = confy[(oy+2) * SpinSize + ox];
  sz20 = confz[(oy+2) * SpinSize + ox];
  sx21 = confx[(oy+2) * SpinSize + ox+1];
  sy21 = confy[(oy+2) * SpinSize + ox+1];
  sz21 = confz[(oy+2) * SpinSize + ox+1];
  sx22 = confx[(oy+2) * SpinSize + ox+2];
  sy22 = confy[(oy+2) * SpinSize + ox+2];
  sz22 = confz[(oy+2) * SpinSize + ox+2];
  sx23 = confx[(oy+2) * SpinSize + ox+3];
  sy23 = confy[(oy+2) * SpinSize + ox+3];
  sz23 = confz[(oy+2) * SpinSize + ox+3];
  sx30 = confx[(oy+3) * SpinSize + ox];
  sy30 = confy[(oy+3) * SpinSize + ox];
  sz30 = confz[(oy+3) * SpinSize + ox];
  sx31 = confx[(oy+3) * SpinSize + ox+1];
  sy31 = confy[(oy+3) * SpinSize + ox+1];
  sz31 = confz[(oy+3) * SpinSize + ox+1];
  sx32 = confx[(oy+3) * SpinSize + ox+2];
  sy32 = confy[(oy+3) * SpinSize + ox+2];
  sz32 = confz[(oy+3) * SpinSize + ox+2];
  sx33 = confx[(oy+3) * SpinSize + ox+3];
  sy33 = confy[(oy+3) * SpinSize + ox+3];
  sz33 = confz[(oy+3) * SpinSize + ox+3];
  corr[ty * SpinSize + tx] += sx00 * confx[ fy0 * SpinSize + fx0] + sy00 * confy[ fy0 * SpinSize + fx0] + sz00 * confz[ fy0 * SpinSize + fx0] +
                              sx01 * confx[ fy0 * SpinSize + fx1] + sy01 * confy[ fy0 * SpinSize + fx1] + sz01 * confz[ fy0 * SpinSize + fx1] +
                              sx02 * confx[ fy0 * SpinSize + fx2] + sy02 * confy[ fy0 * SpinSize + fx2] + sz02 * confz[ fy0 * SpinSize + fx2] +
                              sx03 * confx[ fy0 * SpinSize + fx3] + sy03 * confy[ fy0 * SpinSize + fx3] + sz03 * confz[ fy0 * SpinSize + fx3] +
                              sx10 * confx[ fy1 * SpinSize + fx0] + sy10 * confy[ fy1 * SpinSize + fx0] + sz10 * confz[ fy1 * SpinSize + fx0] +
                              sx11 * confx[ fy1 * SpinSize + fx1] + sy11 * confy[ fy1 * SpinSize + fx1] + sz11 * confz[ fy1 * SpinSize + fx1] +
                              sx12 * confx[ fy1 * SpinSize + fx2] + sy12 * confy[ fy1 * SpinSize + fx2] + sz12 * confz[ fy1 * SpinSize + fx2] +
                              sx13 * confx[ fy1 * SpinSize + fx3] + sy13 * confy[ fy1 * SpinSize + fx3] + sz13 * confz[ fy1 * SpinSize + fx3] +
                              sx20 * confx[ fy2 * SpinSize + fx0] + sy20 * confy[ fy2 * SpinSize + fx0] + sz20 * confz[ fy2 * SpinSize + fx0] +
                              sx21 * confx[ fy2 * SpinSize + fx1] + sy21 * confy[ fy2 * SpinSize + fx1] + sz21 * confz[ fy2 * SpinSize + fx1] +
                              sx22 * confx[ fy2 * SpinSize + fx2] + sy22 * confy[ fy2 * SpinSize + fx2] + sz22 * confz[ fy2 * SpinSize + fx2] +
                              sx23 * confx[ fy2 * SpinSize + fx3] + sy23 * confy[ fy2 * SpinSize + fx3] + sz23 * confz[ fy2 * SpinSize + fx3] +
                              sx30 * confx[ fy3 * SpinSize + fx0] + sy30 * confy[ fy3 * SpinSize + fx0] + sz30 * confz[ fy3 * SpinSize + fx0] +
                              sx31 * confx[ fy3 * SpinSize + fx1] + sy31 * confy[ fy3 * SpinSize + fx1] + sz31 * confz[ fy3 * SpinSize + fx1] +
                              sx32 * confx[ fy3 * SpinSize + fx2] + sy32 * confy[ fy3 * SpinSize + fx2] + sz32 * confz[ fy3 * SpinSize + fx2] +
                              sx33 * confx[ fy3 * SpinSize + fx3] + sy33 * confy[ fy3 * SpinSize + fx3] + sz33 * confz[ fy3 * SpinSize + fx3] ;
  corr[ty * SpinSize + tx+1] += sx00 * confx[ fy0 * SpinSize + fx1] + sy00 * confy[ fy0 * SpinSize + fx1] + sz00 * confz[ fy0 * SpinSize + fx1] +
                                sx01 * confx[ fy0 * SpinSize + fx2] + sy01 * confy[ fy0 * SpinSize + fx2] + sz01 * confz[ fy0 * SpinSize + fx2] +
                                sx02 * confx[ fy0 * SpinSize + fx3] + sy02 * confy[ fy0 * SpinSize + fx3] + sz02 * confz[ fy0 * SpinSize + fx3] +
                                sx03 * confx[ fy0 * SpinSize + fx4] + sy03 * confy[ fy0 * SpinSize + fx4] + sz03 * confz[ fy0 * SpinSize + fx4] +
                                sx10 * confx[ fy1 * SpinSize + fx1] + sy10 * confy[ fy1 * SpinSize + fx1] + sz10 * confz[ fy1 * SpinSize + fx1] +
                                sx11 * confx[ fy1 * SpinSize + fx2] + sy11 * confy[ fy1 * SpinSize + fx2] + sz11 * confz[ fy1 * SpinSize + fx2] +
                                sx12 * confx[ fy1 * SpinSize + fx3] + sy12 * confy[ fy1 * SpinSize + fx3] + sz12 * confz[ fy1 * SpinSize + fx3] +
                                sx13 * confx[ fy1 * SpinSize + fx4] + sy13 * confy[ fy1 * SpinSize + fx4] + sz13 * confz[ fy1 * SpinSize + fx4] +
                                sx20 * confx[ fy2 * SpinSize + fx1] + sy20 * confy[ fy2 * SpinSize + fx1] + sz20 * confz[ fy2 * SpinSize + fx1] +
                                sx21 * confx[ fy2 * SpinSize + fx2] + sy21 * confy[ fy2 * SpinSize + fx2] + sz21 * confz[ fy2 * SpinSize + fx2] +
                                sx22 * confx[ fy2 * SpinSize + fx3] + sy22 * confy[ fy2 * SpinSize + fx3] + sz22 * confz[ fy2 * SpinSize + fx3] +
                                sx23 * confx[ fy2 * SpinSize + fx4] + sy23 * confy[ fy2 * SpinSize + fx4] + sz23 * confz[ fy2 * SpinSize + fx4] +
                                sx30 * confx[ fy3 * SpinSize + fx1] + sy30 * confy[ fy3 * SpinSize + fx1] + sz30 * confz[ fy3 * SpinSize + fx1] +
                                sx31 * confx[ fy3 * SpinSize + fx2] + sy31 * confy[ fy3 * SpinSize + fx2] + sz31 * confz[ fy3 * SpinSize + fx2] +
                                sx32 * confx[ fy3 * SpinSize + fx3] + sy32 * confy[ fy3 * SpinSize + fx3] + sz32 * confz[ fy3 * SpinSize + fx3] +
                                sx33 * confx[ fy3 * SpinSize + fx4] + sy33 * confy[ fy3 * SpinSize + fx4] + sz33 * confz[ fy3 * SpinSize + fx4] ;
  corr[(ty+1) * SpinSize + tx] += sx00 * confx[ fy1 * SpinSize + fx0] + sy00 * confy[ fy1 * SpinSize + fx0] + sz00 * confz[ fy1 * SpinSize + fx0] +
                                  sx01 * confx[ fy1 * SpinSize + fx1] + sy01 * confy[ fy1 * SpinSize + fx1] + sz01 * confz[ fy1 * SpinSize + fx1] +
                                  sx02 * confx[ fy1 * SpinSize + fx2] + sy02 * confy[ fy1 * SpinSize + fx2] + sz02 * confz[ fy1 * SpinSize + fx2] +
                                  sx03 * confx[ fy1 * SpinSize + fx3] + sy03 * confy[ fy1 * SpinSize + fx3] + sz03 * confz[ fy1 * SpinSize + fx3] +
                                  sx10 * confx[ fy2 * SpinSize + fx0] + sy10 * confy[ fy2 * SpinSize + fx0] + sz10 * confz[ fy2 * SpinSize + fx0] +
                                  sx11 * confx[ fy2 * SpinSize + fx1] + sy11 * confy[ fy2 * SpinSize + fx1] + sz11 * confz[ fy2 * SpinSize + fx1] +
                                  sx12 * confx[ fy2 * SpinSize + fx2] + sy12 * confy[ fy2 * SpinSize + fx2] + sz12 * confz[ fy2 * SpinSize + fx2] +
                                  sx13 * confx[ fy2 * SpinSize + fx3] + sy13 * confy[ fy2 * SpinSize + fx3] + sz13 * confz[ fy2 * SpinSize + fx3] +
                                  sx20 * confx[ fy3 * SpinSize + fx0] + sy20 * confy[ fy3 * SpinSize + fx0] + sz20 * confz[ fy3 * SpinSize + fx0] +
                                  sx21 * confx[ fy3 * SpinSize + fx1] + sy21 * confy[ fy3 * SpinSize + fx1] + sz21 * confz[ fy3 * SpinSize + fx1] +
                                  sx22 * confx[ fy3 * SpinSize + fx2] + sy22 * confy[ fy3 * SpinSize + fx2] + sz22 * confz[ fy3 * SpinSize + fx2] +
                                  sx23 * confx[ fy3 * SpinSize + fx3] + sy23 * confy[ fy3 * SpinSize + fx3] + sz23 * confz[ fy3 * SpinSize + fx3] +
                                  sx30 * confx[ fy4 * SpinSize + fx0] + sy30 * confy[ fy4 * SpinSize + fx0] + sz30 * confz[ fy4 * SpinSize + fx0] +
                                  sx31 * confx[ fy4 * SpinSize + fx1] + sy31 * confy[ fy4 * SpinSize + fx1] + sz31 * confz[ fy4 * SpinSize + fx1] +
                                  sx32 * confx[ fy4 * SpinSize + fx2] + sy32 * confy[ fy4 * SpinSize + fx2] + sz32 * confz[ fy4 * SpinSize + fx2] +
                                  sx33 * confx[ fy4 * SpinSize + fx3] + sy33 * confy[ fy4 * SpinSize + fx3] + sz33 * confz[ fy4 * SpinSize + fx3] ;
  corr[(ty+1) * SpinSize + tx+1] += sx00 * confx[ fy1 * SpinSize + fx1] + sy00 * confy[ fy1 * SpinSize + fx1] + sz00 * confz[ fy1 * SpinSize + fx1] +
                                    sx01 * confx[ fy1 * SpinSize + fx2] + sy01 * confy[ fy1 * SpinSize + fx2] + sz01 * confz[ fy1 * SpinSize + fx2] +
                                    sx02 * confx[ fy1 * SpinSize + fx3] + sy02 * confy[ fy1 * SpinSize + fx3] + sz02 * confz[ fy1 * SpinSize + fx3] +
                                    sx03 * confx[ fy1 * SpinSize + fx4] + sy03 * confy[ fy1 * SpinSize + fx4] + sz03 * confz[ fy1 * SpinSize + fx4] +
                                    sx10 * confx[ fy2 * SpinSize + fx1] + sy10 * confy[ fy2 * SpinSize + fx1] + sz10 * confz[ fy2 * SpinSize + fx1] +
                                    sx11 * confx[ fy2 * SpinSize + fx2] + sy11 * confy[ fy2 * SpinSize + fx2] + sz11 * confz[ fy2 * SpinSize + fx2] +
                                    sx12 * confx[ fy2 * SpinSize + fx3] + sy12 * confy[ fy2 * SpinSize + fx3] + sz12 * confz[ fy2 * SpinSize + fx3] +
                                    sx13 * confx[ fy2 * SpinSize + fx4] + sy13 * confy[ fy2 * SpinSize + fx4] + sz13 * confz[ fy2 * SpinSize + fx4] +
                                    sx20 * confx[ fy3 * SpinSize + fx1] + sy20 * confy[ fy3 * SpinSize + fx1] + sz20 * confz[ fy3 * SpinSize + fx1] +
                                    sx21 * confx[ fy3 * SpinSize + fx2] + sy21 * confy[ fy3 * SpinSize + fx2] + sz21 * confz[ fy3 * SpinSize + fx2] +
                                    sx22 * confx[ fy3 * SpinSize + fx3] + sy22 * confy[ fy3 * SpinSize + fx3] + sz22 * confz[ fy3 * SpinSize + fx3] +
                                    sx23 * confx[ fy3 * SpinSize + fx4] + sy23 * confy[ fy3 * SpinSize + fx4] + sz23 * confz[ fy3 * SpinSize + fx4] +
                                    sx30 * confx[ fy4 * SpinSize + fx1] + sy30 * confy[ fy4 * SpinSize + fx1] + sz30 * confz[ fy4 * SpinSize + fx1] +
                                    sx31 * confx[ fy4 * SpinSize + fx2] + sy31 * confy[ fy4 * SpinSize + fx2] + sz31 * confz[ fy4 * SpinSize + fx2] +
                                    sx32 * confx[ fy4 * SpinSize + fx3] + sy32 * confy[ fy4 * SpinSize + fx3] + sz32 * confz[ fy4 * SpinSize + fx3] +
                                    sx33 * confx[ fy4 * SpinSize + fx4] + sy33 * confy[ fy4 * SpinSize + fx4] + sz33 * confz[ fy4 * SpinSize + fx4] ;
	__syncthreads();
}

__global__ void sumcorr(double *DSum_corr, const float *corr, int *DTo){
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int ty_pt =(DTo[blockIdx.x / BN]) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	//calculate all the final position first
	DSum_corr[ty_pt * SpinSize + tx] += corr[ty * SpinSize + tx]/SpinSize/SpinSize;
	DSum_corr[ty_pt * SpinSize + tx+1] += corr[ty * SpinSize + tx+1]/SpinSize/SpinSize;
	DSum_corr[(ty_pt + 1) * SpinSize + tx] += corr[(ty + 1) * SpinSize + tx]/SpinSize/SpinSize;
	DSum_corr[(ty_pt + 1) * SpinSize + tx+1] += corr[(ty + 1) * SpinSize + tx+1]/SpinSize/SpinSize;
	__syncthreads();
}
__global__ void avgcorr(double *DSum_corr, double N_corr){
	/*****************************************************************
	Set ( original_i, original_j) as our original point.
	for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
	use the periodic condition to keep the index positive.
	We need to sum over different (original_i, original_j) to get the correlation.
	*****************************************************************/
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	//calculate all the final position first
	DSum_corr[ty * SpinSize + tx] = DSum_corr[ty * SpinSize + tx]/N_corr;
	DSum_corr[ty * SpinSize + tx+1] = DSum_corr[ty * SpinSize + tx+1]/N_corr;
	DSum_corr[(ty + 1) * SpinSize + tx] = DSum_corr[(ty + 1) * SpinSize + tx]/N_corr;
	DSum_corr[(ty + 1) * SpinSize + tx+1] = DSum_corr[(ty + 1) * SpinSize + tx+1]/N_corr;
	__syncthreads();
}
