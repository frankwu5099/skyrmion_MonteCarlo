
#include <hip/hip_runtime.h>
#ifdef SQ
#include "measurements.cuh"
__global__ void getcorr2D(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
	/*****************************************************************
          !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
	Set ( original_i, original_j) as our original point.
	for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
	use the periodic condition to keep the index positive.
	We need to sum over different (original_i, original_j) to get the correlation.
	*****************************************************************/
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int ox = original_i;
	const int oy =(blockIdx.x / BN) * SpinSize + original_j;
	//const int txp = tx +1 ;
	//const int typ = ty +1 ;
	//const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	float sx00, sy00, sz00, sx01, sy01, sz01, sx02, sy02, sz02, sx03, sy03, sz03,
        sx10, sy10, sz10, sx11, sy11, sz11, sx12, sy12, sz12, sx13, sy13, sz13,
        sx20, sy20, sz20, sx21, sy21, sz21, sx22, sy22, sz22, sx23, sy23, sz23,
        sx30, sy30, sz30, sx31, sy31, sz31, sx32, sy32, sz32, sx33, sy33, sz33;
	int fx0, fy0,
	    fx1, fy1,
	    fx2, fy2,
	    fx3, fy3,
	    fx4, fy4; //from o to f
	//calculate all the final position first

	fx0 = (tx + original_i) % SpinSize;
	fx1 = (tx + original_i + 1) % SpinSize;
	fx2 = (tx + original_i + 2) % SpinSize;
	fx3 = (tx + original_i + 3) % SpinSize;
	fx4 = (tx + original_i + 4) % SpinSize;

	if((ty % SpinSize + original_j) >= SpinSize)	fy0 = ty + original_j - SpinSize;
	else  fy0 = ty + original_j;
	if((ty % SpinSize + original_j + 1) >= SpinSize)	fy1 = ty + original_j + 1 - SpinSize;
	else  fy1 = ty + original_j + 1;
	if((ty % SpinSize + original_j + 2) >= SpinSize)	fy2 = ty + original_j + 2 - SpinSize;
	else  fy2 = ty + original_j + 2;
	if((ty % SpinSize + original_j + 3) >= SpinSize)	fy3 = ty + original_j + 3 - SpinSize;
	else  fy3 = ty + original_j + 3;
	if((ty % SpinSize + original_j + 4) >= SpinSize)	fy4 = ty + original_j + 4 - SpinSize;
	else  fy4 = ty + original_j + 4;

	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[coo2D(oy,ox)];
  sy00 = confy[coo2D(oy,ox)];
  sz00 = confz[coo2D(oy,ox)];
  sx01 = confx[coo2D(oy,ox+1)];
  sy01 = confy[coo2D(oy,ox+1)];
  sz01 = confz[coo2D(oy,ox+1)];
  sx02 = confx[coo2D(oy,ox+2)];
  sy02 = confy[coo2D(oy,ox+2)];
  sz02 = confz[coo2D(oy,ox+2)];
  sx03 = confx[coo2D(oy,ox+3)];
  sy03 = confy[coo2D(oy,ox+3)];
  sz03 = confz[coo2D(oy,ox+3)];
  sx10 = confx[coo2D(oy+1,ox)];
  sy10 = confy[coo2D(oy+1,ox)];
  sz10 = confz[coo2D(oy+1,ox)];
  sx11 = confx[coo2D(oy+1,ox+1)];
  sy11 = confy[coo2D(oy+1,ox+1)];
  sz11 = confz[coo2D(oy+1,ox+1)];
  sx12 = confx[coo2D(oy+1,ox+2)];
  sy12 = confy[coo2D(oy+1,ox+2)];
  sz12 = confz[coo2D(oy+1,ox+2)];
  sx13 = confx[coo2D(oy+1,ox+3)];
  sy13 = confy[coo2D(oy+1,ox+3)];
  sz13 = confz[coo2D(oy+1,ox+3)];
  sx20 = confx[coo2D(oy+2,ox)];
  sy20 = confy[coo2D(oy+2,ox)];
  sz20 = confz[coo2D(oy+2,ox)];
  sx21 = confx[coo2D(oy+2,ox+1)];
  sy21 = confy[coo2D(oy+2,ox+1)];
  sz21 = confz[coo2D(oy+2,ox+1)];
  sx22 = confx[coo2D(oy+2,ox+2)];
  sy22 = confy[coo2D(oy+2,ox+2)];
  sz22 = confz[coo2D(oy+2,ox+2)];
  sx23 = confx[coo2D(oy+2,ox+3)];
  sy23 = confy[coo2D(oy+2,ox+3)];
  sz23 = confz[coo2D(oy+2,ox+3)];
  sx30 = confx[coo2D(oy+3,ox)];
  sy30 = confy[coo2D(oy+3,ox)];
  sz30 = confz[coo2D(oy+3,ox)];
  sx31 = confx[coo2D(oy+3,ox+1)];
  sy31 = confy[coo2D(oy+3,ox+1)];
  sz31 = confz[coo2D(oy+3,ox+1)];
  sx32 = confx[coo2D(oy+3,ox+2)];
  sy32 = confy[coo2D(oy+3,ox+2)];
  sz32 = confz[coo2D(oy+3,ox+2)];
  sx33 = confx[coo2D(oy+3,ox+3)];
  sy33 = confy[coo2D(oy+3,ox+3)];
  sz33 = confz[coo2D(oy+3,ox+3)];
  corr[coo2D(ty,tx)] += sx00 * confx[coo2D( fy0,fx0)] + sy00 * confy[coo2D( fy0,fx0)] + sz00 * confz[coo2D( fy0,fx0)] +
                              sx01 * confx[coo2D( fy0,fx1)] + sy01 * confy[coo2D( fy0,fx1)] + sz01 * confz[coo2D( fy0,fx1)] +
                              sx02 * confx[coo2D( fy0,fx2)] + sy02 * confy[coo2D( fy0,fx2)] + sz02 * confz[coo2D( fy0,fx2)] +
                              sx03 * confx[coo2D( fy0,fx3)] + sy03 * confy[coo2D( fy0,fx3)] + sz03 * confz[coo2D( fy0,fx3)] +
                              sx10 * confx[coo2D( fy1,fx0)] + sy10 * confy[coo2D( fy1,fx0)] + sz10 * confz[coo2D( fy1,fx0)] +
                              sx11 * confx[coo2D( fy1,fx1)] + sy11 * confy[coo2D( fy1,fx1)] + sz11 * confz[coo2D( fy1,fx1)] +
                              sx12 * confx[coo2D( fy1,fx2)] + sy12 * confy[coo2D( fy1,fx2)] + sz12 * confz[coo2D( fy1,fx2)] +
                              sx13 * confx[coo2D( fy1,fx3)] + sy13 * confy[coo2D( fy1,fx3)] + sz13 * confz[coo2D( fy1,fx3)] +
                              sx20 * confx[coo2D( fy2,fx0)] + sy20 * confy[coo2D( fy2,fx0)] + sz20 * confz[coo2D( fy2,fx0)] +
                              sx21 * confx[coo2D( fy2,fx1)] + sy21 * confy[coo2D( fy2,fx1)] + sz21 * confz[coo2D( fy2,fx1)] +
                              sx22 * confx[coo2D( fy2,fx2)] + sy22 * confy[coo2D( fy2,fx2)] + sz22 * confz[coo2D( fy2,fx2)] +
                              sx23 * confx[coo2D( fy2,fx3)] + sy23 * confy[coo2D( fy2,fx3)] + sz23 * confz[coo2D( fy2,fx3)] +
                              sx30 * confx[coo2D( fy3,fx0)] + sy30 * confy[coo2D( fy3,fx0)] + sz30 * confz[coo2D( fy3,fx0)] +
                              sx31 * confx[coo2D( fy3,fx1)] + sy31 * confy[coo2D( fy3,fx1)] + sz31 * confz[coo2D( fy3,fx1)] +
                              sx32 * confx[coo2D( fy3,fx2)] + sy32 * confy[coo2D( fy3,fx2)] + sz32 * confz[coo2D( fy3,fx2)] +
                              sx33 * confx[coo2D( fy3,fx3)] + sy33 * confy[coo2D( fy3,fx3)] + sz33 * confz[coo2D( fy3,fx3)] ;
  corr[coo2D(ty,tx+1)] += sx00 * confx[coo2D( fy0,fx1)] + sy00 * confy[coo2D( fy0,fx1)] + sz00 * confz[coo2D( fy0,fx1)] +
                                sx01 * confx[coo2D( fy0,fx2)] + sy01 * confy[coo2D( fy0,fx2)] + sz01 * confz[coo2D( fy0,fx2)] +
                                sx02 * confx[coo2D( fy0,fx3)] + sy02 * confy[coo2D( fy0,fx3)] + sz02 * confz[coo2D( fy0,fx3)] +
                                sx03 * confx[coo2D( fy0,fx4)] + sy03 * confy[coo2D( fy0,fx4)] + sz03 * confz[coo2D( fy0,fx4)] +
                                sx10 * confx[coo2D( fy1,fx1)] + sy10 * confy[coo2D( fy1,fx1)] + sz10 * confz[coo2D( fy1,fx1)] +
                                sx11 * confx[coo2D( fy1,fx2)] + sy11 * confy[coo2D( fy1,fx2)] + sz11 * confz[coo2D( fy1,fx2)] +
                                sx12 * confx[coo2D( fy1,fx3)] + sy12 * confy[coo2D( fy1,fx3)] + sz12 * confz[coo2D( fy1,fx3)] +
                                sx13 * confx[coo2D( fy1,fx4)] + sy13 * confy[coo2D( fy1,fx4)] + sz13 * confz[coo2D( fy1,fx4)] +
                                sx20 * confx[coo2D( fy2,fx1)] + sy20 * confy[coo2D( fy2,fx1)] + sz20 * confz[coo2D( fy2,fx1)] +
                                sx21 * confx[coo2D( fy2,fx2)] + sy21 * confy[coo2D( fy2,fx2)] + sz21 * confz[coo2D( fy2,fx2)] +
                                sx22 * confx[coo2D( fy2,fx3)] + sy22 * confy[coo2D( fy2,fx3)] + sz22 * confz[coo2D( fy2,fx3)] +
                                sx23 * confx[coo2D( fy2,fx4)] + sy23 * confy[coo2D( fy2,fx4)] + sz23 * confz[coo2D( fy2,fx4)] +
                                sx30 * confx[coo2D( fy3,fx1)] + sy30 * confy[coo2D( fy3,fx1)] + sz30 * confz[coo2D( fy3,fx1)] +
                                sx31 * confx[coo2D( fy3,fx2)] + sy31 * confy[coo2D( fy3,fx2)] + sz31 * confz[coo2D( fy3,fx2)] +
                                sx32 * confx[coo2D( fy3,fx3)] + sy32 * confy[coo2D( fy3,fx3)] + sz32 * confz[coo2D( fy3,fx3)] +
                                sx33 * confx[coo2D( fy3,fx4)] + sy33 * confy[coo2D( fy3,fx4)] + sz33 * confz[coo2D( fy3,fx4)] ;
  corr[coo2D((ty+1),tx)] += sx00 * confx[coo2D( fy1,fx0)] + sy00 * confy[coo2D( fy1,fx0)] + sz00 * confz[coo2D( fy1,fx0)] +
                                  sx01 * confx[coo2D( fy1,fx1)] + sy01 * confy[coo2D( fy1,fx1)] + sz01 * confz[coo2D( fy1,fx1)] +
                                  sx02 * confx[coo2D( fy1,fx2)] + sy02 * confy[coo2D( fy1,fx2)] + sz02 * confz[coo2D( fy1,fx2)] +
                                  sx03 * confx[coo2D( fy1,fx3)] + sy03 * confy[coo2D( fy1,fx3)] + sz03 * confz[coo2D( fy1,fx3)] +
                                  sx10 * confx[coo2D( fy2,fx0)] + sy10 * confy[coo2D( fy2,fx0)] + sz10 * confz[coo2D( fy2,fx0)] +
                                  sx11 * confx[coo2D( fy2,fx1)] + sy11 * confy[coo2D( fy2,fx1)] + sz11 * confz[coo2D( fy2,fx1)] +
                                  sx12 * confx[coo2D( fy2,fx2)] + sy12 * confy[coo2D( fy2,fx2)] + sz12 * confz[coo2D( fy2,fx2)] +
                                  sx13 * confx[coo2D( fy2,fx3)] + sy13 * confy[coo2D( fy2,fx3)] + sz13 * confz[coo2D( fy2,fx3)] +
                                  sx20 * confx[coo2D( fy3,fx0)] + sy20 * confy[coo2D( fy3,fx0)] + sz20 * confz[coo2D( fy3,fx0)] +
                                  sx21 * confx[coo2D( fy3,fx1)] + sy21 * confy[coo2D( fy3,fx1)] + sz21 * confz[coo2D( fy3,fx1)] +
                                  sx22 * confx[coo2D( fy3,fx2)] + sy22 * confy[coo2D( fy3,fx2)] + sz22 * confz[coo2D( fy3,fx2)] +
                                  sx23 * confx[coo2D( fy3,fx3)] + sy23 * confy[coo2D( fy3,fx3)] + sz23 * confz[coo2D( fy3,fx3)] +
                                  sx30 * confx[coo2D( fy4,fx0)] + sy30 * confy[coo2D( fy4,fx0)] + sz30 * confz[coo2D( fy4,fx0)] +
                                  sx31 * confx[coo2D( fy4,fx1)] + sy31 * confy[coo2D( fy4,fx1)] + sz31 * confz[coo2D( fy4,fx1)] +
                                  sx32 * confx[coo2D( fy4,fx2)] + sy32 * confy[coo2D( fy4,fx2)] + sz32 * confz[coo2D( fy4,fx2)] +
                                  sx33 * confx[coo2D( fy4,fx3)] + sy33 * confy[coo2D( fy4,fx3)] + sz33 * confz[coo2D( fy4,fx3)] ;
  corr[coo2D((ty+1),tx+1)] += sx00 * confx[coo2D( fy1,fx1)] + sy00 * confy[coo2D( fy1,fx1)] + sz00 * confz[coo2D( fy1,fx1)] +
                                    sx01 * confx[coo2D( fy1,fx2)] + sy01 * confy[coo2D( fy1,fx2)] + sz01 * confz[coo2D( fy1,fx2)] +
                                    sx02 * confx[coo2D( fy1,fx3)] + sy02 * confy[coo2D( fy1,fx3)] + sz02 * confz[coo2D( fy1,fx3)] +
                                    sx03 * confx[coo2D( fy1,fx4)] + sy03 * confy[coo2D( fy1,fx4)] + sz03 * confz[coo2D( fy1,fx4)] +
                                    sx10 * confx[coo2D( fy2,fx1)] + sy10 * confy[coo2D( fy2,fx1)] + sz10 * confz[coo2D( fy2,fx1)] +
                                    sx11 * confx[coo2D( fy2,fx2)] + sy11 * confy[coo2D( fy2,fx2)] + sz11 * confz[coo2D( fy2,fx2)] +
                                    sx12 * confx[coo2D( fy2,fx3)] + sy12 * confy[coo2D( fy2,fx3)] + sz12 * confz[coo2D( fy2,fx3)] +
                                    sx13 * confx[coo2D( fy2,fx4)] + sy13 * confy[coo2D( fy2,fx4)] + sz13 * confz[coo2D( fy2,fx4)] +
                                    sx20 * confx[coo2D( fy3,fx1)] + sy20 * confy[coo2D( fy3,fx1)] + sz20 * confz[coo2D( fy3,fx1)] +
                                    sx21 * confx[coo2D( fy3,fx2)] + sy21 * confy[coo2D( fy3,fx2)] + sz21 * confz[coo2D( fy3,fx2)] +
                                    sx22 * confx[coo2D( fy3,fx3)] + sy22 * confy[coo2D( fy3,fx3)] + sz22 * confz[coo2D( fy3,fx3)] +
                                    sx23 * confx[coo2D( fy3,fx4)] + sy23 * confy[coo2D( fy3,fx4)] + sz23 * confz[coo2D( fy3,fx4)] +
                                    sx30 * confx[coo2D( fy4,fx1)] + sy30 * confy[coo2D( fy4,fx1)] + sz30 * confz[coo2D( fy4,fx1)] +
                                    sx31 * confx[coo2D( fy4,fx2)] + sy31 * confy[coo2D( fy4,fx2)] + sz31 * confz[coo2D( fy4,fx2)] +
                                    sx32 * confx[coo2D( fy4,fx3)] + sy32 * confy[coo2D( fy4,fx3)] + sz32 * confz[coo2D( fy4,fx3)] +
                                    sx33 * confx[coo2D( fy4,fx4)] + sy33 * confy[coo2D( fy4,fx4)] + sz33 * confz[coo2D( fy4,fx4)] ;
	__syncthreads();
}

__global__ void sumcorr2D(double *DSum_corr, const float *corr, int *DTo){
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int ty_pt =(DTo[blockIdx.x / BN]) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	//calculate all the final position first
	DSum_corr[coo2D(ty_pt,tx)] += corr[coo2D(ty,tx)]/SpinSize/SpinSize;
	DSum_corr[coo2D(ty_pt,tx+1)] += corr[coo2D(ty,tx+1)]/SpinSize/SpinSize;
	DSum_corr[coo2D((ty_pt + 1),tx)] += corr[coo2D((ty + 1),tx)]/SpinSize/SpinSize;
	DSum_corr[coo2D((ty_pt + 1),tx+1)] += corr[coo2D((ty + 1),tx+1)]/SpinSize/SpinSize;
	__syncthreads();
}
__global__ void avgcorr2D(double *DSum_corr, double N_corr){
	/*****************************************************************
	Set ( original_i, original_j) as our original point.
	for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
	use the periodic condition to keep the index positive.
	We need to sum over different (original_i, original_j) to get the correlation.
	*****************************************************************/
	//Energy variables
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	//calculate all the final position first
	DSum_corr[coo2D(ty,tx)] = DSum_corr[coo2D(ty,tx)]/N_corr;
	DSum_corr[coo2D(ty,tx+1)] = DSum_corr[coo2D(ty,tx+1)]/N_corr;
	DSum_corr[coo2D((ty + 1),tx)] = DSum_corr[coo2D((ty + 1),tx)]/N_corr;
	DSum_corr[coo2D((ty + 1),tx+1)] = DSum_corr[coo2D((ty + 1),tx+1)]/N_corr;
	__syncthreads();
}
#endif
