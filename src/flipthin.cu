#include "hip/hip_runtime.h"
#ifndef UPDATE_H
#define UPDATE_H
#define "update.cuh"
#endif
__global__ void flipTLBRthin(float *confx, float *confy, float *confz, unsigned int *rngState, float* Hs, float invT){
	//Energy variables
	__shared__ unsigned rngShmem[BlockSize_x * BlockSize_y * 4];
	unsigned rngRegs[WarpStandard_REG_COUNT];
	WarpStandard_LoadState(rngState, rngRegs, rngShmem);
	float H = Hs[blockIdx.x / BN];
	unsigned int r;
	float du;	//-dE
	float sx, sy, sz;
	float th, phi;
	float hx, hy, hz;
	//float norm;
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize * SpinSize_z + 2 * (((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	int i, j, ib, jb, k;
	//----------Spin flip at the top and left corner of each thread sqare----------
	i = tx;
  j = ty;
  ib = (i + SpinSize - 1) % SpinSize;
	if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
	else			jb = j - 1;
	//Spin flip!
	//first layer
	k = 0;
  hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, jb , i)]\
     + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, jb , i)]\
     + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, jb , i)]\
     + BZPxy * confy[coo(k+1, j, i)] + BZPxx * confx[coo(k+1, j, i)];
  hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, jb , i)]\
     + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, jb , i)]\
     + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, jb , i)]\
     + BZPyx * confx[coo(k+1, j, i)] + BZPyy * confy[coo(k+1, j, i)];
  hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, jb , i)]\
     + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, jb , i)]\
     + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, jb , i)] + H\
     + BZPzz * confz[coo(k+1, j, i)];
  du =- confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
	du += sx * hx + sy * hy + sz * hz - A * sz * sz;
	r = WarpStandard_Generate(rngRegs, rngShmem);
	if(du >= 0){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
	}
	else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
	}

	__syncthreads();

	for (k = 1;k < SpinSize_z - 1; k++){//middle layers
    hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, jb , i)]\
       + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, jb , i)]\
       + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, jb , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZMxy * confy[coo(k-1, j, i)] + BZPxx * confx[coo(k+1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, jb , i)]\
       + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, jb , i)]\
       + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, jb , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZMyx * confx[coo(k-1, j, i)] + BZPyy * confy[coo(k+1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, jb , i)]\
       + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, jb , i)]\
       + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, jb , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)] + BZMzz * confz[coo(k-1, j, i)];
    du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
    r = WarpStandard_Generate(rngRegs, rngShmem);
    sz = r * NORM - 1;
    th = asin( sz );
    r = WarpStandard_Generate(rngRegs, rngShmem);
    phi = r*TOPI;
    sz = sin( th );
    sx = cos( th ) * cos( phi );
    sy = cos( th ) * sin( phi );
    du += sx * hx + sy * hy + sz * hz - A * sz * sz;
    r = WarpStandard_Generate(rngRegs, rngShmem);
    if(du >= 0){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }
    else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }

    __syncthreads();
  }//end middle layers

  //last layer
  k = SpinSize_z - 1;
  hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, jb , i)]\
     + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, jb , i)]\
     + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, jb , i)]\
     + BZMxy * confy[coo(k-1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
  hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, jb , i)]\
     + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, jb , i)]\
     + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, jb , i)]\
     + BZMyx * confx[coo(k-1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
  hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, jb , i)]\
     + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, jb , i)]\
     + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, jb , i)] + H\
     + BZMzz * confz[coo(k-1, j, i)];
  du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
	du += sx * hx + sy * hy + sz * hz - A * sz * sz;
	r = WarpStandard_Generate(rngRegs, rngShmem);
	if(du >= 0){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
	}
	else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
	}

	__syncthreads();

	//----------Spin flip at the bottom and right corner of each thread sqare----------
	i = tx + 1;
	j = ty + 1;
	ib = (i + 1) % SpinSize;
	if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
	else					jb = j + 1;
	//Spin flip!
	//first layer
	k = 0;
  hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, i-1)] + BYMxx * confx[coo(k, j-1 , i)]\
     + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, i-1)] + BYMxy * confy[coo(k, j-1 , i)]\
     + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, i-1)] + BYMxz * confz[coo(k, j-1 , i)]\
     + BZPxy * confy[coo(k+1, j, i)] + BZPxx * confx[coo(k+1, j, i)];
  hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, i-1)] + BYMyx * confx[coo(k, j-1 , i)]\
     + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, i-1)] + BYMyy * confy[coo(k, j-1 , i)]\
     + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, i-1)] + BYMyz * confz[coo(k, j-1 , i)]\
     + BZPyx * confx[coo(k+1, j, i)] + BZPyy * confy[coo(k+1, j, i)];
  hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, i-1)] + BYMzx * confx[coo(k, j-1 , i)]\
     + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, i-1)] + BYMzy * confy[coo(k, j-1 , i)]\
     + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, i-1)] + BYMzz * confz[coo(k, j-1 , i)] + H\
     + BZPzz * confz[coo(k+1, j, i)];
  du =- confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
  }

  __syncthreads();

	for (k = 1;k < SpinSize_z - 1; k++){//middle layers
    hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, i-1)] + BYMxx * confx[coo(k, j-1 , i)]\
       + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, i-1)] + BYMxy * confy[coo(k, j-1 , i)]\
       + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, i-1)] + BYMxz * confz[coo(k, j-1 , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZMxy * confy[coo(k-1, j, i)] + BZPxx * confx[coo(k+1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, i-1)] + BYMyx * confx[coo(k, j-1 , i)]\
       + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, i-1)] + BYMyy * confy[coo(k, j-1 , i)]\
       + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, i-1)] + BYMyz * confz[coo(k, j-1 , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZMyx * confx[coo(k-1, j, i)] + BZPyy * confy[coo(k+1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, i-1)] + BYMzx * confx[coo(k, j-1 , i)]\
       + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, i-1)] + BYMzy * confy[coo(k, j-1 , i)]\
       + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, i-1)] + BYMzz * confz[coo(k, j-1 , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)] + BZMzz * confz[coo(k-1, j, i)];
    du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
    r = WarpStandard_Generate(rngRegs, rngShmem);
    sz = r * NORM - 1;
    th = asin( sz );
    r = WarpStandard_Generate(rngRegs, rngShmem);
    phi = r*TOPI;
    sz = sin( th );
    sx = cos( th ) * cos( phi );
    sy = cos( th ) * sin( phi );
    du += sx * hx + sy * hy + sz * hz - A * sz * sz;
    r = WarpStandard_Generate(rngRegs, rngShmem);
    if(du >= 0){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }
    else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }

    __syncthreads();
  }//end middle layers

	//last layer
	k = SpinSize_z - 1;
  hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, i-1)] + BYMxx * confx[coo(k, j-1 , i)]\
     + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, i-1)] + BYMxy * confy[coo(k, j-1 , i)]\
     + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, i-1)] + BYMxz * confz[coo(k, j-1 , i)]\
     + BZMxy * confy[coo(k-1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
  hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, i-1)] + BYMyx * confx[coo(k, j-1 , i)]\
     + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, i-1)] + BYMyy * confy[coo(k, j-1 , i)]\
     + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, i-1)] + BYMyz * confz[coo(k, j-1 , i)]\
     + BZMyx * confx[coo(k-1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
  hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, i-1)] + BYMzx * confx[coo(k, j-1 , i)]\
     + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, i-1)] + BYMzy * confy[coo(k, j-1 , i)]\
     + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, i-1)] + BYMzz * confz[coo(k, j-1 , i)] + H\
     + BZMzz * confz[coo(k-1, j, i)];
  du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
		confx[coo(k, j, i)] = sx;
		confy[coo(k, j, i)] = sy;
		confz[coo(k, j, i)] = sz;
  }

  __syncthreads();


	//Load random number back to global memory
	WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}

__global__ void flipBLTRthin(float *confx, float *confy, float *confz, unsigned int *rngState, float* Hs, float invT){
	//Energy variables
	__shared__ unsigned rngShmem[BlockSize_x * BlockSize_y * 4];
	unsigned rngRegs[WarpStandard_REG_COUNT];
	WarpStandard_LoadState(rngState, rngRegs, rngShmem);
	float H = Hs[blockIdx.x / BN];
	unsigned int r;
	float du;	//-dE
	float sx, sy, sz;
	float th,phi;
	float hx, hy, hz;
	//float norm;
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);// % BlockSize_y;
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty = (blockIdx.x / BN) * SpinSize * SpinSize_z + 2 * (((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	int i, j, ib, jb, k;
	//----------Spin flip at the bottom and left corner of each thread sqare----------
	i = tx;
	j = ty + 1;
	ib = (i + SpinSize - 1) % SpinSize;
	if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
	else					jb = j + 1;
	//Spin flip!
	//first layer
	k = 0;
    hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, j-1 , i)]\
       + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, j-1 , i)]\
       + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, j-1 , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZPxx * confx[coo(k+1, j, i)];
    hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, j-1 , i)]\
       + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, j-1 , i)]\
       + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, j-1 , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZPyy * confy[coo(k+1, j, i)];
    hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, j-1 , i)]\
       + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, j-1 , i)]\
       + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, j-1 , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)];
  du =- confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }

  __syncthreads();

	for (k = 1;k < SpinSize_z - 1; k++){//middle layers
    hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, j-1 , i)]\
       + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, j-1 , i)]\
       + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, j-1 , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZMxy * confy[coo(k-1, j, i)] + BZPxx * confx[coo(k+1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, j-1 , i)]\
       + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, j-1 , i)]\
       + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, j-1 , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZMyx * confx[coo(k-1, j, i)] + BZPyy * confy[coo(k+1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, j-1 , i)]\
       + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, j-1 , i)]\
       + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, j-1 , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)] + BZMzz * confz[coo(k-1, j, i)];
    du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
    r = WarpStandard_Generate(rngRegs, rngShmem);
    sz = r * NORM - 1;
    th = asin( sz );
    r = WarpStandard_Generate(rngRegs, rngShmem);
    phi = r*TOPI;
    sz = sin( th );
    sx = cos( th ) * cos( phi );
    sy = cos( th ) * sin( phi );
    du += sx * hx + sy * hy + sz * hz - A * sz * sz;
    r = WarpStandard_Generate(rngRegs, rngShmem);
    if(du >= 0){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }
    else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }

    __syncthreads();
  }//end middle layers
  //last layer
  k = SpinSize_z - 1;
    hx = BXPxx * confx[coo(k, j, i+1)] + BYPxx * confx[coo(k, jb, i)] + BXMxx * confx[coo(k, j, ib )] + BYMxx * confx[coo(k, j-1 , i)]\
       + BXPxy * confy[coo(k, j, i+1)] + BYPxy * confy[coo(k, jb, i)] + BXMxy * confy[coo(k, j, ib )] + BYMxy * confy[coo(k, j-1 , i)]\
       + BXPxz * confz[coo(k, j, i+1)] + BYPxz * confz[coo(k, jb, i)] + BXMxz * confz[coo(k, j, ib )] + BYMxz * confz[coo(k, j-1 , i)]\
       + BZMxy * confy[coo(k-1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, i+1)] + BYPyx * confx[coo(k, jb, i)] + BXMyx * confx[coo(k, j, ib )] + BYMyx * confx[coo(k, j-1 , i)]\
       + BXPyy * confy[coo(k, j, i+1)] + BYPyy * confy[coo(k, jb, i)] + BXMyy * confy[coo(k, j, ib )] + BYMyy * confy[coo(k, j-1 , i)]\
       + BXPyz * confz[coo(k, j, i+1)] + BYPyz * confz[coo(k, jb, i)] + BXMyz * confz[coo(k, j, ib )] + BYMyz * confz[coo(k, j-1 , i)]\
       + BZMyx * confx[coo(k-1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, i+1)] + BYPzx * confx[coo(k, jb, i)] + BXMzx * confx[coo(k, j, ib )] + BYMzx * confx[coo(k, j-1 , i)]\
       + BXPzy * confy[coo(k, j, i+1)] + BYPzy * confy[coo(k, jb, i)] + BXMzy * confy[coo(k, j, ib )] + BYMzy * confy[coo(k, j-1 , i)]\
       + BXPzz * confz[coo(k, j, i+1)] + BYPzz * confz[coo(k, jb, i)] + BXMzz * confz[coo(k, j, ib )] + BYMzz * confz[coo(k, j-1 , i)] + H\
       + BZMzz * confz[coo(k-1, j, i)];
  du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }

  __syncthreads();

	//----------Spin flip at the top and right corner of each thread sqare----------
	i = tx + 1;
	j = ty;
	ib = (i + 1) % SpinSize;
	if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
	else			jb = j - 1;
	//Spin flip!
	k = 0;
    hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, i-1 )] + BYMxx * confx[coo(k, jb , i)]\
       + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, i-1 )] + BYMxy * confy[coo(k, jb , i)]\
       + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, i-1 )] + BYMxz * confz[coo(k, jb , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZPxx * confx[coo(k+1, j, i)];
    hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, i-1 )] + BYMyx * confx[coo(k, jb , i)]\
       + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, i-1 )] + BYMyy * confy[coo(k, jb , i)]\
       + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, i-1 )] + BYMyz * confz[coo(k, jb , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZPyy * confy[coo(k+1, j, i)];
    hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, i-1 )] + BYMzx * confx[coo(k, jb , i)]\
       + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, i-1 )] + BYMzy * confy[coo(k, jb , i)]\
       + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, i-1 )] + BYMzz * confz[coo(k, jb , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)];
  du =- confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }

  __syncthreads();
	for (k = 1;k < SpinSize_z - 1; k++){//middle layers
    hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, i-1 )] + BYMxx * confx[coo(k, jb , i)]\
       + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, i-1 )] + BYMxy * confy[coo(k, jb , i)]\
       + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, i-1 )] + BYMxz * confz[coo(k, jb , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZMxy * confy[coo(k-1, j, i)] + BZPxx * confx[coo(k+1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, i-1 )] + BYMyx * confx[coo(k, jb , i)]\
       + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, i-1 )] + BYMyy * confy[coo(k, jb , i)]\
       + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, i-1 )] + BYMyz * confz[coo(k, jb , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZMyx * confx[coo(k-1, j, i)] + BZPyy * confy[coo(k+1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, i-1 )] + BYMzx * confx[coo(k, jb , i)]\
       + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, i-1 )] + BYMzy * confy[coo(k, jb , i)]\
       + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, i-1 )] + BYMzz * confz[coo(k, jb , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)] + BZMzz * confz[coo(k-1, j, i)];
    du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
    r = WarpStandard_Generate(rngRegs, rngShmem);
    sz = r * NORM - 1;
    th = asin( sz );
    r = WarpStandard_Generate(rngRegs, rngShmem);
    phi = r*TOPI;
    sz = sin( th );
    sx = cos( th ) * cos( phi );
    sy = cos( th ) * sin( phi );
    du += sx * hx + sy * hy + sz * hz - A * sz * sz;
    r = WarpStandard_Generate(rngRegs, rngShmem);
    if(du >= 0){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }
    else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
      confx[coo(k, j, i)] = sx;
      confy[coo(k, j, i)] = sy;
      confz[coo(k, j, i)] = sz;
    }

    __syncthreads();
  }//end middle layers
  //last layer
  k = SpinSize_z - 1;
    hx = BXPxx * confx[coo(k, j, ib)] + BYPxx * confx[coo(k, j+1, i)] + BXMxx * confx[coo(k, j, i-1 )] + BYMxx * confx[coo(k, jb , i)]\
       + BXPxy * confy[coo(k, j, ib)] + BYPxy * confy[coo(k, j+1, i)] + BXMxy * confy[coo(k, j, i-1 )] + BYMxy * confy[coo(k, jb , i)]\
       + BXPxz * confz[coo(k, j, ib)] + BYPxz * confz[coo(k, j+1, i)] + BXMxz * confz[coo(k, j, i-1 )] + BYMxz * confz[coo(k, jb , i)]\
       + BZPxy * confy[coo(k+1, j, i)] + BZMxy * confy[coo(k-1, j, i)] + BZPxx * confx[coo(k+1, j, i)] + BZMxx * confx[coo(k-1, j, i)];
    hy = BXPyx * confx[coo(k, j, ib)] + BYPyx * confx[coo(k, j+1, i)] + BXMyx * confx[coo(k, j, i-1 )] + BYMyx * confx[coo(k, jb , i)]\
       + BXPyy * confy[coo(k, j, ib)] + BYPyy * confy[coo(k, j+1, i)] + BXMyy * confy[coo(k, j, i-1 )] + BYMyy * confy[coo(k, jb , i)]\
       + BXPyz * confz[coo(k, j, ib)] + BYPyz * confz[coo(k, j+1, i)] + BXMyz * confz[coo(k, j, i-1 )] + BYMyz * confz[coo(k, jb , i)]\
       + BZPyx * confx[coo(k+1, j, i)] + BZMyx * confx[coo(k-1, j, i)] + BZPyy * confy[coo(k+1, j, i)] + BZMyy * confy[coo(k-1, j, i)];
    hz = BXPzx * confx[coo(k, j, ib)] + BYPzx * confx[coo(k, j+1, i)] + BXMzx * confx[coo(k, j, i-1 )] + BYMzx * confx[coo(k, jb , i)]\
       + BXPzy * confy[coo(k, j, ib)] + BYPzy * confy[coo(k, j+1, i)] + BXMzy * confy[coo(k, j, i-1 )] + BYMzy * confy[coo(k, jb , i)]\
       + BXPzz * confz[coo(k, j, ib)] + BYPzz * confz[coo(k, j+1, i)] + BXMzz * confz[coo(k, j, i-1 )] + BYMzz * confz[coo(k, jb , i)] + H\
       + BZPzz * confz[coo(k+1, j, i)] + BZMzz * confz[coo(k-1, j, i)];
    du -= confx[coo(k, j, i)] * hx - confy[coo(k, j, i)] * hy - confz[coo(k, j, i)] * hz + A * confz[coo(k, j, i)] * confz[coo(k, j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo(k, j, i)] = sx;
    confy[coo(k, j, i)] = sy;
    confz[coo(k, j, i)] = sz;
  }

  __syncthreads();

	//Load random number back to global memory
	WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}
