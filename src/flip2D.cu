
#include <hip/hip_runtime.h>
#ifdef SQ
#include "updates.cuh"
__global__ void flipTLBR_2D(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  extern __shared__ unsigned rngShmem[];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the top and left corner of each thread sqare----------
  i = tx;
  j = ty;
  ib = (i + SpinSize - 1) % SpinSize;
  if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, i+1)] + BYPxx * confx[coo2D((j+1), i)] + BXMxx * confx[coo2D(j, ib)] + BYMxx * confx[coo2D(jb, i)]\
     + BXPxy * confy[coo2D(j, i+1)] + BYPxy * confy[coo2D((j+1), i)] + BXMxy * confy[coo2D(j, ib)] + BYMxy * confy[coo2D(jb, i)]\
     + BXPxz * confz[coo2D(j, i+1)] + BYPxz * confz[coo2D((j+1), i)] + BXMxz * confz[coo2D(j, ib)] + BYMxz * confz[coo2D(jb, i)];
  hy = BXPyx * confx[coo2D(j, i+1)] + BYPyx * confx[coo2D((j+1), i)] + BXMyx * confx[coo2D(j, ib)] + BYMyx * confx[coo2D(jb, i)]\
     + BXPyy * confy[coo2D(j, i+1)] + BYPyy * confy[coo2D((j+1), i)] + BXMyy * confy[coo2D(j, ib)] + BYMyy * confy[coo2D(jb, i)]\
     + BXPyz * confz[coo2D(j, i+1)] + BYPyz * confz[coo2D((j+1), i)] + BXMyz * confz[coo2D(j, ib)] + BYMyz * confz[coo2D(jb, i)];
  hz = BXPzx * confx[coo2D(j, i+1)] + BYPzx * confx[coo2D((j+1), i)] + BXMzx * confx[coo2D(j, ib)] + BYMzx * confx[coo2D(jb, i)]\
     + BXPzy * confy[coo2D(j, i+1)] + BYPzy * confy[coo2D((j+1), i)] + BXMzy * confy[coo2D(j, ib)] + BYMzy * confy[coo2D(jb, i)]\
     + BXPzz * confz[coo2D(j, i+1)] + BYPzz * confz[coo2D((j+1), i)] + BXMzz * confz[coo2D(j, ib)] + BYMzz * confz[coo2D(jb, i)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the bottom and right corner of each thread sqare----------
  i = tx + 1;
  j = ty + 1;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D(jb, i)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D((j-1), i)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D(jb, i)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D((j-1), i)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D(jb, i)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D((j-1), i)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D(jb, i)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D((j-1), i)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D(jb, i)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D((j-1), i)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D(jb, i)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D((j-1), i)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D(jb, i)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D((j-1), i)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D(jb, i)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D((j-1), i)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D(jb, i)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D((j-1), i)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}



__global__ void flipBLTR_2D(float *confx, float *confy, float *confz, unsigned int *rngState, float* Pparameters, float Cparameter){
  //Energy variables
  extern __shared__ unsigned rngShmem[];
  unsigned rngRegs[WarpStandard_REG_COUNT];
  WarpStandard_LoadState(rngState, rngRegs, rngShmem);
  float Pparameter = Pparameters[blockIdx.x / BN];
  unsigned int r;
  float du;	//-dE
  float sx, sy, sz;
  float th,phi;
  float hx, hy, hz;
  //float norm;
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);// % BlockSize_y;
  const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty = (blockIdx.x / BN) * SpinSize + 2 * (((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  int i, j, ib, jb;
  //----------Spin flip at the bottom and left corner of each thread sqare----------
  i = tx;
  j = ty + 1;
  ib = (i + SpinSize - 1) % SpinSize;
  if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
  else					jb = j + 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, i+1)] + BYPxx * confx[coo2D(jb, i)] + BXMxx * confx[coo2D(j, ib)] + BYMxx * confx[coo2D((j-1), i)]\
     + BXPxy * confy[coo2D(j, i+1)] + BYPxy * confy[coo2D(jb, i)] + BXMxy * confy[coo2D(j, ib)] + BYMxy * confy[coo2D((j-1), i)]\
     + BXPxz * confz[coo2D(j, i+1)] + BYPxz * confz[coo2D(jb, i)] + BXMxz * confz[coo2D(j, ib)] + BYMxz * confz[coo2D((j-1), i)];
  hy = BXPyx * confx[coo2D(j, i+1)] + BYPyx * confx[coo2D(jb, i)] + BXMyx * confx[coo2D(j, ib)] + BYMyx * confx[coo2D((j-1), i)]\
     + BXPyy * confy[coo2D(j, i+1)] + BYPyy * confy[coo2D(jb, i)] + BXMyy * confy[coo2D(j, ib)] + BYMyy * confy[coo2D((j-1), i)]\
     + BXPyz * confz[coo2D(j, i+1)] + BYPyz * confz[coo2D(jb, i)] + BXMyz * confz[coo2D(j, ib)] + BYMyz * confz[coo2D((j-1), i)];
  hz = BXPzx * confx[coo2D(j, i+1)] + BYPzx * confx[coo2D(jb, i)] + BXMzx * confx[coo2D(j, ib)] + BYMzx * confx[coo2D((j-1), i)]\
     + BXPzy * confy[coo2D(j, i+1)] + BYPzy * confy[coo2D(jb, i)] + BXMzy * confy[coo2D(j, ib)] + BYMzy * confy[coo2D((j-1), i)]\
     + BXPzz * confz[coo2D(j, i+1)] + BYPzz * confz[coo2D(jb, i)] + BXMzz * confz[coo2D(j, ib)] + BYMzz * confz[coo2D((j-1), i)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //----------Spin flip at the top and right corner of each thread sqare----------
  i = tx + 1;
  j = ty;
  ib = (i + 1) % SpinSize;
  if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
  else			jb = j - 1;
  //Spin flip!
  hx = BXPxx * confx[coo2D(j, ib)] + BYPxx * confx[coo2D((j+1), i)] + BXMxx * confx[coo2D(j, i-1)] + BYMxx * confx[coo2D(jb, i)]\
     + BXPxy * confy[coo2D(j, ib)] + BYPxy * confy[coo2D((j+1), i)] + BXMxy * confy[coo2D(j, i-1)] + BYMxy * confy[coo2D(jb, i)]\
     + BXPxz * confz[coo2D(j, ib)] + BYPxz * confz[coo2D((j+1), i)] + BXMxz * confz[coo2D(j, i-1)] + BYMxz * confz[coo2D(jb, i)];
  hy = BXPyx * confx[coo2D(j, ib)] + BYPyx * confx[coo2D((j+1), i)] + BXMyx * confx[coo2D(j, i-1)] + BYMyx * confx[coo2D(jb, i)]\
     + BXPyy * confy[coo2D(j, ib)] + BYPyy * confy[coo2D((j+1), i)] + BXMyy * confy[coo2D(j, i-1)] + BYMyy * confy[coo2D(jb, i)]\
     + BXPyz * confz[coo2D(j, ib)] + BYPyz * confz[coo2D((j+1), i)] + BXMyz * confz[coo2D(j, i-1)] + BYMyz * confz[coo2D(jb, i)];
  hz = BXPzx * confx[coo2D(j, ib)] + BYPzx * confx[coo2D((j+1), i)] + BXMzx * confx[coo2D(j, i-1)] + BYMzx * confx[coo2D(jb, i)]\
     + BXPzy * confy[coo2D(j, ib)] + BYPzy * confy[coo2D((j+1), i)] + BXMzy * confy[coo2D(j, i-1)] + BYMzy * confy[coo2D(jb, i)]\
     + BXPzz * confz[coo2D(j, ib)] + BYPzz * confz[coo2D((j+1), i)] + BXMzz * confz[coo2D(j, i-1)] + BYMzz * confz[coo2D(jb, i)] + H;
  du = - confx[coo2D(j, i)] * hx - confy[coo2D(j, i)] * hy - confz[coo2D(j, i)] * hz + A * confz[coo2D(j, i)] * confz[coo2D(j, i)];
  r = WarpStandard_Generate(rngRegs, rngShmem);
  sz = r * NORM - 1;
  th = asin( sz );
  r = WarpStandard_Generate(rngRegs, rngShmem);
  phi = r*TOPI;
  sz = sin( th );
  sx = cos( th ) * cos( phi );
  sy = cos( th ) * sin( phi );
  du += sx * hx + sy * hy + sz * hz - A * sz * sz;
  r = WarpStandard_Generate(rngRegs, rngShmem);
  if(du >= 0){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }
  else if((unsigned int)(exp(du * invT) * UINT_MAX) > r){
    confx[coo2D(j, i)] = sx;
    confy[coo2D(j, i)] = sy;
    confz[coo2D(j, i)] = sz;
  }

  __syncthreads();

  //Load random number back to global memory
  WarpStandard_SaveState(rngRegs, rngShmem, rngState);
}
#endif
