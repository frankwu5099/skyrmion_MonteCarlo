#include "hip/hip_runtime.h"
#include "measurements.cuh"


measurements::measurements(char * indir, int Parallel_num, unsigned int binSize){
  measurement_num = 19;
  //raw_memmory = operator new[] (measurement_num * sizeof(measurement));
  strcpy(names[0], "E");
  strcpy(names[1], "M");
  strcpy(names[2], "Chern");
  strcpy(names[3], "E2");
  strcpy(names[4], "E4");
  strcpy(names[5], "M2");
  strcpy(names[6], "M4");
  strcpy(names[7], "Mz2");
  strcpy(names[8], "Mz4");
  strcpy(names[9], "Chern2");
  strcpy(names[10], "Chern4");
  strcpy(names[11], "Nematic2");
  strcpy(names[12], "Nematic4");
  strcpy(names[13], "Mz");
  strcpy(names[14], "EMz");
  strcpy(names[15], "Nematic");
  strcpy(names[16], "Mzi2");
  strcpy(names[17], "Hy");
  strcpy(names[18], "Iy2");
  norms[0] = double(binSize) * H_N;
  norms[1] = double(binSize) * H_N;
  norms[2] = double(binSize) * 2;
  norms[3] = double(binSize) * H_N * H_N;
  norms[4] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[5] = double(binSize) * H_N * H_N;
  norms[6] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[7] = double(binSize) * H_N * H_N;
  norms[8] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[9] = double(binSize) * H_N * H_N * 2 * 2;
  norms[10] = double(binSize) * H_N * H_N * H_N * H_N * 2 * 2 * 2 * 2;
  norms[11] = double(binSize) * H_N * H_N;
  norms[12] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[13] = double(binSize) * H_N;
  norms[14] = double(binSize) * H_N * H_N;
  norms[15] = double(binSize) * H_N;
  norms[16] = double(binSize) * H_N;
  norms[17] = double(binSize) * H_N;
  norms[18] = double(binSize) * H_N * H_N;
  O.reserve(measurement_num);
  for (int i =0 ; i< measurement_num; i++){
    O.push_back(measurement(indir, names[i], norms[i], Parallel_num));
    O[O.size() - 1].fp = fopen(O[O.size() - 1].fn, "w");
  }
  data_num = Parallel_num;
  data_num_s = Parallel_num/StreamN;
  Out_mem_size = Parallel_num * MEASURE_NUM * H_BN * sizeof(double);
  Out_mem_size_s = data_num_s * MEASURE_NUM * H_BN * sizeof(double);
  printf("%u\n", Out_mem_size);
  Hout = (double*)calloc(Parallel_num * MEASURE_NUM * H_BN, sizeof(double));
  Dout = (double**)calloc(StreamN, sizeof(double*));
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&Dout[gpu_i], Out_mem_size_s));
  }
  EHistogram = (unsigned int*) calloc(Parallel_num * Slice_NUM, sizeof(unsigned int));
  ChernHistogram = (unsigned int*) calloc(Parallel_num * Slice_CNUM, sizeof(unsigned int));
  hist_start = 0;
}


measurements::~measurements(){
  printf("measure free begin!\n");
  fflush(stdout);
  for (int i =0 ; i< measurement_num; i++){
    fclose(O[i].fp);
  }
  free(Hout);
  free(EHistogram);
  free(ChernHistogram);
  //CudaSafeCall(hipFree(Dout));
  printf("measure free succeed!\n");
  fflush(stdout);
}



void measurements::virtual_measure(float** Dconfx, float** Dconfy, float** Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E;
  static double Mz;
  int gpu_i;
	printf("device start : %d, #streams = %d\n" , device_0, StreamN);
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CAL(Dconfx[gpu_i], Dconfy[gpu_i], Dconfz[gpu_i], Dout[gpu_i], stream[gpu_i]);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  }
  CudaCheckError();
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(Hout+gpu_i * data_num_s * MEASURE_NUM * H_BN, Dout[gpu_i], Out_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));//Async, stream[gpu_i]
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0;
    Mz = 0;
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
  }
}



void measurements::measure(float** Dconfx, float** Dconfy, float** Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E, E2;
  static double Mx, My, Mz, Chern, M2, Mz2, Chern2;
  static double Mxx, Myy, Mxy, eta, Mzi2, Hy, Iy;
  //static double spinQ1x_r, spinQ1y_r, spinQ1z_r, spinQ1x_i, spinQ1y_i, spinQ1z_i;
  //static double spinQ2x_r, spinQ2y_r, spinQ2z_r, spinQ2x_i, spinQ2y_i, spinQ2z_i;
  int gpu_i;
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CAL(Dconfx[gpu_i], Dconfy[gpu_i], Dconfz[gpu_i], Dout[gpu_i], stream[gpu_i]);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  }
  CudaCheckError();
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(Hout + gpu_i * data_num_s * MEASURE_NUM * H_BN, Dout[gpu_i], Out_mem_size_s, hipMemcpyDeviceToHost, stream[gpu_i]));//Async, stream[gpu_i]
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }


  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0, E2 = 0;
    Mx = 0, My = 0, Mz = 0, Chern = 0;
    Mxx = 0, Myy = 0, Mxy = 0, Iy = 0, Hy = 0, Mzi2 = 0;
    /*
    spinQ1x_r = 0, spinQ1y_r = 0, spinQ1z_r = 0;
    spinQ1x_i = 0, spinQ1y_i = 0, spinQ1z_i = 0;
    spinQ2x_r = 0, spinQ2y_r = 0, spinQ2z_r = 0;
    spinQ2x_i = 0, spinQ2y_i = 0, spinQ2z_i = 0;
    */
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = H_BN; j < 2 * H_BN; j++)
      Mx += Hout[raw_off + j];
    for(int j = 2 * H_BN; j < 3 * H_BN; j++)
      My += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    for(int j = 4 * H_BN; j < 5 * H_BN; j++)
      Chern += Hout[raw_off + j];
    for(int j = 5 * H_BN; j < 6 * H_BN; j++)
      Mxx += Hout[raw_off + j];
    for(int j = 6 * H_BN; j < 7 * H_BN; j++)
      Myy += Hout[raw_off + j];
    for(int j = 7 * H_BN; j < 8 * H_BN; j++)
      Mxy += Hout[raw_off + j];
    for(int j = 8 * H_BN; j < 9 * H_BN; j++)
      Mzi2 += Hout[raw_off + j];
    for(int j = 9 * H_BN; j < 10 * H_BN; j++)
      Hy += Hout[raw_off + j];
    for(int j =10 * H_BN; j < 11 * H_BN; j++)
      Iy += Hout[raw_off + j];
    /*
    for(int j = 5 * H_BN; j < 6 * H_BN; j++)
      spinQ1x_r += Hout[raw_off + j];
    for(int j = 6 * H_BN; j < 7 * H_BN; j++)
      spinQ1y_r += Hout[raw_off + j];
    for(int j = 7 * H_BN; j < 8 * H_BN; j++)
      spinQ1z_r += Hout[raw_off + j];
    for(int j = 8 * H_BN; j < 9 * H_BN; j++)
      spinQ1x_i += Hout[raw_off + j];
    for(int j = 9 * H_BN; j < 10 * H_BN; j++)
      spinQ1y_i += Hout[raw_off + j];
    for(int j = 10 * H_BN; j < 11 * H_BN; j++)
      spinQ1z_i += Hout[raw_off + j];
    for(int j = 11 * H_BN; j < 12 * H_BN; j++)
      spinQ2x_r += Hout[raw_off + j];
    for(int j = 12 * H_BN; j < 13 * H_BN; j++)
      spinQ2y_r += Hout[raw_off + j];
    for(int j = 13 * H_BN; j < 14 * H_BN; j++)
      spinQ2z_r += Hout[raw_off + j];
    for(int j = 14 * H_BN; j < 15 * H_BN; j++)
      spinQ2x_i += Hout[raw_off + j];
    for(int j = 15 * H_BN; j < 16 * H_BN; j++)
      spinQ2y_i += Hout[raw_off + j];
    for(int j = 16 * H_BN; j < 17 * H_BN; j++)
      spinQ2z_i += Hout[raw_off + j];
    */
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
    O[0].outdata[Ho[t]] += E;
    M2 = Mx * Mx + My * My + Mz * Mz;
    Mz2 = Mz * Mz;
    Chern2 = Chern * Chern;
    E2 = E * E;
    eta = sqrt(Mxx*Mxx + Myy*Myy - 2*Mxx*Myy + 4*Mxy*Mxy);
    O[1].outdata[Ho[t]] += sqrt(M2);
    O[2].outdata[Ho[t]] += Chern;
    O[3].outdata[Ho[t]] += E2;
    O[5].outdata[Ho[t]] += M2;
    O[4].outdata[Ho[t]] += E2 * E2;
    O[6].outdata[Ho[t]] += M2 * M2;
    O[7].outdata[Ho[t]] += Mz2;
    O[8].outdata[Ho[t]] += Mz2 * Mz2;
    O[9].outdata[Ho[t]] += Chern2;
    O[10].outdata[Ho[t]] += Chern2 * Chern2;
    //close the output of spinQ
    //O[11].outdata[Ho[t]] += spinQ1x_r * spinQ1x_r + spinQ1y_r * spinQ1y_r + spinQ1z_r * spinQ1z_r\
			    + spinQ1x_i * spinQ1x_i + spinQ1y_i * spinQ1y_i + spinQ1z_i * spinQ1z_i;
    //O[12].outdata[Ho[t]] += spinQ2x_r * spinQ2x_r + spinQ2y_r * spinQ2y_r + spinQ2z_r * spinQ2z_r\
			    + spinQ2x_i * spinQ2x_i + spinQ2y_i * spinQ2y_i + spinQ2z_i * spinQ2z_i;
    O[13].outdata[Ho[t]] += Mz;
    O[14].outdata[Ho[t]] += E*Mz;
    O[15].outdata[Ho[t]] += eta;
    O[11].outdata[Ho[t]] += (Mxx*Mxx + Myy*Myy - 2*Mxx*Myy + 4*Mxy*Mxy);
    O[12].outdata[Ho[t]] += (Mxx*Mxx + Myy*Myy - 2*Mxx*Myy + 4*Mxy*Mxy)*(Mxx*Mxx + Myy*Myy - 2*Mxx*Myy + 4*Mxy*Mxy);
    O[16].outdata[Ho[t]] += Mzi2;
    O[17].outdata[Ho[t]] += Hy;
    O[18].outdata[Ho[t]] += Iy*Iy;
    E /= H_N;
    eta /= H_N;
    if (hist_start > 0){
      if ((E<E_highest)&&(E>E_lowest)) EHistogram[Ho[t]*Slice_NUM+int(Slice_NUM*((E-E_lowest)/(E_highest-E_lowest)))] +=1;
      if ((eta<Chern_highest)&&(eta>Chern_lowest)) ChernHistogram[Ho[t]*Slice_CNUM+int(Slice_CNUM*((eta-Chern_lowest)/(Chern_highest-Chern_lowest)))] +=1;
    }
  }
}

measurement::measurement(char* indir, char* Oname, double normin, int Parallel_num){
  strcpy(name, Oname);
  strcpy(dir, indir);
  data_num = Parallel_num;
  norm = normin;
  data_mem_size = data_num * sizeof(double);
  sprintf(fn, "%s/%s", dir, name);
  outdata = (double*)calloc(data_num, sizeof(double));
}


measurement::~measurement(){
  printf("measuresingle free begin!\n");
  fflush(stdout);
  printf("measuresingle free succeed!\n");
  fflush(stdout);
}


void measurement::normalize_and_save_and_reset(){
  for (int t = 0; t < data_num; t++)
    outdata[t] = outdata[t]/norm;

  fwrite(outdata, sizeof(double), data_num, fp);

  for (int t = 0; t < data_num; t++)
    outdata[t] = 0.0;//memset????
}

void measurements::normalize_and_save_and_reset(){
  for (int i = 0; i < measurement_num; i++)
    O[i].normalize_and_save_and_reset();
}


//========================== corr part ==============================



correlation::correlation(int Pnum, char* _Corrfn){
  data_num = Pnum;
  data_num_s = Pnum/StreamN;
  Spin_mem_size_s = data_num_s * H_N * sizeof(float);
  Spin_mem_size_p_s = data_num_s * H_Nplane * sizeof(float);
  Spin_mem_size_d_s = data_num_s * H_Nplane * sizeof(double);
  Spin_mem_size_d = data_num * H_Nplane * sizeof(double);
  corrcount = 0;
  HSum = (double*)malloc(Spin_mem_size_d);
  Dcorr = (float**)calloc(StreamN, sizeof(float*));
  DSum = (double**)calloc(StreamN, sizeof(double*));
  DPo = (int**)calloc(StreamN, sizeof(int*));

  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&Dcorr[gpu_i], Spin_mem_size_p_s));
    CudaSafeCall(hipMalloc((void**)&DSum[gpu_i], Spin_mem_size_d_s));
    CudaSafeCall(hipMalloc((void**)&DPo[gpu_i], data_num_s * sizeof(int)));
  }

  strcpy(Corrfn, _Corrfn);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
  for(int i = 0; i < H_Nplane * data_num; i++){
    HSum[i] = 0.0; //initialize
  }
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpy(DSum[gpu_i], HSum + gpu_i * data_num_s * H_Nplane, Spin_mem_size_d_s, hipMemcpyHostToDevice));
  }
}


void correlation::extract(std::vector<int>& Ho, configuration &CONF){//in &Ho[0]
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpy(DPo[gpu_i], &Ho[gpu_i * data_num_s], data_num_s * sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemset(Dcorr[gpu_i], 0, Spin_mem_size_p_s));
  }
#ifndef TRI
  for (int labelx = 0; labelx < H_SpinSize; labelx += 4){
    for (int labely = 0; labely < H_SpinSize; labely += 4){
      for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
        GETCORR(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], Dcorr[gpu_i], labelx, labely, stream[gpu_i]);
      }
    }
  }
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    sumcorr<<<grid, block, 0, stream[gpu_i]>>>(DSum[gpu_i], Dcorr[gpu_i], DPo[gpu_i]);
  }
  CudaCheckError();
#endif
#ifdef TRI
  for (int labelx = 0; labelx < H_SpinSize; labelx += 3){
    for (int labely = 0; labely < H_SpinSize; labely += 3){
      for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
        GETCORR(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], Dcorr[gpu_i], labelx, labely, stream[gpu_i]);
      }
    }
  }
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    sumcorrTRI<<<grid, block, 0, stream[gpu_i]>>>(DSum[gpu_i], Dcorr[gpu_i], DPo[gpu_i]);
  }
  CudaCheckError();
#endif
  corrcount += 1;
}


void correlation::avg_write_reset(std::vector<int>& Ho){
  int gpu_i;
#ifdef TRI
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    avgcorrTRI<<<grid, block, 0, stream[gpu_i]>>>(DSum[gpu_i], double(corrcount));
  }
  CudaCheckError();
#endif
#ifndef TRI
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    avgcorr<<<grid, block, 0, stream[gpu_i]>>>(DSum[gpu_i], double(corrcount));
  }
  CudaCheckError();
#endif
	for (int j  = 0; j < data_num_s; j++){
		for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
			hipSetDevice(device_0 + gpu_i);
			CudaSafeCall(hipMemcpyAsync(HSum + (Ho[j + gpu_i*data_num_s]) * H_Nplane, &DSum[gpu_i][j * H_Nplane], H_Nplane * sizeof(double), hipMemcpyDeviceToHost, stream[gpu_i]));
		}
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipDeviceSynchronize();
  }
  write(Corrfd, HSum, Spin_mem_size_d);
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemset(DSum[gpu_i], 0, Spin_mem_size_d_s));
  }
  corrcount = 0;
}

void correlation::changefile(char* _Corrfn){
  close(Corrfd);
  strcpy(Corrfn, _Corrfn);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
}

correlation::~correlation(){
  close(Corrfd);
  free(HSum);
  for (int gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipFree(this->Dcorr[gpu_i]));
    CudaSafeCall(hipFree(this->DPo[gpu_i]));//
    CudaSafeCall(hipFree(this->DSum[gpu_i]));
  }
}

