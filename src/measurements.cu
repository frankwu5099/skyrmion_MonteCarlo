#include "hip/hip_runtime.h"
#include "measurements.cuh"


measurements::measurements(char * indir, int Parallel_num, unsigned int binSize){
  measurement_num = 7;
  //raw_memmory = operator new[] (measurement_num * sizeof(measurement));
  strcpy(names[0], "E");
  strcpy(names[1], "M");
  strcpy(names[2], "Chern");
  strcpy(names[3], "E2");
  strcpy(names[4], "E4");
  strcpy(names[5], "M2");
  strcpy(names[6], "M4");
  norms[0] = binSize * H_N;
  norms[1] = binSize * H_N;
  norms[2] = binSize;
  norms[3] = binSize * H_N * H_N;
  norms[4] = binSize * H_N * H_N * H_N * H_N;
  norms[5] = binSize * H_N * H_N;
  norms[6] = binSize * H_N * H_N * H_N * H_N;
  O.reserve(measurement_num);
  for (int i =0 ; i< measurement_num; i++){
    O.push_back(measurement(indir, names[i], norms[i], Parallel_num));
    O[O.size() - 1].fp = fopen(O[O.size() - 1].fn, "w");
  }
  data_num = Parallel_num;
  Out_mem_size = Parallel_num * MEASURE_NUM * H_BN * sizeof(double);
  printf("%u\n", Out_mem_size);
  Hout = (double*)malloc(Out_mem_size);
  CudaSafeCall(hipMalloc(&Dout, Out_mem_size));
}


measurements::~measurements(){
  printf("measure free begin!\n");
  fflush(stdout);
  for (int i =0 ; i< 7; i++){
    fclose(O[i].fp);
  }
  free(Hout);
  //CudaSafeCall(hipFree(Dout));
  printf("measure free succeed!\n");
  fflush(stdout);
}



void measurements::virtual_measure(float* Dconfx, float* Dconfy, float* Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E;
  static double Mz;
  CAL(Dconfx, Dconfy, Dconfz, Dout);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost));

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0;
    Mz = 0;
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
  }
}



void measurements::measure(float* Dconfx, float* Dconfy, float* Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E, E2;
  static double Mx, My, Mz, Chern, M2;
  CAL(Dconfx, Dconfy, Dconfz, Dout);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost));

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0, E2 = 0;
    Mx = 0, My = 0, Mz = 0, Chern = 0;
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = H_BN; j < 2 * H_BN; j++)
      Mx += Hout[raw_off + j];
    for(int j = 2 * H_BN; j < 3 * H_BN; j++)
      My += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    for(int j = 4 * H_BN; j < 5 * H_BN; j++)
      Chern += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
    O[0].outdata[Ho[t]] += E;
    M2 = Mx * Mx + My * My + Mz * Mz;
    E2 = E * E;
    O[1].outdata[Ho[t]] += sqrt(M2);
    O[2].outdata[Ho[t]] += Chern;
    O[3].outdata[Ho[t]] += E2;
    O[5].outdata[Ho[t]] += M2;
    O[4].outdata[Ho[t]] += E2 * E2;
    O[6].outdata[Ho[t]] += M2 * M2;
  }
}

measurement::measurement(char* indir, char* Oname, int normin, int Parallel_num){
  strcpy(name, Oname);
  strcpy(dir, indir);
  data_num = Parallel_num;
  norm = normin;
  data_mem_size = data_num * sizeof(double);
  sprintf(fn, "%s/%s", dir, name);
  outdata = (double*)calloc(data_num, sizeof(double));
}


measurement::~measurement(){
  printf("measuresingle free begin!\n");
  fflush(stdout);
  printf("measuresingle free succeed!\n");
  fflush(stdout);
}


void measurement::normalize_and_save_and_reset(){
  for (int t = 0; t < data_num; t++)
    outdata[t] = outdata[t]/norm;

  fwrite(outdata, sizeof(double), data_num, fp);

  for (int t = 0; t < data_num; t++)
    outdata[t] = 0.0;//memset????
}

void measurements::normalize_and_save_and_reset(){
  for (int i = 0; i < measurement_num; i++)
    O[i].normalize_and_save_and_reset();
}


//========================== corr part ==============================



correlation::correlation(int Pnum, char* dir){
  data_num = Pnum;
  Spin_mem_size = Pnum * H_N * sizeof(float);
  Spin_mem_size_p = Pnum * H_Nplane * sizeof(float);
  Spin_mem_size_d = Pnum * H_Nplane * sizeof(double);
  corrcount = 0;
  HSum = (double*)malloc(Spin_mem_size_d);

  CudaSafeCall(hipMalloc((void**)&Dcorr, Spin_mem_size_p));

  CudaSafeCall(hipMalloc((void**)&DSum, Spin_mem_size_d));
  CudaSafeCall(hipMalloc((void**)&DPo, Pnum * sizeof(int)));
  sprintf(Corrfn, "%s/Corr", dir);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
  for(int i = 0; i < H_Nplane * data_num; i++){
    HSum[i] = 0.0; //initialize
  }
  CudaSafeCall(hipMemcpy(DSum, HSum, Spin_mem_size_d, hipMemcpyHostToDevice));
}


void correlation::extract(std::vector<int>& Ho, configuration &CONF){//in &Ho[0]
  CudaSafeCall(hipMemcpy(DPo, &Ho[0], data_num * sizeof(int), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemset(Dcorr, 0, Spin_mem_size_p));
#ifndef TRI
  for (int labelx = 0; labelx < H_SpinSize; labelx += 4){
    for (int labely = 0; labely < H_SpinSize; labely += 4){
      GETCORR(CONF.Dx, CONF.Dy, CONF.Dz, Dcorr, labelx, labely);
    }
  }
  sumcorr<<<grid, block>>>(DSum, Dcorr, DPo);
  CudaCheckError();
#endif
#ifdef TRI
  for (int labelx = 0; labelx < H_SpinSize; labelx += 3){
    for (int labely = 0; labely < H_SpinSize; labely += 3){
      GETCORR(CONF.Dx, CONF.Dy, CONF.Dz, Dcorr, labelx, labely);
    }
  }
  sumcorrTRI<<<grid, block>>>(DSum, Dcorr, DPo);
  CudaCheckError();
#endif
  corrcount += 1;
}


void correlation::avg_write_reset(){
#ifdef TRI
  avgcorrTRI<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
#ifndef TRI
  avgcorr<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
  CudaSafeCall(hipMemcpy(HSum, DSum, Spin_mem_size_d, hipMemcpyDeviceToHost));
  write(Corrfd, HSum, Spin_mem_size_d);
  CudaSafeCall(hipMemset(DSum, 0, Spin_mem_size_d));
}



correlation::~correlation(){
  printf("corr %d\n", Corrfd);
  close(Corrfd);
  free(HSum);
  //CudaSafeCall(hipFree(this->Dcorr));
  //CudaSafeCall(hipFree(this->DPo));//
  //CudaSafeCall(hipFree(this->DSum));
}

