#include "hip/hip_runtime.h"
#include "measurements.cuh"


measurements::measurements(char * indir, int Parallel_num, unsigned int binSize){
  measurement_num = 14;
  //raw_memmory = operator new[] (measurement_num * sizeof(measurement));
  strcpy(names[0], "E");
  strcpy(names[1], "M");
  strcpy(names[2], "Chern");
  strcpy(names[3], "E2");
  strcpy(names[4], "E4");
  strcpy(names[5], "M2");
  strcpy(names[6], "M4");
  strcpy(names[7], "Mz2");
  strcpy(names[8], "Mz4");
  strcpy(names[9], "Chern2");
  strcpy(names[10], "Chern4");
  strcpy(names[11], "SQ1");
  strcpy(names[12], "SQ2");
  strcpy(names[13], "Mz");
  norms[0] = double(binSize) * H_N;
  norms[1] = double(binSize) * H_N;
  norms[2] = double(binSize) * 2;
  norms[3] = double(binSize) * H_N * H_N;
  norms[4] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[5] = double(binSize) * H_N * H_N;
  norms[6] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[7] = double(binSize) * H_N * H_N;
  norms[8] = double(binSize) * H_N * H_N * H_N * H_N;
  norms[9] = double(binSize) * H_N * H_N * 2 * 2;
  norms[10] = double(binSize) * H_N * H_N * H_N * H_N * 2 * 2 * 2 * 2;
  norms[11] = double(binSize) * H_N * H_N;
  norms[12] = double(binSize) * H_N * H_N;
  norms[13] = double(binSize) * H_N;
  O.reserve(measurement_num);
  for (int i =0 ; i< measurement_num; i++){
    O.push_back(measurement(indir, names[i], norms[i], Parallel_num));
    O[O.size() - 1].fp = fopen(O[O.size() - 1].fn, "w");
  }
  data_num = Parallel_num;
  Out_mem_size = Parallel_num * MEASURE_NUM * H_BN * sizeof(double);
  printf("%u\n", Out_mem_size);
  Hout = (double*)malloc(Out_mem_size);
  CudaSafeCall(hipMalloc(&Dout, Out_mem_size));
  EHistogram = (unsigned int*) calloc(Parallel_num * Slice_NUM, sizeof(unsigned int));
  ChernHistogram = (unsigned int*) calloc(Parallel_num * Slice_CNUM, sizeof(unsigned int));
}


measurements::~measurements(){
  printf("measure free begin!\n");
  fflush(stdout);
  for (int i =0 ; i< measurement_num; i++){
    fclose(O[i].fp);
  }
  free(Hout);
  free(EHistogram);
  free(ChernHistogram);
  //CudaSafeCall(hipFree(Dout));
  printf("measure free succeed!\n");
  fflush(stdout);
}



void measurements::virtual_measure(float* Dconfx, float* Dconfy, float* Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E;
  static double Mz;
  CAL(Dconfx, Dconfy, Dconfz, Dout);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost));

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0;
    Mz = 0;
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
  }
}



void measurements::measure(float* Dconfx, float* Dconfy, float* Dconfz, std::vector<int>& Ho, double* Ms, double* Es, float* HHs){
  static int raw_off;
  static double E, E2;
  static double Mx, My, Mz, Chern, M2, Mz2, Chern2;
  static double spinQ1x_r, spinQ1y_r, spinQ1z_r, spinQ1x_i, spinQ1y_i, spinQ1z_i;
  static double spinQ2x_r, spinQ2y_r, spinQ2z_r, spinQ2x_i, spinQ2y_i, spinQ2z_i;
  CAL(Dconfx, Dconfy, Dconfz, Dout);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost));

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * H_BN;
    E = 0, E2 = 0;
    Mx = 0, My = 0, Mz = 0, Chern = 0;
    spinQ1x_r = 0, spinQ1y_r = 0, spinQ1z_r = 0;
    spinQ1x_i = 0, spinQ1y_i = 0, spinQ1z_i = 0;
    spinQ2x_r = 0, spinQ2y_r = 0, spinQ2z_r = 0;
    spinQ2x_i = 0, spinQ2y_i = 0, spinQ2z_i = 0;
    for(int j = 0; j < H_BN; j++)
      E += Hout[raw_off + j];
    for(int j = H_BN; j < 2 * H_BN; j++)
      Mx += Hout[raw_off + j];
    for(int j = 2 * H_BN; j < 3 * H_BN; j++)
      My += Hout[raw_off + j];
    for(int j = 3 * H_BN; j < 4 * H_BN; j++)
      Mz += Hout[raw_off + j];
    for(int j = 4 * H_BN; j < 5 * H_BN; j++)
      Chern += Hout[raw_off + j];
    for(int j = 5 * H_BN; j < 6 * H_BN; j++)
      spinQ1x_r += Hout[raw_off + j];
    for(int j = 6 * H_BN; j < 7 * H_BN; j++)
      spinQ1y_r += Hout[raw_off + j];
    for(int j = 7 * H_BN; j < 8 * H_BN; j++)
      spinQ1z_r += Hout[raw_off + j];
    for(int j = 8 * H_BN; j < 9 * H_BN; j++)
      spinQ1x_i += Hout[raw_off + j];
    for(int j = 9 * H_BN; j < 10 * H_BN; j++)
      spinQ1y_i += Hout[raw_off + j];
    for(int j = 10 * H_BN; j < 11 * H_BN; j++)
      spinQ1z_i += Hout[raw_off + j];
    for(int j = 11 * H_BN; j < 12 * H_BN; j++)
      spinQ2x_r += Hout[raw_off + j];
    for(int j = 12 * H_BN; j < 13 * H_BN; j++)
      spinQ2y_r += Hout[raw_off + j];
    for(int j = 13 * H_BN; j < 14 * H_BN; j++)
      spinQ2z_r += Hout[raw_off + j];
    for(int j = 14 * H_BN; j < 15 * H_BN; j++)
      spinQ2x_i += Hout[raw_off + j];
    for(int j = 15 * H_BN; j < 16 * H_BN; j++)
      spinQ2y_i += Hout[raw_off + j];
    for(int j = 16 * H_BN; j < 17 * H_BN; j++)
      spinQ2z_i += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    Es[Ho[t]] = E;	//Es is the energies in order of temperature set
    O[0].outdata[Ho[t]] += E;
    M2 = Mx * Mx + My * My + Mz * Mz;
    Mz2 = Mz * Mz;
    Chern2 = Chern * Chern;
    E2 = E * E;
    O[1].outdata[Ho[t]] += sqrt(M2);
    O[2].outdata[Ho[t]] += Chern;
    O[3].outdata[Ho[t]] += E2;
    O[5].outdata[Ho[t]] += M2;
    O[4].outdata[Ho[t]] += E2 * E2;
    O[6].outdata[Ho[t]] += M2 * M2;
    O[7].outdata[Ho[t]] += Mz2;
    O[8].outdata[Ho[t]] += Mz2 * Mz2;
    O[9].outdata[Ho[t]] += Chern2;
    O[10].outdata[Ho[t]] += Chern2 * Chern2;
    O[11].outdata[Ho[t]] += spinQ1x_r * spinQ1x_r + spinQ1y_r * spinQ1y_r + spinQ1z_r * spinQ1z_r\
			    + spinQ1x_i * spinQ1x_i + spinQ1y_i * spinQ1y_i + spinQ1z_i * spinQ1z_i;
    O[12].outdata[Ho[t]] += spinQ2x_r * spinQ2x_r + spinQ2y_r * spinQ2y_r + spinQ2z_r * spinQ2z_r\
			    + spinQ2x_i * spinQ2x_i + spinQ2y_i * spinQ2y_i + spinQ2z_i * spinQ2z_i;
    O[13].outdata[Ho[t]] += Mz;
    E /= H_N;
    if ((E<E_highest)&&(E>E_lowest)) EHistogram[Ho[t]*Slice_NUM+int(Slice_NUM*((E-E_lowest)/(E_highest-E_lowest)))] +=1;
    if ((-Chern<Chern_highest)&&(-Chern>Chern_lowest)) ChernHistogram[Ho[t]*Slice_CNUM+int(Slice_CNUM*((Chern-Chern_lowest)/(Chern_highest-Chern_lowest)))] +=1;
  }
}

measurement::measurement(char* indir, char* Oname, double normin, int Parallel_num){
  strcpy(name, Oname);
  strcpy(dir, indir);
  data_num = Parallel_num;
  norm = normin;
  data_mem_size = data_num * sizeof(double);
  sprintf(fn, "%s/%s", dir, name);
  outdata = (double*)calloc(data_num, sizeof(double));
}


measurement::~measurement(){
  printf("measuresingle free begin!\n");
  fflush(stdout);
  printf("measuresingle free succeed!\n");
  fflush(stdout);
}


void measurement::normalize_and_save_and_reset(){
  for (int t = 0; t < data_num; t++)
    outdata[t] = outdata[t]/norm;

  fwrite(outdata, sizeof(double), data_num, fp);

  for (int t = 0; t < data_num; t++)
    outdata[t] = 0.0;//memset????
}

void measurements::normalize_and_save_and_reset(){
  for (int i = 0; i < measurement_num; i++)
    O[i].normalize_and_save_and_reset();
}


//========================== corr part ==============================



correlation::correlation(int Pnum, char* _Corrfn){
  data_num = Pnum;
  Spin_mem_size = Pnum * H_N * sizeof(float);
  Spin_mem_size_p = Pnum * H_Nplane * sizeof(float);
  Spin_mem_size_d = Pnum * H_Nplane * sizeof(double);
  corrcount = 0;
  HSum = (double*)malloc(Spin_mem_size_d);

  CudaSafeCall(hipMalloc((void**)&Dcorr, Spin_mem_size_p));
  CudaSafeCall(hipMalloc((void**)&skyr_den, Spin_mem_size_p));

  CudaSafeCall(hipMalloc((void**)&DSum, Spin_mem_size_d));
  CudaSafeCall(hipMalloc((void**)&DPo, Pnum * sizeof(int)));
  strcpy(Corrfn, _Corrfn);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
  for(int i = 0; i < H_Nplane * data_num; i++){
    HSum[i] = 0.0; //initialize
  }
  CudaSafeCall(hipMemcpy(DSum, HSum, Spin_mem_size_d, hipMemcpyHostToDevice));
}


void correlation::extract(std::vector<int>& Ho, configuration &CONF){//in &Ho[0]
  CudaSafeCall(hipMemcpy(DPo, &Ho[0], data_num * sizeof(int), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemset(Dcorr, 0, Spin_mem_size_p));
#ifndef TRI
  for (int labelx = 0; labelx < H_SpinSize; labelx += 4){
    for (int labely = 0; labely < H_SpinSize; labely += 4){
      GETCORR(CONF.Dx, CONF.Dy, CONF.Dz, Dcorr, labelx, labely);
    }
  }
  sumcorr<<<grid, block>>>(DSum, Dcorr, DPo);
  CudaCheckError();
#endif
#ifdef TRI

  GETSKYRDEN(CONF.Dx, CONF.Dy, CONF.Dz, skyr_den);
  for (int labelx = 0; labelx < H_SpinSize; labelx += 3){
    for (int labely = 0; labely < H_SpinSize; labely += 3){
      GETCORR(skyr_den, Dcorr, labelx, labely);
    }
  }
  sumcorrTRI<<<grid, block>>>(DSum, Dcorr, DPo);
  CudaCheckError();
#endif
  corrcount += 1;
}


void correlation::avg_write_reset(){
#ifdef TRI
  avgcorrTRI<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
#ifndef TRI
  avgcorr<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
  CudaSafeCall(hipMemcpy(HSum, DSum, Spin_mem_size_d, hipMemcpyDeviceToHost));
  write(Corrfd, HSum, Spin_mem_size_d);
  CudaSafeCall(hipMemset(DSum, 0, Spin_mem_size_d));
  corrcount = 0;
}

void correlation::changefile(char* _Corrfn){
  close(Corrfd);
  strcpy(Corrfn, _Corrfn);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
}

correlation::~correlation(){
  close(Corrfd);
  free(HSum);
  CudaSafeCall(hipFree(this->Dcorr));
  CudaSafeCall(hipFree(this->skyr_den));
  CudaSafeCall(hipFree(this->DPo));//
  CudaSafeCall(hipFree(this->DSum));
}

