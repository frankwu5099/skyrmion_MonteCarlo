#include "hip/hip_runtime.h"
#include "measurements.cuh"


measurements::measurements(char * indir, int Parallel_num, unsigned int binSize){
  measurement_num = 7;
  void* raw_memmory = operator new[] (measurement_num * sizeof(measurement));
  strcpy(names[0], "E");
  strcpy(names[1], "M");
  strcpy(names[2], "Chern");
  strcpy(names[3], "E2");
  strcpy(names[4], "E4");
  strcpy(names[5], "M2");
  strcpy(names[6], "M4");
  norms[0] = binSize * N;
  norms[1] = binSize * N;
  norms[2] = binSize;
  norms[3] = binSize * N * N;
  norms[4] = binSize * N * N * N * N;
  norms[5] = binSize * N * N;
  norms[6] = binSize * N * N * N * N;
  O = static_cast<measurement*>(raw_memmory);
  for (int i =0 ; i< measurement_num; i++){
    new (&O[i])measurement(indir, names[i], norms[i], Parallel_num);
  }
  data_num = Parallel_num;
  Out_mem_size = Parallel_num * MEASURE_NUM * BN * sizeof(double);
  printf("%u\n", Out_mem_size);
  Hout = (double*)malloc(Out_mem_size);
  CudaSafeCall(hipMalloc(&Dout, Out_mem_size));
  operator delete[] (raw_memmory);
}


measurements::~measurements(){
  for (int i =0 ; i< 7; i++){
    O[i].~measurement();
  }
  free(Hout);
  CudaSafeCall(hipFree(Dout));
}



void measurements::measure(float* Dconfx, float* Dconfy, float* Dconfz, std::vector<int>& Ho, double* Ms, float* HHs){
  static int raw_off;
  static double E, E2;
  static double Mx, My, Mz, Chern, M2;
  CAL(Dconfx, Dconfy, Dconfz, Dout);//cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout);
  CudaCheckError();
  CudaSafeCall(hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost));

  for(int t = 0; t < data_num; t++){
    raw_off = t * MEASURE_NUM * BN;
    E = 0, E2 = 0;
    Mx = 0, My = 0, Mz = 0, Chern = 0;
    for(int j = 0; j < BN; j++)
      E += Hout[raw_off + j];
    for(int j = BN; j < 2 * BN; j++)
      Mx += Hout[raw_off + j];
    for(int j = 2 * BN; j < 3 * BN; j++)
      My += Hout[raw_off + j];
    for(int j = 3 * BN; j < 4 * BN; j++)
      Mz += Hout[raw_off + j];
    for(int j = 4 * BN; j < 5 * BN; j++)
      Chern += Hout[raw_off + j];
    Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
    E = E - HHs[t] * Mz;
    O[0].outdata[Ho[t]] += E;
    M2 = Mx * Mx + My * My + Mz * Mz;
    E2 = E * E;
    O[1].outdata[Ho[t]] += sqrt(M2);
    O[2].outdata[Ho[t]] += Chern;
    O[3].outdata[Ho[t]] += E2;
    O[5].outdata[Ho[t]] += M2;
    O[4].outdata[Ho[t]] += E2 * E2;
    O[6].outdata[Ho[t]] += M2 * M2;
  }
}

measurement::measurement(char* indir, char* Oname, int normin, int Parallel_num){
  strcpy(Oname, name);
  strcpy(indir, dir);
  data_num = Parallel_num;
  norm = normin;
  data_mem_size = data_num * sizeof(double);
  sprintf(fn, "%s/%s", dir, name);
  fd = open(fn, O_CREAT | O_WRONLY, 0644); //watch out, there might be some problem
  outdata = (double*)calloc(data_num, sizeof(double));
}


measurement::~measurement(){
  close(fd);
}


void measurement::normalize_and_save_and_reset(){
  for (int t = 0; t < data_num; t++)
    outdata[t] = outdata[t]/norm;

  write(fd, outdata, data_mem_size);

  for (int t = 0; t < data_num; t++)
    outdata[t] = 0.0;//memset????
}

void measurements::normalize_and_save_and_reset(){
  for (int i = 0; i < measurement_num; i++)
    O[i].normalize_and_save_and_reset();
}


//========================== corr part ==============================



correlation::correlation(int Pnum, char* dir){
  data_num = Pnum;
  Spin_mem_size = Pnum * N * sizeof(float);
  Spin_mem_size_p = Pnum * Nplane * sizeof(float);
  Spin_mem_size_d = Pnum * Nplane * sizeof(double);
  corrcount = 0;
  HSum = (double*)malloc(Spin_mem_size_d);

  CudaSafeCall(hipMalloc((void**)&D, Spin_mem_size_p));

  CudaSafeCall(hipMalloc((void**)&DSum, Spin_mem_size_d));
  CudaSafeCall(hipMalloc((void**)&DPo, Pnum * sizeof(int)));
  sprintf(Corrfn, "%s/Corr", dir);
  Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
  for(int i = 0; i < Nplane * data_num; i++){
    HSum[i] = 0.0; //initialize
  }
  CudaSafeCall(hipMemcpy(DSum, HSum, Spin_mem_size_d, hipMemcpyHostToDevice));
}


void correlation::extract(std::vector<int>* Ho, configuration &CONF){//in &Ho[0]
  CudaSafeCall(hipMemcpy(DPo, Ho, data_num * sizeof(int), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemset(D, 0, Spin_mem_size));
#ifndef TRI
  for (int labelx = 0; labelx < SpinSize; labelx += 4){
    for (int labely = 0; labely < SpinSize; labely += 4){
      GETCORR(CONF.Dx, CONF.Dy, CONF.Dz, D, labelx, labely);
    }
  }
  sumcorr<<<grid, block>>>(DSum, D, DPo);
  CudaCheckError();
#endif
#ifdef TRI
  for (int labelx = 0; labelx < SpinSize; labelx += 3){
    for (int labely = 0; labely < SpinSize; labely += 3){
      GETCORR(CONF.Dx, CONF.Dy, CONF.Dz, D, labelx, labely);
    }
  }
  sumcorrTRI<<<grid, block>>>(DSum, D, DPo);
  CudaCheckError();
#endif
  corrcount += 1;
}


void correlation::avg_write_reset(){
#ifdef TRI
  avgcorrTRI<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
#ifndef TRI
  avgcorr<<<grid, block>>>(DSum, double(corrcount));
  CudaCheckError();
#endif
  CudaSafeCall(hipMemcpy(HSum, DSum, Spin_mem_size_d, hipMemcpyDeviceToHost));
  write(Corrfd, HSum, Spin_mem_size_d);
  CudaSafeCall(hipMemset((void*)DSum, 0, Spin_mem_size_d));
}



correlation::~correlation(){
  close(Corrfd);
  free(HSum);
  CudaSafeCall(hipFree(D));
  CudaSafeCall(hipFree(DPo));//
  CudaSafeCall(hipFree(DSum));
}
/*
   char Efn[128];
   char Mfn[128];
   char Chernfn[128];
   char E2fn[128];
   char E4fn[128];
   char M2fn[128];
   char M4fn[128];
   sprintf(Efn, "%s/E", dir);
   sprintf(Mfn, "%s/M", dir);
   sprintf(Chernfn, "%s/Chern", dir);
   sprintf(E2fn, "%s/E2", dir);
   sprintf(E4fn, "%s/E4", dir);
   sprintf(M2fn, "%s/M2", dir);
   sprintf(M4fn, "%s/M4", dir);
   int Efd = open(Efn, O_CREAT | O_WRONLY, 0644);
   int Mfd = open(Mfn, O_CREAT | O_WRONLY, 0644);
   int Chernfd = open(Chernfn, O_CREAT | O_WRONLY, 0644);
   int E2fd = open(E2fn, O_CREAT | O_WRONLY, 0644);
   int E4fd = open(E4fn, O_CREAT | O_WRONLY, 0644);
   int M2fd = open(M2fn, O_CREAT | O_WRONLY, 0644);
   int M4fd = open(M4fn, O_CREAT | O_WRONLY, 0644);
   double *Eout = (double*)malloc(data_mem_size);
   double *Mout = (double*)malloc(data_mem_size);
   double *Chernout = (double*)malloc(data_mem_size);
   double *E2out = (double*)malloc(data_mem_size);
   double *E4out = (double*)malloc(data_mem_size);
   double *M2out = (double*)malloc(data_mem_size);
   double *M4out = (double*)malloc(data_mem_size);
   double *binE = (double*)calloc(Hnum, sizeof(double));
   double *binM = (double*)calloc(Hnum, sizeof(double));
   double *binChern = (double*)calloc(Hnum, sizeof(double));
   double *binE2 = (double*)calloc(Hnum, sizeof(double));
   double *binM2 = (double*)calloc(Hnum, sizeof(double));
   double *binE4 = (double*)calloc(Hnum, sizeof(double));
   double *binM4 = (double*)calloc(Hnum, sizeof(double));
   for(int t = 0; t < Hnum; t++){
   Eout[t] = (double)binE[t] / BIN_SZ / N;
   Mout[t] = (double)binM[t] / BIN_SZ / N;
   Chernout[t] = (double)binChern[t] / BIN_SZ;
   E2out[t] = (double)binE2[t] / BIN_SZ / N / N;
   E4out[t] = (double)binE4[t] / BIN_SZ / N / N / N / N;
   M2out[t] = (double)binM2[t] / BIN_SZ / N / N;
   M4out[t] = (double)binM4[t] / BIN_SZ / N / N / N / N;
//Helout[t] = (double)binHel[t] / BIN_SZ / N;
}
write(Efd, Eout, data_mem_size);
write(Mfd, Mout, data_mem_size);
write(Chernfd, Chernout, data_mem_size);
write(E2fd, E2out, data_mem_size);
write(E4fd, E4out, data_mem_size);
write(M2fd, M2out, data_mem_size);
write(M4fd, M4out, data_mem_size);
close(Efd);
close(Mfd);
close(Chernfd);
close(E2fd);
close(E4fd);
close(M2fd);
close(M4fd);
free(Eout);
free(Mout);
free(Chernout);
free(E2out);
free(E4out);
free(M2out);
free(M4out);
 */

