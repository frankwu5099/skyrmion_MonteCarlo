#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__constant__ unsigned int corr_SpinSize;
__constant__ unsigned int corr_SpinSize_z;
__constant__ unsigned int corr_BlockSize_x;
__constant__ unsigned int corr_BlockSize_y;
__constant__ unsigned int corr_GridSize_x;
__constant__ unsigned int corr_GridSize_y;
__constant__ unsigned int corr_N;
__constant__ unsigned int corr_Nplane;
__constant__ unsigned int corr_TN;
__constant__ unsigned int corr_BN;
void move_params_device_corr(){
  hipMemcpyToSymbol(HIP_SYMBOL(corr_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_N ), &H_N , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_Nplane ), &H_Nplane , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BN), &H_BN, sizeof(unsigned int));
}
__global__ void skyr_den_gen(const float *confx, const float *confy, const float *confz, float *skyr_den){
	//Energy variables
	const int x = threadIdx.x % (corr_BlockSize_x);
	const int y = (threadIdx.x / corr_BlockSize_x);
	const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
	const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	const int txp2 = tx +2 ;
	const int typ2 = ty +2 ;
	int typ3 = ty +3 ;
	int txp3 = (tx + 3) % corr_SpinSize;
	//const int ty = 2 * ((blockIdx.x / cals_BN) * cals_SpinSize + ((blockIdx.x % cals_BN) / cals_GridSize_x) * cals_BlockSize_y + y);
	const int dataoff = (blockIdx.x / corr_BN) * MEASURE_NUM * corr_BN;
	int bx, by, tx_ty = tx + (ty % corr_SpinSize);
	float Ax, Ay, Az, Bx, By, Bz, Cx, Cy, Cz;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + corr_SpinSize - 1) % corr_SpinSize;
	if((ty % corr_SpinSize) == 0)	by = ty + corr_SpinSize - 1;
	else				by = ty - 1;
	if((typ3 % corr_SpinSize) == 0)	typ3 = typ3 - corr_SpinSize;
	//try to measure Chern number
	Ax = confx[corr_coo2D(ty, tx)];
	Ay = confy[corr_coo2D(ty, tx)];
	Az = confz[corr_coo2D(ty, tx)];
	Bx = confx[corr_coo2D(by, bx)];
	By = confy[corr_coo2D(by, bx)];
	Bz = confz[corr_coo2D(by, bx)];
	Cx = confx[corr_coo2D(ty, bx)];
	Cy = confy[corr_coo2D(ty, bx)];
	Cz = confz[corr_coo2D(ty, bx)];
	skyr_den[corr_coo2D(ty, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(by, tx)];
	Cy = confy[corr_coo2D(by, tx)];
	Cz = confz[corr_coo2D(by, tx)];
	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(ty, txp)];
	By = confy[corr_coo2D(ty, txp)];
	Bz = confz[corr_coo2D(ty, txp)];
	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(ty, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(by, txp)];
	Ay = confy[corr_coo2D(by, txp)];
	Az = confz[corr_coo2D(by, txp)];
	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(ty, txp2)];
	Cy = confy[corr_coo2D(ty, txp2)];
	Cz = confz[corr_coo2D(ty, txp2)];
	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(ty, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(by, txp2)];
	By = confy[corr_coo2D(by, txp2)];
	Bz = confz[corr_coo2D(by, txp2)];
	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(ty, txp3)];
	Ay = confy[corr_coo2D(ty, txp3)];
	Az = confz[corr_coo2D(ty, txp3)];
	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(1,0)
	Ax = confx[corr_coo2D(typ, tx)];
	Ay = confy[corr_coo2D(typ, tx)];
	Az = confz[corr_coo2D(typ, tx)];
	Bx = confx[corr_coo2D(ty, bx)];
	By = confy[corr_coo2D(ty, bx)];
	Bz = confz[corr_coo2D(ty, bx)];
	Cx = confx[corr_coo2D(typ, bx)];
	Cy = confy[corr_coo2D(typ, bx)];
	Cz = confz[corr_coo2D(typ, bx)];
	skyr_den[corr_coo2D(typ, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(ty, tx)];
	Cy = confy[corr_coo2D(ty, tx)];
	Cz = confz[corr_coo2D(ty, tx)];
	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(typ, txp)];
	By = confy[corr_coo2D(typ, txp)];
	Bz = confz[corr_coo2D(typ, txp)];
	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(ty, txp)];
	Ay = confy[corr_coo2D(ty, txp)];
	Az = confz[corr_coo2D(ty, txp)];
	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ, txp2)];
	Cy = confy[corr_coo2D(typ, txp2)];
	Cz = confz[corr_coo2D(typ, txp2)];
	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(ty, txp2)];
	By = confy[corr_coo2D(ty, txp2)];
	Bz = confz[corr_coo2D(ty, txp2)];
	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(typ, txp3)];
	Ay = confy[corr_coo2D(typ, txp3)];
	Az = confz[corr_coo2D(typ, txp3)];
	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(ty, txp3)];
	Cy = confy[corr_coo2D(ty, txp3)];
	Cz = confz[corr_coo2D(ty, txp3)];
	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(2,0)
	Ax = confx[corr_coo2D(typ2, tx)];
	Ay = confy[corr_coo2D(typ2, tx)];
	Az = confz[corr_coo2D(typ2, tx)];
	Bx = confx[corr_coo2D(typ, bx)];
	By = confy[corr_coo2D(typ, bx)];
	Bz = confz[corr_coo2D(typ, bx)];
	Cx = confx[corr_coo2D(typ2, bx)];
	Cy = confy[corr_coo2D(typ2, bx)];
	Cz = confz[corr_coo2D(typ2, bx)];
	skyr_den[corr_coo2D(typ2, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ, tx)];
	Cy = confy[corr_coo2D(typ, tx)];
	Cz = confz[corr_coo2D(typ, tx)];
	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(typ2, txp)];
	By = confy[corr_coo2D(typ2, txp)];
	Bz = confz[corr_coo2D(typ2, txp)];
	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(typ, txp)];
	Ay = confy[corr_coo2D(typ, txp)];
	Az = confz[corr_coo2D(typ, txp)];
	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ2, txp2)];
	Cy = confy[corr_coo2D(typ2, txp2)];
	Cz = confz[corr_coo2D(typ2, txp2)];
	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(typ, txp2)];
	By = confy[corr_coo2D(typ, txp2)];
	Bz = confz[corr_coo2D(typ, txp2)];
	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(typ2, txp3)];
	Ay = confy[corr_coo2D(typ2, txp3)];
	Az = confz[corr_coo2D(typ2, txp3)];
	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ, txp3)];
	Cy = confy[corr_coo2D(typ, txp3)];
	Cz = confz[corr_coo2D(typ, txp3)];
	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(3,0)
	Ax = confx[corr_coo2D(typ3, tx)];
	Ay = confy[corr_coo2D(typ3, tx)];
	Az = confz[corr_coo2D(typ3, tx)];
	Bx = confx[corr_coo2D(typ2, bx)];
	By = confy[corr_coo2D(typ2, bx)];
	Bz = confz[corr_coo2D(typ2, bx)];
	Cx = confx[corr_coo2D(typ2, tx)];
	Cy = confy[corr_coo2D(typ2, tx)];
	Cz = confz[corr_coo2D(typ2, tx)];
	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(typ3, txp)];
	By = confy[corr_coo2D(typ3, txp)];
	Bz = confz[corr_coo2D(typ3, txp)];
	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(typ2, txp)];
	Ay = confy[corr_coo2D(typ2, txp)];
	Az = confz[corr_coo2D(typ2, txp)];
	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ3, txp2)];
	Cy = confy[corr_coo2D(typ3, txp2)];
	Cz = confz[corr_coo2D(typ3, txp2)];
	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[corr_coo2D(typ2, txp2)];
	By = confy[corr_coo2D(typ2, txp2)];
	Bz = confz[corr_coo2D(typ2, txp2)];
	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Ax = confx[corr_coo2D(typ3, txp3)];
	Ay = confy[corr_coo2D(typ3, txp3)];
	Az = confz[corr_coo2D(typ3, txp3)];
	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Cx = confx[corr_coo2D(typ2, txp3)];
	Cy = confy[corr_coo2D(typ2, txp3)];
	Cz = confz[corr_coo2D(typ2, txp3)];
	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	__syncthreads();
	skyr_den[corr_coo2D(ty, tx)] /= 3.0;
	skyr_den[corr_coo2D(ty, txp)] /= 3.0;
	skyr_den[corr_coo2D(ty, txp2)] /= 3.0;
	skyr_den[corr_coo2D(typ, tx)] /= 3.0;
	skyr_den[corr_coo2D(typ, txp)] /= 3.0;
	skyr_den[corr_coo2D(typ, txp2)] /= 3.0;
	skyr_den[corr_coo2D(typ2, tx)] /= 3.0;
	skyr_den[corr_coo2D(typ2, txp)] /= 3.0;
	skyr_den[corr_coo2D(typ2, txp2)] /= 3.0;
	__syncthreads();
}
//__global__ void skyr_den_gen(const float *confx, const float *confy, const float *confz, float *skyr_den){
//	//Energy variables
//	const int x = threadIdx.x % (corr_BlockSize_x);
//	const int y = (threadIdx.x / corr_BlockSize_x);
//	const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
//	const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
//	const int txp = tx +1 ;
//	const int typ = ty +1 ;
//	const int txp2 = tx +2 ;
//	const int typ2 = ty +2 ;
//	//const int ty = 2 * ((blockIdx.x / cals_BN) * cals_SpinSize + ((blockIdx.x % cals_BN) / cals_GridSize_x) * cals_BlockSize_y + y);
//	const int dataoff = (blockIdx.x / corr_BN) * MEASURE_NUM * corr_BN;
//	int bx, by, tx_ty = tx + (ty % corr_SpinSize);
//	float Ax, Ay, Az, Bx, By, Bz, Cx, Cy, Cz;
//	//-----Calculate the energy of each spin pairs in the system-----
//	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.
//
//	bx = (tx + corr_SpinSize - 1) % corr_SpinSize;
//	if((ty % corr_SpinSize) == 0)	by = ty + corr_SpinSize - 1;
//	else				by = ty - 1;
//	//try to measure Chern number
//	//(0,0)
//	Ax = confx[corr_coo2D(ty, tx)];
//	Ay = confy[corr_coo2D(ty, tx)];
//	Az = confz[corr_coo2D(ty, tx)];
//	Bx = confx[corr_coo2D(ty, bx)];
//	By = confy[corr_coo2D(ty, bx)];
//	Bz = confz[corr_coo2D(ty, bx)];
//	Cx = confx[corr_coo2D(by, bx)];
//	Cy = confy[corr_coo2D(by, bx)];
//	Cz = confz[corr_coo2D(by, bx)];
//	skyr_den[corr_coo2D(ty, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(by, tx)];
//	By = confy[corr_coo2D(by, tx)];
//	Bz = confz[corr_coo2D(by, tx)];
//	skyr_den[corr_coo2D(ty, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(1,0)
//	Ax = confx[corr_coo2D(typ, tx)];
//	Ay = confy[corr_coo2D(typ, tx)];
//	Az = confz[corr_coo2D(typ, tx)];
//	Bx = confx[corr_coo2D(typ, bx)];
//	By = confy[corr_coo2D(typ, bx)];
//	Bz = confz[corr_coo2D(typ, bx)];
//	Cx = confx[corr_coo2D(ty, bx)];
//	Cy = confy[corr_coo2D(ty, bx)];
//	Cz = confz[corr_coo2D(ty, bx)];
//	skyr_den[corr_coo2D(typ, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(ty, tx)];
//	By = confy[corr_coo2D(ty, tx)];
//	Bz = confz[corr_coo2D(ty, tx)];
//	skyr_den[corr_coo2D(typ, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(2,0)
//	Ax = confx[corr_coo2D(typ2, tx)];
//	Ay = confy[corr_coo2D(typ2, tx)];
//	Az = confz[corr_coo2D(typ2, tx)];
//	Bx = confx[corr_coo2D(typ2, bx)];
//	By = confy[corr_coo2D(typ2, bx)];
//	Bz = confz[corr_coo2D(typ2, bx)];
//	Cx = confx[corr_coo2D(typ, bx)];
//	Cy = confy[corr_coo2D(typ, bx)];
//	Cz = confz[corr_coo2D(typ, bx)];
//	skyr_den[corr_coo2D(typ2, tx)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(typ, tx)];
//	By = confy[corr_coo2D(typ, tx)];
//	Bz = confz[corr_coo2D(typ, tx)];
//	skyr_den[corr_coo2D(typ2, tx)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(0,1)
//	Ax = confx[corr_coo2D(ty, txp)];
//	Ay = confy[corr_coo2D(ty, txp)];
//	Az = confz[corr_coo2D(ty, txp)];
//	Bx = confx[corr_coo2D(ty, tx)];
//	By = confy[corr_coo2D(ty, tx)];
//	Bz = confz[corr_coo2D(ty, tx)];
//	Cx = confx[corr_coo2D(by, tx)];
//	Cy = confy[corr_coo2D(by, tx)];
//	Cz = confz[corr_coo2D(by, tx)];
//	skyr_den[corr_coo2D(ty, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(by, txp)];
//	By = confy[corr_coo2D(by, txp)];
//	Bz = confz[corr_coo2D(by, txp)];
//	skyr_den[corr_coo2D(ty, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(1,1)
//	Ax = confx[corr_coo2D(typ, txp)];
//	Ay = confy[corr_coo2D(typ, txp)];
//	Az = confz[corr_coo2D(typ, txp)];
//	Bx = confx[corr_coo2D(typ, tx)];
//	By = confy[corr_coo2D(typ, tx)];
//	Bz = confz[corr_coo2D(typ, tx)];
//	Cx = confx[corr_coo2D(ty, tx)];
//	Cy = confy[corr_coo2D(ty, tx)];
//	Cz = confz[corr_coo2D(ty, tx)];
//	skyr_den[corr_coo2D(typ, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(ty, txp)];
//	By = confy[corr_coo2D(ty, txp)];
//	Bz = confz[corr_coo2D(ty, txp)];
//	skyr_den[corr_coo2D(typ, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(2,1)
//	Ax = confx[corr_coo2D(typ2, txp)];
//	Ay = confy[corr_coo2D(typ2, txp)];
//	Az = confz[corr_coo2D(typ2, txp)];
//	Bx = confx[corr_coo2D(typ2, tx)];
//	By = confy[corr_coo2D(typ2, tx)];
//	Bz = confz[corr_coo2D(typ2, tx)];
//	Cx = confx[corr_coo2D(typ, tx)];
//	Cy = confy[corr_coo2D(typ, tx)];
//	Cz = confz[corr_coo2D(typ, tx)];
//	skyr_den[corr_coo2D(typ2, txp)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(typ, txp)];
//	By = confy[corr_coo2D(typ, txp)];
//	Bz = confz[corr_coo2D(typ, txp)];
//	skyr_den[corr_coo2D(typ2, txp)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(0,2)
//	Ax = confx[corr_coo2D(ty, txp2)];
//	Ay = confy[corr_coo2D(ty, txp2)];
//	Az = confz[corr_coo2D(ty, txp2)];
//	Bx = confx[corr_coo2D(ty, txp)];
//	By = confy[corr_coo2D(ty, txp)];
//	Bz = confz[corr_coo2D(ty, txp)];
//	Cx = confx[corr_coo2D(by, txp)];
//	Cy = confy[corr_coo2D(by, txp)];
//	Cz = confz[corr_coo2D(by, txp)];
//	skyr_den[corr_coo2D(ty, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(by, txp2)];
//	By = confy[corr_coo2D(by, txp2)];
//	Bz = confz[corr_coo2D(by, txp2)];
//	skyr_den[corr_coo2D(ty, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(1,2)
//	Ax = confx[corr_coo2D(typ, txp2)];
//	Ay = confy[corr_coo2D(typ, txp2)];
//	Az = confz[corr_coo2D(typ, txp2)];
//	Bx = confx[corr_coo2D(typ, txp)];
//	By = confy[corr_coo2D(typ, txp)];
//	Bz = confz[corr_coo2D(typ, txp)];
//	Cx = confx[corr_coo2D(ty, txp)];
//	Cy = confy[corr_coo2D(ty, txp)];
//	Cz = confz[corr_coo2D(ty, txp)];
//	skyr_den[corr_coo2D(typ, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(ty, txp2)];
//	By = confy[corr_coo2D(ty, txp2)];
//	Bz = confz[corr_coo2D(ty, txp2)];
//	skyr_den[corr_coo2D(typ, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	//(2,2)
//	Ax = confx[corr_coo2D(typ2, txp2)];
//	Ay = confy[corr_coo2D(typ2, txp2)];
//	Az = confz[corr_coo2D(typ2, txp2)];
//	Bx = confx[corr_coo2D(typ2, txp)];
//	By = confy[corr_coo2D(typ2, txp)];
//	Bz = confz[corr_coo2D(typ2, txp)];
//	Cx = confx[corr_coo2D(typ, txp)];
//	Cy = confy[corr_coo2D(typ, txp)];
//	Cz = confz[corr_coo2D(typ, txp)];
//	skyr_den[corr_coo2D(typ2, txp2)] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	Bx = confx[corr_coo2D(typ, txp2)];
//	By = confy[corr_coo2D(typ, txp2)];
//	Bz = confz[corr_coo2D(typ, txp2)];
//	skyr_den[corr_coo2D(typ2, txp2)] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
//	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
//	__syncthreads();
//}
__global__ void getcorrTRI(const float *confx, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / corr_BN) * corr_SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sx00, sx01, sx02,
        sx10, sx11, sx12,
        sx20, sx21, sx22;
  int fx0, fy0,
      fx1, fy1,
      fx2, fy2,
      fx3, fy3,//from o to f
      fx4, fy4;
  //calculate all the final position first

  fx0 = (tx + original_i) % corr_SpinSize;
  fx1 = (tx + original_i + 1) % corr_SpinSize;
  fx2 = (tx + original_i + 2) % corr_SpinSize;
  fx3 = (tx + original_i + 3) % corr_SpinSize;
  fx4 = (tx + original_i + 4) % corr_SpinSize;

  if((ty % corr_SpinSize + original_j) >= corr_SpinSize)	fy0 = ty + original_j - corr_SpinSize;
  else  fy0 = ty + original_j;
  if((ty % corr_SpinSize + original_j + 1) >= corr_SpinSize)	fy1 = ty + original_j + 1 - corr_SpinSize;
  else  fy1 = ty + original_j + 1;
  if((ty % corr_SpinSize + original_j + 2) >= corr_SpinSize)	fy2 = ty + original_j + 2 - corr_SpinSize;
  else  fy2 = ty + original_j + 2;
  if((ty % corr_SpinSize + original_j + 3) >= corr_SpinSize)	fy3 = ty + original_j + 3 - corr_SpinSize;
  else  fy3 = ty + original_j + 3;
  if((ty % corr_SpinSize + original_j + 4) >= corr_SpinSize)	fy4 = ty + original_j + 4 - corr_SpinSize;
  else  fy4 = ty + original_j + 4;

  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[corr_coo2D(oy,ox)];
  sx01 = confx[corr_coo2D(oy,ox+1)];
  sx02 = confx[corr_coo2D(oy,ox+2)];
  sx10 = confx[corr_coo2D(oy+1,ox)];
  sx11 = confx[corr_coo2D(oy+1,ox+1)];
  sx12 = confx[corr_coo2D(oy+1,ox+2)];
  sx20 = confx[corr_coo2D(oy+2,ox)];
  sx21 = confx[corr_coo2D(oy+2,ox+1)];
  sx22 = confx[corr_coo2D(oy+2,ox+2)];
  corr[corr_coo2D(ty,tx)] += sx00 * confx[corr_coo2D( fy0,fx0)] +
                        sx01 * confx[corr_coo2D( fy0,fx1)] +
                        sx02 * confx[corr_coo2D( fy0,fx2)] +
                        sx10 * confx[corr_coo2D( fy1,fx0)] +
                        sx11 * confx[corr_coo2D( fy1,fx1)] +
                        sx12 * confx[corr_coo2D( fy1,fx2)] +
                        sx20 * confx[corr_coo2D( fy2,fx0)] +
                        sx21 * confx[corr_coo2D( fy2,fx1)] +
                        sx22 * confx[corr_coo2D( fy2,fx2)] ;
  corr[corr_coo2D(ty,tx+1)] += sx00 * confx[corr_coo2D( fy0,fx1)] +
                          sx01 * confx[corr_coo2D( fy0,fx2)] +
                          sx02 * confx[corr_coo2D( fy0,fx3)] +
                          sx10 * confx[corr_coo2D( fy1,fx1)] +
                          sx11 * confx[corr_coo2D( fy1,fx2)] +
                          sx12 * confx[corr_coo2D( fy1,fx3)] +
                          sx20 * confx[corr_coo2D( fy2,fx1)] +
                          sx21 * confx[corr_coo2D( fy2,fx2)] +
                          sx22 * confx[corr_coo2D( fy2,fx3)] ;
  corr[corr_coo2D(ty,tx+2)] += sx00 * confx[corr_coo2D( fy0,fx2)] +
                          sx01 * confx[corr_coo2D( fy0,fx3)] +
                          sx02 * confx[corr_coo2D( fy0,fx4)] +
                          sx10 * confx[corr_coo2D( fy1,fx2)] +
                          sx11 * confx[corr_coo2D( fy1,fx3)] +
                          sx12 * confx[corr_coo2D( fy1,fx4)] +
                          sx20 * confx[corr_coo2D( fy2,fx2)] +
                          sx21 * confx[corr_coo2D( fy2,fx3)] +
                          sx22 * confx[corr_coo2D( fy2,fx4)] ;
  corr[corr_coo2D((ty+1),tx)] += sx00 * confx[corr_coo2D( fy1,fx0)] +
                            sx01 * confx[corr_coo2D( fy1,fx1)] +
                            sx02 * confx[corr_coo2D( fy1,fx2)] +
                            sx10 * confx[corr_coo2D( fy2,fx0)] +
                            sx11 * confx[corr_coo2D( fy2,fx1)] +
                            sx12 * confx[corr_coo2D( fy2,fx2)] +
                            sx20 * confx[corr_coo2D( fy3,fx0)] +
                            sx21 * confx[corr_coo2D( fy3,fx1)] +
                            sx22 * confx[corr_coo2D( fy3,fx2)] ;
  corr[corr_coo2D((ty+1),tx+1)] += sx00 * confx[corr_coo2D( fy1,fx1)] +
                              sx01 * confx[corr_coo2D( fy1,fx2)] +
                              sx02 * confx[corr_coo2D( fy1,fx3)] +
                              sx10 * confx[corr_coo2D( fy2,fx1)] +
                              sx11 * confx[corr_coo2D( fy2,fx2)] +
                              sx12 * confx[corr_coo2D( fy2,fx3)] +
                              sx20 * confx[corr_coo2D( fy3,fx1)] +
                              sx21 * confx[corr_coo2D( fy3,fx2)] +
                              sx22 * confx[corr_coo2D( fy3,fx3)] ;
  corr[corr_coo2D((ty+1),tx+2)] += sx00 * confx[corr_coo2D( fy1,fx2)] +
                              sx01 * confx[corr_coo2D( fy1,fx3)] +
                              sx02 * confx[corr_coo2D( fy1,fx4)] +
                              sx10 * confx[corr_coo2D( fy2,fx2)] +
                              sx11 * confx[corr_coo2D( fy2,fx3)] +
                              sx12 * confx[corr_coo2D( fy2,fx4)] +
                              sx20 * confx[corr_coo2D( fy3,fx2)] +
                              sx21 * confx[corr_coo2D( fy3,fx3)] +
                              sx22 * confx[corr_coo2D( fy3,fx4)] ;
  corr[corr_coo2D((ty+2),tx)] += sx00 * confx[corr_coo2D( fy2,fx0)] +
                            sx01 * confx[corr_coo2D( fy2,fx1)] +
                            sx02 * confx[corr_coo2D( fy2,fx2)] +
                            sx10 * confx[corr_coo2D( fy3,fx0)] +
                            sx11 * confx[corr_coo2D( fy3,fx1)] +
                            sx12 * confx[corr_coo2D( fy3,fx2)] +
                            sx20 * confx[corr_coo2D( fy4,fx0)] +
                            sx21 * confx[corr_coo2D( fy4,fx1)] +
                            sx22 * confx[corr_coo2D( fy4,fx2)] ;
  corr[corr_coo2D((ty+2),tx+1)] += sx00 * confx[corr_coo2D( fy2,fx1)] +
                              sx01 * confx[corr_coo2D( fy2,fx2)] +
                              sx02 * confx[corr_coo2D( fy2,fx3)] +
                              sx10 * confx[corr_coo2D( fy3,fx1)] +
                              sx11 * confx[corr_coo2D( fy3,fx2)] +
                              sx12 * confx[corr_coo2D( fy3,fx3)] +
                              sx20 * confx[corr_coo2D( fy4,fx1)] +
                              sx21 * confx[corr_coo2D( fy4,fx2)] +
                              sx22 * confx[corr_coo2D( fy4,fx3)] ;
  corr[corr_coo2D((ty+2),tx+2)] += sx00 * confx[corr_coo2D( fy2,fx2)] +
                              sx01 * confx[corr_coo2D( fy2,fx3)] +
                              sx02 * confx[corr_coo2D( fy2,fx4)] +
                              sx10 * confx[corr_coo2D( fy3,fx2)] +
                              sx11 * confx[corr_coo2D( fy3,fx3)] +
                              sx12 * confx[corr_coo2D( fy3,fx4)] +
                              sx20 * confx[corr_coo2D( fy4,fx2)] +
                              sx21 * confx[corr_coo2D( fy4,fx3)] +
                              sx22 * confx[corr_coo2D( fy4,fx4)] ;
  __syncthreads();
}
__global__ void getFTskyr(const float *confx, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / corr_BN) * corr_SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sx00, sx01, sx02,
        sx10, sx11, sx12,
        sx20, sx21, sx22;
  float fx0, fy0, fx1, fy1, fx2, fy2;
  int ox0 = (ox+0)%corr_SpinSize;
  int ox1 = (ox+1)%corr_SpinSize;
  int ox2 = (ox+2)%corr_SpinSize;
  int ox3 = (ox+3)%corr_SpinSize;
  int oy0 = (oy+0)%corr_SpinSize;
  int oy1 = (oy+1)%corr_SpinSize;
  int oy2 = (oy+2)%corr_SpinSize;
  int oy3 = (oy+3)%corr_SpinSize;
  float sinshift = (tx>=(corr_SpinSize/2))?(3.0/4.0*TWOPI):0.0;
  //calculate all the final position first

  fx0 = (tx) % (corr_SpinSize/2);
  fx1 = (tx + 1) % (corr_SpinSize/2);
  fx2 = (tx + 2) % (corr_SpinSize/2);
  fx0 *= TWOPI/float(corr_SpinSize);
  fx1 *= TWOPI/float(corr_SpinSize);
  fx2 *= TWOPI/float(corr_SpinSize);

  fy0 = (ty) % corr_SpinSize;
  fy1 = (ty + 1) % corr_SpinSize;
  fy2 = (ty + 2) % corr_SpinSize;
  fy0 *= TWOPI/float(corr_SpinSize);
  fy1 *= TWOPI/float(corr_SpinSize);
  fy2 *= TWOPI/float(corr_SpinSize);


  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[corr_coo2D(oy  , ox  )];
  sx01 = confx[corr_coo2D(oy  , ox+1)];
  sx02 = confx[corr_coo2D(oy  , ox+2)];
  sx10 = confx[corr_coo2D(oy+1, ox  )];
  sx11 = confx[corr_coo2D(oy+1, ox+1)];
  sx12 = confx[corr_coo2D(oy+1, ox+2)];
  sx20 = confx[corr_coo2D(oy+2, ox  )];
  sx21 = confx[corr_coo2D(oy+2, ox+1)];
  sx22 = confx[corr_coo2D(oy+2, ox+2)];
  corr[corr_coo2D(ty,tx)] +=  sx00 * cosf(sinshift + fy0*float(oy0) + fx0*float(ox0)) +
			      sx01 * cosf(sinshift + fy0*float(oy0) + fx0*float(ox1)) +
			      sx02 * cosf(sinshift + fy0*float(oy0) + fx0*float(ox2)) +
			      sx10 * cosf(sinshift + fy0*float(oy1) + fx0*float(ox0)) +
			      sx11 * cosf(sinshift + fy0*float(oy1) + fx0*float(ox1)) +
			      sx12 * cosf(sinshift + fy0*float(oy1) + fx0*float(ox2)) +
			      sx20 * cosf(sinshift + fy0*float(oy2) + fx0*float(ox0)) +
			      sx21 * cosf(sinshift + fy0*float(oy2) + fx0*float(ox1)) +
			      sx22 * cosf(sinshift + fy0*float(oy2) + fx0*float(ox2)) ;
  corr[corr_coo2D(ty,tx+1)] += sx00 * cosf(sinshift + fy0*float(oy0) + fx1*float(ox0)) +
			       sx01 * cosf(sinshift + fy0*float(oy0) + fx1*float(ox1)) +
			       sx02 * cosf(sinshift + fy0*float(oy0) + fx1*float(ox2)) +
			       sx10 * cosf(sinshift + fy0*float(oy1) + fx1*float(ox0)) +
			       sx11 * cosf(sinshift + fy0*float(oy1) + fx1*float(ox1)) +
			       sx12 * cosf(sinshift + fy0*float(oy1) + fx1*float(ox2)) +
			       sx20 * cosf(sinshift + fy0*float(oy2) + fx1*float(ox0)) +
			       sx21 * cosf(sinshift + fy0*float(oy2) + fx1*float(ox1)) +
			       sx22 * cosf(sinshift + fy0*float(oy2) + fx1*float(ox2)) ;
  corr[corr_coo2D(ty,tx+2)] += sx00 * cosf(sinshift + fy0*float(oy0) + fx2*float(ox0)) +
			       sx01 * cosf(sinshift + fy0*float(oy0) + fx2*float(ox1)) +
			       sx02 * cosf(sinshift + fy0*float(oy0) + fx2*float(ox2)) +
			       sx10 * cosf(sinshift + fy0*float(oy1) + fx2*float(ox0)) +
			       sx11 * cosf(sinshift + fy0*float(oy1) + fx2*float(ox1)) +
			       sx12 * cosf(sinshift + fy0*float(oy1) + fx2*float(ox2)) +
			       sx20 * cosf(sinshift + fy0*float(oy2) + fx2*float(ox0)) +
			       sx21 * cosf(sinshift + fy0*float(oy2) + fx2*float(ox1)) +
			       sx22 * cosf(sinshift + fy0*float(oy2) + fx2*float(ox2)) ;
  corr[corr_coo2D((ty+1),tx)] += sx00 * cosf(sinshift + fy1*float(oy0) + fx0*float(ox0)) +
			 	 sx01 * cosf(sinshift + fy1*float(oy1) + fx0*float(ox1)) +
			 	 sx02 * cosf(sinshift + fy1*float(oy2) + fx0*float(ox2)) +
			 	 sx10 * cosf(sinshift + fy1*float(oy0) + fx0*float(ox0)) +
			 	 sx11 * cosf(sinshift + fy1*float(oy1) + fx0*float(ox1)) +
			 	 sx12 * cosf(sinshift + fy1*float(oy2) + fx0*float(ox2)) +
			 	 sx20 * cosf(sinshift + fy1*float(oy0) + fx0*float(ox0)) +
			 	 sx21 * cosf(sinshift + fy1*float(oy1) + fx0*float(ox1)) +
			 	 sx22 * cosf(sinshift + fy1*float(oy2) + fx0*float(ox2)) ;
  corr[corr_coo2D((ty+1),tx+1)] += sx00 * cosf(sinshift + fy1*float(oy0) + fx1*float(ox0)) +
				   sx01 * cosf(sinshift + fy1*float(oy1) + fx1*float(ox1)) +
				   sx02 * cosf(sinshift + fy1*float(oy2) + fx1*float(ox2)) +
				   sx10 * cosf(sinshift + fy1*float(oy0) + fx1*float(ox0)) +
				   sx11 * cosf(sinshift + fy1*float(oy1) + fx1*float(ox1)) +
				   sx12 * cosf(sinshift + fy1*float(oy2) + fx1*float(ox2)) +
				   sx20 * cosf(sinshift + fy1*float(oy0) + fx1*float(ox0)) +
				   sx21 * cosf(sinshift + fy1*float(oy1) + fx1*float(ox1)) +
				   sx22 * cosf(sinshift + fy1*float(oy2) + fx1*float(ox2)) ;
  corr[corr_coo2D((ty+1),tx+2)] += sx00 * cosf(sinshift + fy1*float(oy0) + fx2*float(ox0)) +
				   sx01 * cosf(sinshift + fy1*float(oy1) + fx2*float(ox1)) +
				   sx02 * cosf(sinshift + fy1*float(oy2) + fx2*float(ox2)) +
				   sx10 * cosf(sinshift + fy1*float(oy0) + fx2*float(ox0)) +
				   sx11 * cosf(sinshift + fy1*float(oy1) + fx2*float(ox1)) +
				   sx12 * cosf(sinshift + fy1*float(oy2) + fx2*float(ox2)) +
				   sx20 * cosf(sinshift + fy1*float(oy0) + fx2*float(ox0)) +
				   sx21 * cosf(sinshift + fy1*float(oy1) + fx2*float(ox1)) +
				   sx22 * cosf(sinshift + fy1*float(oy2) + fx2*float(ox2)) ;
  corr[corr_coo2D((ty+2),tx)] += sx00 * cosf(sinshift + fy2*float(oy0) + fx0*float(ox0)) +
				 sx01 * cosf(sinshift + fy2*float(oy1) + fx0*float(ox1)) +
				 sx02 * cosf(sinshift + fy2*float(oy2) + fx0*float(ox2)) +
				 sx10 * cosf(sinshift + fy2*float(oy0) + fx0*float(ox0)) +
				 sx11 * cosf(sinshift + fy2*float(oy1) + fx0*float(ox1)) +
				 sx12 * cosf(sinshift + fy2*float(oy2) + fx0*float(ox2)) +
				 sx20 * cosf(sinshift + fy2*float(oy0) + fx0*float(ox0)) +
				 sx21 * cosf(sinshift + fy2*float(oy1) + fx0*float(ox1)) +
				 sx22 * cosf(sinshift + fy2*float(oy2) + fx0*float(ox2)) ;
  corr[corr_coo2D((ty+2),tx+1)] += sx00 * cosf(sinshift + fy2*float(oy0) + fx1*float(ox0)) +
				   sx01 * cosf(sinshift + fy2*float(oy1) + fx1*float(ox1)) +
				   sx02 * cosf(sinshift + fy2*float(oy2) + fx1*float(ox2)) +
				   sx10 * cosf(sinshift + fy2*float(oy0) + fx1*float(ox0)) +
				   sx11 * cosf(sinshift + fy2*float(oy1) + fx1*float(ox1)) +
				   sx12 * cosf(sinshift + fy2*float(oy2) + fx1*float(ox2)) +
				   sx20 * cosf(sinshift + fy2*float(oy0) + fx1*float(ox0)) +
				   sx21 * cosf(sinshift + fy2*float(oy1) + fx1*float(ox1)) +
				   sx22 * cosf(sinshift + fy2*float(oy2) + fx1*float(ox2)) ;
  corr[corr_coo2D((ty+2),tx+2)] += sx00 * cosf(sinshift + fy2*float(oy0) + fx2*float(ox0)) +
				   sx01 * cosf(sinshift + fy2*float(oy1) + fx2*float(ox1)) +
				   sx02 * cosf(sinshift + fy2*float(oy2) + fx2*float(ox2)) +
				   sx10 * cosf(sinshift + fy2*float(oy0) + fx2*float(ox0)) +
				   sx11 * cosf(sinshift + fy2*float(oy1) + fx2*float(ox1)) +
				   sx12 * cosf(sinshift + fy2*float(oy2) + fx2*float(ox2)) +
				   sx20 * cosf(sinshift + fy2*float(oy0) + fx2*float(ox0)) +
				   sx21 * cosf(sinshift + fy2*float(oy1) + fx2*float(ox1)) +
				   sx22 * cosf(sinshift + fy2*float(oy2) + fx2*float(ox2)) ;
  __syncthreads();
}

__global__ void sumcorrTRI(double *DSum_corr, const float *corr, int *DTo){
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ty_pt =(DTo[blockIdx.x / corr_BN]) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  int tx_1 = tx%(corr_SpinSize/2);
  int tx_2 = tx%(corr_SpinSize/2);
  //calculate all the final position first
  DSum_corr[corr_coo2D(ty_pt,tx)] += sqrt(corr[corr_coo2D(ty,tx_2)]*corr[corr_coo2D(ty,tx_2)]+corr[corr_coo2D(ty,tx_1)]*corr[corr_coo2D(ty,tx_1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D(ty_pt,tx+1)] += sqrt(corr[corr_coo2D(ty,tx_2+1)]*corr[corr_coo2D(ty,tx_2+1)]+corr[corr_coo2D(ty,tx_1+1)]*corr[corr_coo2D(ty,tx_1+1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D(ty_pt,tx+2)] += sqrt(corr[corr_coo2D(ty,tx_2+2)]*corr[corr_coo2D(ty,tx_2+2)]+corr[corr_coo2D(ty,tx_1+2)]*corr[corr_coo2D(ty,tx_1+2)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 1),tx)] += sqrt(corr[corr_coo2D((ty + 1),tx_2)]*corr[corr_coo2D((ty + 1),tx_2)]+corr[corr_coo2D((ty + 1),tx_1)]*corr[corr_coo2D((ty + 1),tx_1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 1),tx+1)] += sqrt(corr[corr_coo2D((ty + 1),tx_2+1)]*corr[corr_coo2D((ty + 1),tx_2+1)]+corr[corr_coo2D((ty + 1),tx_1+1)]*corr[corr_coo2D((ty + 1),tx_1+1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 1),tx+2)] += sqrt(corr[corr_coo2D((ty + 1),tx_2+2)]*corr[corr_coo2D((ty + 1),tx_2+2)]+corr[corr_coo2D((ty + 1),tx_1+2)]*corr[corr_coo2D((ty + 1),tx_1+2)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 2),tx)] += sqrt(corr[corr_coo2D((ty + 2),tx_2)]*corr[corr_coo2D((ty + 2),tx_2)]+corr[corr_coo2D((ty + 2),tx_1)]*corr[corr_coo2D((ty + 2),tx_1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 2),tx+1)] += sqrt(corr[corr_coo2D((ty + 2),tx_2+1)]*corr[corr_coo2D((ty + 2),tx_2+1)]+corr[corr_coo2D((ty + 2),tx_1+1)]*corr[corr_coo2D((ty + 2),tx_1+1)])/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty_pt + 2),tx+2)] += sqrt(corr[corr_coo2D((ty + 2),tx_2+2)]*corr[corr_coo2D((ty + 2),tx_2+2)]+corr[corr_coo2D((ty + 2),tx_1+2)]*corr[corr_coo2D((ty + 2),tx_1+2)])/corr_SpinSize/corr_SpinSize;
  __syncthreads();
}

__global__ void getcorrTRI_z(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / corr_BN) * corr_SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sz00, sz01, sz02,
        sz10, sz11, sz12,
        sz20, sz21, sz22;
  int fx0, fy0,
      fx1, fy1,
      fx2, fy2,
      fx3, fy3,//from o to f
      fx4, fy4;
  //calculate all the final position first

  fx0 = (tx + original_i) % corr_SpinSize;
  fx1 = (tx + original_i + 1) % corr_SpinSize;
  fx2 = (tx + original_i + 2) % corr_SpinSize;
  fx3 = (tx + original_i + 3) % corr_SpinSize;
  fx4 = (tx + original_i + 4) % corr_SpinSize;

  if((ty % corr_SpinSize + original_j) >= corr_SpinSize)	fy0 = ty + original_j - corr_SpinSize;
  else  fy0 = ty + original_j;
  if((ty % corr_SpinSize + original_j + 1) >= corr_SpinSize)	fy1 = ty + original_j + 1 - corr_SpinSize;
  else  fy1 = ty + original_j + 1;
  if((ty % corr_SpinSize + original_j + 2) >= corr_SpinSize)	fy2 = ty + original_j + 2 - corr_SpinSize;
  else  fy2 = ty + original_j + 2;
  if((ty % corr_SpinSize + original_j + 3) >= corr_SpinSize)	fy3 = ty + original_j + 3 - corr_SpinSize;
  else  fy3 = ty + original_j + 3;
  if((ty % corr_SpinSize + original_j + 4) >= corr_SpinSize)	fy4 = ty + original_j + 4 - corr_SpinSize;
  else  fy4 = ty + original_j + 4;

  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sz00 = confz[corr_coo2D(oy,ox)];
  sz01 = confz[corr_coo2D(oy,ox+1)];
  sz02 = confz[corr_coo2D(oy,ox+2)];
  sz10 = confz[corr_coo2D(oy+1,ox)];
  sz11 = confz[corr_coo2D(oy+1,ox+1)];
  sz12 = confz[corr_coo2D(oy+1,ox+2)];
  sz20 = confz[corr_coo2D(oy+2,ox)];
  sz21 = confz[corr_coo2D(oy+2,ox+1)];
  sz22 = confz[corr_coo2D(oy+2,ox+2)];
  corr[corr_coo2D(ty,tx)] += sz00 * confz[corr_coo2D( fy0,fx0)] +
                        sz01 * confz[corr_coo2D( fy0,fx1)] +
                        sz02 * confz[corr_coo2D( fy0,fx2)] +
                        sz10 * confz[corr_coo2D( fy1,fx0)] +
                        sz11 * confz[corr_coo2D( fy1,fx1)] +
                        sz12 * confz[corr_coo2D( fy1,fx2)] +
                        sz20 * confz[corr_coo2D( fy2,fx0)] +
                        sz21 * confz[corr_coo2D( fy2,fx1)] +
                        sz22 * confz[corr_coo2D( fy2,fx2)];
  corr[corr_coo2D(ty,tx+1)] += sz00 * confz[corr_coo2D( fy0,fx1)] +
                          sz01 * confz[corr_coo2D( fy0,fx2)] +
                          sz02 * confz[corr_coo2D( fy0,fx3)] +
                          sz10 * confz[corr_coo2D( fy1,fx1)] +
                          sz11 * confz[corr_coo2D( fy1,fx2)] +
                          sz12 * confz[corr_coo2D( fy1,fx3)] +
                          sz20 * confz[corr_coo2D( fy2,fx1)] +
                          sz21 * confz[corr_coo2D( fy2,fx2)] +
                          sz22 * confz[corr_coo2D( fy2,fx3)];
  corr[corr_coo2D(ty,tx+2)] += sz00 * confz[corr_coo2D( fy0,fx2)] +
                          sz01 * confz[corr_coo2D( fy0,fx3)] +
                          sz02 * confz[corr_coo2D( fy0,fx4)] +
                          sz10 * confz[corr_coo2D( fy1,fx2)] +
                          sz11 * confz[corr_coo2D( fy1,fx3)] +
                          sz12 * confz[corr_coo2D( fy1,fx4)] +
                          sz20 * confz[corr_coo2D( fy2,fx2)] +
                          sz21 * confz[corr_coo2D( fy2,fx3)] +
                          sz22 * confz[corr_coo2D( fy2,fx4)];
  corr[corr_coo2D((ty+1),tx)] += sz00 * confz[corr_coo2D( fy1,fx0)] +
                            sz01 * confz[corr_coo2D( fy1,fx1)] +
                            sz02 * confz[corr_coo2D( fy1,fx2)] +
                            sz10 * confz[corr_coo2D( fy2,fx0)] +
                            sz11 * confz[corr_coo2D( fy2,fx1)] +
                            sz12 * confz[corr_coo2D( fy2,fx2)] +
                            sz20 * confz[corr_coo2D( fy3,fx0)] +
                            sz21 * confz[corr_coo2D( fy3,fx1)] +
                            sz22 * confz[corr_coo2D( fy3,fx2)];
  corr[corr_coo2D((ty+1),tx+1)] += sz00 * confz[corr_coo2D( fy1,fx1)] +
                              sz01 * confz[corr_coo2D( fy1,fx2)] +
                              sz02 * confz[corr_coo2D( fy1,fx3)] +
                              sz10 * confz[corr_coo2D( fy2,fx1)] +
                              sz11 * confz[corr_coo2D( fy2,fx2)] +
                              sz12 * confz[corr_coo2D( fy2,fx3)] +
                              sz20 * confz[corr_coo2D( fy3,fx1)] +
                              sz21 * confz[corr_coo2D( fy3,fx2)] +
                              sz22 * confz[corr_coo2D( fy3,fx3)];
  corr[corr_coo2D((ty+1),tx+2)] += sz00 * confz[corr_coo2D( fy1,fx2)] +
                              sz01 * confz[corr_coo2D( fy1,fx3)] +
                              sz02 * confz[corr_coo2D( fy1,fx4)] +
                              sz10 * confz[corr_coo2D( fy2,fx2)] +
                              sz11 * confz[corr_coo2D( fy2,fx3)] +
                              sz12 * confz[corr_coo2D( fy2,fx4)] +
                              sz20 * confz[corr_coo2D( fy3,fx2)] +
                              sz21 * confz[corr_coo2D( fy3,fx3)] +
                              sz22 * confz[corr_coo2D( fy3,fx4)];
  corr[corr_coo2D((ty+2),tx)] += sz00 * confz[corr_coo2D( fy2,fx0)] +
                            sz01 * confz[corr_coo2D( fy2,fx1)] +
                            sz02 * confz[corr_coo2D( fy2,fx2)] +
                            sz10 * confz[corr_coo2D( fy3,fx0)] +
                            sz11 * confz[corr_coo2D( fy3,fx1)] +
                            sz12 * confz[corr_coo2D( fy3,fx2)] +
                            sz20 * confz[corr_coo2D( fy4,fx0)] +
                            sz21 * confz[corr_coo2D( fy4,fx1)] +
                            sz22 * confz[corr_coo2D( fy4,fx2)];
  corr[corr_coo2D((ty+2),tx+1)] += sz00 * confz[corr_coo2D( fy2,fx1)] +
                              sz01 * confz[corr_coo2D( fy2,fx2)] +
                              sz02 * confz[corr_coo2D( fy2,fx3)] +
                              sz10 * confz[corr_coo2D( fy3,fx1)] +
                              sz11 * confz[corr_coo2D( fy3,fx2)] +
                              sz12 * confz[corr_coo2D( fy3,fx3)] +
                              sz20 * confz[corr_coo2D( fy4,fx1)] +
                              sz21 * confz[corr_coo2D( fy4,fx2)] +
                              sz22 * confz[corr_coo2D( fy4,fx3)];
  corr[corr_coo2D((ty+2),tx+2)] += sz00 * confz[corr_coo2D( fy2,fx2)] +
                              sz01 * confz[corr_coo2D( fy2,fx3)] +
                              sz02 * confz[corr_coo2D( fy2,fx4)] +
                              sz10 * confz[corr_coo2D( fy3,fx2)] +
                              sz11 * confz[corr_coo2D( fy3,fx3)] +
                              sz12 * confz[corr_coo2D( fy3,fx4)] +
                              sz20 * confz[corr_coo2D( fy4,fx2)] +
                              sz21 * confz[corr_coo2D( fy4,fx3)] +
                              sz22 * confz[corr_coo2D( fy4,fx4)];
  __syncthreads();
}

__global__ void avgcorrTRI(double *DSum_corr, double N_corr){
  /*****************************************************************
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  //calculate all the final position first
  DSum_corr[corr_coo2D(ty,tx)] = DSum_corr[corr_coo2D(ty,tx)]/N_corr;
  DSum_corr[corr_coo2D(ty,tx+1)] = DSum_corr[corr_coo2D(ty,tx+1)]/N_corr;
  DSum_corr[corr_coo2D(ty,tx+2)] = DSum_corr[corr_coo2D(ty,tx+2)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx)] = DSum_corr[corr_coo2D((ty + 1),tx)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx+1)] = DSum_corr[corr_coo2D((ty + 1),tx+1)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx+2)] = DSum_corr[corr_coo2D((ty + 1),tx+2)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx)] = DSum_corr[corr_coo2D((ty + 2),tx)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx+1)] = DSum_corr[corr_coo2D((ty + 2),tx+1)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx+2)] = DSum_corr[corr_coo2D((ty + 2),tx+2)]/N_corr;
  __syncthreads();
}
#endif
