#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__constant__ unsigned int corr_SpinSize;
__constant__ unsigned int corr_SpinSize_z;
__constant__ unsigned int corr_BlockSize_x;
__constant__ unsigned int corr_BlockSize_y;
__constant__ unsigned int corr_GridSize_x;
__constant__ unsigned int corr_GridSize_y;
__constant__ unsigned int corr_N;
__constant__ unsigned int corr_Nplane;
__constant__ unsigned int corr_TN;
__constant__ unsigned int corr_BN;
void move_params_device_corr(){
  hipMemcpyToSymbol(HIP_SYMBOL(corr_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_N ), &H_N , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_Nplane ), &H_Nplane , sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(corr_BN), &H_BN, sizeof(unsigned int));
}
__global__ void getcorrTRI(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / corr_BN) * corr_SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sx00, sy00, sz00, sx01, sy01, sz01, sx02, sy02, sz02,
        sx10, sy10, sz10, sx11, sy11, sz11, sx12, sy12, sz12,
        sx20, sy20, sz20, sx21, sy21, sz21, sx22, sy22, sz22;
  int fx0, fy0,
      fx1, fy1,
      fx2, fy2,
      fx3, fy3,//from o to f
      fx4, fy4;
  //calculate all the final position first

  fx0 = (tx + original_i) % corr_SpinSize;
  fx1 = (tx + original_i + 1) % corr_SpinSize;
  fx2 = (tx + original_i + 2) % corr_SpinSize;
  fx3 = (tx + original_i + 3) % corr_SpinSize;
  fx4 = (tx + original_i + 4) % corr_SpinSize;

  if((ty % corr_SpinSize + original_j) >= corr_SpinSize)	fy0 = ty + original_j - corr_SpinSize;
  else  fy0 = ty + original_j;
  if((ty % corr_SpinSize + original_j + 1) >= corr_SpinSize)	fy1 = ty + original_j + 1 - corr_SpinSize;
  else  fy1 = ty + original_j + 1;
  if((ty % corr_SpinSize + original_j + 2) >= corr_SpinSize)	fy2 = ty + original_j + 2 - corr_SpinSize;
  else  fy2 = ty + original_j + 2;
  if((ty % corr_SpinSize + original_j + 3) >= corr_SpinSize)	fy3 = ty + original_j + 3 - corr_SpinSize;
  else  fy3 = ty + original_j + 3;
  if((ty % corr_SpinSize + original_j + 4) >= corr_SpinSize)	fy4 = ty + original_j + 4 - corr_SpinSize;
  else  fy4 = ty + original_j + 4;

  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[corr_coo2D(oy,ox)];
  sy00 = confy[corr_coo2D(oy,ox)];
  sz00 = confz[corr_coo2D(oy,ox)];
  sx01 = confx[corr_coo2D(oy,ox+1)];
  sy01 = confy[corr_coo2D(oy,ox+1)];
  sz01 = confz[corr_coo2D(oy,ox+1)];
  sx02 = confx[corr_coo2D(oy,ox+2)];
  sy02 = confy[corr_coo2D(oy,ox+2)];
  sz02 = confz[corr_coo2D(oy,ox+2)];
  sx10 = confx[corr_coo2D(oy+1,ox)];
  sy10 = confy[corr_coo2D(oy+1,ox)];
  sz10 = confz[corr_coo2D(oy+1,ox)];
  sx11 = confx[corr_coo2D(oy+1,ox+1)];
  sy11 = confy[corr_coo2D(oy+1,ox+1)];
  sz11 = confz[corr_coo2D(oy+1,ox+1)];
  sx12 = confx[corr_coo2D(oy+1,ox+2)];
  sy12 = confy[corr_coo2D(oy+1,ox+2)];
  sz12 = confz[corr_coo2D(oy+1,ox+2)];
  sx20 = confx[corr_coo2D(oy+2,ox)];
  sy20 = confy[corr_coo2D(oy+2,ox)];
  sz20 = confz[corr_coo2D(oy+2,ox)];
  sx21 = confx[corr_coo2D(oy+2,ox+1)];
  sy21 = confy[corr_coo2D(oy+2,ox+1)];
  sz21 = confz[corr_coo2D(oy+2,ox+1)];
  sx22 = confx[corr_coo2D(oy+2,ox+2)];
  sy22 = confy[corr_coo2D(oy+2,ox+2)];
  sz22 = confz[corr_coo2D(oy+2,ox+2)];
  corr[corr_coo2D(ty,tx)] += sx00 * confx[corr_coo2D( fy0,fx0)] + sy00 * confy[corr_coo2D( fy0,fx0)] + sz00 * confz[corr_coo2D( fy0,fx0)] +
                        sx01 * confx[corr_coo2D( fy0,fx1)] + sy01 * confy[corr_coo2D( fy0,fx1)] + sz01 * confz[corr_coo2D( fy0,fx1)] +
                        sx02 * confx[corr_coo2D( fy0,fx2)] + sy02 * confy[corr_coo2D( fy0,fx2)] + sz02 * confz[corr_coo2D( fy0,fx2)] +
                        sx10 * confx[corr_coo2D( fy1,fx0)] + sy10 * confy[corr_coo2D( fy1,fx0)] + sz10 * confz[corr_coo2D( fy1,fx0)] +
                        sx11 * confx[corr_coo2D( fy1,fx1)] + sy11 * confy[corr_coo2D( fy1,fx1)] + sz11 * confz[corr_coo2D( fy1,fx1)] +
                        sx12 * confx[corr_coo2D( fy1,fx2)] + sy12 * confy[corr_coo2D( fy1,fx2)] + sz12 * confz[corr_coo2D( fy1,fx2)] +
                        sx20 * confx[corr_coo2D( fy2,fx0)] + sy20 * confy[corr_coo2D( fy2,fx0)] + sz20 * confz[corr_coo2D( fy2,fx0)] +
                        sx21 * confx[corr_coo2D( fy2,fx1)] + sy21 * confy[corr_coo2D( fy2,fx1)] + sz21 * confz[corr_coo2D( fy2,fx1)] +
                        sx22 * confx[corr_coo2D( fy2,fx2)] + sy22 * confy[corr_coo2D( fy2,fx2)] + sz22 * confz[corr_coo2D( fy2,fx2)];
  corr[corr_coo2D(ty,tx+1)] += sx00 * confx[corr_coo2D( fy0,fx1)] + sy00 * confy[corr_coo2D( fy0,fx1)] + sz00 * confz[corr_coo2D( fy0,fx1)] +
                          sx01 * confx[corr_coo2D( fy0,fx2)] + sy01 * confy[corr_coo2D( fy0,fx2)] + sz01 * confz[corr_coo2D( fy0,fx2)] +
                          sx02 * confx[corr_coo2D( fy0,fx3)] + sy02 * confy[corr_coo2D( fy0,fx3)] + sz02 * confz[corr_coo2D( fy0,fx3)] +
                          sx10 * confx[corr_coo2D( fy1,fx1)] + sy10 * confy[corr_coo2D( fy1,fx1)] + sz10 * confz[corr_coo2D( fy1,fx1)] +
                          sx11 * confx[corr_coo2D( fy1,fx2)] + sy11 * confy[corr_coo2D( fy1,fx2)] + sz11 * confz[corr_coo2D( fy1,fx2)] +
                          sx12 * confx[corr_coo2D( fy1,fx3)] + sy12 * confy[corr_coo2D( fy1,fx3)] + sz12 * confz[corr_coo2D( fy1,fx3)] +
                          sx20 * confx[corr_coo2D( fy2,fx1)] + sy20 * confy[corr_coo2D( fy2,fx1)] + sz20 * confz[corr_coo2D( fy2,fx1)] +
                          sx21 * confx[corr_coo2D( fy2,fx2)] + sy21 * confy[corr_coo2D( fy2,fx2)] + sz21 * confz[corr_coo2D( fy2,fx2)] +
                          sx22 * confx[corr_coo2D( fy2,fx3)] + sy22 * confy[corr_coo2D( fy2,fx3)] + sz22 * confz[corr_coo2D( fy2,fx3)];
  corr[corr_coo2D(ty,tx+2)] += sx00 * confx[corr_coo2D( fy0,fx2)] + sy00 * confy[corr_coo2D( fy0,fx2)] + sz00 * confz[corr_coo2D( fy0,fx2)] +
                          sx01 * confx[corr_coo2D( fy0,fx3)] + sy01 * confy[corr_coo2D( fy0,fx3)] + sz01 * confz[corr_coo2D( fy0,fx3)] +
                          sx02 * confx[corr_coo2D( fy0,fx4)] + sy02 * confy[corr_coo2D( fy0,fx4)] + sz02 * confz[corr_coo2D( fy0,fx4)] +
                          sx10 * confx[corr_coo2D( fy1,fx2)] + sy10 * confy[corr_coo2D( fy1,fx2)] + sz10 * confz[corr_coo2D( fy1,fx2)] +
                          sx11 * confx[corr_coo2D( fy1,fx3)] + sy11 * confy[corr_coo2D( fy1,fx3)] + sz11 * confz[corr_coo2D( fy1,fx3)] +
                          sx12 * confx[corr_coo2D( fy1,fx4)] + sy12 * confy[corr_coo2D( fy1,fx4)] + sz12 * confz[corr_coo2D( fy1,fx4)] +
                          sx20 * confx[corr_coo2D( fy2,fx2)] + sy20 * confy[corr_coo2D( fy2,fx2)] + sz20 * confz[corr_coo2D( fy2,fx2)] +
                          sx21 * confx[corr_coo2D( fy2,fx3)] + sy21 * confy[corr_coo2D( fy2,fx3)] + sz21 * confz[corr_coo2D( fy2,fx3)] +
                          sx22 * confx[corr_coo2D( fy2,fx4)] + sy22 * confy[corr_coo2D( fy2,fx4)] + sz22 * confz[corr_coo2D( fy2,fx4)];
  corr[corr_coo2D((ty+1),tx)] += sx00 * confx[corr_coo2D( fy1,fx0)] + sy00 * confy[corr_coo2D( fy1,fx0)] + sz00 * confz[corr_coo2D( fy1,fx0)] +
                            sx01 * confx[corr_coo2D( fy1,fx1)] + sy01 * confy[corr_coo2D( fy1,fx1)] + sz01 * confz[corr_coo2D( fy1,fx1)] +
                            sx02 * confx[corr_coo2D( fy1,fx2)] + sy02 * confy[corr_coo2D( fy1,fx2)] + sz02 * confz[corr_coo2D( fy1,fx2)] +
                            sx10 * confx[corr_coo2D( fy2,fx0)] + sy10 * confy[corr_coo2D( fy2,fx0)] + sz10 * confz[corr_coo2D( fy2,fx0)] +
                            sx11 * confx[corr_coo2D( fy2,fx1)] + sy11 * confy[corr_coo2D( fy2,fx1)] + sz11 * confz[corr_coo2D( fy2,fx1)] +
                            sx12 * confx[corr_coo2D( fy2,fx2)] + sy12 * confy[corr_coo2D( fy2,fx2)] + sz12 * confz[corr_coo2D( fy2,fx2)] +
                            sx20 * confx[corr_coo2D( fy3,fx0)] + sy20 * confy[corr_coo2D( fy3,fx0)] + sz20 * confz[corr_coo2D( fy3,fx0)] +
                            sx21 * confx[corr_coo2D( fy3,fx1)] + sy21 * confy[corr_coo2D( fy3,fx1)] + sz21 * confz[corr_coo2D( fy3,fx1)] +
                            sx22 * confx[corr_coo2D( fy3,fx2)] + sy22 * confy[corr_coo2D( fy3,fx2)] + sz22 * confz[corr_coo2D( fy3,fx2)];
  corr[corr_coo2D((ty+1),tx+1)] += sx00 * confx[corr_coo2D( fy1,fx1)] + sy00 * confy[corr_coo2D( fy1,fx1)] + sz00 * confz[corr_coo2D( fy1,fx1)] +
                              sx01 * confx[corr_coo2D( fy1,fx2)] + sy01 * confy[corr_coo2D( fy1,fx2)] + sz01 * confz[corr_coo2D( fy1,fx2)] +
                              sx02 * confx[corr_coo2D( fy1,fx3)] + sy02 * confy[corr_coo2D( fy1,fx3)] + sz02 * confz[corr_coo2D( fy1,fx3)] +
                              sx10 * confx[corr_coo2D( fy2,fx1)] + sy10 * confy[corr_coo2D( fy2,fx1)] + sz10 * confz[corr_coo2D( fy2,fx1)] +
                              sx11 * confx[corr_coo2D( fy2,fx2)] + sy11 * confy[corr_coo2D( fy2,fx2)] + sz11 * confz[corr_coo2D( fy2,fx2)] +
                              sx12 * confx[corr_coo2D( fy2,fx3)] + sy12 * confy[corr_coo2D( fy2,fx3)] + sz12 * confz[corr_coo2D( fy2,fx3)] +
                              sx20 * confx[corr_coo2D( fy3,fx1)] + sy20 * confy[corr_coo2D( fy3,fx1)] + sz20 * confz[corr_coo2D( fy3,fx1)] +
                              sx21 * confx[corr_coo2D( fy3,fx2)] + sy21 * confy[corr_coo2D( fy3,fx2)] + sz21 * confz[corr_coo2D( fy3,fx2)] +
                              sx22 * confx[corr_coo2D( fy3,fx3)] + sy22 * confy[corr_coo2D( fy3,fx3)] + sz22 * confz[corr_coo2D( fy3,fx3)];
  corr[corr_coo2D((ty+1),tx+2)] += sx00 * confx[corr_coo2D( fy1,fx2)] + sy00 * confy[corr_coo2D( fy1,fx2)] + sz00 * confz[corr_coo2D( fy1,fx2)] +
                              sx01 * confx[corr_coo2D( fy1,fx3)] + sy01 * confy[corr_coo2D( fy1,fx3)] + sz01 * confz[corr_coo2D( fy1,fx3)] +
                              sx02 * confx[corr_coo2D( fy1,fx4)] + sy02 * confy[corr_coo2D( fy1,fx4)] + sz02 * confz[corr_coo2D( fy1,fx4)] +
                              sx10 * confx[corr_coo2D( fy2,fx2)] + sy10 * confy[corr_coo2D( fy2,fx2)] + sz10 * confz[corr_coo2D( fy2,fx2)] +
                              sx11 * confx[corr_coo2D( fy2,fx3)] + sy11 * confy[corr_coo2D( fy2,fx3)] + sz11 * confz[corr_coo2D( fy2,fx3)] +
                              sx12 * confx[corr_coo2D( fy2,fx4)] + sy12 * confy[corr_coo2D( fy2,fx4)] + sz12 * confz[corr_coo2D( fy2,fx4)] +
                              sx20 * confx[corr_coo2D( fy3,fx2)] + sy20 * confy[corr_coo2D( fy3,fx2)] + sz20 * confz[corr_coo2D( fy3,fx2)] +
                              sx21 * confx[corr_coo2D( fy3,fx3)] + sy21 * confy[corr_coo2D( fy3,fx3)] + sz21 * confz[corr_coo2D( fy3,fx3)] +
                              sx22 * confx[corr_coo2D( fy3,fx4)] + sy22 * confy[corr_coo2D( fy3,fx4)] + sz22 * confz[corr_coo2D( fy3,fx4)];
  corr[corr_coo2D((ty+2),tx)] += sx00 * confx[corr_coo2D( fy2,fx0)] + sy00 * confy[corr_coo2D( fy2,fx0)] + sz00 * confz[corr_coo2D( fy2,fx0)] +
                            sx01 * confx[corr_coo2D( fy2,fx1)] + sy01 * confy[corr_coo2D( fy2,fx1)] + sz01 * confz[corr_coo2D( fy2,fx1)] +
                            sx02 * confx[corr_coo2D( fy2,fx2)] + sy02 * confy[corr_coo2D( fy2,fx2)] + sz02 * confz[corr_coo2D( fy2,fx2)] +
                            sx10 * confx[corr_coo2D( fy3,fx0)] + sy10 * confy[corr_coo2D( fy3,fx0)] + sz10 * confz[corr_coo2D( fy3,fx0)] +
                            sx11 * confx[corr_coo2D( fy3,fx1)] + sy11 * confy[corr_coo2D( fy3,fx1)] + sz11 * confz[corr_coo2D( fy3,fx1)] +
                            sx12 * confx[corr_coo2D( fy3,fx2)] + sy12 * confy[corr_coo2D( fy3,fx2)] + sz12 * confz[corr_coo2D( fy3,fx2)] +
                            sx20 * confx[corr_coo2D( fy4,fx0)] + sy20 * confy[corr_coo2D( fy4,fx0)] + sz20 * confz[corr_coo2D( fy4,fx0)] +
                            sx21 * confx[corr_coo2D( fy4,fx1)] + sy21 * confy[corr_coo2D( fy4,fx1)] + sz21 * confz[corr_coo2D( fy4,fx1)] +
                            sx22 * confx[corr_coo2D( fy4,fx2)] + sy22 * confy[corr_coo2D( fy4,fx2)] + sz22 * confz[corr_coo2D( fy4,fx2)];
  corr[corr_coo2D((ty+2),tx+1)] += sx00 * confx[corr_coo2D( fy2,fx1)] + sy00 * confy[corr_coo2D( fy2,fx1)] + sz00 * confz[corr_coo2D( fy2,fx1)] +
                              sx01 * confx[corr_coo2D( fy2,fx2)] + sy01 * confy[corr_coo2D( fy2,fx2)] + sz01 * confz[corr_coo2D( fy2,fx2)] +
                              sx02 * confx[corr_coo2D( fy2,fx3)] + sy02 * confy[corr_coo2D( fy2,fx3)] + sz02 * confz[corr_coo2D( fy2,fx3)] +
                              sx10 * confx[corr_coo2D( fy3,fx1)] + sy10 * confy[corr_coo2D( fy3,fx1)] + sz10 * confz[corr_coo2D( fy3,fx1)] +
                              sx11 * confx[corr_coo2D( fy3,fx2)] + sy11 * confy[corr_coo2D( fy3,fx2)] + sz11 * confz[corr_coo2D( fy3,fx2)] +
                              sx12 * confx[corr_coo2D( fy3,fx3)] + sy12 * confy[corr_coo2D( fy3,fx3)] + sz12 * confz[corr_coo2D( fy3,fx3)] +
                              sx20 * confx[corr_coo2D( fy4,fx1)] + sy20 * confy[corr_coo2D( fy4,fx1)] + sz20 * confz[corr_coo2D( fy4,fx1)] +
                              sx21 * confx[corr_coo2D( fy4,fx2)] + sy21 * confy[corr_coo2D( fy4,fx2)] + sz21 * confz[corr_coo2D( fy4,fx2)] +
                              sx22 * confx[corr_coo2D( fy4,fx3)] + sy22 * confy[corr_coo2D( fy4,fx3)] + sz22 * confz[corr_coo2D( fy4,fx3)];
  corr[corr_coo2D((ty+2),tx+2)] += sx00 * confx[corr_coo2D( fy2,fx2)] + sy00 * confy[corr_coo2D( fy2,fx2)] + sz00 * confz[corr_coo2D( fy2,fx2)] +
                              sx01 * confx[corr_coo2D( fy2,fx3)] + sy01 * confy[corr_coo2D( fy2,fx3)] + sz01 * confz[corr_coo2D( fy2,fx3)] +
                              sx02 * confx[corr_coo2D( fy2,fx4)] + sy02 * confy[corr_coo2D( fy2,fx4)] + sz02 * confz[corr_coo2D( fy2,fx4)] +
                              sx10 * confx[corr_coo2D( fy3,fx2)] + sy10 * confy[corr_coo2D( fy3,fx2)] + sz10 * confz[corr_coo2D( fy3,fx2)] +
                              sx11 * confx[corr_coo2D( fy3,fx3)] + sy11 * confy[corr_coo2D( fy3,fx3)] + sz11 * confz[corr_coo2D( fy3,fx3)] +
                              sx12 * confx[corr_coo2D( fy3,fx4)] + sy12 * confy[corr_coo2D( fy3,fx4)] + sz12 * confz[corr_coo2D( fy3,fx4)] +
                              sx20 * confx[corr_coo2D( fy4,fx2)] + sy20 * confy[corr_coo2D( fy4,fx2)] + sz20 * confz[corr_coo2D( fy4,fx2)] +
                              sx21 * confx[corr_coo2D( fy4,fx3)] + sy21 * confy[corr_coo2D( fy4,fx3)] + sz21 * confz[corr_coo2D( fy4,fx3)] +
                              sx22 * confx[corr_coo2D( fy4,fx4)] + sy22 * confy[corr_coo2D( fy4,fx4)] + sz22 * confz[corr_coo2D( fy4,fx4)];
  __syncthreads();
}

__global__ void sumcorrTRI(double *DSum_corr, const float *corr, int *DTo){
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
	//const int ty_pt =(DTo[blockIdx.x / corr_BN]) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  //calculate all the final position first
  DSum_corr[corr_coo2D(ty,tx)] += corr[corr_coo2D(ty,tx)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D(ty,tx+1)] += corr[corr_coo2D(ty,tx+1)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D(ty,tx+2)] += corr[corr_coo2D(ty,tx+2)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 1),tx)] += corr[corr_coo2D((ty + 1),tx)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 1),tx+1)] += corr[corr_coo2D((ty + 1),tx+1)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 1),tx+2)] += corr[corr_coo2D((ty + 1),tx+2)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 2),tx)] += corr[corr_coo2D((ty + 2),tx)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 2),tx+1)] += corr[corr_coo2D((ty + 2),tx+1)]/corr_SpinSize/corr_SpinSize;
  DSum_corr[corr_coo2D((ty + 2),tx+2)] += corr[corr_coo2D((ty + 2),tx+2)]/corr_SpinSize/corr_SpinSize;
  __syncthreads();
}

__global__ void getcorrTRI_z(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / corr_BN) * corr_SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sz00, sz01, sz02,
        sz10, sz11, sz12,
        sz20, sz21, sz22;
  int fx0, fy0,
      fx1, fy1,
      fx2, fy2,
      fx3, fy3,//from o to f
      fx4, fy4;
  //calculate all the final position first

  fx0 = (tx + original_i) % corr_SpinSize;
  fx1 = (tx + original_i + 1) % corr_SpinSize;
  fx2 = (tx + original_i + 2) % corr_SpinSize;
  fx3 = (tx + original_i + 3) % corr_SpinSize;
  fx4 = (tx + original_i + 4) % corr_SpinSize;

  if((ty % corr_SpinSize + original_j) >= corr_SpinSize)	fy0 = ty + original_j - corr_SpinSize;
  else  fy0 = ty + original_j;
  if((ty % corr_SpinSize + original_j + 1) >= corr_SpinSize)	fy1 = ty + original_j + 1 - corr_SpinSize;
  else  fy1 = ty + original_j + 1;
  if((ty % corr_SpinSize + original_j + 2) >= corr_SpinSize)	fy2 = ty + original_j + 2 - corr_SpinSize;
  else  fy2 = ty + original_j + 2;
  if((ty % corr_SpinSize + original_j + 3) >= corr_SpinSize)	fy3 = ty + original_j + 3 - corr_SpinSize;
  else  fy3 = ty + original_j + 3;
  if((ty % corr_SpinSize + original_j + 4) >= corr_SpinSize)	fy4 = ty + original_j + 4 - corr_SpinSize;
  else  fy4 = ty + original_j + 4;

  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sz00 = confz[corr_coo2D(oy,ox)];
  sz01 = confz[corr_coo2D(oy,ox+1)];
  sz02 = confz[corr_coo2D(oy,ox+2)];
  sz10 = confz[corr_coo2D(oy+1,ox)];
  sz11 = confz[corr_coo2D(oy+1,ox+1)];
  sz12 = confz[corr_coo2D(oy+1,ox+2)];
  sz20 = confz[corr_coo2D(oy+2,ox)];
  sz21 = confz[corr_coo2D(oy+2,ox+1)];
  sz22 = confz[corr_coo2D(oy+2,ox+2)];
  corr[corr_coo2D(ty,tx)] += sz00 * confz[corr_coo2D( fy0,fx0)] +
                        sz01 * confz[corr_coo2D( fy0,fx1)] +
                        sz02 * confz[corr_coo2D( fy0,fx2)] +
                        sz10 * confz[corr_coo2D( fy1,fx0)] +
                        sz11 * confz[corr_coo2D( fy1,fx1)] +
                        sz12 * confz[corr_coo2D( fy1,fx2)] +
                        sz20 * confz[corr_coo2D( fy2,fx0)] +
                        sz21 * confz[corr_coo2D( fy2,fx1)] +
                        sz22 * confz[corr_coo2D( fy2,fx2)];
  corr[corr_coo2D(ty,tx+1)] += sz00 * confz[corr_coo2D( fy0,fx1)] +
                          sz01 * confz[corr_coo2D( fy0,fx2)] +
                          sz02 * confz[corr_coo2D( fy0,fx3)] +
                          sz10 * confz[corr_coo2D( fy1,fx1)] +
                          sz11 * confz[corr_coo2D( fy1,fx2)] +
                          sz12 * confz[corr_coo2D( fy1,fx3)] +
                          sz20 * confz[corr_coo2D( fy2,fx1)] +
                          sz21 * confz[corr_coo2D( fy2,fx2)] +
                          sz22 * confz[corr_coo2D( fy2,fx3)];
  corr[corr_coo2D(ty,tx+2)] += sz00 * confz[corr_coo2D( fy0,fx2)] +
                          sz01 * confz[corr_coo2D( fy0,fx3)] +
                          sz02 * confz[corr_coo2D( fy0,fx4)] +
                          sz10 * confz[corr_coo2D( fy1,fx2)] +
                          sz11 * confz[corr_coo2D( fy1,fx3)] +
                          sz12 * confz[corr_coo2D( fy1,fx4)] +
                          sz20 * confz[corr_coo2D( fy2,fx2)] +
                          sz21 * confz[corr_coo2D( fy2,fx3)] +
                          sz22 * confz[corr_coo2D( fy2,fx4)];
  corr[corr_coo2D((ty+1),tx)] += sz00 * confz[corr_coo2D( fy1,fx0)] +
                            sz01 * confz[corr_coo2D( fy1,fx1)] +
                            sz02 * confz[corr_coo2D( fy1,fx2)] +
                            sz10 * confz[corr_coo2D( fy2,fx0)] +
                            sz11 * confz[corr_coo2D( fy2,fx1)] +
                            sz12 * confz[corr_coo2D( fy2,fx2)] +
                            sz20 * confz[corr_coo2D( fy3,fx0)] +
                            sz21 * confz[corr_coo2D( fy3,fx1)] +
                            sz22 * confz[corr_coo2D( fy3,fx2)];
  corr[corr_coo2D((ty+1),tx+1)] += sz00 * confz[corr_coo2D( fy1,fx1)] +
                              sz01 * confz[corr_coo2D( fy1,fx2)] +
                              sz02 * confz[corr_coo2D( fy1,fx3)] +
                              sz10 * confz[corr_coo2D( fy2,fx1)] +
                              sz11 * confz[corr_coo2D( fy2,fx2)] +
                              sz12 * confz[corr_coo2D( fy2,fx3)] +
                              sz20 * confz[corr_coo2D( fy3,fx1)] +
                              sz21 * confz[corr_coo2D( fy3,fx2)] +
                              sz22 * confz[corr_coo2D( fy3,fx3)];
  corr[corr_coo2D((ty+1),tx+2)] += sz00 * confz[corr_coo2D( fy1,fx2)] +
                              sz01 * confz[corr_coo2D( fy1,fx3)] +
                              sz02 * confz[corr_coo2D( fy1,fx4)] +
                              sz10 * confz[corr_coo2D( fy2,fx2)] +
                              sz11 * confz[corr_coo2D( fy2,fx3)] +
                              sz12 * confz[corr_coo2D( fy2,fx4)] +
                              sz20 * confz[corr_coo2D( fy3,fx2)] +
                              sz21 * confz[corr_coo2D( fy3,fx3)] +
                              sz22 * confz[corr_coo2D( fy3,fx4)];
  corr[corr_coo2D((ty+2),tx)] += sz00 * confz[corr_coo2D( fy2,fx0)] +
                            sz01 * confz[corr_coo2D( fy2,fx1)] +
                            sz02 * confz[corr_coo2D( fy2,fx2)] +
                            sz10 * confz[corr_coo2D( fy3,fx0)] +
                            sz11 * confz[corr_coo2D( fy3,fx1)] +
                            sz12 * confz[corr_coo2D( fy3,fx2)] +
                            sz20 * confz[corr_coo2D( fy4,fx0)] +
                            sz21 * confz[corr_coo2D( fy4,fx1)] +
                            sz22 * confz[corr_coo2D( fy4,fx2)];
  corr[corr_coo2D((ty+2),tx+1)] += sz00 * confz[corr_coo2D( fy2,fx1)] +
                              sz01 * confz[corr_coo2D( fy2,fx2)] +
                              sz02 * confz[corr_coo2D( fy2,fx3)] +
                              sz10 * confz[corr_coo2D( fy3,fx1)] +
                              sz11 * confz[corr_coo2D( fy3,fx2)] +
                              sz12 * confz[corr_coo2D( fy3,fx3)] +
                              sz20 * confz[corr_coo2D( fy4,fx1)] +
                              sz21 * confz[corr_coo2D( fy4,fx2)] +
                              sz22 * confz[corr_coo2D( fy4,fx3)];
  corr[corr_coo2D((ty+2),tx+2)] += sz00 * confz[corr_coo2D( fy2,fx2)] +
                              sz01 * confz[corr_coo2D( fy2,fx3)] +
                              sz02 * confz[corr_coo2D( fy2,fx4)] +
                              sz10 * confz[corr_coo2D( fy3,fx2)] +
                              sz11 * confz[corr_coo2D( fy3,fx3)] +
                              sz12 * confz[corr_coo2D( fy3,fx4)] +
                              sz20 * confz[corr_coo2D( fy4,fx2)] +
                              sz21 * confz[corr_coo2D( fy4,fx3)] +
                              sz22 * confz[corr_coo2D( fy4,fx4)];
  __syncthreads();
}

__global__ void avgcorrTRI(double *DSum_corr, double N_corr){
  /*****************************************************************
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (corr_BlockSize_x);
  const int y = (threadIdx.x / corr_BlockSize_x);
  const int tx = 3 * (((blockIdx.x % corr_BN) % corr_GridSize_x) * corr_BlockSize_x + x);
  const int ty =(blockIdx.x / corr_BN) * corr_SpinSize +  3 * ((((blockIdx.x % corr_BN) / corr_GridSize_x) % corr_GridSize_y) * corr_BlockSize_y + y);
  //calculate all the final position first
  DSum_corr[corr_coo2D(ty,tx)] = DSum_corr[corr_coo2D(ty,tx)]/N_corr;
  DSum_corr[corr_coo2D(ty,tx+1)] = DSum_corr[corr_coo2D(ty,tx+1)]/N_corr;
  DSum_corr[corr_coo2D(ty,tx+2)] = DSum_corr[corr_coo2D(ty,tx+2)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx)] = DSum_corr[corr_coo2D((ty + 1),tx)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx+1)] = DSum_corr[corr_coo2D((ty + 1),tx+1)]/N_corr;
  DSum_corr[corr_coo2D((ty + 1),tx+2)] = DSum_corr[corr_coo2D((ty + 1),tx+2)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx)] = DSum_corr[corr_coo2D((ty + 2),tx)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx+1)] = DSum_corr[corr_coo2D((ty + 2),tx+1)]/N_corr;
  DSum_corr[corr_coo2D((ty + 2),tx+2)] = DSum_corr[corr_coo2D((ty + 2),tx+2)]/N_corr;
  __syncthreads();
}
#endif
