#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__global__ void getcorrTRI(const float *confx, const float *confy, const float *confz, float *corr, int original_i, int original_j){
  /*****************************************************************
    !!!!!!!!!!!!!!! It can be used for square lattice and triangular lattice.
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  const int ox = original_i;
  const int oy =(blockIdx.x / BN) * SpinSize + original_j;
  //const int txp = tx +1 ;
  //const int typ = ty +1 ;
  //const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
  float sx00, sy00, sz00, sx01, sy01, sz01, sx02, sy02, sz02,
        sx10, sy10, sz10, sx11, sy11, sz11, sx12, sy12, sz12,
        sx20, sy20, sz20, sx21, sy21, sz21, sx22, sy22, sz22;
  int fx0, fy0,
      fx1, fy1,
      fx2, fy2,
      fx3, fy3,//from o to f
      fx4, fy4; 
  //calculate all the final position first

  fx0 = (tx + original_i) % SpinSize;
  fx1 = (tx + original_i + 1) % SpinSize;
  fx2 = (tx + original_i + 2) % SpinSize;
  fx3 = (tx + original_i + 3) % SpinSize;
  fx4 = (tx + original_i + 4) % SpinSize;

  if((ty % SpinSize + original_j) >= SpinSize)	fy0 = ty + original_j - SpinSize;
  else  fy0 = ty + original_j;
  if((ty % SpinSize + original_j + 1) >= SpinSize)	fy1 = ty + original_j + 1 - SpinSize;
  else  fy1 = ty + original_j + 1;
  if((ty % SpinSize + original_j + 2) >= SpinSize)	fy2 = ty + original_j + 2 - SpinSize;
  else  fy2 = ty + original_j + 2;
  if((ty % SpinSize + original_j + 3) >= SpinSize)	fy3 = ty + original_j + 3 - SpinSize;
  else  fy3 = ty + original_j + 3;
  if((ty % SpinSize + original_j + 4) >= SpinSize)	fy4 = ty + original_j + 4 - SpinSize;
  else  fy4 = ty + original_j + 4;

  //Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
  sx00 = confx[coo2D(oy,ox)];
  sy00 = confy[coo2D(oy,ox)];
  sz00 = confz[coo2D(oy,ox)];
  sx01 = confx[coo2D(oy,ox+1)];
  sy01 = confy[coo2D(oy,ox+1)];
  sz01 = confz[coo2D(oy,ox+1)];
  sx02 = confx[coo2D(oy,ox+2)];
  sy02 = confy[coo2D(oy,ox+2)];
  sz02 = confz[coo2D(oy,ox+2)];
  sx10 = confx[coo2D(oy+1,ox)];
  sy10 = confy[coo2D(oy+1,ox)];
  sz10 = confz[coo2D(oy+1,ox)];
  sx11 = confx[coo2D(oy+1,ox+1)];
  sy11 = confy[coo2D(oy+1,ox+1)];
  sz11 = confz[coo2D(oy+1,ox+1)];
  sx12 = confx[coo2D(oy+1,ox+2)];
  sy12 = confy[coo2D(oy+1,ox+2)];
  sz12 = confz[coo2D(oy+1,ox+2)];
  sx20 = confx[coo2D(oy+2,ox)];
  sy20 = confy[coo2D(oy+2,ox)];
  sz20 = confz[coo2D(oy+2,ox)];
  sx21 = confx[coo2D(oy+2,ox+1)];
  sy21 = confy[coo2D(oy+2,ox+1)];
  sz21 = confz[coo2D(oy+2,ox+1)];
  sx22 = confx[coo2D(oy+2,ox+2)];
  sy22 = confy[coo2D(oy+2,ox+2)];
  sz22 = confz[coo2D(oy+2,ox+2)];
  corr[coo2D(ty,tx)] += sx00 * confx[coo2D( fy0,fx0)] + sy00 * confy[coo2D( fy0,fx0)] + sz00 * confz[coo2D( fy0,fx0)] +
                        sx01 * confx[coo2D( fy0,fx1)] + sy01 * confy[coo2D( fy0,fx1)] + sz01 * confz[coo2D( fy0,fx1)] +
                        sx02 * confx[coo2D( fy0,fx2)] + sy02 * confy[coo2D( fy0,fx2)] + sz02 * confz[coo2D( fy0,fx2)] +
                        sx10 * confx[coo2D( fy1,fx0)] + sy10 * confy[coo2D( fy1,fx0)] + sz10 * confz[coo2D( fy1,fx0)] +
                        sx11 * confx[coo2D( fy1,fx1)] + sy11 * confy[coo2D( fy1,fx1)] + sz11 * confz[coo2D( fy1,fx1)] +
                        sx12 * confx[coo2D( fy1,fx2)] + sy12 * confy[coo2D( fy1,fx2)] + sz12 * confz[coo2D( fy1,fx2)] +
                        sx20 * confx[coo2D( fy2,fx0)] + sy20 * confy[coo2D( fy2,fx0)] + sz20 * confz[coo2D( fy2,fx0)] +
                        sx21 * confx[coo2D( fy2,fx1)] + sy21 * confy[coo2D( fy2,fx1)] + sz21 * confz[coo2D( fy2,fx1)] +
                        sx22 * confx[coo2D( fy2,fx2)] + sy22 * confy[coo2D( fy2,fx2)] + sz22 * confz[coo2D( fy2,fx2)];
  corr[coo2D(ty,tx+1)] += sx00 * confx[coo2D( fy0,fx1)] + sy00 * confy[coo2D( fy0,fx1)] + sz00 * confz[coo2D( fy0,fx1)] +
                          sx01 * confx[coo2D( fy0,fx2)] + sy01 * confy[coo2D( fy0,fx2)] + sz01 * confz[coo2D( fy0,fx2)] +
                          sx02 * confx[coo2D( fy0,fx3)] + sy02 * confy[coo2D( fy0,fx3)] + sz02 * confz[coo2D( fy0,fx3)] +
                          sx10 * confx[coo2D( fy1,fx1)] + sy10 * confy[coo2D( fy1,fx1)] + sz10 * confz[coo2D( fy1,fx1)] +
                          sx11 * confx[coo2D( fy1,fx2)] + sy11 * confy[coo2D( fy1,fx2)] + sz11 * confz[coo2D( fy1,fx2)] +
                          sx12 * confx[coo2D( fy1,fx3)] + sy12 * confy[coo2D( fy1,fx3)] + sz12 * confz[coo2D( fy1,fx3)] +
                          sx20 * confx[coo2D( fy2,fx1)] + sy20 * confy[coo2D( fy2,fx1)] + sz20 * confz[coo2D( fy2,fx1)] +
                          sx21 * confx[coo2D( fy2,fx2)] + sy21 * confy[coo2D( fy2,fx2)] + sz21 * confz[coo2D( fy2,fx2)] +
                          sx22 * confx[coo2D( fy2,fx3)] + sy22 * confy[coo2D( fy2,fx3)] + sz22 * confz[coo2D( fy2,fx3)];
  corr[coo2D(ty,tx+2)] += sx00 * confx[coo2D( fy0,fx2)] + sy00 * confy[coo2D( fy0,fx2)] + sz00 * confz[coo2D( fy0,fx2)] +
                          sx01 * confx[coo2D( fy0,fx3)] + sy01 * confy[coo2D( fy0,fx3)] + sz01 * confz[coo2D( fy0,fx3)] +
                          sx02 * confx[coo2D( fy0,fx4)] + sy02 * confy[coo2D( fy0,fx4)] + sz02 * confz[coo2D( fy0,fx4)] +
                          sx10 * confx[coo2D( fy1,fx2)] + sy10 * confy[coo2D( fy1,fx2)] + sz10 * confz[coo2D( fy1,fx2)] +
                          sx11 * confx[coo2D( fy1,fx3)] + sy11 * confy[coo2D( fy1,fx3)] + sz11 * confz[coo2D( fy1,fx3)] +
                          sx12 * confx[coo2D( fy1,fx4)] + sy12 * confy[coo2D( fy1,fx4)] + sz12 * confz[coo2D( fy1,fx4)] +
                          sx20 * confx[coo2D( fy2,fx2)] + sy20 * confy[coo2D( fy2,fx2)] + sz20 * confz[coo2D( fy2,fx2)] +
                          sx21 * confx[coo2D( fy2,fx3)] + sy21 * confy[coo2D( fy2,fx3)] + sz21 * confz[coo2D( fy2,fx3)] +
                          sx22 * confx[coo2D( fy2,fx4)] + sy22 * confy[coo2D( fy2,fx4)] + sz22 * confz[coo2D( fy2,fx4)];
  corr[coo2D((ty+1),tx)] += sx00 * confx[coo2D( fy1,fx0)] + sy00 * confy[coo2D( fy1,fx0)] + sz00 * confz[coo2D( fy1,fx0)] +
                            sx01 * confx[coo2D( fy1,fx1)] + sy01 * confy[coo2D( fy1,fx1)] + sz01 * confz[coo2D( fy1,fx1)] +
                            sx02 * confx[coo2D( fy1,fx2)] + sy02 * confy[coo2D( fy1,fx2)] + sz02 * confz[coo2D( fy1,fx2)] +
                            sx10 * confx[coo2D( fy2,fx0)] + sy10 * confy[coo2D( fy2,fx0)] + sz10 * confz[coo2D( fy2,fx0)] +
                            sx11 * confx[coo2D( fy2,fx1)] + sy11 * confy[coo2D( fy2,fx1)] + sz11 * confz[coo2D( fy2,fx1)] +
                            sx12 * confx[coo2D( fy2,fx2)] + sy12 * confy[coo2D( fy2,fx2)] + sz12 * confz[coo2D( fy2,fx2)] +
                            sx20 * confx[coo2D( fy3,fx0)] + sy20 * confy[coo2D( fy3,fx0)] + sz20 * confz[coo2D( fy3,fx0)] +
                            sx21 * confx[coo2D( fy3,fx1)] + sy21 * confy[coo2D( fy3,fx1)] + sz21 * confz[coo2D( fy3,fx1)] +
                            sx22 * confx[coo2D( fy3,fx2)] + sy22 * confy[coo2D( fy3,fx2)] + sz22 * confz[coo2D( fy3,fx2)];
  corr[coo2D((ty+1),tx+1)] += sx00 * confx[coo2D( fy1,fx1)] + sy00 * confy[coo2D( fy1,fx1)] + sz00 * confz[coo2D( fy1,fx1)] +
                              sx01 * confx[coo2D( fy1,fx2)] + sy01 * confy[coo2D( fy1,fx2)] + sz01 * confz[coo2D( fy1,fx2)] +
                              sx02 * confx[coo2D( fy1,fx3)] + sy02 * confy[coo2D( fy1,fx3)] + sz02 * confz[coo2D( fy1,fx3)] +
                              sx10 * confx[coo2D( fy2,fx1)] + sy10 * confy[coo2D( fy2,fx1)] + sz10 * confz[coo2D( fy2,fx1)] +
                              sx11 * confx[coo2D( fy2,fx2)] + sy11 * confy[coo2D( fy2,fx2)] + sz11 * confz[coo2D( fy2,fx2)] +
                              sx12 * confx[coo2D( fy2,fx3)] + sy12 * confy[coo2D( fy2,fx3)] + sz12 * confz[coo2D( fy2,fx3)] +
                              sx20 * confx[coo2D( fy3,fx1)] + sy20 * confy[coo2D( fy3,fx1)] + sz20 * confz[coo2D( fy3,fx1)] +
                              sx21 * confx[coo2D( fy3,fx2)] + sy21 * confy[coo2D( fy3,fx2)] + sz21 * confz[coo2D( fy3,fx2)] +
                              sx22 * confx[coo2D( fy3,fx3)] + sy22 * confy[coo2D( fy3,fx3)] + sz22 * confz[coo2D( fy3,fx3)];
  corr[coo2D((ty+1),tx+2)] += sx00 * confx[coo2D( fy1,fx2)] + sy00 * confy[coo2D( fy1,fx2)] + sz00 * confz[coo2D( fy1,fx2)] +
                              sx01 * confx[coo2D( fy1,fx3)] + sy01 * confy[coo2D( fy1,fx3)] + sz01 * confz[coo2D( fy1,fx3)] +
                              sx02 * confx[coo2D( fy1,fx4)] + sy02 * confy[coo2D( fy1,fx4)] + sz02 * confz[coo2D( fy1,fx4)] +
                              sx10 * confx[coo2D( fy2,fx2)] + sy10 * confy[coo2D( fy2,fx2)] + sz10 * confz[coo2D( fy2,fx2)] +
                              sx11 * confx[coo2D( fy2,fx3)] + sy11 * confy[coo2D( fy2,fx3)] + sz11 * confz[coo2D( fy2,fx3)] +
                              sx12 * confx[coo2D( fy2,fx4)] + sy12 * confy[coo2D( fy2,fx4)] + sz12 * confz[coo2D( fy2,fx4)] +
                              sx20 * confx[coo2D( fy3,fx2)] + sy20 * confy[coo2D( fy3,fx2)] + sz20 * confz[coo2D( fy3,fx2)] +
                              sx21 * confx[coo2D( fy3,fx3)] + sy21 * confy[coo2D( fy3,fx3)] + sz21 * confz[coo2D( fy3,fx3)] +
                              sx22 * confx[coo2D( fy3,fx4)] + sy22 * confy[coo2D( fy3,fx4)] + sz22 * confz[coo2D( fy3,fx4)];
  corr[coo2D((ty+2),tx)] += sx00 * confx[coo2D( fy2,fx0)] + sy00 * confy[coo2D( fy2,fx0)] + sz00 * confz[coo2D( fy2,fx0)] +
                            sx01 * confx[coo2D( fy2,fx1)] + sy01 * confy[coo2D( fy2,fx1)] + sz01 * confz[coo2D( fy2,fx1)] +
                            sx02 * confx[coo2D( fy2,fx2)] + sy02 * confy[coo2D( fy2,fx2)] + sz02 * confz[coo2D( fy2,fx2)] +
                            sx10 * confx[coo2D( fy3,fx0)] + sy10 * confy[coo2D( fy3,fx0)] + sz10 * confz[coo2D( fy3,fx0)] +
                            sx11 * confx[coo2D( fy3,fx1)] + sy11 * confy[coo2D( fy3,fx1)] + sz11 * confz[coo2D( fy3,fx1)] +
                            sx12 * confx[coo2D( fy3,fx2)] + sy12 * confy[coo2D( fy3,fx2)] + sz12 * confz[coo2D( fy3,fx2)] +
                            sx20 * confx[coo2D( fy4,fx0)] + sy20 * confy[coo2D( fy4,fx0)] + sz20 * confz[coo2D( fy4,fx0)] +
                            sx21 * confx[coo2D( fy4,fx1)] + sy21 * confy[coo2D( fy4,fx1)] + sz21 * confz[coo2D( fy4,fx1)] +
                            sx22 * confx[coo2D( fy4,fx2)] + sy22 * confy[coo2D( fy4,fx2)] + sz22 * confz[coo2D( fy4,fx2)];
  corr[coo2D((ty+2),tx+1)] += sx00 * confx[coo2D( fy2,fx1)] + sy00 * confy[coo2D( fy2,fx1)] + sz00 * confz[coo2D( fy2,fx1)] +
                              sx01 * confx[coo2D( fy2,fx2)] + sy01 * confy[coo2D( fy2,fx2)] + sz01 * confz[coo2D( fy2,fx2)] +
                              sx02 * confx[coo2D( fy2,fx3)] + sy02 * confy[coo2D( fy2,fx3)] + sz02 * confz[coo2D( fy2,fx3)] +
                              sx10 * confx[coo2D( fy3,fx1)] + sy10 * confy[coo2D( fy3,fx1)] + sz10 * confz[coo2D( fy3,fx1)] +
                              sx11 * confx[coo2D( fy3,fx2)] + sy11 * confy[coo2D( fy3,fx2)] + sz11 * confz[coo2D( fy3,fx2)] +
                              sx12 * confx[coo2D( fy3,fx3)] + sy12 * confy[coo2D( fy3,fx3)] + sz12 * confz[coo2D( fy3,fx3)] +
                              sx20 * confx[coo2D( fy4,fx1)] + sy20 * confy[coo2D( fy4,fx1)] + sz20 * confz[coo2D( fy4,fx1)] +
                              sx21 * confx[coo2D( fy4,fx2)] + sy21 * confy[coo2D( fy4,fx2)] + sz21 * confz[coo2D( fy4,fx2)] +
                              sx22 * confx[coo2D( fy4,fx3)] + sy22 * confy[coo2D( fy4,fx3)] + sz22 * confz[coo2D( fy4,fx3)];
  corr[coo2D((ty+2),tx+2)] += sx00 * confx[coo2D( fy2,fx2)] + sy00 * confy[coo2D( fy2,fx2)] + sz00 * confz[coo2D( fy2,fx2)] +
                              sx01 * confx[coo2D( fy2,fx3)] + sy01 * confy[coo2D( fy2,fx3)] + sz01 * confz[coo2D( fy2,fx3)] +
                              sx02 * confx[coo2D( fy2,fx4)] + sy02 * confy[coo2D( fy2,fx4)] + sz02 * confz[coo2D( fy2,fx4)] +
                              sx10 * confx[coo2D( fy3,fx2)] + sy10 * confy[coo2D( fy3,fx2)] + sz10 * confz[coo2D( fy3,fx2)] +
                              sx11 * confx[coo2D( fy3,fx3)] + sy11 * confy[coo2D( fy3,fx3)] + sz11 * confz[coo2D( fy3,fx3)] +
                              sx12 * confx[coo2D( fy3,fx4)] + sy12 * confy[coo2D( fy3,fx4)] + sz12 * confz[coo2D( fy3,fx4)] +
                              sx20 * confx[coo2D( fy4,fx2)] + sy20 * confy[coo2D( fy4,fx2)] + sz20 * confz[coo2D( fy4,fx2)] +
                              sx21 * confx[coo2D( fy4,fx3)] + sy21 * confy[coo2D( fy4,fx3)] + sz21 * confz[coo2D( fy4,fx3)] +
                              sx22 * confx[coo2D( fy4,fx4)] + sy22 * confy[coo2D( fy4,fx4)] + sz22 * confz[coo2D( fy4,fx4)];
  __syncthreads();
}

__global__ void sumcorrTRI(double *DSum_corr, const float *corr, int *DTo){
  //Energy variables
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  const int ty_pt =(DTo[blockIdx.x / BN]) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  //calculate all the final position first
  DSum_corr[coo2D(ty_pt,tx)] += corr[coo2D(ty,tx)]/SpinSize/SpinSize;
  DSum_corr[coo2D(ty_pt,tx+1)] += corr[coo2D(ty,tx+1)]/SpinSize/SpinSize;
  DSum_corr[coo2D(ty_pt,tx+2)] += corr[coo2D(ty,tx+2)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 1),tx)] += corr[coo2D((ty + 1),tx)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 1),tx+1)] += corr[coo2D((ty + 1),tx+1)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 1),tx+2)] += corr[coo2D((ty + 1),tx+2)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 2),tx)] += corr[coo2D((ty + 2),tx)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 2),tx+1)] += corr[coo2D((ty + 2),tx+1)]/SpinSize/SpinSize;
  DSum_corr[coo2D((ty_pt + 2),tx+2)] += corr[coo2D((ty + 2),tx+2)]/SpinSize/SpinSize;
  __syncthreads();
}
__global__ void avgcorrTRI(double *DSum_corr, double N_corr){
  /*****************************************************************
    Set ( original_i, original_j) as our original point.
    for tx_o , ty_o in 2x2 block of (original_i, original_j):
    corr[i - tx_o][j - ty_o] <-  the correlation between  and  (i, j)
    corr[   tx   ][   ty   ]
    use the periodic condition to keep the index positive.
    We need to sum over different (original_i, original_j) to get the correlation.
   *****************************************************************/
  //Energy variables
  const int x = threadIdx.x % (BlockSize_x);
  const int y = (threadIdx.x / BlockSize_x);
  const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
  const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
  //calculate all the final position first
  DSum_corr[coo2D(ty,tx)] = DSum_corr[coo2D(ty,tx)]/N_corr;
  DSum_corr[coo2D(ty,tx+1)] = DSum_corr[coo2D(ty,tx+1)]/N_corr;
  DSum_corr[coo2D(ty,tx+2)] = DSum_corr[coo2D(ty,tx+2)]/N_corr;
  DSum_corr[coo2D((ty + 1),tx)] = DSum_corr[coo2D((ty + 1),tx)]/N_corr;
  DSum_corr[coo2D((ty + 1),tx+1)] = DSum_corr[coo2D((ty + 1),tx+1)]/N_corr;
  DSum_corr[coo2D((ty + 1),tx+2)] = DSum_corr[coo2D((ty + 1),tx+2)]/N_corr;
  DSum_corr[coo2D((ty + 2),tx)] = DSum_corr[coo2D((ty + 2),tx)]/N_corr;
  DSum_corr[coo2D((ty + 2),tx+1)] = DSum_corr[coo2D((ty + 2),tx+1)]/N_corr;
  DSum_corr[coo2D((ty + 2),tx+2)] = DSum_corr[coo2D((ty + 2),tx+2)]/N_corr;
  __syncthreads();
}
#endif
