#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__global__ void calTRI(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	__shared__ double sD[BlockSize_y][BlockSize_x];
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 3 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  3 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	const int txp2 = tx +2 ;
	const int typ2 = ty +2 ;
	//const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	const int dataoff = (blockIdx.x / BN) * MEASURE_NUM * BN;
	int bx, by;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + SpinSize - 1) % SpinSize;
	if((ty % SpinSize) == 0)	by = ty + SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.

	//0,0
	sD[y][x] = -confx[coo2D(ty, tx)] * ( BXMxx * confx[coo2D(ty, bx)] + BYMxx * confx[coo2D(by, tx)] + BWMxx * confx[coo2D(by, bx)])\
	           -confx[coo2D(ty, tx)] * ( BXMxy * confy[coo2D(ty, bx)] + BYMxy * confy[coo2D(by, tx)] + BWMxy * confy[coo2D(by, bx)])\
	           -confx[coo2D(ty, tx)] * ( BXMxz * confz[coo2D(ty, bx)] + BYMxz * confz[coo2D(by, tx)] + BWMxz * confz[coo2D(by, bx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyx * confx[coo2D(ty, bx)] + BYMyx * confx[coo2D(by, tx)] + BWMyx * confx[coo2D(by, bx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyy * confy[coo2D(ty, bx)] + BYMyy * confy[coo2D(by, tx)] + BWMyy * confy[coo2D(by, bx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyz * confz[coo2D(ty, bx)] + BYMyz * confz[coo2D(by, tx)] + BWMyz * confz[coo2D(by, bx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzx * confx[coo2D(ty, bx)] + BYMzx * confx[coo2D(by, tx)] + BWMzx * confx[coo2D(by, bx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzy * confy[coo2D(ty, bx)] + BYMzy * confy[coo2D(by, tx)] + BWMzy * confy[coo2D(by, bx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzz * confz[coo2D(ty, bx)] + BYMzz * confz[coo2D(by, tx)] + BWMzz * confz[coo2D(by, bx)] - A * confz[coo2D(ty, tx)]);
	//1,0
	sD[y][x] -= confx[coo2D(typ, tx)] * ( BXMxx * confx[coo2D(typ, bx)] + BYMxx * confx[coo2D(ty, tx)] + BWMxx * confx[coo2D(ty, bx)])\
		         +confx[coo2D(typ, tx)] * ( BXMxy * confy[coo2D(typ, bx)] + BYMxy * confy[coo2D(ty, tx)] + BWMxy * confy[coo2D(ty, bx)])\
		         +confx[coo2D(typ, tx)] * ( BXMxz * confz[coo2D(typ, bx)] + BYMxz * confz[coo2D(ty, tx)] + BWMxz * confz[coo2D(ty, bx)])\
		         +confy[coo2D(typ, tx)] * ( BXMyx * confx[coo2D(typ, bx)] + BYMyx * confx[coo2D(ty, tx)] + BWMyx * confx[coo2D(ty, bx)])\
		         +confy[coo2D(typ, tx)] * ( BXMyy * confy[coo2D(typ, bx)] + BYMyy * confy[coo2D(ty, tx)] + BWMyy * confy[coo2D(ty, bx)])\
		         +confy[coo2D(typ, tx)] * ( BXMyz * confz[coo2D(typ, bx)] + BYMyz * confz[coo2D(ty, tx)] + BWMyz * confz[coo2D(ty, bx)])\
		         +confz[coo2D(typ, tx)] * ( BXMzx * confx[coo2D(typ, bx)] + BYMzx * confx[coo2D(ty, tx)] + BWMzx * confx[coo2D(ty, bx)])\
		         +confz[coo2D(typ, tx)] * ( BXMzy * confy[coo2D(typ, bx)] + BYMzy * confy[coo2D(ty, tx)] + BWMzy * confy[coo2D(ty, bx)])\
		         +confz[coo2D(typ, tx)] * ( BXMzz * confz[coo2D(typ, bx)] + BYMzz * confz[coo2D(ty, tx)] + BWMzz * confz[coo2D(ty, bx)] - A * confz[coo2D((ty+1), tx)]);
	//2,0
	sD[y][x] -= confx[coo2D(typ2, tx)] * ( BXMxx * confx[coo2D(typ2, bx)] + BYMxx * confx[coo2D(typ, tx)] + BWMxx * confx[coo2D(typ, bx)])\
		         +confx[coo2D(typ2, tx)] * ( BXMxy * confy[coo2D(typ2, bx)] + BYMxy * confy[coo2D(typ, tx)] + BWMxy * confy[coo2D(typ, bx)])\
		         +confx[coo2D(typ2, tx)] * ( BXMxz * confz[coo2D(typ2, bx)] + BYMxz * confz[coo2D(typ, tx)] + BWMxz * confz[coo2D(typ, bx)])\
		         +confy[coo2D(typ2, tx)] * ( BXMyx * confx[coo2D(typ2, bx)] + BYMyx * confx[coo2D(typ, tx)] + BWMyx * confx[coo2D(typ, bx)])\
		         +confy[coo2D(typ2, tx)] * ( BXMyy * confy[coo2D(typ2, bx)] + BYMyy * confy[coo2D(typ, tx)] + BWMyy * confy[coo2D(typ, bx)])\
		         +confy[coo2D(typ2, tx)] * ( BXMyz * confz[coo2D(typ2, bx)] + BYMyz * confz[coo2D(typ, tx)] + BWMyz * confz[coo2D(typ, bx)])\
		         +confz[coo2D(typ2, tx)] * ( BXMzx * confx[coo2D(typ2, bx)] + BYMzx * confx[coo2D(typ, tx)] + BWMzx * confx[coo2D(typ, bx)])\
		         +confz[coo2D(typ2, tx)] * ( BXMzy * confy[coo2D(typ2, bx)] + BYMzy * confy[coo2D(typ, tx)] + BWMzy * confy[coo2D(typ, bx)])\
		         +confz[coo2D(typ2, tx)] * ( BXMzz * confz[coo2D(typ2, bx)] + BYMzz * confz[coo2D(typ, tx)] + BWMzz * confz[coo2D(typ, bx)] - A * confz[coo2D((ty+1), tx)]);
	//0,1
	sD[y][x] -= confx[coo2D(ty, txp)] * ( BXMxx * confx[coo2D(ty, tx)] + BYMxx * confx[coo2D(by, txp)] + BWMxx * confx[coo2D(by, tx)])\
		         +confx[coo2D(ty, txp)] * ( BXMxy * confy[coo2D(ty, tx)] + BYMxy * confy[coo2D(by, txp)] + BWMxy * confy[coo2D(by, tx)])\
		         +confx[coo2D(ty, txp)] * ( BXMxz * confz[coo2D(ty, tx)] + BYMxz * confz[coo2D(by, txp)] + BWMxz * confz[coo2D(by, tx)])\
		         +confy[coo2D(ty, txp)] * ( BXMyx * confx[coo2D(ty, tx)] + BYMyx * confx[coo2D(by, txp)] + BWMyx * confx[coo2D(by, tx)])\
		         +confy[coo2D(ty, txp)] * ( BXMyy * confy[coo2D(ty, tx)] + BYMyy * confy[coo2D(by, txp)] + BWMyy * confy[coo2D(by, tx)])\
		         +confy[coo2D(ty, txp)] * ( BXMyz * confz[coo2D(ty, tx)] + BYMyz * confz[coo2D(by, txp)] + BWMyz * confz[coo2D(by, tx)])\
		         +confz[coo2D(ty, txp)] * ( BXMzx * confx[coo2D(ty, tx)] + BYMzx * confx[coo2D(by, txp)] + BWMzx * confx[coo2D(by, tx)])\
		         +confz[coo2D(ty, txp)] * ( BXMzy * confy[coo2D(ty, tx)] + BYMzy * confy[coo2D(by, txp)] + BWMzy * confy[coo2D(by, tx)])\
		         +confz[coo2D(ty, txp)] * ( BXMzz * confz[coo2D(ty, tx)] + BYMzz * confz[coo2D(by, txp)] + BWMzz * confz[coo2D(by, tx)] - A * confz[coo2D(ty, tx+1)]);
	//1,1
	sD[y][x] -= confx[coo2D(typ, txp)] * ( BXMxx * confx[coo2D(typ, tx)] + BYMxx * confx[coo2D(ty, txp)] + BWMxx * confx[coo2D(ty, tx)])\
		         +confx[coo2D(typ, txp)] * ( BXMxy * confy[coo2D(typ, tx)] + BYMxy * confy[coo2D(ty, txp)] + BWMxy * confy[coo2D(ty, tx)])\
		         +confx[coo2D(typ, txp)] * ( BXMxz * confz[coo2D(typ, tx)] + BYMxz * confz[coo2D(ty, txp)] + BWMxz * confz[coo2D(ty, tx)])\
		         +confy[coo2D(typ, txp)] * ( BXMyx * confx[coo2D(typ, tx)] + BYMyx * confx[coo2D(ty, txp)] + BWMyx * confx[coo2D(ty, tx)])\
		         +confy[coo2D(typ, txp)] * ( BXMyy * confy[coo2D(typ, tx)] + BYMyy * confy[coo2D(ty, txp)] + BWMyy * confy[coo2D(ty, tx)])\
		         +confy[coo2D(typ, txp)] * ( BXMyz * confz[coo2D(typ, tx)] + BYMyz * confz[coo2D(ty, txp)] + BWMyz * confz[coo2D(ty, tx)])\
		         +confz[coo2D(typ, txp)] * ( BXMzx * confx[coo2D(typ, tx)] + BYMzx * confx[coo2D(ty, txp)] + BWMzx * confx[coo2D(ty, tx)])\
		         +confz[coo2D(typ, txp)] * ( BXMzy * confy[coo2D(typ, tx)] + BYMzy * confy[coo2D(ty, txp)] + BWMzy * confy[coo2D(ty, tx)])\
		         +confz[coo2D(typ, txp)] * ( BXMzz * confz[coo2D(typ, tx)] + BYMzz * confz[coo2D(ty, txp)] + BWMzz * confz[coo2D(ty, tx)] - A * confz[coo2D(ty, tx+1)]);
	//2,1
	sD[y][x] -= confx[coo2D(typ2, txp)] * ( BXMxx * confx[coo2D(typ2, tx)] + BYMxx * confx[coo2D(typ, txp)] + BWMxx * confx[coo2D(typ, tx)])\
		         +confx[coo2D(typ2, txp)] * ( BXMxy * confy[coo2D(typ2, tx)] + BYMxy * confy[coo2D(typ, txp)] + BWMxy * confy[coo2D(typ, tx)])\
		         +confx[coo2D(typ2, txp)] * ( BXMxz * confz[coo2D(typ2, tx)] + BYMxz * confz[coo2D(typ, txp)] + BWMxz * confz[coo2D(typ, tx)])\
		         +confy[coo2D(typ2, txp)] * ( BXMyx * confx[coo2D(typ2, tx)] + BYMyx * confx[coo2D(typ, txp)] + BWMyx * confx[coo2D(typ, tx)])\
		         +confy[coo2D(typ2, txp)] * ( BXMyy * confy[coo2D(typ2, tx)] + BYMyy * confy[coo2D(typ, txp)] + BWMyy * confy[coo2D(typ, tx)])\
		         +confy[coo2D(typ2, txp)] * ( BXMyz * confz[coo2D(typ2, tx)] + BYMyz * confz[coo2D(typ, txp)] + BWMyz * confz[coo2D(typ, tx)])\
		         +confz[coo2D(typ2, txp)] * ( BXMzx * confx[coo2D(typ2, tx)] + BYMzx * confx[coo2D(typ, txp)] + BWMzx * confx[coo2D(typ, tx)])\
		         +confz[coo2D(typ2, txp)] * ( BXMzy * confy[coo2D(typ2, tx)] + BYMzy * confy[coo2D(typ, txp)] + BWMzy * confy[coo2D(typ, tx)])\
		         +confz[coo2D(typ2, txp)] * ( BXMzz * confz[coo2D(typ2, tx)] + BYMzz * confz[coo2D(typ, txp)] + BWMzz * confz[coo2D(typ, tx)] - A * confz[coo2D(ty, tx+1)]);
	//0,2
	sD[y][x] -= confx[coo2D(ty, txp2)] * ( BXMxx * confx[coo2D(ty, txp)] + BYMxx * confx[coo2D(by, txp2)] + BWMxx * confx[coo2D(by, txp)])\
		         +confx[coo2D(ty, txp2)] * ( BXMxy * confy[coo2D(ty, txp)] + BYMxy * confy[coo2D(by, txp2)] + BWMxy * confy[coo2D(by, txp)])\
		         +confx[coo2D(ty, txp2)] * ( BXMxz * confz[coo2D(ty, txp)] + BYMxz * confz[coo2D(by, txp2)] + BWMxz * confz[coo2D(by, txp)])\
		         +confy[coo2D(ty, txp2)] * ( BXMyx * confx[coo2D(ty, txp)] + BYMyx * confx[coo2D(by, txp2)] + BWMyx * confx[coo2D(by, txp)])\
		         +confy[coo2D(ty, txp2)] * ( BXMyy * confy[coo2D(ty, txp)] + BYMyy * confy[coo2D(by, txp2)] + BWMyy * confy[coo2D(by, txp)])\
		         +confy[coo2D(ty, txp2)] * ( BXMyz * confz[coo2D(ty, txp)] + BYMyz * confz[coo2D(by, txp2)] + BWMyz * confz[coo2D(by, txp)])\
		         +confz[coo2D(ty, txp2)] * ( BXMzx * confx[coo2D(ty, txp)] + BYMzx * confx[coo2D(by, txp2)] + BWMzx * confx[coo2D(by, txp)])\
		         +confz[coo2D(ty, txp2)] * ( BXMzy * confy[coo2D(ty, txp)] + BYMzy * confy[coo2D(by, txp2)] + BWMzy * confy[coo2D(by, txp)])\
		         +confz[coo2D(ty, txp2)] * ( BXMzz * confz[coo2D(ty, txp)] + BYMzz * confz[coo2D(by, txp2)] + BWMzz * confz[coo2D(by, txp)] - A * confz[coo2D(ty, tx+1)]);
	//1,2
	sD[y][x] -= confx[coo2D(typ, txp2)] * ( BXMxx * confx[coo2D(typ, txp)] + BYMxx * confx[coo2D(ty, txp2)] + BWMxx * confx[coo2D(ty, txp)])\
		         +confx[coo2D(typ, txp2)] * ( BXMxy * confy[coo2D(typ, txp)] + BYMxy * confy[coo2D(ty, txp2)] + BWMxy * confy[coo2D(ty, txp)])\
		         +confx[coo2D(typ, txp2)] * ( BXMxz * confz[coo2D(typ, txp)] + BYMxz * confz[coo2D(ty, txp2)] + BWMxz * confz[coo2D(ty, txp)])\
		         +confy[coo2D(typ, txp2)] * ( BXMyx * confx[coo2D(typ, txp)] + BYMyx * confx[coo2D(ty, txp2)] + BWMyx * confx[coo2D(ty, txp)])\
		         +confy[coo2D(typ, txp2)] * ( BXMyy * confy[coo2D(typ, txp)] + BYMyy * confy[coo2D(ty, txp2)] + BWMyy * confy[coo2D(ty, txp)])\
		         +confy[coo2D(typ, txp2)] * ( BXMyz * confz[coo2D(typ, txp)] + BYMyz * confz[coo2D(ty, txp2)] + BWMyz * confz[coo2D(ty, txp)])\
		         +confz[coo2D(typ, txp2)] * ( BXMzx * confx[coo2D(typ, txp)] + BYMzx * confx[coo2D(ty, txp2)] + BWMzx * confx[coo2D(ty, txp)])\
		         +confz[coo2D(typ, txp2)] * ( BXMzy * confy[coo2D(typ, txp)] + BYMzy * confy[coo2D(ty, txp2)] + BWMzy * confy[coo2D(ty, txp)])\
		         +confz[coo2D(typ, txp2)] * ( BXMzz * confz[coo2D(typ, txp)] + BYMzz * confz[coo2D(ty, txp2)] + BWMzz * confz[coo2D(ty, txp)] - A * confz[coo2D(ty, tx+1)]);
	//2,2
	sD[y][x] -= confx[coo2D(typ2, txp2)] * ( BXMxx * confx[coo2D(typ2, txp)] + BYMxx * confx[coo2D(typ, txp2)] + BWMxx * confx[coo2D(typ, txp)])\
		         +confx[coo2D(typ2, txp2)] * ( BXMxy * confy[coo2D(typ2, txp)] + BYMxy * confy[coo2D(typ, txp2)] + BWMxy * confy[coo2D(typ, txp)])\
		         +confx[coo2D(typ2, txp2)] * ( BXMxz * confz[coo2D(typ2, txp)] + BYMxz * confz[coo2D(typ, txp2)] + BWMxz * confz[coo2D(typ, txp)])\
		         +confy[coo2D(typ2, txp2)] * ( BXMyx * confx[coo2D(typ2, txp)] + BYMyx * confx[coo2D(typ, txp2)] + BWMyx * confx[coo2D(typ, txp)])\
		         +confy[coo2D(typ2, txp2)] * ( BXMyy * confy[coo2D(typ2, txp)] + BYMyy * confy[coo2D(typ, txp2)] + BWMyy * confy[coo2D(typ, txp)])\
		         +confy[coo2D(typ2, txp2)] * ( BXMyz * confz[coo2D(typ2, txp)] + BYMyz * confz[coo2D(typ, txp2)] + BWMyz * confz[coo2D(typ, txp)])\
		         +confz[coo2D(typ2, txp2)] * ( BXMzx * confx[coo2D(typ2, txp)] + BYMzx * confx[coo2D(typ, txp2)] + BWMzx * confx[coo2D(typ, txp)])\
		         +confz[coo2D(typ2, txp2)] * ( BXMzy * confy[coo2D(typ2, txp)] + BYMzy * confy[coo2D(typ, txp2)] + BWMzy * confy[coo2D(typ, txp)])\
		         +confz[coo2D(typ2, txp2)] * ( BXMzz * confz[coo2D(typ2, txp)] + BYMzz * confz[coo2D(typ, txp2)] + BWMzz * confz[coo2D(typ, txp)] - A * confz[coo2D(ty, tx+1)]);
	__syncthreads();


	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y<BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y<BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y<BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(y==0 && x==0)
		out[dataoff + (blockIdx.x % BN)] = sD[0][0];
	__syncthreads();
	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[y][x]  = confx[coo2D(ty, tx)];
	sD[y][x] += confx[coo2D(typ, tx)];
	sD[y][x] += confx[coo2D(typ2, tx)];
	sD[y][x] += confx[coo2D(ty, txp)];
	sD[y][x] += confx[coo2D(typ, txp)];
	sD[y][x] += confx[coo2D(typ2, txp)];
	sD[y][x] += confx[coo2D(ty, txp2)];
	sD[y][x] += confx[coo2D(typ, txp2)];
	sD[y][x] += confx[coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[y][x]  = confy[coo2D(ty, tx)];
	sD[y][x] += confy[coo2D(typ, tx)];
	sD[y][x] += confy[coo2D(typ2, tx)];
	sD[y][x] += confy[coo2D(ty, txp)];
	sD[y][x] += confy[coo2D(typ, txp)];
	sD[y][x] += confy[coo2D(typ2, txp)];
	sD[y][x] += confy[coo2D(ty, txp2)];
	sD[y][x] += confy[coo2D(typ, txp2)];
	sD[y][x] += confy[coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 2 * BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[y][x]  = confy[coo2D(ty, tx)];
	sD[y][x] += confy[coo2D(typ, tx)];
	sD[y][x] += confy[coo2D(typ2, tx)];
	sD[y][x] += confy[coo2D(ty, txp)];
	sD[y][x] += confy[coo2D(typ, txp)];
	sD[y][x] += confy[coo2D(typ2, txp)];
	sD[y][x] += confy[coo2D(ty, txp2)];
	sD[y][x] += confy[coo2D(typ, txp2)];
	sD[y][x] += confy[coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 3 * BN] = sD[0][0];
	__syncthreads();
	//try to measure Chern number
	//(0,0)
	sD[y][x]  = confx[coo2D(ty, tx)] * (
	 (confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])*(2*confz[coo2D(ty, tx)]-confz[coo2D(by, tx)]-confz[coo2D(by, bx)])
	-(confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])*(2*confy[coo2D(ty, tx)]-confy[coo2D(by, tx)]-confy[coo2D(by, bx)])
	)+confy[coo2D(ty, tx)] * (
	 (confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])*(2*confx[coo2D(ty, tx)]-confx[coo2D(by, tx)]-confx[coo2D(by, bx)])
	-(confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])*(2*confz[coo2D(ty, tx)]-confz[coo2D(by, tx)]-confz[coo2D(by, bx)])
	)+confz[coo2D(ty, tx)] * (
	 (confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])*(2*confy[coo2D(ty, tx)]-confy[coo2D(by, tx)]-confy[coo2D(by, bx)])
	-(confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])*(2*confx[coo2D(ty, tx)]-confx[coo2D(by, tx)]-confx[coo2D(by, bx)])
	);
	//(1,0)
	sD[y][x] += confx[coo2D(typ, tx)] * (
	 (confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])*(2*confz[coo2D(typ, tx)]-confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])
	-(confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])*(2*confy[coo2D(typ, tx)]-confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])
	)+confy[coo2D(typ, tx)]*(
	 (confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])*(2*confx[coo2D(typ, tx)]-confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])
	-(confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])*(2*confz[coo2D(typ, tx)]-confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])
	)+confz[coo2D(typ, tx)] * (
	 (confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])*(2*confy[coo2D(typ, tx)]-confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])
	-(confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])*(2*confx[coo2D(typ, tx)]-confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])
	);
	//(2,0)
	sD[y][x] += confx[coo2D(typ2, tx)] * (
	 (confy[coo2D(typ2, tx)]-confy[coo2D(typ2, bx)])*(2*confz[coo2D(typ2, tx)]-confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])
	-(confz[coo2D(typ2, tx)]-confz[coo2D(typ2, bx)])*(2*confy[coo2D(typ2, tx)]-confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])
	)+confy[coo2D(typ2, tx)]*(
	 (confz[coo2D(typ2, tx)]-confz[coo2D(typ2, bx)])*(2*confx[coo2D(typ2, tx)]-confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])
	-(confx[coo2D(typ2, tx)]-confx[coo2D(typ2, bx)])*(2*confz[coo2D(typ2, tx)]-confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])
	)+confz[coo2D(typ2, tx)] * (
	 (confx[coo2D(typ2, tx)]-confx[coo2D(typ2, bx)])*(2*confy[coo2D(typ2, tx)]-confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])
	-(confy[coo2D(typ2, tx)]-confy[coo2D(typ2, bx)])*(2*confx[coo2D(typ2, tx)]-confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])
	);
	//(0,1)
	sD[y][x] += confx[coo2D(ty, txp)] * (
	 (confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])*(2*confz[coo2D(ty, txp)]-confz[coo2D(by, txp)]-confz[coo2D(by, tx)])
	-(confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])*(2*confy[coo2D(ty, txp)]-confy[coo2D(by, txp)]-confy[coo2D(by, tx)])
	)+confy[coo2D(ty, txp)]*(                                                                                        
	 (confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])*(2*confx[coo2D(ty, txp)]-confx[coo2D(by, txp)]-confx[coo2D(by, tx)])
	-(confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])*(2*confz[coo2D(ty, txp)]-confz[coo2D(by, txp)]-confz[coo2D(by, tx)])
	)+confz[coo2D(ty, txp)] * (                                                                                      
	 (confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])*(2*confy[coo2D(ty, txp)]-confy[coo2D(by, txp)]-confy[coo2D(by, tx)])
	-(confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])*(2*confx[coo2D(ty, txp)]-confx[coo2D(by, txp)]-confx[coo2D(by, tx)])
	);
	//(1,1)
	sD[y][x] += confx[coo2D(typ, txp)] * (
	 (confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])*(2*confz[coo2D(typ, txp)]-confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])
	-(confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])*(2*confy[coo2D(typ, txp)]-confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])
	)+confy[coo2D(typ, txp)]*(
	 (confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])*(2*confx[coo2D(typ, txp)]-confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])
	-(confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])*(2*confz[coo2D(typ, txp)]-confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])
	)+confz[coo2D(typ, txp)] * (
	 (confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])*(2*confy[coo2D(typ, txp)]-confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])
	-(confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])*(2*confx[coo2D(typ, txp)]-confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])
	);
	//(2,1)
	sD[y][x] += confx[coo2D(typ2, txp)] * (
	 (confy[coo2D(typ2, txp)]-confy[coo2D(typ2, tx)])*(2*confz[coo2D(typ2, txp)]-confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])
	-(confz[coo2D(typ2, txp)]-confz[coo2D(typ2, tx)])*(2*confy[coo2D(typ2, txp)]-confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])
	)+confy[coo2D(typ2, txp)]*(
	 (confz[coo2D(typ2, txp)]-confz[coo2D(typ2, tx)])*(2*confx[coo2D(typ2, txp)]-confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])
	-(confx[coo2D(typ2, txp)]-confx[coo2D(typ2, tx)])*(2*confz[coo2D(typ2, txp)]-confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])
	)+confz[coo2D(typ2, txp)] * (
	 (confx[coo2D(typ2, txp)]-confx[coo2D(typ2, tx)])*(2*confy[coo2D(typ2, txp)]-confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])
	-(confy[coo2D(typ2, txp)]-confy[coo2D(typ2, tx)])*(2*confx[coo2D(typ2, txp)]-confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])
	);
	//(0,2)
	sD[y][x] += confx[coo2D(ty, txp2)] * (
	 (confy[coo2D(ty, txp2)]-confy[coo2D(ty, txp)])*(2*confz[coo2D(ty, txp2)]-confz[coo2D(by, txp2)]-confz[coo2D(by, txp)])
	-(confz[coo2D(ty, txp2)]-confz[coo2D(ty, txp)])*(2*confy[coo2D(ty, txp2)]-confy[coo2D(by, txp2)]-confy[coo2D(by, txp)])
	)+confy[coo2D(ty, txp2)]*(                                                                                          
	 (confz[coo2D(ty, txp2)]-confz[coo2D(ty, txp)])*(2*confx[coo2D(ty, txp2)]-confx[coo2D(by, txp2)]-confx[coo2D(by, txp)])
	-(confx[coo2D(ty, txp2)]-confx[coo2D(ty, txp)])*(2*confz[coo2D(ty, txp2)]-confz[coo2D(by, txp2)]-confz[coo2D(by, txp)])
	)+confz[coo2D(ty, txp2)] * (                                                                                        
	 (confx[coo2D(ty, txp2)]-confx[coo2D(ty, txp)])*(2*confy[coo2D(ty, txp2)]-confy[coo2D(by, txp2)]-confy[coo2D(by, txp)])
	-(confy[coo2D(ty, txp2)]-confy[coo2D(ty, txp)])*(2*confx[coo2D(ty, txp2)]-confx[coo2D(by, txp2)]-confx[coo2D(by, txp)])
	);
	//(1,2)
	sD[y][x] += confx[coo2D(typ, txp2)] * (
	 (confy[coo2D(typ, txp2)]-confy[coo2D(typ, txp)])*(2*confz[coo2D(typ, txp2)]-confz[coo2D(ty, txp2)]-confz[coo2D(ty, txp)])
	-(confz[coo2D(typ, txp2)]-confz[coo2D(typ, txp)])*(2*confy[coo2D(typ, txp2)]-confy[coo2D(ty, txp2)]-confy[coo2D(ty, txp)])
	)+confy[coo2D(typ, txp2)]*(                                                                                            
	 (confz[coo2D(typ, txp2)]-confz[coo2D(typ, txp)])*(2*confx[coo2D(typ, txp2)]-confx[coo2D(ty, txp2)]-confx[coo2D(ty, txp)])
	-(confx[coo2D(typ, txp2)]-confx[coo2D(typ, txp)])*(2*confz[coo2D(typ, txp2)]-confz[coo2D(ty, txp2)]-confz[coo2D(ty, txp)])
	)+confz[coo2D(typ, txp2)] * (                                                                                          
	 (confx[coo2D(typ, txp2)]-confx[coo2D(typ, txp)])*(2*confy[coo2D(typ, txp2)]-confy[coo2D(ty, txp2)]-confy[coo2D(ty, txp)])
	-(confy[coo2D(typ, txp2)]-confy[coo2D(typ, txp)])*(2*confx[coo2D(typ, txp2)]-confx[coo2D(ty, txp2)]-confx[coo2D(ty, txp)])
	);
	//(2,2)
	sD[y][x] += confx[coo2D(typ2, txp2)] * (
	 (confy[coo2D(typ2, txp2)]-confy[coo2D(typ2, txp)])*(2*confz[coo2D(typ2, txp2)]-confz[coo2D(typ, txp2)]-confz[coo2D(typ, txp)])
	-(confz[coo2D(typ2, txp2)]-confz[coo2D(typ2, txp)])*(2*confy[coo2D(typ2, txp2)]-confy[coo2D(typ, txp2)]-confy[coo2D(typ, txp)])
	)+confy[coo2D(typ2, txp2)]*( 
	 (confz[coo2D(typ2, txp2)]-confz[coo2D(typ2, txp)])*(2*confx[coo2D(typ2, txp2)]-confx[coo2D(typ, txp2)]-confx[coo2D(typ, txp)])
	-(confx[coo2D(typ2, txp2)]-confx[coo2D(typ2, txp)])*(2*confz[coo2D(typ2, txp2)]-confz[coo2D(typ, txp2)]-confz[coo2D(typ, txp)])
	)+confz[coo2D(typ2, txp2)] * (
	 (confx[coo2D(typ2, txp2)]-confx[coo2D(typ2, txp)])*(2*confy[coo2D(typ2, txp2)]-confy[coo2D(typ, txp2)]-confy[coo2D(typ, txp)])
	-(confy[coo2D(typ2, txp2)]-confy[coo2D(typ2, txp)])*(2*confx[coo2D(typ2, txp2)]-confx[coo2D(typ, txp2)]-confx[coo2D(typ, txp)])
	);
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 4 * BN] = sD[0][0];
	__syncthreads();
}
#endif
