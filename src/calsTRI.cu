#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__constant__ unsigned int cals_SpinSize;
__constant__ unsigned int cals_SpinSize_z;
__constant__ unsigned int cals_BlockSize_x;
__constant__ unsigned int cals_BlockSize_y;
__constant__ unsigned int cals_GridSize_x;
__constant__ unsigned int cals_GridSize_y;
__constant__ unsigned int cals_TN;
__constant__ unsigned int cals_BN;
__constant__ float Q1x;
__constant__ float Q1y;
__constant__ float Q2x;
__constant__ float Q2y;
__constant__ float cals_A; //(0.0)
__constant__ float cBXPyz;
__constant__ float cBYPyz;
__constant__ float cBWPyz;
__constant__ float cBXMyz;
__constant__ float cBYMyz;
__constant__ float cBWMyz;
__constant__ float cBXPzy;
__constant__ float cBYPzy;
__constant__ float cBWPzy;
__constant__ float cBXMzy;
__constant__ float cBYMzy;
__constant__ float cBWMzy;
__constant__ float cBXPxz;
__constant__ float cBYPxz;
__constant__ float cBWPxz;
__constant__ float cBXMxz;
__constant__ float cBYMxz;
__constant__ float cBWMxz;
__constant__ float cBXPzx;
__constant__ float cBYPzx;
__constant__ float cBWPzx;
__constant__ float cBXMzx;
__constant__ float cBYMzx;
__constant__ float cBWMzx;
void move_params_device_cals(){
  float tmpp;
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BN), &H_BN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_A ), &H_A , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q1x ), &H_Q1x , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q1y ), &H_Q1y , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q2x ), &H_Q2x , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q2y ), &H_Q2y , sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPzy), &tmpp, sizeof(float));
  tmpp =  (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPzy), &tmpp, sizeof(float));
  tmpp =  (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPzy), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMzy), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMzy), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMzy), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPzx), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMzx), &tmpp, sizeof(float));
}
__global__ void calTRI(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	extern __shared__ double sD[];
	const int x = threadIdx.x % (cals_BlockSize_x);
	const int y = (threadIdx.x / cals_BlockSize_x);
	const int tx = 3 * (((blockIdx.x % cals_BN) % cals_GridSize_x) * cals_BlockSize_x + x);
	const int ty =(blockIdx.x / cals_BN) * cals_SpinSize +  3 * ((((blockIdx.x % cals_BN) / cals_GridSize_x) % cals_GridSize_y) * cals_BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	const int txp2 = tx +2 ;
	const int typ2 = ty +2 ;
	//const int ty = 2 * ((blockIdx.x / cals_BN) * cals_SpinSize + ((blockIdx.x % cals_BN) / cals_GridSize_x) * cals_BlockSize_y + y);
	const int dataoff = (blockIdx.x / cals_BN) * MEASURE_NUM * cals_BN;
	int bx, by, tx_ty = tx + (ty % cals_SpinSize);
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + cals_SpinSize - 1) % cals_SpinSize;
	if((ty % cals_SpinSize) == 0)	by = ty + cals_SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.

	//0,0
	sD[threadIdx.x] = -confx[cals_coo2D(ty, tx)] * ( BXMxx * confx[cals_coo2D(ty, bx)] + BYMxx * confx[cals_coo2D(by, tx)] + BWMxx * confx[cals_coo2D(by, bx)])\
	           -confx[cals_coo2D(ty, tx)] * ( BXMxy * confy[cals_coo2D(ty, bx)] + BYMxy * confy[cals_coo2D(by, tx)] + BWMxy * confy[cals_coo2D(by, bx)])\
	           -confx[cals_coo2D(ty, tx)] * ( cBXMxz * confz[cals_coo2D(ty, bx)] + cBYMxz * confz[cals_coo2D(by, tx)] + cBWMxz * confz[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( BXMyx * confx[cals_coo2D(ty, bx)] + BYMyx * confx[cals_coo2D(by, tx)] + BWMyx * confx[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( BXMyy * confy[cals_coo2D(ty, bx)] + BYMyy * confy[cals_coo2D(by, tx)] + BWMyy * confy[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( cBXMyz * confz[cals_coo2D(ty, bx)] + cBYMyz * confz[cals_coo2D(by, tx)] + cBWMyz * confz[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( cBXMzx * confx[cals_coo2D(ty, bx)] + cBYMzx * confx[cals_coo2D(by, tx)] + cBWMzx * confx[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( cBXMzy * confy[cals_coo2D(ty, bx)] + cBYMzy * confy[cals_coo2D(by, tx)] + cBWMzy * confy[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( BXMzz * confz[cals_coo2D(ty, bx)] + BYMzz * confz[cals_coo2D(by, tx)] + BWMzz * confz[cals_coo2D(by, bx)] - cals_A * confz[cals_coo2D(ty, tx)]);
	//1,0
	sD[threadIdx.x] -= confx[cals_coo2D(typ, tx)] * ( BXMxx * confx[cals_coo2D(typ, bx)] + BYMxx * confx[cals_coo2D(ty, tx)] + BWMxx * confx[cals_coo2D(ty, bx)])\
		         +confx[cals_coo2D(typ, tx)] * ( BXMxy * confy[cals_coo2D(typ, bx)] + BYMxy * confy[cals_coo2D(ty, tx)] + BWMxy * confy[cals_coo2D(ty, bx)])\
		         +confx[cals_coo2D(typ, tx)] * ( cBXMxz * confz[cals_coo2D(typ, bx)] + cBYMxz * confz[cals_coo2D(ty, tx)] + cBWMxz * confz[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( BXMyx * confx[cals_coo2D(typ, bx)] + BYMyx * confx[cals_coo2D(ty, tx)] + BWMyx * confx[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( BXMyy * confy[cals_coo2D(typ, bx)] + BYMyy * confy[cals_coo2D(ty, tx)] + BWMyy * confy[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( cBXMyz * confz[cals_coo2D(typ, bx)] + cBYMyz * confz[cals_coo2D(ty, tx)] + cBWMyz * confz[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( cBXMzx * confx[cals_coo2D(typ, bx)] + cBYMzx * confx[cals_coo2D(ty, tx)] + cBWMzx * confx[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( cBXMzy * confy[cals_coo2D(typ, bx)] + cBYMzy * confy[cals_coo2D(ty, tx)] + cBWMzy * confy[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( BXMzz * confz[cals_coo2D(typ, bx)] + BYMzz * confz[cals_coo2D(ty, tx)] + BWMzz * confz[cals_coo2D(ty, bx)] - cals_A * confz[cals_coo2D((ty+1), tx)]);
	//2,0
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, tx)] * ( BXMxx * confx[cals_coo2D(typ2, bx)] + BYMxx * confx[cals_coo2D(typ, tx)] + BWMxx * confx[cals_coo2D(typ, bx)])\
		         +confx[cals_coo2D(typ2, tx)] * ( BXMxy * confy[cals_coo2D(typ2, bx)] + BYMxy * confy[cals_coo2D(typ, tx)] + BWMxy * confy[cals_coo2D(typ, bx)])\
		         +confx[cals_coo2D(typ2, tx)] * ( cBXMxz * confz[cals_coo2D(typ2, bx)] + cBYMxz * confz[cals_coo2D(typ, tx)] + cBWMxz * confz[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( BXMyx * confx[cals_coo2D(typ2, bx)] + BYMyx * confx[cals_coo2D(typ, tx)] + BWMyx * confx[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( BXMyy * confy[cals_coo2D(typ2, bx)] + BYMyy * confy[cals_coo2D(typ, tx)] + BWMyy * confy[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( cBXMyz * confz[cals_coo2D(typ2, bx)] + cBYMyz * confz[cals_coo2D(typ, tx)] + cBWMyz * confz[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( cBXMzx * confx[cals_coo2D(typ2, bx)] + cBYMzx * confx[cals_coo2D(typ, tx)] + cBWMzx * confx[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( cBXMzy * confy[cals_coo2D(typ2, bx)] + cBYMzy * confy[cals_coo2D(typ, tx)] + cBWMzy * confy[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( BXMzz * confz[cals_coo2D(typ2, bx)] + BYMzz * confz[cals_coo2D(typ, tx)] + BWMzz * confz[cals_coo2D(typ, bx)] - cals_A * confz[cals_coo2D((ty+1), tx)]);
	//0,1
	sD[threadIdx.x] -= confx[cals_coo2D(ty, txp)] * ( BXMxx * confx[cals_coo2D(ty, tx)] + BYMxx * confx[cals_coo2D(by, txp)] + BWMxx * confx[cals_coo2D(by, tx)])\
		         +confx[cals_coo2D(ty, txp)] * ( BXMxy * confy[cals_coo2D(ty, tx)] + BYMxy * confy[cals_coo2D(by, txp)] + BWMxy * confy[cals_coo2D(by, tx)])\
		         +confx[cals_coo2D(ty, txp)] * ( cBXMxz * confz[cals_coo2D(ty, tx)] + cBYMxz * confz[cals_coo2D(by, txp)] + cBWMxz * confz[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( BXMyx * confx[cals_coo2D(ty, tx)] + BYMyx * confx[cals_coo2D(by, txp)] + BWMyx * confx[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( BXMyy * confy[cals_coo2D(ty, tx)] + BYMyy * confy[cals_coo2D(by, txp)] + BWMyy * confy[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( cBXMyz * confz[cals_coo2D(ty, tx)] + cBYMyz * confz[cals_coo2D(by, txp)] + cBWMyz * confz[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( cBXMzx * confx[cals_coo2D(ty, tx)] + cBYMzx * confx[cals_coo2D(by, txp)] + cBWMzx * confx[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( cBXMzy * confy[cals_coo2D(ty, tx)] + cBYMzy * confy[cals_coo2D(by, txp)] + cBWMzy * confy[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( BXMzz * confz[cals_coo2D(ty, tx)] + BYMzz * confz[cals_coo2D(by, txp)] + BWMzz * confz[cals_coo2D(by, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//1,1
	sD[threadIdx.x] -= confx[cals_coo2D(typ, txp)] * ( BXMxx * confx[cals_coo2D(typ, tx)] + BYMxx * confx[cals_coo2D(ty, txp)] + BWMxx * confx[cals_coo2D(ty, tx)])\
		         +confx[cals_coo2D(typ, txp)] * ( BXMxy * confy[cals_coo2D(typ, tx)] + BYMxy * confy[cals_coo2D(ty, txp)] + BWMxy * confy[cals_coo2D(ty, tx)])\
		         +confx[cals_coo2D(typ, txp)] * ( cBXMxz * confz[cals_coo2D(typ, tx)] + cBYMxz * confz[cals_coo2D(ty, txp)] + cBWMxz * confz[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( BXMyx * confx[cals_coo2D(typ, tx)] + BYMyx * confx[cals_coo2D(ty, txp)] + BWMyx * confx[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( BXMyy * confy[cals_coo2D(typ, tx)] + BYMyy * confy[cals_coo2D(ty, txp)] + BWMyy * confy[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( cBXMyz * confz[cals_coo2D(typ, tx)] + cBYMyz * confz[cals_coo2D(ty, txp)] + cBWMyz * confz[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( cBXMzx * confx[cals_coo2D(typ, tx)] + cBYMzx * confx[cals_coo2D(ty, txp)] + cBWMzx * confx[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( cBXMzy * confy[cals_coo2D(typ, tx)] + cBYMzy * confy[cals_coo2D(ty, txp)] + cBWMzy * confy[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( BXMzz * confz[cals_coo2D(typ, tx)] + BYMzz * confz[cals_coo2D(ty, txp)] + BWMzz * confz[cals_coo2D(ty, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//2,1
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, txp)] * ( BXMxx * confx[cals_coo2D(typ2, tx)] + BYMxx * confx[cals_coo2D(typ, txp)] + BWMxx * confx[cals_coo2D(typ, tx)])\
		         +confx[cals_coo2D(typ2, txp)] * ( BXMxy * confy[cals_coo2D(typ2, tx)] + BYMxy * confy[cals_coo2D(typ, txp)] + BWMxy * confy[cals_coo2D(typ, tx)])\
		         +confx[cals_coo2D(typ2, txp)] * ( cBXMxz * confz[cals_coo2D(typ2, tx)] + cBYMxz * confz[cals_coo2D(typ, txp)] + cBWMxz * confz[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( BXMyx * confx[cals_coo2D(typ2, tx)] + BYMyx * confx[cals_coo2D(typ, txp)] + BWMyx * confx[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( BXMyy * confy[cals_coo2D(typ2, tx)] + BYMyy * confy[cals_coo2D(typ, txp)] + BWMyy * confy[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( cBXMyz * confz[cals_coo2D(typ2, tx)] + cBYMyz * confz[cals_coo2D(typ, txp)] + cBWMyz * confz[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( cBXMzx * confx[cals_coo2D(typ2, tx)] + cBYMzx * confx[cals_coo2D(typ, txp)] + cBWMzx * confx[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( cBXMzy * confy[cals_coo2D(typ2, tx)] + cBYMzy * confy[cals_coo2D(typ, txp)] + cBWMzy * confy[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( BXMzz * confz[cals_coo2D(typ2, tx)] + BYMzz * confz[cals_coo2D(typ, txp)] + BWMzz * confz[cals_coo2D(typ, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//0,2
	sD[threadIdx.x] -= confx[cals_coo2D(ty, txp2)] * ( BXMxx * confx[cals_coo2D(ty, txp)] + BYMxx * confx[cals_coo2D(by, txp2)] + BWMxx * confx[cals_coo2D(by, txp)])\
		         +confx[cals_coo2D(ty, txp2)] * ( BXMxy * confy[cals_coo2D(ty, txp)] + BYMxy * confy[cals_coo2D(by, txp2)] + BWMxy * confy[cals_coo2D(by, txp)])\
		         +confx[cals_coo2D(ty, txp2)] * ( cBXMxz * confz[cals_coo2D(ty, txp)] + cBYMxz * confz[cals_coo2D(by, txp2)] + cBWMxz * confz[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( BXMyx * confx[cals_coo2D(ty, txp)] + BYMyx * confx[cals_coo2D(by, txp2)] + BWMyx * confx[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( BXMyy * confy[cals_coo2D(ty, txp)] + BYMyy * confy[cals_coo2D(by, txp2)] + BWMyy * confy[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( cBXMyz * confz[cals_coo2D(ty, txp)] + cBYMyz * confz[cals_coo2D(by, txp2)] + cBWMyz * confz[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( cBXMzx * confx[cals_coo2D(ty, txp)] + cBYMzx * confx[cals_coo2D(by, txp2)] + cBWMzx * confx[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( cBXMzy * confy[cals_coo2D(ty, txp)] + cBYMzy * confy[cals_coo2D(by, txp2)] + cBWMzy * confy[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( BXMzz * confz[cals_coo2D(ty, txp)] + BYMzz * confz[cals_coo2D(by, txp2)] + BWMzz * confz[cals_coo2D(by, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//1,2
	sD[threadIdx.x] -= confx[cals_coo2D(typ, txp2)] * ( BXMxx * confx[cals_coo2D(typ, txp)] + BYMxx * confx[cals_coo2D(ty, txp2)] + BWMxx * confx[cals_coo2D(ty, txp)])\
		         +confx[cals_coo2D(typ, txp2)] * ( BXMxy * confy[cals_coo2D(typ, txp)] + BYMxy * confy[cals_coo2D(ty, txp2)] + BWMxy * confy[cals_coo2D(ty, txp)])\
		         +confx[cals_coo2D(typ, txp2)] * ( cBXMxz * confz[cals_coo2D(typ, txp)] + cBYMxz * confz[cals_coo2D(ty, txp2)] + cBWMxz * confz[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( BXMyx * confx[cals_coo2D(typ, txp)] + BYMyx * confx[cals_coo2D(ty, txp2)] + BWMyx * confx[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( BXMyy * confy[cals_coo2D(typ, txp)] + BYMyy * confy[cals_coo2D(ty, txp2)] + BWMyy * confy[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( cBXMyz * confz[cals_coo2D(typ, txp)] + cBYMyz * confz[cals_coo2D(ty, txp2)] + cBWMyz * confz[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( cBXMzx * confx[cals_coo2D(typ, txp)] + cBYMzx * confx[cals_coo2D(ty, txp2)] + cBWMzx * confx[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( cBXMzy * confy[cals_coo2D(typ, txp)] + cBYMzy * confy[cals_coo2D(ty, txp2)] + cBWMzy * confy[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( BXMzz * confz[cals_coo2D(typ, txp)] + BYMzz * confz[cals_coo2D(ty, txp2)] + BWMzz * confz[cals_coo2D(ty, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//2,2
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, txp2)] * ( BXMxx * confx[cals_coo2D(typ2, txp)] + BYMxx * confx[cals_coo2D(typ, txp2)] + BWMxx * confx[cals_coo2D(typ, txp)])\
		         +confx[cals_coo2D(typ2, txp2)] * ( BXMxy * confy[cals_coo2D(typ2, txp)] + BYMxy * confy[cals_coo2D(typ, txp2)] + BWMxy * confy[cals_coo2D(typ, txp)])\
		         +confx[cals_coo2D(typ2, txp2)] * ( cBXMxz * confz[cals_coo2D(typ2, txp)] + cBYMxz * confz[cals_coo2D(typ, txp2)] + cBWMxz * confz[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( BXMyx * confx[cals_coo2D(typ2, txp)] + BYMyx * confx[cals_coo2D(typ, txp2)] + BWMyx * confx[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( BXMyy * confy[cals_coo2D(typ2, txp)] + BYMyy * confy[cals_coo2D(typ, txp2)] + BWMyy * confy[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( cBXMyz * confz[cals_coo2D(typ2, txp)] + cBYMyz * confz[cals_coo2D(typ, txp2)] + cBWMyz * confz[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( cBXMzx * confx[cals_coo2D(typ2, txp)] + cBYMzx * confx[cals_coo2D(typ, txp2)] + cBWMzx * confx[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( cBXMzy * confy[cals_coo2D(typ2, txp)] + cBYMzy * confy[cals_coo2D(typ, txp2)] + cBWMzy * confy[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( BXMzz * confz[cals_coo2D(typ2, txp)] + BYMzz * confz[cals_coo2D(typ, txp2)] + BWMzz * confz[cals_coo2D(typ, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	__syncthreads();


	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN)] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 2*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confz[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confz[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 3*cals_BN] = sD[0];
	__syncthreads();

	//try to measure Chern number
	//(0,0)
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)] * (
	 (confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])*(2*confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(by, tx)]-confz[cals_coo2D(by, bx)])
	-(confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])*(2*confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(by, tx)]-confy[cals_coo2D(by, bx)])
	)+confy[cals_coo2D(ty, tx)] * (
	 (confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])*(2*confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(by, tx)]-confx[cals_coo2D(by, bx)])
	-(confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])*(2*confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(by, tx)]-confz[cals_coo2D(by, bx)])
	)+confz[cals_coo2D(ty, tx)] * (
	 (confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])*(2*confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(by, tx)]-confy[cals_coo2D(by, bx)])
	-(confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])*(2*confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(by, tx)]-confx[cals_coo2D(by, bx)])
	);
	//(1,0)
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)] * (
	 (confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])*(2*confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])
	-(confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])*(2*confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])
	)+confy[cals_coo2D(typ, tx)]*(
	 (confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])*(2*confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])
	-(confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])*(2*confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])
	)+confz[cals_coo2D(typ, tx)] * (
	 (confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])*(2*confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])
	-(confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])*(2*confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])
	);
	//(2,0)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)] * (
	 (confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ2, bx)])*(2*confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])
	-(confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ2, bx)])*(2*confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])
	)+confy[cals_coo2D(typ2, tx)]*(
	 (confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ2, bx)])*(2*confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])
	-(confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ2, bx)])*(2*confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])
	)+confz[cals_coo2D(typ2, tx)] * (
	 (confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ2, bx)])*(2*confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])
	-(confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ2, bx)])*(2*confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])
	);
	//(0,1)
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)] * (
	 (confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])*(2*confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(by, txp)]-confz[cals_coo2D(by, tx)])
	-(confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])*(2*confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(by, txp)]-confy[cals_coo2D(by, tx)])
	)+confy[cals_coo2D(ty, txp)]*(
	 (confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])*(2*confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(by, txp)]-confx[cals_coo2D(by, tx)])
	-(confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])*(2*confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(by, txp)]-confz[cals_coo2D(by, tx)])
	)+confz[cals_coo2D(ty, txp)] * (
	 (confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])*(2*confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(by, txp)]-confy[cals_coo2D(by, tx)])
	-(confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])*(2*confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(by, txp)]-confx[cals_coo2D(by, tx)])
	);
	//(1,1)
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)] * (
	 (confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])*(2*confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])
	-(confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])*(2*confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])
	)+confy[cals_coo2D(typ, txp)]*(
	 (confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])*(2*confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])
	-(confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])*(2*confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])
	)+confz[cals_coo2D(typ, txp)] * (
	 (confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])*(2*confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])
	-(confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])*(2*confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])
	);
	//(2,1)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)] * (
	 (confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ2, tx)])*(2*confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])
	-(confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ2, tx)])*(2*confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])
	)+confy[cals_coo2D(typ2, txp)]*(
	 (confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ2, tx)])*(2*confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])
	-(confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ2, tx)])*(2*confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])
	)+confz[cals_coo2D(typ2, txp)] * (
	 (confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ2, tx)])*(2*confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])
	-(confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ2, tx)])*(2*confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])
	);
	//(0,2)
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)] * (
	 (confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])*(2*confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(by, txp2)]-confz[cals_coo2D(by, txp)])
	-(confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])*(2*confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(by, txp2)]-confy[cals_coo2D(by, txp)])
	)+confy[cals_coo2D(ty, txp2)]*(
	 (confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])*(2*confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(by, txp2)]-confx[cals_coo2D(by, txp)])
	-(confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])*(2*confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(by, txp2)]-confz[cals_coo2D(by, txp)])
	)+confz[cals_coo2D(ty, txp2)] * (
	 (confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])*(2*confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(by, txp2)]-confy[cals_coo2D(by, txp)])
	-(confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])*(2*confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(by, txp2)]-confx[cals_coo2D(by, txp)])
	);
	//(1,2)
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)] * (
	 (confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])*(2*confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])
	-(confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])*(2*confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])
	)+confy[cals_coo2D(typ, txp2)]*(
	 (confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])*(2*confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])
	-(confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])*(2*confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])
	)+confz[cals_coo2D(typ, txp2)] * (
	 (confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])*(2*confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])
	-(confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])*(2*confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])
	);
	//(2,2)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * (
	 (confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ2, txp)])*(2*confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])
	-(confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ2, txp)])*(2*confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])
	)+confy[cals_coo2D(typ2, txp2)]*(
	 (confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ2, txp)])*(2*confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])
	-(confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ2, txp)])*(2*confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])
	)+confz[cals_coo2D(typ2, txp2)] * (
	 (confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ2, txp)])*(2*confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])
	-(confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ2, txp)])*(2*confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])
	);
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 4*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 5*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 6*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo2D(ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 7*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 8*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 9*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo2D(ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 10*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 11*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 12*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo2D(ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 13*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 14*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 15*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo2D(ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo2D(ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo2D(typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo2D(typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 16*cals_BN] = sD[0];
	__syncthreads();
}
#endif
