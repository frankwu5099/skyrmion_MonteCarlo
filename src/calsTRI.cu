#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__constant__ unsigned int cals_SpinSize;
__constant__ unsigned int cals_SpinSize_z;
__constant__ unsigned int cals_BlockSize_x;
__constant__ unsigned int cals_BlockSize_y;
__constant__ unsigned int cals_GridSize_x;
__constant__ unsigned int cals_GridSize_y;
__constant__ unsigned int cals_TN;
__constant__ unsigned int cals_BN;
__constant__ float cals_A; //(0.0)
void move_params_device_cals(){
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BN), &H_BN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_A ), &H_A , sizeof(float));
}
__global__ void calTRI(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	extern __shared__ double sD[];
	const int x = threadIdx.x % (cals_BlockSize_x);
	const int y = (threadIdx.x / cals_BlockSize_x);
	const int tx = 3 * (((blockIdx.x % cals_BN) % cals_GridSize_x) * cals_BlockSize_x + x);
	const int ty =(blockIdx.x / cals_BN) * cals_SpinSize +  3 * ((((blockIdx.x % cals_BN) / cals_GridSize_x) % cals_GridSize_y) * cals_BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	const int txp2 = tx +2 ;
	const int typ2 = ty +2 ;
	//const int ty = 2 * ((blockIdx.x / cals_BN) * cals_SpinSize + ((blockIdx.x % cals_BN) / cals_GridSize_x) * cals_BlockSize_y + y);
	const int dataoff = (blockIdx.x / cals_BN) * MEASURE_NUM * cals_BN;
	int bx, by;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + cals_SpinSize - 1) % cals_SpinSize;
	if((ty % cals_SpinSize) == 0)	by = ty + cals_SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.

	//0,0
	sD[threadIdx.x] = -confx[cals_coo2D(ty, tx)] * ( BXMxx * confx[cals_coo2D(ty, bx)] + BYMxx * confx[cals_coo2D(by, tx)] + BWMxx * confx[cals_coo2D(by, bx)])\
	           -confx[cals_coo2D(ty, tx)] * ( BXMxy * confy[cals_coo2D(ty, bx)] + BYMxy * confy[cals_coo2D(by, tx)] + BWMxy * confy[cals_coo2D(by, bx)])\
	           -confx[cals_coo2D(ty, tx)] * ( BXMxz * confz[cals_coo2D(ty, bx)] + BYMxz * confz[cals_coo2D(by, tx)] + BWMxz * confz[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( BXMyx * confx[cals_coo2D(ty, bx)] + BYMyx * confx[cals_coo2D(by, tx)] + BWMyx * confx[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( BXMyy * confy[cals_coo2D(ty, bx)] + BYMyy * confy[cals_coo2D(by, tx)] + BWMyy * confy[cals_coo2D(by, bx)])\
		         -confy[cals_coo2D(ty, tx)] * ( BXMyz * confz[cals_coo2D(ty, bx)] + BYMyz * confz[cals_coo2D(by, tx)] + BWMyz * confz[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( BXMzx * confx[cals_coo2D(ty, bx)] + BYMzx * confx[cals_coo2D(by, tx)] + BWMzx * confx[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( BXMzy * confy[cals_coo2D(ty, bx)] + BYMzy * confy[cals_coo2D(by, tx)] + BWMzy * confy[cals_coo2D(by, bx)])\
		         -confz[cals_coo2D(ty, tx)] * ( BXMzz * confz[cals_coo2D(ty, bx)] + BYMzz * confz[cals_coo2D(by, tx)] + BWMzz * confz[cals_coo2D(by, bx)] - cals_A * confz[cals_coo2D(ty, tx)]);
	//1,0
	sD[threadIdx.x] -= confx[cals_coo2D(typ, tx)] * ( BXMxx * confx[cals_coo2D(typ, bx)] + BYMxx * confx[cals_coo2D(ty, tx)] + BWMxx * confx[cals_coo2D(ty, bx)])\
		         +confx[cals_coo2D(typ, tx)] * ( BXMxy * confy[cals_coo2D(typ, bx)] + BYMxy * confy[cals_coo2D(ty, tx)] + BWMxy * confy[cals_coo2D(ty, bx)])\
		         +confx[cals_coo2D(typ, tx)] * ( BXMxz * confz[cals_coo2D(typ, bx)] + BYMxz * confz[cals_coo2D(ty, tx)] + BWMxz * confz[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( BXMyx * confx[cals_coo2D(typ, bx)] + BYMyx * confx[cals_coo2D(ty, tx)] + BWMyx * confx[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( BXMyy * confy[cals_coo2D(typ, bx)] + BYMyy * confy[cals_coo2D(ty, tx)] + BWMyy * confy[cals_coo2D(ty, bx)])\
		         +confy[cals_coo2D(typ, tx)] * ( BXMyz * confz[cals_coo2D(typ, bx)] + BYMyz * confz[cals_coo2D(ty, tx)] + BWMyz * confz[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( BXMzx * confx[cals_coo2D(typ, bx)] + BYMzx * confx[cals_coo2D(ty, tx)] + BWMzx * confx[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( BXMzy * confy[cals_coo2D(typ, bx)] + BYMzy * confy[cals_coo2D(ty, tx)] + BWMzy * confy[cals_coo2D(ty, bx)])\
		         +confz[cals_coo2D(typ, tx)] * ( BXMzz * confz[cals_coo2D(typ, bx)] + BYMzz * confz[cals_coo2D(ty, tx)] + BWMzz * confz[cals_coo2D(ty, bx)] - cals_A * confz[cals_coo2D((ty+1), tx)]);
	//2,0
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, tx)] * ( BXMxx * confx[cals_coo2D(typ2, bx)] + BYMxx * confx[cals_coo2D(typ, tx)] + BWMxx * confx[cals_coo2D(typ, bx)])\
		         +confx[cals_coo2D(typ2, tx)] * ( BXMxy * confy[cals_coo2D(typ2, bx)] + BYMxy * confy[cals_coo2D(typ, tx)] + BWMxy * confy[cals_coo2D(typ, bx)])\
		         +confx[cals_coo2D(typ2, tx)] * ( BXMxz * confz[cals_coo2D(typ2, bx)] + BYMxz * confz[cals_coo2D(typ, tx)] + BWMxz * confz[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( BXMyx * confx[cals_coo2D(typ2, bx)] + BYMyx * confx[cals_coo2D(typ, tx)] + BWMyx * confx[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( BXMyy * confy[cals_coo2D(typ2, bx)] + BYMyy * confy[cals_coo2D(typ, tx)] + BWMyy * confy[cals_coo2D(typ, bx)])\
		         +confy[cals_coo2D(typ2, tx)] * ( BXMyz * confz[cals_coo2D(typ2, bx)] + BYMyz * confz[cals_coo2D(typ, tx)] + BWMyz * confz[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( BXMzx * confx[cals_coo2D(typ2, bx)] + BYMzx * confx[cals_coo2D(typ, tx)] + BWMzx * confx[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( BXMzy * confy[cals_coo2D(typ2, bx)] + BYMzy * confy[cals_coo2D(typ, tx)] + BWMzy * confy[cals_coo2D(typ, bx)])\
		         +confz[cals_coo2D(typ2, tx)] * ( BXMzz * confz[cals_coo2D(typ2, bx)] + BYMzz * confz[cals_coo2D(typ, tx)] + BWMzz * confz[cals_coo2D(typ, bx)] - cals_A * confz[cals_coo2D((ty+1), tx)]);
	//0,1
	sD[threadIdx.x] -= confx[cals_coo2D(ty, txp)] * ( BXMxx * confx[cals_coo2D(ty, tx)] + BYMxx * confx[cals_coo2D(by, txp)] + BWMxx * confx[cals_coo2D(by, tx)])\
		         +confx[cals_coo2D(ty, txp)] * ( BXMxy * confy[cals_coo2D(ty, tx)] + BYMxy * confy[cals_coo2D(by, txp)] + BWMxy * confy[cals_coo2D(by, tx)])\
		         +confx[cals_coo2D(ty, txp)] * ( BXMxz * confz[cals_coo2D(ty, tx)] + BYMxz * confz[cals_coo2D(by, txp)] + BWMxz * confz[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( BXMyx * confx[cals_coo2D(ty, tx)] + BYMyx * confx[cals_coo2D(by, txp)] + BWMyx * confx[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( BXMyy * confy[cals_coo2D(ty, tx)] + BYMyy * confy[cals_coo2D(by, txp)] + BWMyy * confy[cals_coo2D(by, tx)])\
		         +confy[cals_coo2D(ty, txp)] * ( BXMyz * confz[cals_coo2D(ty, tx)] + BYMyz * confz[cals_coo2D(by, txp)] + BWMyz * confz[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( BXMzx * confx[cals_coo2D(ty, tx)] + BYMzx * confx[cals_coo2D(by, txp)] + BWMzx * confx[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( BXMzy * confy[cals_coo2D(ty, tx)] + BYMzy * confy[cals_coo2D(by, txp)] + BWMzy * confy[cals_coo2D(by, tx)])\
		         +confz[cals_coo2D(ty, txp)] * ( BXMzz * confz[cals_coo2D(ty, tx)] + BYMzz * confz[cals_coo2D(by, txp)] + BWMzz * confz[cals_coo2D(by, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//1,1
	sD[threadIdx.x] -= confx[cals_coo2D(typ, txp)] * ( BXMxx * confx[cals_coo2D(typ, tx)] + BYMxx * confx[cals_coo2D(ty, txp)] + BWMxx * confx[cals_coo2D(ty, tx)])\
		         +confx[cals_coo2D(typ, txp)] * ( BXMxy * confy[cals_coo2D(typ, tx)] + BYMxy * confy[cals_coo2D(ty, txp)] + BWMxy * confy[cals_coo2D(ty, tx)])\
		         +confx[cals_coo2D(typ, txp)] * ( BXMxz * confz[cals_coo2D(typ, tx)] + BYMxz * confz[cals_coo2D(ty, txp)] + BWMxz * confz[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( BXMyx * confx[cals_coo2D(typ, tx)] + BYMyx * confx[cals_coo2D(ty, txp)] + BWMyx * confx[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( BXMyy * confy[cals_coo2D(typ, tx)] + BYMyy * confy[cals_coo2D(ty, txp)] + BWMyy * confy[cals_coo2D(ty, tx)])\
		         +confy[cals_coo2D(typ, txp)] * ( BXMyz * confz[cals_coo2D(typ, tx)] + BYMyz * confz[cals_coo2D(ty, txp)] + BWMyz * confz[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( BXMzx * confx[cals_coo2D(typ, tx)] + BYMzx * confx[cals_coo2D(ty, txp)] + BWMzx * confx[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( BXMzy * confy[cals_coo2D(typ, tx)] + BYMzy * confy[cals_coo2D(ty, txp)] + BWMzy * confy[cals_coo2D(ty, tx)])\
		         +confz[cals_coo2D(typ, txp)] * ( BXMzz * confz[cals_coo2D(typ, tx)] + BYMzz * confz[cals_coo2D(ty, txp)] + BWMzz * confz[cals_coo2D(ty, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//2,1
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, txp)] * ( BXMxx * confx[cals_coo2D(typ2, tx)] + BYMxx * confx[cals_coo2D(typ, txp)] + BWMxx * confx[cals_coo2D(typ, tx)])\
		         +confx[cals_coo2D(typ2, txp)] * ( BXMxy * confy[cals_coo2D(typ2, tx)] + BYMxy * confy[cals_coo2D(typ, txp)] + BWMxy * confy[cals_coo2D(typ, tx)])\
		         +confx[cals_coo2D(typ2, txp)] * ( BXMxz * confz[cals_coo2D(typ2, tx)] + BYMxz * confz[cals_coo2D(typ, txp)] + BWMxz * confz[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( BXMyx * confx[cals_coo2D(typ2, tx)] + BYMyx * confx[cals_coo2D(typ, txp)] + BWMyx * confx[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( BXMyy * confy[cals_coo2D(typ2, tx)] + BYMyy * confy[cals_coo2D(typ, txp)] + BWMyy * confy[cals_coo2D(typ, tx)])\
		         +confy[cals_coo2D(typ2, txp)] * ( BXMyz * confz[cals_coo2D(typ2, tx)] + BYMyz * confz[cals_coo2D(typ, txp)] + BWMyz * confz[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( BXMzx * confx[cals_coo2D(typ2, tx)] + BYMzx * confx[cals_coo2D(typ, txp)] + BWMzx * confx[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( BXMzy * confy[cals_coo2D(typ2, tx)] + BYMzy * confy[cals_coo2D(typ, txp)] + BWMzy * confy[cals_coo2D(typ, tx)])\
		         +confz[cals_coo2D(typ2, txp)] * ( BXMzz * confz[cals_coo2D(typ2, tx)] + BYMzz * confz[cals_coo2D(typ, txp)] + BWMzz * confz[cals_coo2D(typ, tx)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//0,2
	sD[threadIdx.x] -= confx[cals_coo2D(ty, txp2)] * ( BXMxx * confx[cals_coo2D(ty, txp)] + BYMxx * confx[cals_coo2D(by, txp2)] + BWMxx * confx[cals_coo2D(by, txp)])\
		         +confx[cals_coo2D(ty, txp2)] * ( BXMxy * confy[cals_coo2D(ty, txp)] + BYMxy * confy[cals_coo2D(by, txp2)] + BWMxy * confy[cals_coo2D(by, txp)])\
		         +confx[cals_coo2D(ty, txp2)] * ( BXMxz * confz[cals_coo2D(ty, txp)] + BYMxz * confz[cals_coo2D(by, txp2)] + BWMxz * confz[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( BXMyx * confx[cals_coo2D(ty, txp)] + BYMyx * confx[cals_coo2D(by, txp2)] + BWMyx * confx[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( BXMyy * confy[cals_coo2D(ty, txp)] + BYMyy * confy[cals_coo2D(by, txp2)] + BWMyy * confy[cals_coo2D(by, txp)])\
		         +confy[cals_coo2D(ty, txp2)] * ( BXMyz * confz[cals_coo2D(ty, txp)] + BYMyz * confz[cals_coo2D(by, txp2)] + BWMyz * confz[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( BXMzx * confx[cals_coo2D(ty, txp)] + BYMzx * confx[cals_coo2D(by, txp2)] + BWMzx * confx[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( BXMzy * confy[cals_coo2D(ty, txp)] + BYMzy * confy[cals_coo2D(by, txp2)] + BWMzy * confy[cals_coo2D(by, txp)])\
		         +confz[cals_coo2D(ty, txp2)] * ( BXMzz * confz[cals_coo2D(ty, txp)] + BYMzz * confz[cals_coo2D(by, txp2)] + BWMzz * confz[cals_coo2D(by, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//1,2
	sD[threadIdx.x] -= confx[cals_coo2D(typ, txp2)] * ( BXMxx * confx[cals_coo2D(typ, txp)] + BYMxx * confx[cals_coo2D(ty, txp2)] + BWMxx * confx[cals_coo2D(ty, txp)])\
		         +confx[cals_coo2D(typ, txp2)] * ( BXMxy * confy[cals_coo2D(typ, txp)] + BYMxy * confy[cals_coo2D(ty, txp2)] + BWMxy * confy[cals_coo2D(ty, txp)])\
		         +confx[cals_coo2D(typ, txp2)] * ( BXMxz * confz[cals_coo2D(typ, txp)] + BYMxz * confz[cals_coo2D(ty, txp2)] + BWMxz * confz[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( BXMyx * confx[cals_coo2D(typ, txp)] + BYMyx * confx[cals_coo2D(ty, txp2)] + BWMyx * confx[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( BXMyy * confy[cals_coo2D(typ, txp)] + BYMyy * confy[cals_coo2D(ty, txp2)] + BWMyy * confy[cals_coo2D(ty, txp)])\
		         +confy[cals_coo2D(typ, txp2)] * ( BXMyz * confz[cals_coo2D(typ, txp)] + BYMyz * confz[cals_coo2D(ty, txp2)] + BWMyz * confz[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( BXMzx * confx[cals_coo2D(typ, txp)] + BYMzx * confx[cals_coo2D(ty, txp2)] + BWMzx * confx[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( BXMzy * confy[cals_coo2D(typ, txp)] + BYMzy * confy[cals_coo2D(ty, txp2)] + BWMzy * confy[cals_coo2D(ty, txp)])\
		         +confz[cals_coo2D(typ, txp2)] * ( BXMzz * confz[cals_coo2D(typ, txp)] + BYMzz * confz[cals_coo2D(ty, txp2)] + BWMzz * confz[cals_coo2D(ty, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	//2,2
	sD[threadIdx.x] -= confx[cals_coo2D(typ2, txp2)] * ( BXMxx * confx[cals_coo2D(typ2, txp)] + BYMxx * confx[cals_coo2D(typ, txp2)] + BWMxx * confx[cals_coo2D(typ, txp)])\
		         +confx[cals_coo2D(typ2, txp2)] * ( BXMxy * confy[cals_coo2D(typ2, txp)] + BYMxy * confy[cals_coo2D(typ, txp2)] + BWMxy * confy[cals_coo2D(typ, txp)])\
		         +confx[cals_coo2D(typ2, txp2)] * ( BXMxz * confz[cals_coo2D(typ2, txp)] + BYMxz * confz[cals_coo2D(typ, txp2)] + BWMxz * confz[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( BXMyx * confx[cals_coo2D(typ2, txp)] + BYMyx * confx[cals_coo2D(typ, txp2)] + BWMyx * confx[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( BXMyy * confy[cals_coo2D(typ2, txp)] + BYMyy * confy[cals_coo2D(typ, txp2)] + BWMyy * confy[cals_coo2D(typ, txp)])\
		         +confy[cals_coo2D(typ2, txp2)] * ( BXMyz * confz[cals_coo2D(typ2, txp)] + BYMyz * confz[cals_coo2D(typ, txp2)] + BWMyz * confz[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( BXMzx * confx[cals_coo2D(typ2, txp)] + BYMzx * confx[cals_coo2D(typ, txp2)] + BWMzx * confx[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( BXMzy * confy[cals_coo2D(typ2, txp)] + BYMzy * confy[cals_coo2D(typ, txp2)] + BWMzy * confy[cals_coo2D(typ, txp)])\
		         +confz[cals_coo2D(typ2, txp2)] * ( BXMzz * confz[cals_coo2D(typ2, txp)] + BYMzz * confz[cals_coo2D(typ, txp2)] + BWMzz * confz[cals_coo2D(typ, txp)] - cals_A * confz[cals_coo2D(ty, tx+1)]);
	__syncthreads();


	//Sum over all elements in each sD
	if(cals_TN>=512){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>=256){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>=128){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN)] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>=512){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>=256){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>=128){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>=512){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>=256){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>=128){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 2*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo2D(ty, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, tx)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp)];
	sD[threadIdx.x] += confy[cals_coo2D(ty, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ, txp2)];
	sD[threadIdx.x] += confy[cals_coo2D(typ2, txp2)];
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>=512){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>=256){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>=128){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 3*cals_BN] = sD[0];
	__syncthreads();

	//try to measure Chern number
	//(0,0)
	sD[threadIdx.x]  = confx[cals_coo2D(ty, tx)] * (
	 (confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])*(2*confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(by, tx)]-confz[cals_coo2D(by, bx)])
	-(confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])*(2*confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(by, tx)]-confy[cals_coo2D(by, bx)])
	)+confy[cals_coo2D(ty, tx)] * (
	 (confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])*(2*confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(by, tx)]-confx[cals_coo2D(by, bx)])
	-(confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])*(2*confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(by, tx)]-confz[cals_coo2D(by, bx)])
	)+confz[cals_coo2D(ty, tx)] * (
	 (confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])*(2*confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(by, tx)]-confy[cals_coo2D(by, bx)])
	-(confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])*(2*confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(by, tx)]-confx[cals_coo2D(by, bx)])
	);
	//(1,0)
	sD[threadIdx.x] += confx[cals_coo2D(typ, tx)] * (
	 (confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])*(2*confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])
	-(confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])*(2*confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])
	)+confy[cals_coo2D(typ, tx)]*(
	 (confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])*(2*confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])
	-(confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])*(2*confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(ty, tx)]-confz[cals_coo2D(ty, bx)])
	)+confz[cals_coo2D(typ, tx)] * (
	 (confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])*(2*confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(ty, tx)]-confy[cals_coo2D(ty, bx)])
	-(confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])*(2*confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(ty, tx)]-confx[cals_coo2D(ty, bx)])
	);
	//(2,0)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, tx)] * (
	 (confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ2, bx)])*(2*confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])
	-(confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ2, bx)])*(2*confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])
	)+confy[cals_coo2D(typ2, tx)]*(
	 (confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ2, bx)])*(2*confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])
	-(confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ2, bx)])*(2*confz[cals_coo2D(typ2, tx)]-confz[cals_coo2D(typ, tx)]-confz[cals_coo2D(typ, bx)])
	)+confz[cals_coo2D(typ2, tx)] * (
	 (confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ2, bx)])*(2*confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ, tx)]-confy[cals_coo2D(typ, bx)])
	-(confy[cals_coo2D(typ2, tx)]-confy[cals_coo2D(typ2, bx)])*(2*confx[cals_coo2D(typ2, tx)]-confx[cals_coo2D(typ, tx)]-confx[cals_coo2D(typ, bx)])
	);
	//(0,1)
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp)] * (
	 (confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])*(2*confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(by, txp)]-confz[cals_coo2D(by, tx)])
	-(confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])*(2*confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(by, txp)]-confy[cals_coo2D(by, tx)])
	)+confy[cals_coo2D(ty, txp)]*(
	 (confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])*(2*confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(by, txp)]-confx[cals_coo2D(by, tx)])
	-(confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])*(2*confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(by, txp)]-confz[cals_coo2D(by, tx)])
	)+confz[cals_coo2D(ty, txp)] * (
	 (confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])*(2*confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(by, txp)]-confy[cals_coo2D(by, tx)])
	-(confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])*(2*confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(by, txp)]-confx[cals_coo2D(by, tx)])
	);
	//(1,1)
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp)] * (
	 (confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])*(2*confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])
	-(confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])*(2*confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])
	)+confy[cals_coo2D(typ, txp)]*(
	 (confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])*(2*confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])
	-(confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])*(2*confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(ty, txp)]-confz[cals_coo2D(ty, tx)])
	)+confz[cals_coo2D(typ, txp)] * (
	 (confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])*(2*confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(ty, txp)]-confy[cals_coo2D(ty, tx)])
	-(confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])*(2*confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(ty, txp)]-confx[cals_coo2D(ty, tx)])
	);
	//(2,1)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp)] * (
	 (confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ2, tx)])*(2*confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])
	-(confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ2, tx)])*(2*confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])
	)+confy[cals_coo2D(typ2, txp)]*(
	 (confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ2, tx)])*(2*confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])
	-(confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ2, tx)])*(2*confz[cals_coo2D(typ2, txp)]-confz[cals_coo2D(typ, txp)]-confz[cals_coo2D(typ, tx)])
	)+confz[cals_coo2D(typ2, txp)] * (
	 (confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ2, tx)])*(2*confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ, txp)]-confy[cals_coo2D(typ, tx)])
	-(confy[cals_coo2D(typ2, txp)]-confy[cals_coo2D(typ2, tx)])*(2*confx[cals_coo2D(typ2, txp)]-confx[cals_coo2D(typ, txp)]-confx[cals_coo2D(typ, tx)])
	);
	//(0,2)
	sD[threadIdx.x] += confx[cals_coo2D(ty, txp2)] * (
	 (confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])*(2*confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(by, txp2)]-confz[cals_coo2D(by, txp)])
	-(confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])*(2*confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(by, txp2)]-confy[cals_coo2D(by, txp)])
	)+confy[cals_coo2D(ty, txp2)]*(
	 (confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])*(2*confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(by, txp2)]-confx[cals_coo2D(by, txp)])
	-(confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])*(2*confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(by, txp2)]-confz[cals_coo2D(by, txp)])
	)+confz[cals_coo2D(ty, txp2)] * (
	 (confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])*(2*confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(by, txp2)]-confy[cals_coo2D(by, txp)])
	-(confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])*(2*confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(by, txp2)]-confx[cals_coo2D(by, txp)])
	);
	//(1,2)
	sD[threadIdx.x] += confx[cals_coo2D(typ, txp2)] * (
	 (confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])*(2*confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])
	-(confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])*(2*confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])
	)+confy[cals_coo2D(typ, txp2)]*(
	 (confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])*(2*confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])
	-(confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])*(2*confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(ty, txp2)]-confz[cals_coo2D(ty, txp)])
	)+confz[cals_coo2D(typ, txp2)] * (
	 (confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])*(2*confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(ty, txp2)]-confy[cals_coo2D(ty, txp)])
	-(confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])*(2*confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(ty, txp2)]-confx[cals_coo2D(ty, txp)])
	);
	//(2,2)
	sD[threadIdx.x] += confx[cals_coo2D(typ2, txp2)] * (
	 (confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ2, txp)])*(2*confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])
	-(confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ2, txp)])*(2*confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])
	)+confy[cals_coo2D(typ2, txp2)]*(
	 (confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ2, txp)])*(2*confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])
	-(confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ2, txp)])*(2*confz[cals_coo2D(typ2, txp2)]-confz[cals_coo2D(typ, txp2)]-confz[cals_coo2D(typ, txp)])
	)+confz[cals_coo2D(typ2, txp2)] * (
	 (confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ2, txp)])*(2*confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ, txp2)]-confy[cals_coo2D(typ, txp)])
	-(confy[cals_coo2D(typ2, txp2)]-confy[cals_coo2D(typ2, txp)])*(2*confx[cals_coo2D(typ2, txp2)]-confx[cals_coo2D(typ, txp2)]-confx[cals_coo2D(typ, txp)])
	);
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>=512){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>=256){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>=128){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 4*cals_BN] = sD[0];
	__syncthreads();
}
#endif
