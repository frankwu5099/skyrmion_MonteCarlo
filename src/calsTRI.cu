#include "hip/hip_runtime.h"
#include "measurements.cuh"
#ifdef TRI
__constant__ unsigned int cals_SpinSize;
__constant__ unsigned int cals_SpinSize_z;
__constant__ unsigned int cals_BlockSize_x;
__constant__ unsigned int cals_BlockSize_y;
__constant__ unsigned int cals_GridSize_x;
__constant__ unsigned int cals_GridSize_y;
__constant__ unsigned int cals_TN;
__constant__ unsigned int cals_BN;
__constant__ unsigned int cals_Nplane;
__constant__ float Q1x;
__constant__ float Q1y;
__constant__ float Q2x;
__constant__ float Q2y;
__constant__ float cals_A; //(0.0)
__constant__ float cBXPyz;
__constant__ float cBYPyz;
__constant__ float cBWPyz;
__constant__ float cBXMyz;
__constant__ float cBYMyz;
__constant__ float cBWMyz;
__constant__ float cBXPzy;
__constant__ float cBYPzy;
__constant__ float cBWPzy;
__constant__ float cBXMzy;
__constant__ float cBYMzy;
__constant__ float cBWMzy;
__constant__ float cBXPxz;
__constant__ float cBYPxz;
__constant__ float cBWPxz;
__constant__ float cBXMxz;
__constant__ float cBYMxz;
__constant__ float cBWMxz;
__constant__ float cBXPzx;
__constant__ float cBYPzx;
__constant__ float cBWPzx;
__constant__ float cBXMzx;
__constant__ float cBYMzx;
__constant__ float cBWMzx;
__constant__ float cBZMxy;
__constant__ float cBZMyx;
__constant__ float cBZPxy;
__constant__ float cBZPyx;
void move_params_device_cals(){
  float tmpp;
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize), &H_SpinSize, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_SpinSize_z), &H_SpinSize_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_x), &H_BlockSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BlockSize_y), &H_BlockSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_x), &H_GridSize_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_GridSize_y), &H_GridSize_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_Nplane), &H_Nplane, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_TN), &H_TN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_BN), &H_BN, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL( cals_A ), &H_A , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q1x ), &H_Q1x , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q1y ), &H_Q1y , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q2x ), &H_Q2x , sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( Q2y ), &H_Q2y , sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMyz), &tmpp, sizeof(float));
  tmpp = (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMyz), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMyz), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPzy), &tmpp, sizeof(float));
  tmpp =  (0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPzy), &tmpp, sizeof(float));
  tmpp =  (-0.5 * DD - sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPzy), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMzy), &tmpp, sizeof(float));
  tmpp = (-0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMzy), &tmpp, sizeof(float));
  tmpp = (0.5 * DD + sqrt3d2 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMzy), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPxz), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMxz), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMxz), &tmpp, sizeof(float));
  tmpp = (-DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXPzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYPzx), &tmpp, sizeof(float));
  tmpp = (-sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWPzx), &tmpp, sizeof(float));
  tmpp = (DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBXMzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD - 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBYMzx), &tmpp, sizeof(float));
  tmpp = (sqrt3d2 * DD + 0.5 * DR);
  hipMemcpyToSymbol(HIP_SYMBOL( cBWMzx), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBZPxy), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBZPyx), &tmpp, sizeof(float));
  tmpp = (-DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBZMxy), &tmpp, sizeof(float));
  tmpp = (DD);
  hipMemcpyToSymbol(HIP_SYMBOL( cBZMyx), &tmpp, sizeof(float));
}
__global__ void calTRI(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	extern __shared__ double sD[];
	const int x = threadIdx.x % (cals_BlockSize_x);
	const int y = (threadIdx.x / cals_BlockSize_x);
	const int tx = 3 * (((blockIdx.x % cals_BN) % cals_GridSize_x) * cals_BlockSize_x + x);
	const int ty =(blockIdx.x / cals_BN) * cals_SpinSize * cals_SpinSize_z +  3 * ((((blockIdx.x % cals_BN) / cals_GridSize_x) % cals_GridSize_y) * cals_BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	const int txp2 = tx +2 ;
	const int typ2 = ty +2 ;
	int z;
	//const int ty = 2 * ((blockIdx.x / cals_BN) * cals_SpinSize + ((blockIdx.x % cals_BN) / cals_GridSize_x) * cals_BlockSize_y + y);
	const int dataoff = (blockIdx.x / cals_BN) * MEASURE_NUM * cals_BN;
	int bx, by, tx_ty = tx + (ty % cals_SpinSize);
	float Ax, Ay, Az, Bx, By, Bz, Cx, Cy, Cz;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + cals_SpinSize - 1) % cals_SpinSize;
	if((ty % cals_SpinSize) == 0)	by = ty + cals_SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.

	z = 0;
	//0,0
	sD[threadIdx.x] = -confx[cals_coo(z, ty, tx)] * ( BXMxx * confx[cals_coo(z, ty, bx)] + BYMxx * confx[cals_coo(z, by, tx)] + BWMxx * confx[cals_coo(z, by, bx)])\
	           -confx[cals_coo(z, ty, tx)] * ( BXMxy * confy[cals_coo(z, ty, bx)] + BYMxy * confy[cals_coo(z, by, tx)] + BWMxy * confy[cals_coo(z, by, bx)])\
	           -confx[cals_coo(z, ty, tx)] * ( cBXMxz * confz[cals_coo(z, ty, bx)] + cBYMxz * confz[cals_coo(z, by, tx)] + cBWMxz * confz[cals_coo(z, by, bx)])\
		         -confy[cals_coo(z, ty, tx)] * ( BXMyx * confx[cals_coo(z, ty, bx)] + BYMyx * confx[cals_coo(z, by, tx)] + BWMyx * confx[cals_coo(z, by, bx)])\
		         -confy[cals_coo(z, ty, tx)] * ( BXMyy * confy[cals_coo(z, ty, bx)] + BYMyy * confy[cals_coo(z, by, tx)] + BWMyy * confy[cals_coo(z, by, bx)])\
		         -confy[cals_coo(z, ty, tx)] * ( cBXMyz * confz[cals_coo(z, ty, bx)] + cBYMyz * confz[cals_coo(z, by, tx)] + cBWMyz * confz[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( cBXMzx * confx[cals_coo(z, ty, bx)] + cBYMzx * confx[cals_coo(z, by, tx)] + cBWMzx * confx[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( cBXMzy * confy[cals_coo(z, ty, bx)] + cBYMzy * confy[cals_coo(z, by, tx)] + cBWMzy * confy[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( BXMzz * confz[cals_coo(z, ty, bx)] + BYMzz * confz[cals_coo(z, by, tx)] + BWMzz * confz[cals_coo(z, by, bx)] - cals_A * confz[cals_coo(z, ty, tx)]);
	//1,0
	sD[threadIdx.x] -= confx[cals_coo(z, typ, tx)] * ( BXMxx * confx[cals_coo(z, typ, bx)] + BYMxx * confx[cals_coo(z, ty, tx)] + BWMxx * confx[cals_coo(z, ty, bx)])\
		         +confx[cals_coo(z, typ, tx)] * ( BXMxy * confy[cals_coo(z, typ, bx)] + BYMxy * confy[cals_coo(z, ty, tx)] + BWMxy * confy[cals_coo(z, ty, bx)])\
		         +confx[cals_coo(z, typ, tx)] * ( cBXMxz * confz[cals_coo(z, typ, bx)] + cBYMxz * confz[cals_coo(z, ty, tx)] + cBWMxz * confz[cals_coo(z, ty, bx)])\
		         +confy[cals_coo(z, typ, tx)] * ( BXMyx * confx[cals_coo(z, typ, bx)] + BYMyx * confx[cals_coo(z, ty, tx)] + BWMyx * confx[cals_coo(z, ty, bx)])\
		         +confy[cals_coo(z, typ, tx)] * ( BXMyy * confy[cals_coo(z, typ, bx)] + BYMyy * confy[cals_coo(z, ty, tx)] + BWMyy * confy[cals_coo(z, ty, bx)])\
		         +confy[cals_coo(z, typ, tx)] * ( cBXMyz * confz[cals_coo(z, typ, bx)] + cBYMyz * confz[cals_coo(z, ty, tx)] + cBWMyz * confz[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( cBXMzx * confx[cals_coo(z, typ, bx)] + cBYMzx * confx[cals_coo(z, ty, tx)] + cBWMzx * confx[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( cBXMzy * confy[cals_coo(z, typ, bx)] + cBYMzy * confy[cals_coo(z, ty, tx)] + cBWMzy * confy[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( BXMzz * confz[cals_coo(z, typ, bx)] + BYMzz * confz[cals_coo(z, ty, tx)] + BWMzz * confz[cals_coo(z, ty, bx)] - cals_A * confz[cals_coo(z, (typ), tx)]);
	//2,0
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, tx)] * ( BXMxx * confx[cals_coo(z, typ2, bx)] + BYMxx * confx[cals_coo(z, typ, tx)] + BWMxx * confx[cals_coo(z, typ, bx)])\
		         +confx[cals_coo(z, typ2, tx)] * ( BXMxy * confy[cals_coo(z, typ2, bx)] + BYMxy * confy[cals_coo(z, typ, tx)] + BWMxy * confy[cals_coo(z, typ, bx)])\
		         +confx[cals_coo(z, typ2, tx)] * ( cBXMxz * confz[cals_coo(z, typ2, bx)] + cBYMxz * confz[cals_coo(z, typ, tx)] + cBWMxz * confz[cals_coo(z, typ, bx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( BXMyx * confx[cals_coo(z, typ2, bx)] + BYMyx * confx[cals_coo(z, typ, tx)] + BWMyx * confx[cals_coo(z, typ, bx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( BXMyy * confy[cals_coo(z, typ2, bx)] + BYMyy * confy[cals_coo(z, typ, tx)] + BWMyy * confy[cals_coo(z, typ, bx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( cBXMyz * confz[cals_coo(z, typ2, bx)] + cBYMyz * confz[cals_coo(z, typ, tx)] + cBWMyz * confz[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( cBXMzx * confx[cals_coo(z, typ2, bx)] + cBYMzx * confx[cals_coo(z, typ, tx)] + cBWMzx * confx[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( cBXMzy * confy[cals_coo(z, typ2, bx)] + cBYMzy * confy[cals_coo(z, typ, tx)] + cBWMzy * confy[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( BXMzz * confz[cals_coo(z, typ2, bx)] + BYMzz * confz[cals_coo(z, typ, tx)] + BWMzz * confz[cals_coo(z, typ, bx)] - cals_A * confz[cals_coo(z, (typ2), tx)]);
	//0,1
	sD[threadIdx.x] -= confx[cals_coo(z, ty, txp)] * ( BXMxx * confx[cals_coo(z, ty, tx)] + BYMxx * confx[cals_coo(z, by, txp)] + BWMxx * confx[cals_coo(z, by, tx)])\
		         +confx[cals_coo(z, ty, txp)] * ( BXMxy * confy[cals_coo(z, ty, tx)] + BYMxy * confy[cals_coo(z, by, txp)] + BWMxy * confy[cals_coo(z, by, tx)])\
		         +confx[cals_coo(z, ty, txp)] * ( cBXMxz * confz[cals_coo(z, ty, tx)] + cBYMxz * confz[cals_coo(z, by, txp)] + cBWMxz * confz[cals_coo(z, by, tx)])\
		         +confy[cals_coo(z, ty, txp)] * ( BXMyx * confx[cals_coo(z, ty, tx)] + BYMyx * confx[cals_coo(z, by, txp)] + BWMyx * confx[cals_coo(z, by, tx)])\
		         +confy[cals_coo(z, ty, txp)] * ( BXMyy * confy[cals_coo(z, ty, tx)] + BYMyy * confy[cals_coo(z, by, txp)] + BWMyy * confy[cals_coo(z, by, tx)])\
		         +confy[cals_coo(z, ty, txp)] * ( cBXMyz * confz[cals_coo(z, ty, tx)] + cBYMyz * confz[cals_coo(z, by, txp)] + cBWMyz * confz[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( cBXMzx * confx[cals_coo(z, ty, tx)] + cBYMzx * confx[cals_coo(z, by, txp)] + cBWMzx * confx[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( cBXMzy * confy[cals_coo(z, ty, tx)] + cBYMzy * confy[cals_coo(z, by, txp)] + cBWMzy * confy[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( BXMzz * confz[cals_coo(z, ty, tx)] + BYMzz * confz[cals_coo(z, by, txp)] + BWMzz * confz[cals_coo(z, by, tx)] - cals_A * confz[cals_coo(z, ty, txp)]);
	//1,1
	sD[threadIdx.x] -= confx[cals_coo(z, typ, txp)] * ( BXMxx * confx[cals_coo(z, typ, tx)] + BYMxx * confx[cals_coo(z, ty, txp)] + BWMxx * confx[cals_coo(z, ty, tx)])\
		         +confx[cals_coo(z, typ, txp)] * ( BXMxy * confy[cals_coo(z, typ, tx)] + BYMxy * confy[cals_coo(z, ty, txp)] + BWMxy * confy[cals_coo(z, ty, tx)])\
		         +confx[cals_coo(z, typ, txp)] * ( cBXMxz * confz[cals_coo(z, typ, tx)] + cBYMxz * confz[cals_coo(z, ty, txp)] + cBWMxz * confz[cals_coo(z, ty, tx)])\
		         +confy[cals_coo(z, typ, txp)] * ( BXMyx * confx[cals_coo(z, typ, tx)] + BYMyx * confx[cals_coo(z, ty, txp)] + BWMyx * confx[cals_coo(z, ty, tx)])\
		         +confy[cals_coo(z, typ, txp)] * ( BXMyy * confy[cals_coo(z, typ, tx)] + BYMyy * confy[cals_coo(z, ty, txp)] + BWMyy * confy[cals_coo(z, ty, tx)])\
		         +confy[cals_coo(z, typ, txp)] * ( cBXMyz * confz[cals_coo(z, typ, tx)] + cBYMyz * confz[cals_coo(z, ty, txp)] + cBWMyz * confz[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( cBXMzx * confx[cals_coo(z, typ, tx)] + cBYMzx * confx[cals_coo(z, ty, txp)] + cBWMzx * confx[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( cBXMzy * confy[cals_coo(z, typ, tx)] + cBYMzy * confy[cals_coo(z, ty, txp)] + cBWMzy * confy[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( BXMzz * confz[cals_coo(z, typ, tx)] + BYMzz * confz[cals_coo(z, ty, txp)] + BWMzz * confz[cals_coo(z, ty, tx)] - cals_A * confz[cals_coo(z, typ, txp)]);
	//2,1
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, txp)] * ( BXMxx * confx[cals_coo(z, typ2, tx)] + BYMxx * confx[cals_coo(z, typ, txp)] + BWMxx * confx[cals_coo(z, typ, tx)])\
		         +confx[cals_coo(z, typ2, txp)] * ( BXMxy * confy[cals_coo(z, typ2, tx)] + BYMxy * confy[cals_coo(z, typ, txp)] + BWMxy * confy[cals_coo(z, typ, tx)])\
		         +confx[cals_coo(z, typ2, txp)] * ( cBXMxz * confz[cals_coo(z, typ2, tx)] + cBYMxz * confz[cals_coo(z, typ, txp)] + cBWMxz * confz[cals_coo(z, typ, tx)])\
		         +confy[cals_coo(z, typ2, txp)] * ( BXMyx * confx[cals_coo(z, typ2, tx)] + BYMyx * confx[cals_coo(z, typ, txp)] + BWMyx * confx[cals_coo(z, typ, tx)])\
		         +confy[cals_coo(z, typ2, txp)] * ( BXMyy * confy[cals_coo(z, typ2, tx)] + BYMyy * confy[cals_coo(z, typ, txp)] + BWMyy * confy[cals_coo(z, typ, tx)])\
		         +confy[cals_coo(z, typ2, txp)] * ( cBXMyz * confz[cals_coo(z, typ2, tx)] + cBYMyz * confz[cals_coo(z, typ, txp)] + cBWMyz * confz[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( cBXMzx * confx[cals_coo(z, typ2, tx)] + cBYMzx * confx[cals_coo(z, typ, txp)] + cBWMzx * confx[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( cBXMzy * confy[cals_coo(z, typ2, tx)] + cBYMzy * confy[cals_coo(z, typ, txp)] + cBWMzy * confy[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( BXMzz * confz[cals_coo(z, typ2, tx)] + BYMzz * confz[cals_coo(z, typ, txp)] + BWMzz * confz[cals_coo(z, typ, tx)] - cals_A * confz[cals_coo(z, typ2, txp)]);
	//0,2
	sD[threadIdx.x] -= confx[cals_coo(z, ty, txp2)] * ( BXMxx * confx[cals_coo(z, ty, txp)] + BYMxx * confx[cals_coo(z, by, txp2)] + BWMxx * confx[cals_coo(z, by, txp)])\
		         +confx[cals_coo(z, ty, txp2)] * ( BXMxy * confy[cals_coo(z, ty, txp)] + BYMxy * confy[cals_coo(z, by, txp2)] + BWMxy * confy[cals_coo(z, by, txp)])\
		         +confx[cals_coo(z, ty, txp2)] * ( cBXMxz * confz[cals_coo(z, ty, txp)] + cBYMxz * confz[cals_coo(z, by, txp2)] + cBWMxz * confz[cals_coo(z, by, txp)])\
		         +confy[cals_coo(z, ty, txp2)] * ( BXMyx * confx[cals_coo(z, ty, txp)] + BYMyx * confx[cals_coo(z, by, txp2)] + BWMyx * confx[cals_coo(z, by, txp)])\
		         +confy[cals_coo(z, ty, txp2)] * ( BXMyy * confy[cals_coo(z, ty, txp)] + BYMyy * confy[cals_coo(z, by, txp2)] + BWMyy * confy[cals_coo(z, by, txp)])\
		         +confy[cals_coo(z, ty, txp2)] * ( cBXMyz * confz[cals_coo(z, ty, txp)] + cBYMyz * confz[cals_coo(z, by, txp2)] + cBWMyz * confz[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( cBXMzx * confx[cals_coo(z, ty, txp)] + cBYMzx * confx[cals_coo(z, by, txp2)] + cBWMzx * confx[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( cBXMzy * confy[cals_coo(z, ty, txp)] + cBYMzy * confy[cals_coo(z, by, txp2)] + cBWMzy * confy[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( BXMzz * confz[cals_coo(z, ty, txp)] + BYMzz * confz[cals_coo(z, by, txp2)] + BWMzz * confz[cals_coo(z, by, txp)] - cals_A * confz[cals_coo(z, ty, txp2)]);
	//1,2
	sD[threadIdx.x] -= confx[cals_coo(z, typ, txp2)] * ( BXMxx * confx[cals_coo(z, typ, txp)] + BYMxx * confx[cals_coo(z, ty, txp2)] + BWMxx * confx[cals_coo(z, ty, txp)])\
		         +confx[cals_coo(z, typ, txp2)] * ( BXMxy * confy[cals_coo(z, typ, txp)] + BYMxy * confy[cals_coo(z, ty, txp2)] + BWMxy * confy[cals_coo(z, ty, txp)])\
		         +confx[cals_coo(z, typ, txp2)] * ( cBXMxz * confz[cals_coo(z, typ, txp)] + cBYMxz * confz[cals_coo(z, ty, txp2)] + cBWMxz * confz[cals_coo(z, ty, txp)])\
		         +confy[cals_coo(z, typ, txp2)] * ( BXMyx * confx[cals_coo(z, typ, txp)] + BYMyx * confx[cals_coo(z, ty, txp2)] + BWMyx * confx[cals_coo(z, ty, txp)])\
		         +confy[cals_coo(z, typ, txp2)] * ( BXMyy * confy[cals_coo(z, typ, txp)] + BYMyy * confy[cals_coo(z, ty, txp2)] + BWMyy * confy[cals_coo(z, ty, txp)])\
		         +confy[cals_coo(z, typ, txp2)] * ( cBXMyz * confz[cals_coo(z, typ, txp)] + cBYMyz * confz[cals_coo(z, ty, txp2)] + cBWMyz * confz[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( cBXMzx * confx[cals_coo(z, typ, txp)] + cBYMzx * confx[cals_coo(z, ty, txp2)] + cBWMzx * confx[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( cBXMzy * confy[cals_coo(z, typ, txp)] + cBYMzy * confy[cals_coo(z, ty, txp2)] + cBWMzy * confy[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( BXMzz * confz[cals_coo(z, typ, txp)] + BYMzz * confz[cals_coo(z, ty, txp2)] + BWMzz * confz[cals_coo(z, ty, txp)] - cals_A * confz[cals_coo(z, typ, txp2)]);
	//2,2
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, txp2)] * ( BXMxx * confx[cals_coo(z, typ2, txp)] + BYMxx * confx[cals_coo(z, typ, txp2)] + BWMxx * confx[cals_coo(z, typ, txp)])\
		         +confx[cals_coo(z, typ2, txp2)] * ( BXMxy * confy[cals_coo(z, typ2, txp)] + BYMxy * confy[cals_coo(z, typ, txp2)] + BWMxy * confy[cals_coo(z, typ, txp)])\
		         +confx[cals_coo(z, typ2, txp2)] * ( cBXMxz * confz[cals_coo(z, typ2, txp)] + cBYMxz * confz[cals_coo(z, typ, txp2)] + cBWMxz * confz[cals_coo(z, typ, txp)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( BXMyx * confx[cals_coo(z, typ2, txp)] + BYMyx * confx[cals_coo(z, typ, txp2)] + BWMyx * confx[cals_coo(z, typ, txp)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( BXMyy * confy[cals_coo(z, typ2, txp)] + BYMyy * confy[cals_coo(z, typ, txp2)] + BWMyy * confy[cals_coo(z, typ, txp)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( cBXMyz * confz[cals_coo(z, typ2, txp)] + cBYMyz * confz[cals_coo(z, typ, txp2)] + cBWMyz * confz[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( cBXMzx * confx[cals_coo(z, typ2, txp)] + cBYMzx * confx[cals_coo(z, typ, txp2)] + cBWMzx * confx[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( cBXMzy * confy[cals_coo(z, typ2, txp)] + cBYMzy * confy[cals_coo(z, typ, txp2)] + cBWMzy * confy[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( BXMzz * confz[cals_coo(z, typ2, txp)] + BYMzz * confz[cals_coo(z, typ, txp2)] + BWMzz * confz[cals_coo(z, typ, txp)] - cals_A * confz[cals_coo(z, typ2, txp2)]);
  for (z = 1; z < cals_SpinSize_z; z++){
	//0,0
	sD[threadIdx.x] = -confx[cals_coo(z, ty, tx)] * ( BXMxx * confx[cals_coo(z, ty, bx)] + BYMxx * confx[cals_coo(z, by, tx)] + BWMxx * confx[cals_coo(z, by, bx)] + BZMxx * confx[cals_coo(z-1, ty, tx)])\
	           -confx[cals_coo(z, ty, tx)] * ( BXMxy * confy[cals_coo(z, ty, bx)] + BYMxy * confy[cals_coo(z, by, tx)] + BWMxy * confy[cals_coo(z, by, bx)] + cBZMxy * confy[cals_coo(z-1, ty, tx)])\
	           -confx[cals_coo(z, ty, tx)] * ( cBXMxz * confz[cals_coo(z, ty, bx)] + cBYMxz * confz[cals_coo(z, by, tx)] + cBWMxz * confz[cals_coo(z, by, bx)])\
		         -confy[cals_coo(z, ty, tx)] * ( BXMyx * confx[cals_coo(z, ty, bx)] + BYMyx * confx[cals_coo(z, by, tx)] + BWMyx * confx[cals_coo(z, by, bx)] + cBZMyx * confx[cals_coo(z-1, ty, tx)])\
		         -confy[cals_coo(z, ty, tx)] * ( BXMyy * confy[cals_coo(z, ty, bx)] + BYMyy * confy[cals_coo(z, by, tx)] + BWMyy * confy[cals_coo(z, by, bx)] + BZMyy * confy[cals_coo(z-1, ty, tx)])\
		         -confy[cals_coo(z, ty, tx)] * ( cBXMyz * confz[cals_coo(z, ty, bx)] + cBYMyz * confz[cals_coo(z, by, tx)] + cBWMyz * confz[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( cBXMzx * confx[cals_coo(z, ty, bx)] + cBYMzx * confx[cals_coo(z, by, tx)] + cBWMzx * confx[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( cBXMzy * confy[cals_coo(z, ty, bx)] + cBYMzy * confy[cals_coo(z, by, tx)] + cBWMzy * confy[cals_coo(z, by, bx)])\
		         -confz[cals_coo(z, ty, tx)] * ( BXMzz * confz[cals_coo(z, ty, bx)] + BYMzz * confz[cals_coo(z, by, tx)] + BWMzz * confz[cals_coo(z, by, bx)] + BZMzz * confz[cals_coo(z-1, ty, tx)] - cals_A * confz[cals_coo(z, ty, tx)]);
	//1,0
	sD[threadIdx.x] -= confx[cals_coo(z, typ, tx)] * ( BXMxx * confx[cals_coo(z, typ, bx)] + BYMxx * confx[cals_coo(z, ty, tx)] + BWMxx * confx[cals_coo(z, ty, bx)] + BZMxx * confx[cals_coo(z-1, typ, tx)])\
		         +confx[cals_coo(z, typ, tx)] * ( BXMxy * confy[cals_coo(z, typ, bx)] + BYMxy * confy[cals_coo(z, ty, tx)] + BWMxy * confy[cals_coo(z, ty, bx)] + cBZMxy * confy[cals_coo(z-1, typ, tx)])\
		         +confx[cals_coo(z, typ, tx)] * ( cBXMxz * confz[cals_coo(z, typ, bx)] + cBYMxz * confz[cals_coo(z, ty, tx)] + cBWMxz * confz[cals_coo(z, ty, bx)])\
		         +confy[cals_coo(z, typ, tx)] * ( BXMyx * confx[cals_coo(z, typ, bx)] + BYMyx * confx[cals_coo(z, ty, tx)] + BWMyx * confx[cals_coo(z, ty, bx)] + cBZMyx * confx[cals_coo(z-1, typ, tx)])\
		         +confy[cals_coo(z, typ, tx)] * ( BXMyy * confy[cals_coo(z, typ, bx)] + BYMyy * confy[cals_coo(z, ty, tx)] + BWMyy * confy[cals_coo(z, ty, bx)] + BZMyy * confy[cals_coo(z-1, typ, tx)])\
		         +confy[cals_coo(z, typ, tx)] * ( cBXMyz * confz[cals_coo(z, typ, bx)] + cBYMyz * confz[cals_coo(z, ty, tx)] + cBWMyz * confz[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( cBXMzx * confx[cals_coo(z, typ, bx)] + cBYMzx * confx[cals_coo(z, ty, tx)] + cBWMzx * confx[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( cBXMzy * confy[cals_coo(z, typ, bx)] + cBYMzy * confy[cals_coo(z, ty, tx)] + cBWMzy * confy[cals_coo(z, ty, bx)])\
		         +confz[cals_coo(z, typ, tx)] * ( BXMzz * confz[cals_coo(z, typ, bx)] + BYMzz * confz[cals_coo(z, ty, tx)] + BWMzz * confz[cals_coo(z, ty, bx)] + BZMzz * confz[cals_coo(z-1, typ, tx)] - cals_A * confz[cals_coo(z, (typ), tx)]);
	//2,0
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, tx)] * ( BXMxx * confx[cals_coo(z, typ2, bx)] + BYMxx * confx[cals_coo(z, typ, tx)] + BWMxx * confx[cals_coo(z, typ, bx)] + BZMxx * confx[cals_coo(z-1, typ2, tx)])\
		         +confx[cals_coo(z, typ2, tx)] * ( BXMxy * confy[cals_coo(z, typ2, bx)] + BYMxy * confy[cals_coo(z, typ, tx)] + BWMxy * confy[cals_coo(z, typ, bx)] + cBZMxy * confy[cals_coo(z-1, typ2, tx)])\
		         +confx[cals_coo(z, typ2, tx)] * ( cBXMxz * confz[cals_coo(z, typ2, bx)] + cBYMxz * confz[cals_coo(z, typ, tx)] + cBWMxz * confz[cals_coo(z, typ, bx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( BXMyx * confx[cals_coo(z, typ2, bx)] + BYMyx * confx[cals_coo(z, typ, tx)] + BWMyx * confx[cals_coo(z, typ, bx)] + cBZMyx * confx[cals_coo(z-1, typ2, tx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( BXMyy * confy[cals_coo(z, typ2, bx)] + BYMyy * confy[cals_coo(z, typ, tx)] + BWMyy * confy[cals_coo(z, typ, bx)] + BZMyy * confy[cals_coo(z-1, typ2, tx)])\
		         +confy[cals_coo(z, typ2, tx)] * ( cBXMyz * confz[cals_coo(z, typ2, bx)] + cBYMyz * confz[cals_coo(z, typ, tx)] + cBWMyz * confz[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( cBXMzx * confx[cals_coo(z, typ2, bx)] + cBYMzx * confx[cals_coo(z, typ, tx)] + cBWMzx * confx[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( cBXMzy * confy[cals_coo(z, typ2, bx)] + cBYMzy * confy[cals_coo(z, typ, tx)] + cBWMzy * confy[cals_coo(z, typ, bx)])\
		         +confz[cals_coo(z, typ2, tx)] * ( BXMzz * confz[cals_coo(z, typ2, bx)] + BYMzz * confz[cals_coo(z, typ, tx)] + BWMzz * confz[cals_coo(z, typ, bx)] + BZMzz * confz[cals_coo(z-1, typ2, tx)] - cals_A * confz[cals_coo(z, (typ2), tx)]);
	//0,1
	sD[threadIdx.x] -= confx[cals_coo(z, ty, txp)] * ( BXMxx * confx[cals_coo(z, ty, tx)] + BYMxx * confx[cals_coo(z, by, txp)] + BWMxx * confx[cals_coo(z, by, tx)] + BZMxx * confx[cals_coo(z-1, ty, txp)])\
		         +confx[cals_coo(z, ty, txp)] * ( BXMxy * confy[cals_coo(z, ty, tx)] + BYMxy * confy[cals_coo(z, by, txp)] + BWMxy * confy[cals_coo(z, by, tx)] + cBZMxy * confy[cals_coo(z-1, ty, txp)])\
		         +confx[cals_coo(z, ty, txp)] * ( cBXMxz * confz[cals_coo(z, ty, tx)] + cBYMxz * confz[cals_coo(z, by, txp)] + cBWMxz * confz[cals_coo(z, by, tx)])\
		         +confy[cals_coo(z, ty, txp)] * ( BXMyx * confx[cals_coo(z, ty, tx)] + BYMyx * confx[cals_coo(z, by, txp)] + BWMyx * confx[cals_coo(z, by, tx)] + cBZMyx * confx[cals_coo(z-1, ty, txp)])\
		         +confy[cals_coo(z, ty, txp)] * ( BXMyy * confy[cals_coo(z, ty, tx)] + BYMyy * confy[cals_coo(z, by, txp)] + BWMyy * confy[cals_coo(z, by, tx)] + BZMyy * confy[cals_coo(z-1, ty, txp)])\
		         +confy[cals_coo(z, ty, txp)] * ( cBXMyz * confz[cals_coo(z, ty, tx)] + cBYMyz * confz[cals_coo(z, by, txp)] + cBWMyz * confz[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( cBXMzx * confx[cals_coo(z, ty, tx)] + cBYMzx * confx[cals_coo(z, by, txp)] + cBWMzx * confx[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( cBXMzy * confy[cals_coo(z, ty, tx)] + cBYMzy * confy[cals_coo(z, by, txp)] + cBWMzy * confy[cals_coo(z, by, tx)])\
		         +confz[cals_coo(z, ty, txp)] * ( BXMzz * confz[cals_coo(z, ty, tx)] + BYMzz * confz[cals_coo(z, by, txp)] + BWMzz * confz[cals_coo(z, by, tx)] + BZMzz * confz[cals_coo(z-1, ty, txp)] - cals_A * confz[cals_coo(z, ty, txp)]);
	//1,1
	sD[threadIdx.x] -= confx[cals_coo(z, typ, txp)] * ( BXMxx * confx[cals_coo(z, typ, tx)] + BYMxx * confx[cals_coo(z, ty, txp)] + BWMxx * confx[cals_coo(z, ty, tx)] + BZMxx * confx[cals_coo(z-1, typ, txp)])\
		         +confx[cals_coo(z, typ, txp)] * ( BXMxy * confy[cals_coo(z, typ, tx)] + BYMxy * confy[cals_coo(z, ty, txp)] + BWMxy * confy[cals_coo(z, ty, tx)] + cBZMxy * confy[cals_coo(z-1, typ, txp)])\
		         +confx[cals_coo(z, typ, txp)] * ( cBXMxz * confz[cals_coo(z, typ, tx)] + cBYMxz * confz[cals_coo(z, ty, txp)] + cBWMxz * confz[cals_coo(z, ty, tx)])\
		         +confy[cals_coo(z, typ, txp)] * ( BXMyx * confx[cals_coo(z, typ, tx)] + BYMyx * confx[cals_coo(z, ty, txp)] + BWMyx * confx[cals_coo(z, ty, tx)] + cBZMyx * confx[cals_coo(z-1, typ, txp)])\
		         +confy[cals_coo(z, typ, txp)] * ( BXMyy * confy[cals_coo(z, typ, tx)] + BYMyy * confy[cals_coo(z, ty, txp)] + BWMyy * confy[cals_coo(z, ty, tx)] + BZMyy * confy[cals_coo(z-1, typ, txp)])\
		         +confy[cals_coo(z, typ, txp)] * ( cBXMyz * confz[cals_coo(z, typ, tx)] + cBYMyz * confz[cals_coo(z, ty, txp)] + cBWMyz * confz[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( cBXMzx * confx[cals_coo(z, typ, tx)] + cBYMzx * confx[cals_coo(z, ty, txp)] + cBWMzx * confx[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( cBXMzy * confy[cals_coo(z, typ, tx)] + cBYMzy * confy[cals_coo(z, ty, txp)] + cBWMzy * confy[cals_coo(z, ty, tx)])\
		         +confz[cals_coo(z, typ, txp)] * ( BXMzz * confz[cals_coo(z, typ, tx)] + BYMzz * confz[cals_coo(z, ty, txp)] + BWMzz * confz[cals_coo(z, ty, tx)] + BZMzz * confz[cals_coo(z-1, typ, txp)] - cals_A * confz[cals_coo(z, typ, txp)]);
	//2,1
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, txp)] * ( BXMxx * confx[cals_coo(z, typ2, tx)] + BYMxx * confx[cals_coo(z, typ, txp)] + BWMxx * confx[cals_coo(z, typ, tx)] + BZMxx * confx[cals_coo(z-1, typ2, txp)])\
		         +confx[cals_coo(z, typ2, txp)] * ( BXMxy * confy[cals_coo(z, typ2, tx)] + BYMxy * confy[cals_coo(z, typ, txp)] + BWMxy * confy[cals_coo(z, typ, tx)] + cBZMxy * confy[cals_coo(z-1, typ2, txp)])\
		         +confx[cals_coo(z, typ2, txp)] * ( cBXMxz * confz[cals_coo(z, typ2, tx)] + cBYMxz * confz[cals_coo(z, typ, txp)] + cBWMxz * confz[cals_coo(z, typ, tx)])\
		         +confy[cals_coo(z, typ2, txp)] * ( BXMyx * confx[cals_coo(z, typ2, tx)] + BYMyx * confx[cals_coo(z, typ, txp)] + BWMyx * confx[cals_coo(z, typ, tx)] + cBZMyx * confx[cals_coo(z-1, typ2, txp)])\
		         +confy[cals_coo(z, typ2, txp)] * ( BXMyy * confy[cals_coo(z, typ2, tx)] + BYMyy * confy[cals_coo(z, typ, txp)] + BWMyy * confy[cals_coo(z, typ, tx)] + BZMyy * confy[cals_coo(z-1, typ2, txp)])\
		         +confy[cals_coo(z, typ2, txp)] * ( cBXMyz * confz[cals_coo(z, typ2, tx)] + cBYMyz * confz[cals_coo(z, typ, txp)] + cBWMyz * confz[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( cBXMzx * confx[cals_coo(z, typ2, tx)] + cBYMzx * confx[cals_coo(z, typ, txp)] + cBWMzx * confx[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( cBXMzy * confy[cals_coo(z, typ2, tx)] + cBYMzy * confy[cals_coo(z, typ, txp)] + cBWMzy * confy[cals_coo(z, typ, tx)])\
		         +confz[cals_coo(z, typ2, txp)] * ( BXMzz * confz[cals_coo(z, typ2, tx)] + BYMzz * confz[cals_coo(z, typ, txp)] + BWMzz * confz[cals_coo(z, typ, tx)] + BZMzz * confz[cals_coo(z-1, typ2, txp)] - cals_A * confz[cals_coo(z, typ2, txp)]);
	//0,2
	sD[threadIdx.x] -= confx[cals_coo(z, ty, txp2)] * ( BXMxx * confx[cals_coo(z, ty, txp)] + BYMxx * confx[cals_coo(z, by, txp2)] + BWMxx * confx[cals_coo(z, by, txp)] + BZMxx * confx[cals_coo(z-1, ty, txp2)])\
		         +confx[cals_coo(z, ty, txp2)] * ( BXMxy * confy[cals_coo(z, ty, txp)] + BYMxy * confy[cals_coo(z, by, txp2)] + BWMxy * confy[cals_coo(z, by, txp)] + cBZMxy * confy[cals_coo(z-1, ty, txp2)])\
		         +confx[cals_coo(z, ty, txp2)] * ( cBXMxz * confz[cals_coo(z, ty, txp)] + cBYMxz * confz[cals_coo(z, by, txp2)] + cBWMxz * confz[cals_coo(z, by, txp)])\
		         +confy[cals_coo(z, ty, txp2)] * ( BXMyx * confx[cals_coo(z, ty, txp)] + BYMyx * confx[cals_coo(z, by, txp2)] + BWMyx * confx[cals_coo(z, by, txp)] + cBZMyx * confx[cals_coo(z-1, ty, txp2)])\
		         +confy[cals_coo(z, ty, txp2)] * ( BXMyy * confy[cals_coo(z, ty, txp)] + BYMyy * confy[cals_coo(z, by, txp2)] + BWMyy * confy[cals_coo(z, by, txp)] + BZMyy * confy[cals_coo(z-1, ty, txp2)])\
		         +confy[cals_coo(z, ty, txp2)] * ( cBXMyz * confz[cals_coo(z, ty, txp)] + cBYMyz * confz[cals_coo(z, by, txp2)] + cBWMyz * confz[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( cBXMzx * confx[cals_coo(z, ty, txp)] + cBYMzx * confx[cals_coo(z, by, txp2)] + cBWMzx * confx[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( cBXMzy * confy[cals_coo(z, ty, txp)] + cBYMzy * confy[cals_coo(z, by, txp2)] + cBWMzy * confy[cals_coo(z, by, txp)])\
		         +confz[cals_coo(z, ty, txp2)] * ( BXMzz * confz[cals_coo(z, ty, txp)] + BYMzz * confz[cals_coo(z, by, txp2)] + BWMzz * confz[cals_coo(z, by, txp)] + BZMzz * confz[cals_coo(z-1, ty, txp2)] - cals_A * confz[cals_coo(z, ty, txp2)]);
	//1,2
	sD[threadIdx.x] -= confx[cals_coo(z, typ, txp2)] * ( BXMxx * confx[cals_coo(z, typ, txp)] + BYMxx * confx[cals_coo(z, ty, txp2)] + BWMxx * confx[cals_coo(z, ty, txp)] + BZMxx * confx[cals_coo(z-1, typ, txp2)])\
		         +confx[cals_coo(z, typ, txp2)] * ( BXMxy * confy[cals_coo(z, typ, txp)] + BYMxy * confy[cals_coo(z, ty, txp2)] + BWMxy * confy[cals_coo(z, ty, txp)] + cBZMxy * confy[cals_coo(z-1, typ, txp2)])\
		         +confx[cals_coo(z, typ, txp2)] * ( cBXMxz * confz[cals_coo(z, typ, txp)] + cBYMxz * confz[cals_coo(z, ty, txp2)] + cBWMxz * confz[cals_coo(z, ty, txp)])\
		         +confy[cals_coo(z, typ, txp2)] * ( BXMyx * confx[cals_coo(z, typ, txp)] + BYMyx * confx[cals_coo(z, ty, txp2)] + BWMyx * confx[cals_coo(z, ty, txp)] + cBZMyx * confx[cals_coo(z-1, typ, txp2)])\
		         +confy[cals_coo(z, typ, txp2)] * ( BXMyy * confy[cals_coo(z, typ, txp)] + BYMyy * confy[cals_coo(z, ty, txp2)] + BWMyy * confy[cals_coo(z, ty, txp)] + BZMyy * confy[cals_coo(z-1, typ, txp2)])\
		         +confy[cals_coo(z, typ, txp2)] * ( cBXMyz * confz[cals_coo(z, typ, txp)] + cBYMyz * confz[cals_coo(z, ty, txp2)] + cBWMyz * confz[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( cBXMzx * confx[cals_coo(z, typ, txp)] + cBYMzx * confx[cals_coo(z, ty, txp2)] + cBWMzx * confx[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( cBXMzy * confy[cals_coo(z, typ, txp)] + cBYMzy * confy[cals_coo(z, ty, txp2)] + cBWMzy * confy[cals_coo(z, ty, txp)])\
		         +confz[cals_coo(z, typ, txp2)] * ( BXMzz * confz[cals_coo(z, typ, txp)] + BYMzz * confz[cals_coo(z, ty, txp2)] + BWMzz * confz[cals_coo(z, ty, txp)] + BZMzz * confz[cals_coo(z-1, typ, txp2)] - cals_A * confz[cals_coo(z, typ, txp2)]);
	//2,2
	sD[threadIdx.x] -= confx[cals_coo(z, typ2, txp2)] * ( BXMxx * confx[cals_coo(z, typ2, txp)] + BYMxx * confx[cals_coo(z, typ, txp2)] + BWMxx * confx[cals_coo(z, typ, txp)] + BZMxx * confx[cals_coo(z-1, typ2, txp2)])\
		         +confx[cals_coo(z, typ2, txp2)] * ( BXMxy * confy[cals_coo(z, typ2, txp)] + BYMxy * confy[cals_coo(z, typ, txp2)] + BWMxy * confy[cals_coo(z, typ, txp)] + cBZMxy * confy[cals_coo(z-1, typ2, txp2)])\
		         +confx[cals_coo(z, typ2, txp2)] * ( cBXMxz * confz[cals_coo(z, typ2, txp)] + cBYMxz * confz[cals_coo(z, typ, txp2)] + cBWMxz * confz[cals_coo(z, typ, txp)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( BXMyx * confx[cals_coo(z, typ2, txp)] + BYMyx * confx[cals_coo(z, typ, txp2)] + BWMyx * confx[cals_coo(z, typ, txp)] + cBZMyx * confx[cals_coo(z-1, typ2, txp2)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( BXMyy * confy[cals_coo(z, typ2, txp)] + BYMyy * confy[cals_coo(z, typ, txp2)] + BWMyy * confy[cals_coo(z, typ, txp)] + BZMyy * confy[cals_coo(z-1, typ2, txp2)])\
		         +confy[cals_coo(z, typ2, txp2)] * ( cBXMyz * confz[cals_coo(z, typ2, txp)] + cBYMyz * confz[cals_coo(z, typ, txp2)] + cBWMyz * confz[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( cBXMzx * confx[cals_coo(z, typ2, txp)] + cBYMzx * confx[cals_coo(z, typ, txp2)] + cBWMzx * confx[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( cBXMzy * confy[cals_coo(z, typ2, txp)] + cBYMzy * confy[cals_coo(z, typ, txp2)] + cBWMzy * confy[cals_coo(z, typ, txp)])\
		         +confz[cals_coo(z, typ2, txp2)] * ( BXMzz * confz[cals_coo(z, typ2, txp)] + BYMzz * confz[cals_coo(z, typ, txp2)] + BWMzz * confz[cals_coo(z, typ, txp)] + BZMzz * confz[cals_coo(z-1, typ2, txp2)] - cals_A * confz[cals_coo(z, typ2, txp2)]);
	}
	__syncthreads();


	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN)] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
  for (z = 0; z < cals_SpinSize_z; z++){
		sD[threadIdx.x]  = confx[cals_coo(z, ty, tx)];
		sD[threadIdx.x] += confx[cals_coo(z, typ, tx)];
		sD[threadIdx.x] += confx[cals_coo(z, typ2, tx)];
		sD[threadIdx.x] += confx[cals_coo(z, ty, txp)];
		sD[threadIdx.x] += confx[cals_coo(z, typ, txp)];
		sD[threadIdx.x] += confx[cals_coo(z, typ2, txp)];
		sD[threadIdx.x] += confx[cals_coo(z, ty, txp2)];
		sD[threadIdx.x] += confx[cals_coo(z, typ, txp2)];
		sD[threadIdx.x] += confx[cals_coo(z, typ2, txp2)];
	}
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
  for (z = 0; z < cals_SpinSize_z; z++){
		sD[threadIdx.x]  = confy[cals_coo(z, ty, tx)];
		sD[threadIdx.x] += confy[cals_coo(z, typ, tx)];
		sD[threadIdx.x] += confy[cals_coo(z, typ2, tx)];
		sD[threadIdx.x] += confy[cals_coo(z, ty, txp)];
		sD[threadIdx.x] += confy[cals_coo(z, typ, txp)];
		sD[threadIdx.x] += confy[cals_coo(z, typ2, txp)];
		sD[threadIdx.x] += confy[cals_coo(z, ty, txp2)];
		sD[threadIdx.x] += confy[cals_coo(z, typ, txp2)];
		sD[threadIdx.x] += confy[cals_coo(z, typ2, txp2)];
	}
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 2*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
  for (z = 0; z < cals_SpinSize_z; z++){
		sD[threadIdx.x]  = confz[cals_coo(z, ty, tx)];
		sD[threadIdx.x] += confz[cals_coo(z, typ, tx)];
		sD[threadIdx.x] += confz[cals_coo(z, typ2, tx)];
		sD[threadIdx.x] += confz[cals_coo(z, ty, txp)];
		sD[threadIdx.x] += confz[cals_coo(z, typ, txp)];
		sD[threadIdx.x] += confz[cals_coo(z, typ2, txp)];
		sD[threadIdx.x] += confz[cals_coo(z, ty, txp2)];
		sD[threadIdx.x] += confz[cals_coo(z, typ, txp2)];
		sD[threadIdx.x] += confz[cals_coo(z, typ2, txp2)];
	}
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 3*cals_BN] = sD[0];
	__syncthreads();

	//try to measure Chern number
	//(0,0)
	z=0;
	Ax = confx[cals_coo(z, ty, tx)];
	Ay = confy[cals_coo(z, ty, tx)];
	Az = confz[cals_coo(z, ty, tx)];
	Bx = confx[cals_coo(z, ty, bx)];
	By = confy[cals_coo(z, ty, bx)];
	Bz = confz[cals_coo(z, ty, bx)];
	Cx = confx[cals_coo(z, by, bx)];
	Cy = confy[cals_coo(z, by, bx)];
	Cz = confz[cals_coo(z, by, bx)];
	sD[threadIdx.x] = 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, by, tx)];
	By = confy[cals_coo(z, by, tx)];
	Bz = confz[cals_coo(z, by, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(1,0)
	Ax = confx[cals_coo(z, typ, tx)];
	Ay = confy[cals_coo(z, typ, tx)];
	Az = confz[cals_coo(z, typ, tx)];
	Bx = confx[cals_coo(z, typ, bx)];
	By = confy[cals_coo(z, typ, bx)];
	Bz = confz[cals_coo(z, typ, bx)];
	Cx = confx[cals_coo(z, ty, bx)];
	Cy = confy[cals_coo(z, ty, bx)];
	Cz = confz[cals_coo(z, ty, bx)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, ty, tx)];
	By = confy[cals_coo(z, ty, tx)];
	Bz = confz[cals_coo(z, ty, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(2,0)
	Ax = confx[cals_coo(z, typ2, tx)];
	Ay = confy[cals_coo(z, typ2, tx)];
	Az = confz[cals_coo(z, typ2, tx)];
	Bx = confx[cals_coo(z, typ2, bx)];
	By = confy[cals_coo(z, typ2, bx)];
	Bz = confz[cals_coo(z, typ2, bx)];
	Cx = confx[cals_coo(z, typ, bx)];
	Cy = confy[cals_coo(z, typ, bx)];
	Cz = confz[cals_coo(z, typ, bx)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, typ, tx)];
	By = confy[cals_coo(z, typ, tx)];
	Bz = confz[cals_coo(z, typ, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(0,1)
	Ax = confx[cals_coo(z, ty, txp)];
	Ay = confy[cals_coo(z, ty, txp)];
	Az = confz[cals_coo(z, ty, txp)];
	Bx = confx[cals_coo(z, ty, tx)];
	By = confy[cals_coo(z, ty, tx)];
	Bz = confz[cals_coo(z, ty, tx)];
	Cx = confx[cals_coo(z, by, tx)];
	Cy = confy[cals_coo(z, by, tx)];
	Cz = confz[cals_coo(z, by, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, by, txp)];
	By = confy[cals_coo(z, by, txp)];
	Bz = confz[cals_coo(z, by, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(1,1)
	Ax = confx[cals_coo(z, typ, txp)];
	Ay = confy[cals_coo(z, typ, txp)];
	Az = confz[cals_coo(z, typ, txp)];
	Bx = confx[cals_coo(z, typ, tx)];
	By = confy[cals_coo(z, typ, tx)];
	Bz = confz[cals_coo(z, typ, tx)];
	Cx = confx[cals_coo(z, ty, tx)];
	Cy = confy[cals_coo(z, ty, tx)];
	Cz = confz[cals_coo(z, ty, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, ty, txp)];
	By = confy[cals_coo(z, ty, txp)];
	Bz = confz[cals_coo(z, ty, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(2,1)
	Ax = confx[cals_coo(z, typ2, txp)];
	Ay = confy[cals_coo(z, typ2, txp)];
	Az = confz[cals_coo(z, typ2, txp)];
	Bx = confx[cals_coo(z, typ2, tx)];
	By = confy[cals_coo(z, typ2, tx)];
	Bz = confz[cals_coo(z, typ2, tx)];
	Cx = confx[cals_coo(z, typ, tx)];
	Cy = confy[cals_coo(z, typ, tx)];
	Cz = confz[cals_coo(z, typ, tx)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, typ, txp)];
	By = confy[cals_coo(z, typ, txp)];
	Bz = confz[cals_coo(z, typ, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(0,2)
	Ax = confx[cals_coo(z, ty, txp2)];
	Ay = confy[cals_coo(z, ty, txp2)];
	Az = confz[cals_coo(z, ty, txp2)];
	Bx = confx[cals_coo(z, ty, txp)];
	By = confy[cals_coo(z, ty, txp)];
	Bz = confz[cals_coo(z, ty, txp)];
	Cx = confx[cals_coo(z, by, txp)];
	Cy = confy[cals_coo(z, by, txp)];
	Cz = confz[cals_coo(z, by, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, by, txp2)];
	By = confy[cals_coo(z, by, txp2)];
	Bz = confz[cals_coo(z, by, txp2)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(1,2)
	Ax = confx[cals_coo(z, typ, txp2)];
	Ay = confy[cals_coo(z, typ, txp2)];
	Az = confz[cals_coo(z, typ, txp2)];
	Bx = confx[cals_coo(z, typ, txp)];
	By = confy[cals_coo(z, typ, txp)];
	Bz = confz[cals_coo(z, typ, txp)];
	Cx = confx[cals_coo(z, ty, txp)];
	Cy = confy[cals_coo(z, ty, txp)];
	Cz = confz[cals_coo(z, ty, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, ty, txp2)];
	By = confy[cals_coo(z, ty, txp2)];
	Bz = confz[cals_coo(z, ty, txp2)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	//(2,2)
	Ax = confx[cals_coo(z, typ2, txp2)];
	Ay = confy[cals_coo(z, typ2, txp2)];
	Az = confz[cals_coo(z, typ2, txp2)];
	Bx = confx[cals_coo(z, typ2, txp)];
	By = confy[cals_coo(z, typ2, txp)];
	Bz = confz[cals_coo(z, typ2, txp)];
	Cx = confx[cals_coo(z, typ, txp)];
	Cy = confy[cals_coo(z, typ, txp)];
	Cz = confz[cals_coo(z, typ, txp)];
	sD[threadIdx.x] += 2*atan((Ax * (By*Cz-Bz*Cy) + Ay * (Bz*Cx-Bx*Cz) + Az * (Bx*Cy-By*Cx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	Bx = confx[cals_coo(z, typ, txp2)];
	By = confy[cals_coo(z, typ, txp2)];
	Bz = confz[cals_coo(z, typ, txp2)];
	sD[threadIdx.x] += 2*atan((Ax * (Cy*Bz-Cz*By) + Ay * (Cz*Bx-Cx*Bz) + Az * (Cx*By-Cy*Bx))/
	  (1.0 + Ax*Bx + Ay*By + Az*Bz + Cx*Bx + Cy*By + Cz*Bz + Ax*Cx + Ay*Cy + Az*Cz));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 4*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo(z, ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 5*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo(z, ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 6*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo(z, ty, tx)]     * cosf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, tx)]    * cosf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, tx)]   * cosf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp)]    * cosf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp)]   * cosf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp)]  * cosf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp2)]   * cosf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp2)]  * cosf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp2)] * cosf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 7*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo(z, ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 8*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo(z, ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 9*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo(z, ty, tx)]     * sinf(Q1x*(tx  ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, tx)]    * sinf(Q1x*(tx  ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, tx)]   * sinf(Q1x*(tx  ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp)]    * sinf(Q1x*(txp ) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp)]   * sinf(Q1x*(txp ) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp)]  * sinf(Q1x*(txp ) + Q1y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp2)]   * sinf(Q1x*(txp2) + Q1y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp2)]  * sinf(Q1x*(txp2) + Q1y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp2)] * sinf(Q1x*(txp2) + Q1y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 10*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo(z, ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 11*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo(z, ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 12*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo(z, ty, tx)]     * cosf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, tx)]    * cosf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, tx)]   * cosf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp)]    * cosf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp)]   * cosf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp)]  * cosf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp2)]   * cosf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp2)]  * cosf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp2)] * cosf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 13*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confx[cals_coo(z, ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confx[cals_coo(z, ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confx[cals_coo(z, typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confx[cals_coo(z, typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 14*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confy[cals_coo(z, ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confy[cals_coo(z, ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confy[cals_coo(z, typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confy[cals_coo(z, typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 15*cals_BN] = sD[0];
	__syncthreads();

	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[threadIdx.x]  = confz[cals_coo(z, ty, tx)]     * sinf(Q2x*(tx  ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, tx)]    * sinf(Q2x*(tx  ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, tx)]   * sinf(Q2x*(tx  ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp)]    * sinf(Q2x*(txp ) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp)]   * sinf(Q2x*(txp ) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp)]  * sinf(Q2x*(txp ) + Q2y*(typ2));
	sD[threadIdx.x] += confz[cals_coo(z, ty, txp2)]   * sinf(Q2x*(txp2) + Q2y*(ty  ));
	sD[threadIdx.x] += confz[cals_coo(z, typ, txp2)]  * sinf(Q2x*(txp2) + Q2y*(typ ));
	sD[threadIdx.x] += confz[cals_coo(z, typ2, txp2)] * sinf(Q2x*(txp2) + Q2y*(typ2));
	__syncthreads();

	//Sum over all elements in each sD
	if(cals_TN>256){
		if((threadIdx.x < 256) && (threadIdx.x+256 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+256];
		}
		__syncthreads();
	}
	if(cals_TN>128){
		if((threadIdx.x < 128) && (threadIdx.x+128 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+128];
		}
		__syncthreads();
	}
	if(cals_TN>64){
		if((threadIdx.x < 64) && (threadIdx.x+64 < cals_TN)){
			sD[threadIdx.x] += sD[threadIdx.x+64];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		sD[threadIdx.x] += sD[threadIdx.x+32];
	}
	__syncthreads();
	if(threadIdx.x < 16){
		sD[threadIdx.x] += sD[threadIdx.x+16];
	}
	__syncthreads();
	if(threadIdx.x < 8){
		sD[threadIdx.x] += sD[threadIdx.x+8];
	}
	__syncthreads();
	if(threadIdx.x < 4){
		sD[threadIdx.x] += sD[threadIdx.x+4];
	}
	__syncthreads();
	if(threadIdx.x < 2){
		sD[threadIdx.x] += sD[threadIdx.x+2];
	}
	__syncthreads();
	if(threadIdx.x < 1){
		sD[threadIdx.x] += sD[threadIdx.x+1];
	}
	__syncthreads();
	if(threadIdx.x == 0)
		out[dataoff + (blockIdx.x % cals_BN) + 16*cals_BN] = sD[0];
	__syncthreads();
}
#endif
