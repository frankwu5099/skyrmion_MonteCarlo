#include "hip/hip_runtime.h"
//Ms PT function writedata  HHs
//Po -> Cpara
//why exchange conf and measurement works
using namespace std;


#include "params.cuh"
#include "updates.cuh"
#include "measurements.cuh"
#include "configuration.cuh"
#include "extend.cu"
#define GET_CORR


unsigned seed = 73;
mt19937 rng(seed);
uniform_01<mt19937> uni01_sampler(rng);
void tempering(double*, double*, int*);
vector< vector<float> > Tls;
vector< vector<float> > Hls;
vector<int> Po;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
vector<int> ivPo;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
unsigned int Pnum;
unsigned int Cnum;
unsigned int Tnum;
unsigned int Hnum;
unsigned int f_CORR;
unsigned int CORR_N;
float Cparameter = 0.8;
int C_i = 0;
void var_examine();



int main(int argc, char *argv[]){
  //call GPU
  read_params(argv[1]);

  if (setDev()==1){
    return 1;
  }
  hipGetLastError();
  CudaCheckError();
  move_params_device_flip();
  move_params_device_cals();
  move_params_device_corr();

  //examine variables
  var_examine();

  // ========================== initialize ===========================

  //begin (read in temperatures)
  unsigned int params_mem_size;

  if(argc > 2){
    float tmpT, tmpH;
    FILE *paramsfp = fopen(argv[2], "r");
    vector<float> tmpTls;
    vector<float> tmpHls;
    fscanf(paramsfp, "%d %d", &Tnum, &Hnum);
    while(fscanf(paramsfp, "%f %f", &tmpT, &tmpH) != EOF){
      if (tmpT < 0){
	Tls.push_back(tmpTls);
	Hls.push_back(tmpHls);
	tmpTls.clear();
	tmpHls.clear();
      }
      else {
	tmpTls.push_back(tmpT);
	tmpHls.push_back(tmpH*(DR*DR + DD*DD));
      }
    }
    fclose(paramsfp);
    Pnum = Tls[0].size();
    Cnum = Tls.size();
    if (Tnum * Hnum != Pnum){
      fprintf(stderr, "wrong temperatures and fields!!!\n");
      exit(0);
    }
    params_mem_size = Pnum * sizeof(float);

    grid = Pnum * H_BN;
  }
  else{
    fprintf(stderr, "Give me a temperature set!!!\n");
    fprintf(stderr, "Give me a field set!!!\n");
    exit(0);
  }
  //end (read in temperatures)


  //invTs is the inverse temperature in order of configurations on GPU.
  for (int i = 0; i < Pnum ; i++){
    Po.push_back(i);
    ivPo.push_back(i);
  }
  float *HHs;
  float *DHs;
  float *invTs;
  float *DinvTs;
  HHs = (float*)malloc(params_mem_size);
  invTs = (float*)malloc(params_mem_size);
  CudaSafeCall(hipMalloc((void**)&DHs, params_mem_size));
  CudaSafeCall(hipMalloc((void**)&DinvTs, params_mem_size));


  //begin (initialize random seeds)
  //Declare sizes
  unsigned int totalRngs = Pnum * H_TN / WarpStandard_K;
  unsigned seedBytes = totalRngs * sizeof(unsigned int) * WarpStandard_STATE_WORDS;
  unsigned int *seedDevice = 0;
  CudaSafeCall(hipMalloc((void **)&seedDevice, seedBytes));
  unsigned int* seedHost = (unsigned int*)malloc(seedBytes);
  srand(seed);
  for(int i = 0; i < seedBytes / sizeof(unsigned int); i++)
    seedHost[i] = uni01_sampler() * UINT_MAX;
  CudaSafeCall(hipMemcpy(seedDevice, seedHost, seedBytes, hipMemcpyHostToDevice));
  //end (initialize random seeds)


  //Set up output data path
  char dir[128];
  sprintf(dir, "Data/L_%d-%s", H_SpinSize, Output);
  mkdir(dir, 0755);
  char Seedfn[128];
  sprintf(Seedfn, "Data/L_%d-%s/seed", H_SpinSize, Output);
  int seedfd = open(Seedfn, O_CREAT | O_WRONLY, 0644);
  write(seedfd, seedHost, seedBytes);
  close(seedfd);


  //MEASUREMENT initialize

  configuration CONF(Pnum, dir);
  measurements MEASURE(dir, Pnum, BIN_SZ); //Tnum for parallel tempering for T
#ifdef GET_CORR
  char Corrfn[128];
  sprintf(Corrfn, "%s/Corrpt", dir);
  correlation CORR(Pnum, Corrfn);
#endif

  StopWatchInterface *timer=NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  //Give initial configuration and settle the systems down to equilibrium states
  CONF.initialize(ORDER);
  int Eqii = 0;//150;
  for(int i = 0; i < Pnum; i++){
    HHs[i] = Hls[0][i];
    invTs[i] = 1.0/Tls[0][i];
  }
  CudaSafeCall(hipMemcpy(DinvTs, invTs, params_mem_size, hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(DHs, HHs, params_mem_size, hipMemcpyHostToDevice));
  double *Ms = (double*)malloc(Pnum * sizeof(double));
  double *Es = (double*)malloc(Pnum * sizeof(double));
  int *accept1 = (int*)calloc((Tnum - 1)*Hnum + Tnum*(Hnum - 1), sizeof(int));
  float cnt = 0;

  for(int i = 0; i < EQUI_N; i++){
    if (i % 10 ==0) printf("%d\n",i);
    SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DHs, DinvTs);
    //Parallel Tempering
    cnt += PTF;
    for(int p = 0; p < int(cnt); p++){
      MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
      tempering(Ms, Es, accept1);
      for(int t = 0; t < Pnum; t++){
	HHs[t] = Hls[0][Po[t]];
	invTs[t] = 1.0/Tls[0][Po[t]];
      }
      CudaSafeCall(hipMemcpy(DinvTs, invTs, params_mem_size, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(DHs, HHs, params_mem_size, hipMemcpyHostToDevice));
    }
    if(int(cnt))
    cnt = 0;
  }

  //Do measurements (annealing)

  int *accept = (int*)calloc(Pnum - 1, sizeof(int));
  for(C_i = 0 ; C_i < Cnum ; C_i ++){

    for(int t = 0; t < Pnum; t++){
      HHs[t] = Hls[C_i][Po[t]];
      invTs[t] = 1.0/Tls[C_i][Po[t]];
    }
    for (int i = 0; i< Pnum-1; i++) accept[i] = 0;
    CudaSafeCall(hipMemcpy(DinvTs, invTs, params_mem_size, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(DHs, HHs, params_mem_size, hipMemcpyHostToDevice));

    for(int i = 0; i < EQUI_Ni; i++){
      SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DHs, DinvTs);
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
	MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
	tempering(Ms, Es, accept1);
	for(int t = 0; t < Pnum; t++){
	  HHs[t] = Hls[C_i][Po[t]];
	  invTs[t] = 1.0/Tls[C_i][Po[t]];
	}
	CudaSafeCall(hipMemcpy(DinvTs, invTs, params_mem_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(DHs, HHs, params_mem_size, hipMemcpyHostToDevice));
      }
      if(int(cnt))
	cnt = 0;
    }
    cnt = 0;
    for(int b = 0; b < BIN_NUM; b++){
      //Take the ensemble average
      for(int i = 0; i < BIN_SZ; i++){
	SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DHs, DinvTs);
	MEASURE.measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
#ifdef GET_CORR
	if ( i % f_CORR==0){
	  CORR.extract(Po, CONF);//==
	}
#endif
	cnt += PTF;
	for(int p = 0; p < int(cnt); p++){
	  tempering(Ms, Es, accept);
	  for(int t = 0; t < Pnum; t++){
	    HHs[t] = Hls[C_i][Po[t]];
	    invTs[t] = 1.0/Tls[C_i][Po[t]];
	  }
	  CudaSafeCall(hipMemcpy(DinvTs, invTs, params_mem_size, hipMemcpyHostToDevice));
	  CudaSafeCall(hipMemcpy(DHs, HHs, params_mem_size, hipMemcpyHostToDevice));
	}
	if(int(cnt))
	  cnt = 0;
      }
      MEASURE.normalize_and_save_and_reset();
    }
    for (int iii = 0 ; iii < Pnum; iii ++){
      ivPo[Po[iii]] = iii;
    }
    CONF.backtoHost();
    CONF.writedata();
#ifdef GET_CORR
    CORR.avg_write_reset();
#endif
  }
#ifdef GET_CORR
  sprintf(Corrfn, "%s/Corr", dir);
  CORR.changefile(Corrfn);
  for(int i = 0; i < CORR_N * f_CORR; i++){
    SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DHs, DinvTs);
    if ( i % f_CORR==0){
      CORR.extract(Po, CONF);//==
    }
  }
  CORR.avg_write_reset();
#endif
  free(Ms);
  sdkStopTimer(&timer);
  double time = 1.0e-3 * sdkGetTimerValue(&timer);


//======================= print details ==========================
  char detailFn[128];
  sprintf(detailFn, "%s/details", dir);
  FILE *detailFp = fopen(detailFn, "w");
  fprintf(detailFp, "elapsed time = %f (sec)\n", time);
  double speed = 0;
  speed = (H_N / time / 1000000000) * ((EQUI_Ni + BIN_SZ * BIN_NUM) * Cnum + EQUI_N) * Pnum;
  fprintf(detailFp, "speed = %f (GHz)\n", speed);
  fprintf(detailFp, "RNG: WarpStandard\n", H_SpinSize);
  fprintf(detailFp, "SpinSize = %d\n", H_SpinSize);
  fprintf(detailFp, "A = %4.3f\n", H_A);
  fprintf(detailFp, "D_Rashba = %4.3f\n", DR);
  fprintf(detailFp, "D_Dresselhaus = %4.3f\n", DD);
  fprintf(detailFp, "BlockSize_x = %d\n", H_BlockSize_x);
  fprintf(detailFp, "BlockSize_y = %d\n", H_BlockSize_y);
  fprintf(detailFp, "GridSize_x = %d\n", H_GridSize_x);
  fprintf(detailFp, "GridSize_y = %d\n", H_GridSize_y);
  fprintf(detailFp, "Bin Size = %d\n", BIN_SZ);
  fprintf(detailFp, "Bin Number = %d\n", BIN_NUM);
  fprintf(detailFp, "Equilibration N = %d\n", EQUI_N);
  fprintf(detailFp, "Equilibration Ni = %d\n", EQUI_Ni);
  fprintf(detailFp, "f_CORR = %d\n", f_CORR);
  fprintf(detailFp, "CORR_N = %d\n", CORR_N);
  fprintf(detailFp, "PT frequency = %3.2f\n", PTF);
  fprintf(detailFp, "Pnum = %d\n", Pnum);
  fprintf(detailFp, "Temperature Set: ");
  for(int i = 0; i < Cnum; i++){
    for(int j = 0; j < Pnum; j++){
      fprintf(detailFp, "%.5f  ", Tls[i][j]);
    }
  }
  fprintf(detailFp, "\n");
  fprintf(detailFp, "field Set: ");
  for(int i = 0; i < Cnum; i++){
    for(int j = 0; j < Pnum; j++){
      fprintf(detailFp, "%.5f  ", Hls[i][j]/(DR*DR + DD*DD));
    }
  }
  for(int i = 0; i < Pnum; i++){
    fprintf(detailFp, "\n");
    fprintf(detailFp, "Po[%d]=%d",i,Po[i]);
  }
  fprintf(detailFp, "\n");
  fprintf(detailFp, "Acceptance rates: ");
  if (PTF != 0 ){
    for(int i = 0; i < Pnum - 1; i++)
      fprintf(detailFp, "%4.3f  ", float(accept[i]) / (BIN_SZ * BIN_NUM * PTF));
  }
  fprintf(detailFp, "\n");
  if (ORDER){
    fprintf(detailFp, "Configurations start from ordered state.\n");
  }
  else {
    fprintf(detailFp, "Configurations start from random state.\n");
  }
  fprintf(detailFp, "Done by Po-Kuan Wu ^_^\n", EQUI_N);
  fclose(detailFp);
//===================== print details end =========================

  //Set free memory
  free(seedHost);
  CudaSafeCall(hipFree(DinvTs));
  CudaSafeCall(hipFree(DHs));
  CudaSafeCall(hipFree(seedDevice));
  //CORR.~correlation();
  //MEASURE.~measurements();
  //CONF.~configuration();
  return 0;
}


//=============================== functions ==================================
void tempering(double *Ms, double *Es, int *accept){
  int map[Pnum];	//map[t] the configuration of t'th temperature
  int i, j, tmp, partT_num = (Tnum - 1) * Hnum;
  double tmpEM;

  for(i = 0; i < Pnum; i++)
    map[Po[i]] = i;

  double delta;
  int flag = 0;
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //T excnange
      if (i < Tnum -1){
	delta = (Es[j * Tnum + i] - Es[j * Tnum + i + 1]) * ((1.0 / Tls[C_i][j*Tnum + i]) - (1.0 / Tls[C_i][j*Tnum +i + 1]));
	if(delta > 0)
	  flag = 1;
	else if(uni01_sampler() < exp(delta))
	  flag = 1;
	if(flag){
	  tmp = Po[map[j * Tnum + i]];
	  Po[map[j * Tnum + i]] = Po[map[j * Tnum + i + 1]];
	  Po[map[j * Tnum + i + 1]] = tmp;
	  tmp = map[j * Tnum + i];
	  map[j * Tnum + i] = map[j * Tnum + i + 1];
	  map[j * Tnum + i + 1] = tmp;
	  tmpEM = Es[j * Tnum + i];
	  Es[j * Tnum + i] = Es[j * Tnum + i + 1];
	  Es[j * Tnum + i + 1] = tmpEM;
	  tmpEM = Ms[j * Tnum + i];
	  Ms[j * Tnum + i] = Ms[j * Tnum + i + 1];
	  Ms[j * Tnum + i + 1] = tmpEM;
	  accept[j * (Tnum - 1) + i] += 1;
	  flag = 0;
	}
      }
      //H excnange
      if (j < Hnum -1){
        delta = (Ms[(j + 1) * Tnum + i] - Ms[j * Tnum + i]) * ( Hls[C_i][j * Tnum + i] - Hls[C_i][(j + 1) * Tnum + i]) / Tls[C_i][j * Tnum + i];
        if(delta > 0)
          flag = 1;
        else if(uni01_sampler() < exp(delta))
          flag = 1;
        if(flag){
          tmp = Po[map[j * Tnum + i]];
          Po[map[j * Tnum + i]] = Po[map[(j + 1) * Tnum + i]];
          Po[map[(j + 1) * Tnum + i]] = tmp;
          tmp = map[j * Tnum + i];
          map[j * Tnum + i] = map[(j + 1) * Tnum + i];
          map[(j + 1) * Tnum + i] = tmp;
          tmpEM = Es[j * Tnum + i];
          Es[j * Tnum + i] = Es[(j + 1) * Tnum + i];
          Es[(j + 1) * Tnum + i] = tmpEM;
          tmpEM = Ms[j * Tnum + i];
          Ms[j * Tnum + i] = Ms[(j + 1) * Tnum + i];
          Ms[(j + 1) * Tnum + i] = tmpEM;
          accept[partT_num + j * Tnum + i] += 1;
          flag = 0;
        }
      }
    }
  }
}

void var_examine(){
#ifndef TRI
  if(H_SpinSize % (H_BlockSize_x * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
#ifdef TRI
  if(H_SpinSize % (H_BlockSize_x * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
#ifndef THIN
  if (H_SpinSize_z != 1){
    fprintf(stderr, "SpinSize_z must be 1 %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
}
