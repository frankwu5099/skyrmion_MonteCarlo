#include "hip/hip_runtime.h"
//Ms PT function writedata  HHs
//Po -> Cpara
//why exchange conf and measurement works
using namespace std;


#include "params.cuh"
#include "updates.cuh"
#include "configuration.cuh"
#include "measurements.cuh"
#include "extend.cu"
#include<hiprand.h>
#define GET_CORR


void tempering_simple(double*, double*, int*, mt19937&);
void tempering(double*, double*, int*, int*, int*, mt19937&);
vector< vector<float> > Tls;
vector< vector<float> > Hls;
vector<int> Po;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
vector<int> ivPo;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
unsigned int Pnum;
unsigned int Cnum;
unsigned int Tnum;
unsigned int Hnum;
unsigned int f_CORR;
unsigned int CORR_N;
float Cparameter = 0.8;
int C_i = 0;
void var_examine();
unsigned seed = 73;



int main(int argc, char *argv[]){
  mt19937 generator(seed);
  uniform_real_distribution<double >uni01;
  //call GPU
  bool json_read = false;
  json configj;
  if (argc > 1){
    configj = read_json(argv[1]);
    json_read = true;
  } 
  else {
    configj = read_json("config.json");
    json_read = true;
  } 

  int deviceNum, gpu_i;
  hipGetDeviceCount(&deviceNum);
  deviceNum = 1;
  device_0 = 0;//setDev(); //use setDev if using multigpu
  StreamN = deviceNum - device_0;
  printf("# of gpus = %d\n", deviceNum);
  printf("# of gpu = %d\n", device_0);
  fflush(stdout);

  if (device_0 == -1){
    return 1;
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipStreamCreate(&stream[gpu_i]);
    hipGetLastError();
    CudaCheckError();
  }
  //note: gpu part

  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    move_params_device_flip();
    move_params_device_cals();
    move_params_device_corr();
  }

  //examine variables
  var_examine();

  // ========================== initialize ===========================

  //begin (read in temperatures)
  unsigned int params_mem_size;

  if  (json_read)
    params_mem_size = Pnum * sizeof(float);
  else if(argc > 2){
    float tmpT, tmpH;
    FILE *paramsfp = fopen(argv[2], "r");
    vector<float> tmpTls;
    vector<float> tmpHls;
    fscanf(paramsfp, "%d %d", &Tnum, &Hnum);
    while(fscanf(paramsfp, "%f %f", &tmpT, &tmpH) != EOF){
      if (tmpT < 0){
        Tls.push_back(tmpTls);
        Hls.push_back(tmpHls);
        tmpTls.clear();
        tmpHls.clear();
      }
      else {
        tmpTls.push_back(tmpT);
        tmpHls.push_back(tmpH*(DR*DR + DD*DD));
      }
    }
    fclose(paramsfp);
    Pnum = Tls[0].size();
    Cnum = Tls.size();
    if (Tnum * Hnum != Pnum){
      fprintf(stderr, "wrong temperatures and fields!!!\n");
      exit(0);
    }
    params_mem_size = Pnum * sizeof(float);

  }
  else{
    fprintf(stderr, "Give me a temperature set!!!\n");
    fprintf(stderr, "Give me a field set!!!\n");
    exit(0);
  }
  //end (read in temperatures)

  if (Pnum%StreamN != 0){
    printf("Fatal error: The number of replicas is not consistent with the number of streams.");
    return 1;
  }
  int Pnum_s = Pnum/StreamN;
  int params_mem_size_s = Pnum_s * sizeof(float);
	grid = Pnum_s * H_BN;


  //invTs is the inverse temperature in order of configurations on GPU.
  for (int i = 0; i < Pnum ; i++){
    Po.push_back(i);
    ivPo.push_back(i);
  }
  float *HHs;
  float **DHs;
  DHs = (float**)calloc(StreamN, sizeof(float*));
  float *invTs;
  float **DinvTs;
  DinvTs = (float**)calloc(StreamN, sizeof(float*));
  HHs = (float*)malloc(params_mem_size);
  invTs = (float*)malloc(params_mem_size);

  //note: gpu part
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&DHs[gpu_i], params_mem_size/StreamN));
    CudaSafeCall(hipMalloc((void**)&DinvTs[gpu_i], params_mem_size/StreamN));
  }

  //begin (initialize random seeds)
  //Declare sizes
  unsigned int totalRngs = Pnum * H_BN * H_TN / WarpStandard_K;
  unsigned seedBytes = totalRngs * sizeof(unsigned int) * WarpStandard_STATE_WORDS;
  unsigned int **seedDevice;
  seedDevice = (unsigned int**)calloc(StreamN, sizeof(unsigned int*));
  //note: gpu part
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void **)&seedDevice[gpu_i], seedBytes/StreamN));
  }
  unsigned int* seedHost = (unsigned int*)malloc(seedBytes);
  srand(seed);
  for(int i = 0; i < seedBytes / sizeof(unsigned int); i++)
    seedHost[i] = uni01(generator) * UINT_MAX;
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(seedDevice[gpu_i], seedHost + (seedBytes/sizeof(unsigned int)/StreamN)*gpu_i, (seedBytes/StreamN), hipMemcpyHostToDevice, stream[gpu_i]));
  }
  //Mtgp
	//Define state:
	
	//Env:
	unsigned int NBlock = grid;
	unsigned int ThreadperBlock = H_TN; // Max limit at 256	
	unsigned int seed = 99;
  unsigned int total_Nthread = Pnum * H_BN * H_TN;
	


  //hiprand
  hiprandState* devStates;
  hipMalloc ( &devStates, total_Nthread*sizeof( hiprandState  )  );
  setup_kernel<<<grid, block>>>(devStates);


	//generate_kernel<<<NBlock,ThreadperBlock>>>(DStates);

	///cleanup

	//hipFree(DStates);
	//hipFree(DParams);
  //end (initialize random seeds)


  //Set up output data path
  char dir[128];
  sprintf(dir, "Data/L_%d-%s", H_SpinSize, Output.c_str());
  mkdir(dir, 0755);
  char Seedfn[128];
  sprintf(Seedfn, "Data/L_%d-%s/seed", H_SpinSize, Output.c_str());

  char config_bak[128];
  sprintf(config_bak, "%s/config.json", dir);
  ofstream oconfig(config_bak);
  oconfig << setw(8) << configj << endl;

  int seedfd = open(Seedfn, O_CREAT | O_WRONLY, 0644);
  write(seedfd, seedHost, seedBytes);
  close(seedfd);


  //MEASUREMENT initialize

  configuration CONF(Pnum, dir);
  measurements MEASURE(dir, Pnum, BIN_SZ); //Tnum for parallel tempering for T
#ifdef GET_CORR
  char Corrfn[128];
  sprintf(Corrfn, "%s/Corrpt", dir);
  correlation CORR(Pnum, Corrfn);
#endif

  StopWatchInterface *timer=NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  //Give initial configuration and settle the systems down to equilibrium states
  CONF.initialize(ORDER,generator);
  int Eqii = 0;//150;
  for(int i = 0; i < Pnum; i++){
    HHs[i] = Hls[0][i];
    invTs[i] = 1.0/Tls[0][i];
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
  }
  double *Ms = (double*)malloc(Pnum * sizeof(double));
  double *Es = (double*)malloc(Pnum * sizeof(double));
  int *accept1 = (int*)calloc((Tnum - 1)*Hnum + Tnum*(Hnum - 1), sizeof(int));
  vector<float> acceptance;  
  int *stay = (int*)calloc(Tnum * Hnum, sizeof(int));
  int *staylargest = (int*)calloc(Tnum * Hnum, sizeof(int));
  int *staytmp = (int*)calloc(Tnum * Hnum, sizeof(int));
  float cnt = 0;
  CONF.backtoHost(); //watch out! it must be compatible with the
  CONF.writedata();

  for(int i = 0; i < EQUI_N; i++){
    if (i % 10 ==0) printf("%d\n",i);
    fflush(stdout);
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    //Parallel Tempering
    cnt += PTF;
    for(int p = 0; p < int(cnt); p++){
      MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
      tempering_simple(Ms, Es, accept1, generator);
      for(int t = 0; t < Pnum; t++){
        HHs[t] = Hls[0][Po[t]];
        invTs[t] = 1.0/Tls[0][Po[t]];
      }
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
        CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
        CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
      }
    }
    if(int(cnt))
    cnt = 0;
  }
  CONF.backtoHost(); //watch out! it must be compatible with the
  CONF.writedata();

  //Do measurements (annealing)

  //int *accept = (int*)calloc(Pnum - 1, sizeof(int));
  int *accept = (int*)calloc((Tnum - 1)*Hnum + Tnum*(Hnum - 1), sizeof(int));
  for(C_i = 0 ; C_i < Cnum ; C_i ++){

    for(int t = 0; t < Pnum; t++){
      HHs[t] = Hls[C_i][Po[t]];
      invTs[t] = 1.0/Tls[C_i][Po[t]];
    }
    for (int i = 0; i< (Tnum - 1)*Hnum + Tnum*(Hnum - 1); i++) accept[i] = 0;
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
      CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    }

    for(int i = 0; i < EQUI_Ni; i++){
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
	hipSetDevice(device_0 + gpu_i);
	SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
      }
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
	hipSetDevice(device_0 + gpu_i);
	SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
      }
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
	hipSetDevice(device_0 + gpu_i);
	SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
      }
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
        MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
        tempering_simple(Ms, Es, accept1, generator);
        for(int t = 0; t < Pnum; t++){
          HHs[t] = Hls[C_i][Po[t]];
          invTs[t] = 1.0/Tls[C_i][Po[t]];
        }
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
          CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
        }
      }
      if(int(cnt))
        cnt = 0;
    }
    cnt = 0;
    for(int b = 0; b < BIN_NUM; b++){
      //Take the ensemble average
      for(int i = 0; i < BIN_SZ; i++){
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
        }
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
        }
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
        }
        if (b>10) MEASURE.hist_start = 1;
        MEASURE.measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
        /*
#ifdef GET_CORR
        if ( i % f_CORR==0){
          CORR.extract(Po, CONF);//==
        }
#endif
*/
        cnt += PTF;
        for(int p = 0; p < int(cnt); p++){
          tempering(Ms, Es, accept, staytmp, stay, generator);
          for(int t = 0; t < Pnum; t++){
            HHs[t] = Hls[C_i][Po[t]];
            invTs[t] = 1.0/Tls[C_i][Po[t]];
            if (stay[Po[t]] > staylargest[Po[t]]){
              CONF.Dominatestateback(Po[t],t);
              staylargest[Po[t]] = stay[Po[t]];
            }
          }
          for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
            hipSetDevice(device_0 + gpu_i);
            CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
            CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
          }
        }
        if(int(cnt))
          cnt = 0;
      }
      MEASURE.normalize_and_save_and_reset();
			CONF.writedata();
			for ( int t = 0; t < Pnum; t++ ){
				staylargest[t] = 0;
			}
#ifdef GET_CORR
			sprintf(Corrfn, "%s/Corr_%d", dir, b);
			CORR.changefile(Corrfn);
			for(int i = 0; i < CORR_N * f_CORR; i++){
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
				}
				if ( i % f_CORR==0){
					CORR.extract(Po, CONF);//==
				}
			}
			CORR.avg_write_reset(Po);
			for(int i = 0; i < 1000; i++){
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
				}
				//Parallel Tempering
				cnt += PTF;
				for(int p = 0; p < int(cnt); p++){
					MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
					tempering_simple(Ms, Es, accept, generator);
					for(int t = 0; t < Pnum; t++){
						HHs[t] = Hls[0][Po[t]];
						invTs[t] = 1.0/Tls[0][Po[t]];
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
						CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
					}
				}
				if(int(cnt))
				cnt = 0;
			}
#endif
    }
    for (int iii = 0 ; iii < Pnum; iii ++){
      ivPo[Po[iii]] = iii;
    }
    //CONF.backtoHost(); //watch out! it must be compatible with the
  }
  char Histfn[128];
  sprintf(Histfn, "%s/%s", dir, "EHistogram");
  FILE *f_hist = fopen(Histfn, "w");
  fwrite(MEASURE.EHistogram, sizeof(unsigned int),Slice_NUM * Pnum, f_hist);
  fclose(f_hist);
  sprintf(Histfn, "%s/%s", dir, "ChernHistogram");
  FILE *f_chist = fopen(Histfn, "w");
  fwrite(MEASURE.ChernHistogram, sizeof(unsigned int), Slice_CNUM * Pnum, f_chist);
  fclose(f_chist);
  free(Ms);
  sdkStopTimer(&timer);
  double time = 1.0e-3 * sdkGetTimerValue(&timer);
    printf("G\n");
    fflush(stdout);


//======================= print details ==========================
  char detailFn[128];
  json detailj, sdetailj, finalstatej;
  sprintf(detailFn, "%s/details.json", dir);
  sdetailj["elapsed_time(s)"] = time;
  double speed = 0;
  speed = (H_N / time / 1000000000) * ((EQUI_Ni + BIN_SZ * BIN_NUM) * Cnum + EQUI_N) * Pnum;
  sdetailj["speed(GHz)"] = speed;
  sdetailj["NumGPU"] = StreamN;
  sdetailj["BlockSize_x"] =  H_BlockSize_x;
  sdetailj["BlockSize_y"] =  H_BlockSize_y;
  sdetailj["GridSize_x"] =  H_GridSize_x;
  sdetailj["GridSize_y"] =  H_GridSize_y;
  finalstatej["final_order"] = Po;
  /*
  for(int i = 0; i < Pnum; i++){
    fprintf(detailFp, "\n");
    fprintf(detailFp, "Po[%d]=%d",i,Po[i]);
  }*/
  if (PTF != 0 ){
    for(int i = 0; i < (Tnum - 1)*Hnum + Tnum*(Hnum - 1); i++)
      acceptance.push_back(float(accept[i]) / (BIN_SZ * BIN_NUM * PTF));
  }
  finalstatej["acceptance_rates"] = acceptance;
  finalstatej["N_histE"] = Slice_NUM;
  finalstatej["E_lowest"]  =  E_lowest;
  finalstatej["E_highest"] =  E_highest;
  finalstatej["N_histChern"] = Slice_CNUM;
  finalstatej["Chern_lowest"] = Chern_lowest;
  finalstatej["Chern_highest"] = Chern_highest;
  detailj["simulation"] = sdetailj;
  detailj["finalstate"] = finalstatej;
  ofstream odetail(detailFn);
  odetail << setw(8) << detailj << endl;

  
//===================== print details end =========================

  //Set free memory
  free(seedHost);
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipFree(DinvTs[gpu_i]));
    CudaSafeCall(hipFree(DHs[gpu_i]));
    CudaSafeCall(hipFree(seedDevice[gpu_i]));
  }
  //CORR.~correlation();
  //MEASURE.~measurements();
  //CONF.~configuration();
  return 0;
}


//=============================== functions ==================================
void tempering_simple(double *Ms, double *Es, int *accept, mt19937 &generator){
  uniform_real_distribution<double >uni01;
  int map[Pnum];	//map[t] the configuration of t'th temperature
  int i, j, tmp, partT_num = (Tnum - 1) * Hnum;
  double tmpEM;

  for(i = 0; i < Pnum; i++)
    map[Po[i]] = i;

  double delta;
  int flag = 0;
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //T excnange
      if (i < Tnum -1){
	delta = (Es[j * Tnum + i] - Es[j * Tnum + i + 1]) * ((1.0 / Tls[C_i][j*Tnum + i]) - (1.0 / Tls[C_i][j*Tnum +i + 1]));
	if(delta > 0)
	  flag = 1;
	else if(uni01(generator) < exp(delta))
	  flag = 1;
	if(flag){
	  tmp = Po[map[j * Tnum + i]];
	  Po[map[j * Tnum + i]] = Po[map[j * Tnum + i + 1]];
	  Po[map[j * Tnum + i + 1]] = tmp;
	  tmp = map[j * Tnum + i];
	  map[j * Tnum + i] = map[j * Tnum + i + 1];
	  map[j * Tnum + i + 1] = tmp;
	  tmpEM = Es[j * Tnum + i];
	  Es[j * Tnum + i] = Es[j * Tnum + i + 1];
	  Es[j * Tnum + i + 1] = tmpEM;
	  tmpEM = Ms[j * Tnum + i];
	  Ms[j * Tnum + i] = Ms[j * Tnum + i + 1];
	  Ms[j * Tnum + i + 1] = tmpEM;
	  accept[j * (Tnum - 1) + i] += 1;
	  flag = 0;
	}
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //H excnange
      if (j < Hnum -1){
        delta = (Ms[(j + 1) * Tnum + i] - Ms[j * Tnum + i]) * ( Hls[C_i][j * Tnum + i] - Hls[C_i][(j + 1) * Tnum + i]) / Tls[C_i][j * Tnum + i];
        if(delta > 0)
          flag = 1;
        else if(uni01(generator) < exp(delta))
          flag = 1;
        if(flag){
          tmp = Po[map[j * Tnum + i]];
          Po[map[j * Tnum + i]] = Po[map[(j + 1) * Tnum + i]];
          Po[map[(j + 1) * Tnum + i]] = tmp;
          tmp = map[j * Tnum + i];
          map[j * Tnum + i] = map[(j + 1) * Tnum + i];
          map[(j + 1) * Tnum + i] = tmp;
          tmpEM = Es[j * Tnum + i];
          Es[j * Tnum + i] = Es[(j + 1) * Tnum + i];
          Es[(j + 1) * Tnum + i] = tmpEM;
          tmpEM = Ms[j * Tnum + i];
          Ms[j * Tnum + i] = Ms[(j + 1) * Tnum + i];
          Ms[(j + 1) * Tnum + i] = tmpEM;
          accept[partT_num + j * Tnum + i] += 1;
          flag = 0;
        }
      }
    }
  }
}

//=============================== functions ==================================
void tempering(double *Ms, double *Es, int *accept, int *staytmp, int *stay, mt19937 &generator){
  uniform_real_distribution<double >uni01;
  int map[Pnum];	//map[t] the configuration of t'th temperature
  int i, j, tmp, partT_num = (Tnum - 1) * Hnum;
  double tmpEM;

  for(i = 0; i < Pnum; i++)
    map[Po[i]] = i;

  double delta;
  int flag = 0;
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
	staytmp[j * Tnum + i] = 1;
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //T excnange
      if (i < Tnum -1){
	delta = (Es[j * Tnum + i] - Es[j * Tnum + i + 1]) * ((1.0 / Tls[C_i][j*Tnum + i]) - (1.0 / Tls[C_i][j*Tnum +i + 1]));
	if(delta > 0)
	  flag = 1;
	else if(uni01(generator) < exp(delta))
	  flag = 1;
	if(flag){
	  tmp = Po[map[j * Tnum + i]];
	  Po[map[j * Tnum + i]] = Po[map[j * Tnum + i + 1]];
	  Po[map[j * Tnum + i + 1]] = tmp;
	  tmp = map[j * Tnum + i];
	  map[j * Tnum + i] = map[j * Tnum + i + 1];
	  map[j * Tnum + i + 1] = tmp;
	  tmpEM = Es[j * Tnum + i];
	  Es[j * Tnum + i] = Es[j * Tnum + i + 1];
	  Es[j * Tnum + i + 1] = tmpEM;
	  tmpEM = Ms[j * Tnum + i];
	  Ms[j * Tnum + i] = Ms[j * Tnum + i + 1];
	  Ms[j * Tnum + i + 1] = tmpEM;
	  accept[j * (Tnum - 1) + i] += 1;
	  flag = 0;
	  staytmp[j * Tnum + i] *= 0;
	  staytmp[j * Tnum + i + 1] *= 0;
	}
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //H excnange
      if (j < Hnum -1){
        delta = (Ms[(j + 1) * Tnum + i] - Ms[j * Tnum + i]) * ( Hls[C_i][j * Tnum + i] - Hls[C_i][(j + 1) * Tnum + i]) / Tls[C_i][j * Tnum + i];
        if(delta > 0)
          flag = 1;
        else if(uni01(generator) < exp(delta))
          flag = 1;
        if(flag){
          tmp = Po[map[j * Tnum + i]];
          Po[map[j * Tnum + i]] = Po[map[(j + 1) * Tnum + i]];
          Po[map[(j + 1) * Tnum + i]] = tmp;
          tmp = map[j * Tnum + i];
          map[j * Tnum + i] = map[(j + 1) * Tnum + i];
          map[(j + 1) * Tnum + i] = tmp;
          tmpEM = Es[j * Tnum + i];
          Es[j * Tnum + i] = Es[(j + 1) * Tnum + i];
          Es[(j + 1) * Tnum + i] = tmpEM;
          tmpEM = Ms[j * Tnum + i];
          Ms[j * Tnum + i] = Ms[(j + 1) * Tnum + i];
          Ms[(j + 1) * Tnum + i] = tmpEM;
          accept[partT_num + j * Tnum + i] += 1;
          flag = 0;
	  staytmp[j * Tnum + i] *= 0;
	  staytmp[(j + 1) * Tnum + i] *= 0;
        }
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      stay[j * Tnum + i] = staytmp[j * Tnum + i]?(stay[j * Tnum + i]+1):0;
    }
  }
}
void var_examine(){
#ifndef TRI
  if(H_SpinSize % (H_BlockSize_x * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
#ifdef TRI
  if(H_SpinSize % (H_BlockSize_x * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
//#ifndef THIN
//  if (H_SpinSize_z != 1){
//    fprintf(stderr, "SpinSize_z must be 1 %d\n", H_BlockSize_y * 2);
//    exit(0);
//  }
//#endif
}
