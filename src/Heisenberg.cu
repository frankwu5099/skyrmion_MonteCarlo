#include "hip/hip_runtime.h"
//Ms PT function writedata  HHs
//why exchange conf and measurement works
using namespace std;

#define BIN_SZ 3000//0//00//
#define BIN_NUM 3//0
#define EQUI_N 20000//0//0//00////16000000

#define ID "skyr_d16AC_testmeasurement_TRI"
#define PTF	(float(0.00))	//Frequency of parallel tempering
#include "params.cuh"
#include "updates.cuh"
#include "measurements.cuh"
#include "configuration.cuh"
#include "extend.cu"
#define EQUI_Ni (4000)//0)
#define GET_CORR
#define f_CORR (500)


unsigned seed = 73;
mt19937 rng(seed);
uniform_01<mt19937> uni01_sampler(rng);
void tempering(double*, int*);
unsigned int block = BlockSize_x * BlockSize_y;
unsigned int grid = 0;
vector<float> Tls;
vector<float> Hls;
vector<int>Po;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
vector<int>ivPo;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
unsigned int Tnum;
unsigned int Hnum;
float Cparameter = 0.8;
void var_examine();



int main(int argc, char *argv[]){
  //call GPU

  if (setDev()==1){
    return 1;
  }
  hipGetLastError();
  CudaCheckError();

  //examine variables
  var_examine();

  // ========================== initialize ===========================

  //begin (read in temperatures)
  unsigned int Temp_mem_size;
  unsigned int H_mem_size;

  if(argc > 2){
    float tmp;
    FILE *Tfp = fopen(argv[1], "r");
    int i = 0;
    while(fscanf(Tfp, "%f", &tmp) != EOF){
      Tls.push_back(tmp);
      i++;
    }
    fclose(Tfp);
    Tnum = Tls.size();
    Temp_mem_size = Tnum * sizeof(float);

    i = 0;
    Tfp = fopen(argv[2], "r");
    while(fscanf(Tfp, "%f", &tmp) != EOF){
      Hls.push_back((DD * DD + DR * DR)*tmp);
      i++;
    }
    fclose(Tfp);
    Hnum = Hls.size();
    H_mem_size = Hnum * sizeof(float);

    grid = Pnum * BN;
  }
  else{
    fprintf(stderr, "Give me a temperature set!!!\n");
    fprintf(stderr, "Give me a field set!!!\n");
    exit(0);
  }
  //end (read in temperatures)


  //invTs is the inverse temperature in order of configurations on GPU.
  for (int i = 0; i < Pnum ; i++){
    Po.push_back(i);
    ivPo.push_back(i);
  }
  float *Cparameters;
  float *HPparameters;
  float *DPparameters;
  Cparameters = (float*)malloc(C_mem_size);
  HPparameters = (float*)malloc(P_mem_size);
  CudaSafeCall(hipMalloc((void**)&DPparameters, P_mem_size));


  //begin (initialize random seeds)
  //Declare sizes
  unsigned int totalRngs = Pnum * TN / WarpStandard_K;
  unsigned seedBytes = totalRngs * sizeof(unsigned int) * WarpStandard_STATE_WORDS;
  unsigned int *seedDevice = 0;
  CudaSafeCall(hipMalloc((void **)&seedDevice, seedBytes));
  unsigned int* seedHost = (unsigned int*)malloc(seedBytes);
  srand(seed);
  for(int i = 0; i < seedBytes / sizeof(unsigned int); i++)
    seedHost[i] = uni01_sampler() * UINT_MAX;
  CudaSafeCall(hipMemcpy(seedDevice, seedHost, seedBytes, hipMemcpyHostToDevice));
  //end (initialize random seeds)


  //Set up output data path
  char dir[128];
  sprintf(dir, "Data/L_%d-%s", SpinSize, ID);
  mkdir(dir, 0755);
  char Seedfn[128];
  sprintf(Seedfn, "Data/L_%d-%s/seed", SpinSize, ID);
  int seedfd = open(Seedfn, O_CREAT | O_WRONLY, 0644);
  write(seedfd, seedHost, seedBytes);
  close(seedfd);


  //MEASUREMENT initialize

  configuration CONF(Pnum, dir);
  measurements MEASURE(dir, Pnum, BIN_SZ); //Tnum for parallel tempering for T
#ifdef GET_CORR
  correlation CORR(Pnum, dir);
#endif

  StopWatchInterface *timer=NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  //Give initial configuration and settle the systems down to equilibrium states
  CONF.initialize(ORDER);
  int Eqii = 0;//150;
  for(int i = 0; i < Hnum; i++)
    HHs[i] = Hls[i];
  for(int i = 0; i < Tnum; i++)
    invTs[i] = 1.0/Tls[i];
  CudaSafeCall(hipMemcpy(DPparameters, HPparameters, P_mem_size, hipMemcpyHostToDevice));
  double *Ms = (double*)malloc(Pnum * sizeof(double));
  int *accept1 = (int*)calloc(Pnum - 1, sizeof(int));
  float cnt = 0;
  Cparameter = Cparameters[Cnum-1];

  for(int i = 0; i < EQUI_N; i++){
    if (i % 10 ==0) printf("%d\n",i);
    SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DPparameters, Cparameter);
    //================================= no PT ==========================================
    /*
       cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout); //, Dcorr);
       hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost);
       for(int t = 0; t < Pnum; t++){
       int raw_off = t * MEASURE_NUM * BN;
       E = 0;
       for(int j = 3 * BN; j < 4 * BN; j++)
       E += Hout[raw_off + j];
       Ms[Ho[t]] = E;	//Es is the energies in order of temperature set
       }
    //Parallel Tempering
    cnt += PTF;
    for(int p = 0; p < int(cnt); p++){
    tempering(Ms, accept1);
    for(int t = 0; t < Pnum; t++) HPparameters[t] = Porder(t);
    hipMemcpy(DPparameters, HPparameters, H_mem_size, hipMemcpyHostToDevice);
    }
    if(int(cnt))
    cnt = 0;
     */
  }

  //Do measurements (annealing)

  int *accept = (int*)calloc(Pnum - 1, sizeof(int));
  for(int C_i = 0 ; C_i < Cnum ; C_i ++){
    for (int i = 0; i< Pnum-1; i++) accept[i] = 0;
    Cparameter = Ccurrent(C_i);
    for(int i = 0; i < EQUI_Ni; i++){
      if (i % 10 ==0) printf("%f : %d\n", Cparameter, i);
      SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DPparameters, Cparameter);
      //======================= no PT ===============================
      /*
	 cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout); //, Dcorr);
	 hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost);
	 for(int t = 0; t < Pnum; t++){
	 int raw_off = t * MEASURE_NUM * BN;
	 E = 0;
	 for(int j = 3 * BN; j < 4 * BN; j++)
	 E += Hout[raw_off + j];
	 Ms[Ho[t]] = E;	//Es is the energies in order of temperature set
	 }
      //Parallel Tempering
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
      tempering(Ms, accept1);
      for(int t = 0; t < Pnum; t++) HPparameters[t] = Porder(t);
      hipMemcpy(DPparameters, HPparameters, P_mem_size, hipMemcpyHostToDevice);
      }
      if(int(cnt))
      cnt = 0;
       */
    }
    cnt = 0;
    for(int b = 0; b < BIN_NUM; b++){
      //Take the ensemble average
      for(int i = 0; i < BIN_SZ; i++){
	SSF(CONF.Dx, CONF.Dy, CONF.Dz, seedDevice, DPparameters, Cparameter);
	MEASURE.measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, HHs);
#ifdef GET_CORR
	if ( i % f_CORR==0){
	  CORR.extract(Po, CONF);//==
	}
#endif
	//Parallel Tempering
	/*
	   cnt += PTF;
	   for(int p = 0; p < int(cnt); p++){
	   tempering(Ms, accept);
	   for(int t = 0; t < Pnum; t++) HPparameters[t] = Porder(t);
	   hipMemcpy(DPparameters, HPparameters, P_mem_size, hipMemcpyHostToDevice);
	   }
	   if(int(cnt))
	   cnt = 0;
	 */
      }
      MEASURE.normalize_and_save_and_reset();
    }
    for (int iii = 0 ; iii < Pnum; iii ++){
      ivPo[Po[iii]] = Po[iii];
    }
    CONF.backtoHost();
    CONF.writedata();
#ifdef GET_CORR
    CORR.avg_write_reset();
#endif
  }
  free(Ms);
  sdkStopTimer(&timer);
  double time = 1.0e-3 * sdkGetTimerValue(&timer);


//======================= print details ==========================
  char detailFn[128];
  sprintf(detailFn, "%s/details", dir);
  FILE *detailFp = fopen(detailFn, "w");
  fprintf(detailFp, "elapsed time = %f (sec)\n", time);
  double speed = 0;
  speed = (N / time / 1000000000) * (BIN_SZ * BIN_NUM + EQUI_N) * Pnum * Cnum;
  fprintf(detailFp, "speed = %f (GHz)\n", speed);
  fprintf(detailFp, "RNG: WarpStandard\n", SpinSize);
  fprintf(detailFp, "SpinSize = %d\n", SpinSize);
  fprintf(detailFp, "A = %4.3f\n", A);
  fprintf(detailFp, "D_Rashba = %4.3f\n", DR);
  fprintf(detailFp, "D_Dresselhaus = %4.3f\n", DD);
  fprintf(detailFp, "BlockSize_x = %d\n", BlockSize_x);
  fprintf(detailFp, "BlockSize_y = %d\n", BlockSize_y);
  fprintf(detailFp, "GridSize_x = %d\n", GridSize_x);
  fprintf(detailFp, "GridSize_y = %d\n", GridSize_y);
  fprintf(detailFp, "Bin Size = %d\n", BIN_SZ);
  fprintf(detailFp, "Bin Number = %d\n", BIN_NUM);
  fprintf(detailFp, "Equilibration N = %d\n", EQUI_N);
  fprintf(detailFp, "Equilibration Ni = %d\n", EQUI_Ni);
  fprintf(detailFp, "f_CORR = %d\n", f_CORR);
  fprintf(detailFp, "PT frequency = %3.2f\n", PTF);
  fprintf(detailFp, "Tnum = %d\n", Tnum);
  fprintf(detailFp, "Temperature Set: ");
  for(int i = 0; i < Tnum; i++){
    fprintf(detailFp, "%.5f  ", Tls[i]);
  }
  fprintf(detailFp, "\nHnum = %d\n", Hnum);
  fprintf(detailFp, "field Set: ");
  for(int i = 0; i < Hnum; i++){
    fprintf(detailFp, "%.5f  ", Hls[i]/(DR*DR + DD*DD));
  }
  for(int i = 0; i < Pnum; i++){
    fprintf(detailFp, "\n");
    fprintf(detailFp, "Po[%d]=%d",i,Po[i]);
  }
  fprintf(detailFp, "\n");
  fprintf(detailFp, "Acceptance rates: ");
  if (PTF != 0 ){
    for(int i = 0; i < Pnum - 1; i++)
      fprintf(detailFp, "%4.3f  ", float(accept[i]) / (BIN_SZ * BIN_NUM * PTF));
  }
  fprintf(detailFp, "\n");
  if (ORDER){
    fprintf(detailFp, "Configurations start from ordered state.\n");
  }
  else {
    fprintf(detailFp, "Configurations start from random state.\n");
  }
  fprintf(detailFp, "Done by Po-Kuan Wu ^_^\n", EQUI_N);
  fclose(detailFp);
//===================== print details end =========================

  //Set free memory
  free(seedHost);
  CudaSafeCall(hipFree(DPparameters));
  CudaSafeCall(hipFree(seedDevice));
  //CORR.~correlation();
  //MEASURE.~measurements();
  //CONF.~configuration();
  return 0;
}


//=============================== functions ==================================
void tempering(double *Ms, int *accept){
  int map[Pnum];	//map[t] the configuration of t'th temperature
  for(int i = 0; i < Pnum; i++){
    map[Po[i]] = i;
  }
  double delta;
  int flag = 0;
  for(int i = 0; i < Pnum - 1; i++){
    delta = exchangecriterion(i);
    if(delta > 0)
      flag = 1;
    else if(uni01_sampler() < exp(delta))
      flag = 1;
    if(flag){
      int tmp = Po[map[i]];
      Po[map[i]] = Po[map[i + 1]];
      Po[map[i + 1]] = tmp;
      tmp = map[i];
      map[i] = map[i + 1];
      map[i + 1] = tmp;
      double tmpE = Ms[i];
      Ms[i] = Ms[i + 1];
      Ms[i + 1] = tmpE;
      accept[i] += 1;
      flag = 0;
    }
  }
}


void var_examine(){
#ifndef TRI
  if(SpinSize % (BlockSize_x * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_x * 2);
    exit(0);
  }
  if(SpinSize % (BlockSize_y * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_y * 2);
    exit(0);
  }
#endif
#ifdef TRI
  if(SpinSize % (BlockSize_x * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_x * 2);
    exit(0);
  }
  if(SpinSize % (BlockSize_y * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_y * 2);
    exit(0);
  }
#endif
#ifndef THIN
  if (SpinSize_z != 1){
    fprintf(stderr, "SpinSize_z must be 1 %d\n", BlockSize_y * 2);
    exit(0);
  }
#endif
}
