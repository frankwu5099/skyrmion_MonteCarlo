#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/time.h>
#include <fcntl.h>
#include <stdint.h>
#include <time.h>
#include <helper_timer.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "WarpStandard.cuh"
using namespace std;
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_01.hpp>
using namespace boost;
unsigned seed = 73;
mt19937 rng(seed);
uniform_01<mt19937> uni01_sampler(rng);
//--------variables for one temperature replica----------
#define SpinSize 32                          //Each thread controls 2 by 2 by 2 spins
#define SpinSize_z 8
#define BlockSize_x 16
#define BlockSize_y 16
#define GridSize_x (SpinSize/BlockSize_x/2)
#define GridSize_y (SpinSize/BlockSize_y/2)
#define N (SpinSize*SpinSize*(SpinSize_z))              //The number of spins of the system + boundary effective spins
#define Nplane (SpinSize*SpinSize)              //The number of spins of the system
#define TN (Nplane / 4)									//The number of needed threads
#define BN (GridSize_x*GridSize_y)       //The number of needed blocks
//---------------------End-------------------------------
#define BIN_SZ 10000//
#define BIN_NUM 20
#define EQUI_N 100000////16000000
#define relax_N 0
#define CORR_L 0
#define MEASURE_NUM 5
#define coo(k, j, i) ((k) * Nplane + (j) * SpinSize + (i))

/*
#define Jxx (-0.8)
#define Jyy (-0.8)
#define Jzz (-1.0)
#define Jxy (0.0)
#define Jyx (0.0)
#define Jyz (0.0)
#define Jzy (0.0)
#define Jxz (0.0)
#define Jzx (0.0)
#define H (6.0)
*/
#define NORM (float(4.656612873077393e-10)) // UINT_MAX * NORM = 2
#define TOPI (float(1.462918078360668e-9))
#define TWOPI (float(6.28318530717956))	//2*pi
/*
#define BXPxx(BD) Jxx
#define BYPxx(BD) Jxx
#define BXMxx(BD) Jxx
#define BYMxx(BD) Jxx
#define BXPyy(BD) Jyy
#define BYPyy(BD) Jyy
#define BXMyy(BD) Jyy
#define BYMyy(BD) Jyy
#define BXPzz(BD) Jzz
#define BYPzz(BD) Jzz
#define BXMzz(BD) Jzz
#define BYMzz(BD) Jzz
#define BXPxy(BD) Jxy
#define BYPxy(BD) Jxy
#define BXMxy(BD) Jxy
#define BYMxy(BD) Jxy
#define BXPyx(BD) Jyx
#define BYPyx(BD) Jyx
#define BXMyx(BD) Jyx
#define BYMyx(BD) Jyx
#define BXPyz(BD) Jyz
#define BYPyz(BD) Jyz
#define BXMyz(BD) Jyz
#define BYMyz(BD) Jyz
#define BXPzy(BD) Jzy
#define BYPzy(BD) Jzy
#define BXMzy(BD) Jzy
#define BYMzy(BD) Jzy
#define BXPzx(BD) Jzx
#define BYPzx(BD) Jzx
#define BXMzx(BD) Jzx
#define BYMzx(BD) Jzx
#define BXPxz(BD) Jxz
#define BYPxz(BD) Jxz
#define BXMxz(BD) Jxz
#define BYMxz(BD) Jxz
*/
#define Hfinal (0.016000)
#define A (0.171573)//(-DR * DR)
#define DR (0.0)//(1.41421)//(0.585786)//(0.8)////(0.4)//(1.02749)
#define DD (0.585786)//(1.41421)//(0.8)////(0.4)//(1.02749)
#define BXPxx (1.000000)
#define BYPxx (1.000000)
#define BZPxx (1.000000)
#define BXMxx (1.000000)
#define BYMxx (1.000000)
#define BZMxx (1.000000)
#define BXPyy (1.000000)
#define BYPyy (1.000000)
#define BZPyy (1.000000)
#define BXMyy (1.000000)
#define BYMyy (1.000000)
#define BZMyy (1.000000)
#define BXPzz (1.000000)
#define BYPzz (1.000000)
#define BZPzz (1.000000)
#define BXMzz (1.000000)
#define BYMzz (1.000000)
#define BZMzz (1.000000)

#define BXPxy (0.000000)
#define BYPxy (0.000000)
#define BXMxy (0.000000)
#define BYMxy (0.000000)
#define BXPyx (0.000000)
#define BYPyx (0.000000)
#define BXMyx (0.000000)
#define BYMyx (0.000000)

#define BZPyx (-DD)
#define BZMyx (DD)
#define BZPxy (DD)
#define BZMxy (-DD)

#define BXPyz (DD)
#define BYPyz (DR)
#define BXMyz (-DD)
#define BYMyz (-DR)
#define BXPzy (-DD)
#define BYPzy (-DR)
#define BXMzy (DD)
#define BYMzy (DR)

#define BXPzx (-DR)
#define BYPzx (DD)
#define BXMzx (DR)
#define BYMzx (-DD)
#define BXPxz (DR)
#define BYPxz (-DD)
#define BXMxz (-DR)
#define BYMxz (DD)
#define ID "skyr_d16z8AO_annealingT_thin"
#define PTF	(float(0.00))	//Frequency of parallel tempering

#include "flip1.cu"
#include "flip2.cu"
//#include "relax1.cu"
//#include "relax2.cu"
#include "cals.cu"
#include "corr.cu"
#include "extend.cu"
#define EQUI_Ni (400000)//)
#define GET_CORR
#define f_CORR (500)

void tempering(double*, int*);
unsigned int block = BlockSize_x * BlockSize_y;
unsigned int grid;
vector<float> Tls;
vector<float> Hls;
vector<int>Ho;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
vector<int>ivHo;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
unsigned int Tnum;
unsigned int Hnum;
float Tcurrent = 0.8;

int main(int argc, char *argv[]){
	//initialize
  if (setDev()==1){
    return 1;

  }
	if(SpinSize % (BlockSize_x * 2) != 0){
		fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_x * 2);
		exit(0);
	}
	if(SpinSize % (BlockSize_y * 2) != 0){
		fprintf(stderr, "SpinSize must be the multiple of %d\n", BlockSize_y * 2);
		exit(0);
	}
	if(CORR_L > SpinSize / 2){
		fprintf(stderr, "The length for correlation measurement must be smaller than half the system size.\n");
		exit(0);
	}
	//read in temperatures
	unsigned int Temp_mem_size;
	unsigned int H_mem_size;
	if(argc > 2){
		float tmp;
		FILE *Tfp = fopen(argv[1], "r");
		int i = 0;
		while(fscanf(Tfp, "%f", &tmp) != EOF){
			Tls.push_back(tmp);
			i++;
		}
		fclose(Tfp);
		Tnum = Tls.size();
		i = 0;
		Temp_mem_size = Tnum * sizeof(float);
		Tfp = fopen(argv[2], "r");
		while(fscanf(Tfp, "%f", &tmp) != EOF){
			Hls.push_back((DD * DD + DR * DR)*tmp);
			Ho.push_back(i);
			ivHo.push_back(i);
			i++;
		}
		fclose(Tfp);
		Hnum = Hls.size();
		H_mem_size = Hnum * sizeof(float);
		grid = Hnum * BN;
	}
	else{
		fprintf(stderr, "Give me a temperature set!!!\n");
		fprintf(stderr, "Give me a field set!!!\n");
		exit(0);
	}
	//invTs is the inverse temperature in order of configurations on GPU.
	float *HHs;
	float *DHs;
	int *DTo;
	hipMalloc((void**)&DTo, Hnum * sizeof(int));
	HHs = (float*)malloc(H_mem_size);
	hipMalloc((void**)&DHs, H_mem_size);
	//Declare sizes
	unsigned int Spin_mem_size = Hnum * N * sizeof(float);
	//unsigned int Corr_mem_size = Tnum * CORR_L * CORR_L * sizeof(double);
	unsigned int Out_mem_size = Hnum * MEASURE_NUM * BN * sizeof(double);
	unsigned int totalRngs = Hnum * TN / WarpStandard_K;
	float *Hconfx = (float*)malloc(Spin_mem_size);
	float *Hconfy = (float*)malloc(Spin_mem_size);
	float *Hconfz = (float*)malloc(Spin_mem_size);
	double *Hout = (double*)malloc(Out_mem_size);
#ifdef GET_CORR
	unsigned int Spin_mem_size_p = Hnum * Nplane * sizeof(float);
	unsigned int Spin_mem_size_d = Hnum * Nplane * sizeof(double);
	int corrcount = 0;
	double *HSum_corr = (double*)malloc(Spin_mem_size_d);
#endif
	//double *Hcorr = (double*)malloc(Corr_mem_size);
	unsigned seedBytes = totalRngs * sizeof(unsigned int) * WarpStandard_STATE_WORDS;
	unsigned int *seedDevice = 0;
	if(hipMalloc((void **)&seedDevice, seedBytes)){
		fprintf(stderr, "Error couldn't allocate state array of size %u\n", seedBytes);
		exit(1);
	}
	unsigned int* seedHost = (unsigned int*)malloc(seedBytes);
	srand(seed);
	for(int i = 0; i < seedBytes / sizeof(unsigned int); i++)
		seedHost[i] = uni01_sampler() * UINT_MAX;
	hipMemcpy(seedDevice, seedHost, seedBytes, hipMemcpyHostToDevice);
	//Allocate device memory
	float *Dconfx;
	float *Dconfy;
	float *Dconfz;
	double *Dout;
	//double *Dcorr;
	if(hipMalloc((void**)&Dconfx, Spin_mem_size)){
		fprintf(stderr, "Error couldn't allocate Device Memory!!\n");
		exit(1);
	}
	if(hipMalloc((void**)&Dconfy, Spin_mem_size)){
		fprintf(stderr, "Error couldn't allocate Device Memory!!\n");
		exit(1);
	}
	if(hipMalloc((void**)&Dconfz, Spin_mem_size)){
		fprintf(stderr, "Error couldn't allocate Device Memory!!\n");
		exit(1);
	}
	hipMalloc((void**)&Dout, Out_mem_size);
	//hipMalloc((void**)&Dcorr, Corr_mem_size);
#ifdef GET_CORR
	float *Dcorr;
	double *DSum_corr;
	if(hipMalloc((void**)&Dcorr, Spin_mem_size_p)){
		fprintf(stderr, "Error couldn't allocate Device Memory!!\n");
		exit(1);
	}

	if(hipMalloc((void**)&DSum_corr, Spin_mem_size_d)){
		fprintf(stderr, "Error couldn't allocate Device Memory!!\n");
		exit(1);
	}
#endif

	//Set up output data path
	char dir[128];
	char conf_dir[128];
	sprintf(dir, "Data/L_%d-%s", SpinSize, ID);
	sprintf(conf_dir, "Conf/L_%d-%s", SpinSize, ID);
	mkdir(dir, 0755);
	mkdir(conf_dir, 0755);
	char Seedfn[128];
	sprintf(Seedfn, "Conf/L_%d-%s/seed", SpinSize, ID);
	int seedfd = open(Seedfn, O_CREAT | O_WRONLY, 0644);
	write(seedfd, seedHost, seedBytes);
	close(seedfd);
	char Efn[128];
	char Mfn[128];
	char Chernfn[128];
	char E2fn[128];
	char E4fn[128];
	char M2fn[128];
	char M4fn[128];
	//char Helfn[128];
	//char Corrfn[128];
	char Confxfn[128];
	char Confyfn[128];
	char Confzfn[128];
	sprintf(Efn, "%s/E", dir);
	sprintf(Mfn, "%s/M", dir);
	sprintf(Chernfn, "%s/Chern", dir);
	sprintf(E2fn, "%s/E2", dir);
	sprintf(E4fn, "%s/E4", dir);
	sprintf(M2fn, "%s/M2", dir);
	sprintf(M4fn, "%s/M4", dir);
	//sprintf(Helfn, "%s/Hel", dir);
	sprintf(Confxfn, "%s/Confx", conf_dir);
	sprintf(Confyfn, "%s/Confy", conf_dir);
	sprintf(Confzfn, "%s/Confz", conf_dir);
#ifdef GET_CORR
  char Corrfn[128];
	sprintf(Corrfn, "%s/Corr", dir);
	int Corrfd = open(Corrfn, O_CREAT | O_WRONLY, 0644);
#endif

	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	//Give initial configuration and settle the systems down to equilibrium states
	double pi = 3.141592653589793;
	double th, phi;
	for(int i = 0; i < N * Hnum; i++){
		th = uni01_sampler() * pi;
		phi = uni01_sampler() * 2 * pi;
		Hconfx[i] = cos(th);
		th = sin(th);
		Hconfy[i] = th * cos(phi);
		Hconfz[i] = th * sin(phi);
    /*
		Hconfx[i] = 0;
		Hconfy[i] = 0;
		Hconfz[i] = 1;
    */
	}
	for(int i = 0; i < Nplane * Hnum; i++){
#ifdef GET_CORR
    HSum_corr[i] = 0.0; //initialize
#endif
	}
	hipMemcpy(Dconfx, Hconfx, Spin_mem_size, hipMemcpyHostToDevice);
	hipMemcpy(Dconfy, Hconfy, Spin_mem_size, hipMemcpyHostToDevice);
	hipMemcpy(Dconfz, Hconfz, Spin_mem_size, hipMemcpyHostToDevice);
#ifdef GET_CORR
	hipMemcpy(DSum_corr, HSum_corr, Spin_mem_size_d, hipMemcpyHostToDevice);
#endif
  int Eqii = 0;//150;
	for(int i = 0; i < Hnum; i++)
		HHs[i] = Hls[i];
	hipMemcpy(DHs, HHs, H_mem_size, hipMemcpyHostToDevice);
  double *Ms = (double*)malloc(Hnum * sizeof(double));
  double E;
	int *accept1 = (int*)calloc(Hnum - 1, sizeof(int));
	float cnt = 0;
  Tcurrent = Tls[Tls.size()-1];

	for(int i = 0; i < EQUI_N; i++){
		flipTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
		flipBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
		/*
		for (int q = 0; q < relax_N; q++){
			relaxTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
			relaxBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
		}*/
		//================================= no PT ==========================================
		/*
    cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout); //, Dcorr);
    hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost);
    for(int t = 0; t < Hnum; t++){
      int raw_off = t * MEASURE_NUM * BN;
      E = 0;
      for(int j = 3 * BN; j < 4 * BN; j++)
        E += Hout[raw_off + j];
      Ms[Ho[t]] = E;	//Es is the energies in order of temperature set
    }
    //Parallel Tempering
    cnt += PTF;
    for(int p = 0; p < int(cnt); p++){
      tempering(Ms, accept1);
      for(int t = 0; t < Hnum; t++) HHs[t] = Hls[Ho[t]];
      hipMemcpy(DHs, HHs, H_mem_size, hipMemcpyHostToDevice);
    }
    if(int(cnt))
      cnt = 0;
    */
	}

	//Do measurements (annealing)
	unsigned int data_mem_size = Hnum * sizeof(double);
	int Efd = open(Efn, O_CREAT | O_WRONLY, 0644);
	int Mfd = open(Mfn, O_CREAT | O_WRONLY, 0644);
	int Chernfd = open(Chernfn, O_CREAT | O_WRONLY, 0644);
	int E2fd = open(E2fn, O_CREAT | O_WRONLY, 0644);
	int E4fd = open(E4fn, O_CREAT | O_WRONLY, 0644);
	int M2fd = open(M2fn, O_CREAT | O_WRONLY, 0644);
	int M4fd = open(M4fn, O_CREAT | O_WRONLY, 0644);
	//int Helfd = open(Helfn, O_CREAT | O_WRONLY, 0644);
	double *Eout = (double*)malloc(data_mem_size);
	double *Mout = (double*)malloc(data_mem_size);
	double *Chernout = (double*)malloc(data_mem_size);
	double *E2out = (double*)malloc(data_mem_size);
	double *E4out = (double*)malloc(data_mem_size);
	double *M2out = (double*)malloc(data_mem_size);
	double *M4out = (double*)malloc(data_mem_size);
	int Confxfd = open(Confxfn, O_CREAT | O_WRONLY, 0644);
	int Confyfd = open(Confyfn, O_CREAT | O_WRONLY, 0644);
	int Confzfd = open(Confzfn, O_CREAT | O_WRONLY, 0644);
	int *accept = (int*)calloc(Hnum - 1, sizeof(int));
	//double *Helout = (double*)malloc(data_mem_size);
	for(int T_i = 0 ; T_i < Tnum ; T_i ++){
		for (int i = 0; i< Hnum-1; i++) accept[i] = 0;
		Tcurrent = Tls [Tls.size() - 1 - T_i];
    for(int i = 0; i < EQUI_Ni; i++){
      flipTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
      flipBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
      /*
      for (int q = 0; q < relax_N; q++){
        relaxTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
        relaxBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
      }*/
      //======================= no PT ===============================
      /*
      cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout); //, Dcorr);
      hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost);
      for(int t = 0; t < Hnum; t++){
        int raw_off = t * MEASURE_NUM * BN;
        E = 0;
        for(int j = 3 * BN; j < 4 * BN; j++)
          E += Hout[raw_off + j];
        Ms[Ho[t]] = E;	//Es is the energies in order of temperature set
      }
      //Parallel Tempering
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
        tempering(Ms, accept1);
        for(int t = 0; t < Hnum; t++) HHs[t] = Hls[Ho[t]];
        hipMemcpy(DHs, HHs, H_mem_size, hipMemcpyHostToDevice);
      }
      if(int(cnt))
        cnt = 0;
      */
    }
    cnt = 0;
    for(int b = 0; b < BIN_NUM; b++){
      //printf("b = %d\n", b);
      //Take the ensemble average
      double E2;
      double Mx, My, Mz, M2, Chern;
      //double Helx, Hely, Helz;
      double *binE = (double*)calloc(Hnum, sizeof(double));
      double *binM = (double*)calloc(Hnum, sizeof(double));
      double *binChern = (double*)calloc(Hnum, sizeof(double));
      double *binE2 = (double*)calloc(Hnum, sizeof(double));
      double *binM2 = (double*)calloc(Hnum, sizeof(double));
      double *binE4 = (double*)calloc(Hnum, sizeof(double));
      double *binM4 = (double*)calloc(Hnum, sizeof(double));
      //double *binHel = (double*)calloc(Tnum, sizeof(double));
      //hipMemset((void*)Dcorr, 0, Corr_mem_size);
      for(int i = 0; i < BIN_SZ; i++){
        flipTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
        flipBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, DHs, 1.0 / Tcurrent);
        /*
        for (int q = 0; q < relax_N; q++){
          relaxTLBR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
          relaxBLTR<<<grid, block>>>(Dconfx, Dconfy, Dconfz, seedDevice, H);
        }*/
        cal<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dout); //, Dcorr);
#ifdef GET_CORR
        if ( i % f_CORR==0){
          hipMemcpy(DTo, &Ho[0], Tnum * sizeof(int), hipMemcpyHostToDevice);
          hipMemset(Dcorr, 0, Spin_mem_size);
          for (int labelx = 0; labelx < SpinSize; labelx += 4){
            for (int labely = 0; labely < SpinSize; labely += 4){
              getcorr<<<grid, block>>>(Dconfx, Dconfy, Dconfz, Dcorr, labelx, labely);
            }
          }
          sumcorr<<<grid, block>>>(DSum_corr, Dcorr, DTo);
          corrcount += 1;
        }
#endif
        hipMemcpy(Hout, Dout, Out_mem_size, hipMemcpyDeviceToHost);
        for(int t = 0; t < Hnum; t++){
          int raw_off = t * MEASURE_NUM * BN;
          E = 0;
          //Helx = 0, Hely = 0, Helz = 0;
          Mx = 0, My = 0, Mz = 0, Chern = 0;
          for(int j = 0; j < BN; j++)
            E += Hout[raw_off + j];
          //for(int j = BN; j < 2 * BN; j++)
          //	Helx += Hout[raw_off + j];
          //for(int j = 2 * BN; j < 3 * BN; j++)
          //	Hely += Hout[raw_off + j];
          //for(int j = 3 * BN; j < 4 * BN; j++)
          //	Helz += Hout[raw_off + j];
          for(int j = BN; j < 2 * BN; j++)
            Mx += Hout[raw_off + j];
          for(int j = 2 * BN; j < 3 * BN; j++)
            My += Hout[raw_off + j];
          for(int j = 3 * BN; j < 4 * BN; j++)
            Mz += Hout[raw_off + j];
          for(int j = 4 * BN; j < 5 * BN; j++)
            Chern += Hout[raw_off + j];
          Ms[Ho[t]] = Mz;	//Es is the energies in order of temperature set
          E = E - HHs[t] * Mz;
          binE[Ho[t]] += E;
          M2 = Mx * Mx + My * My + Mz * Mz;
          E2 = E * E;
          binM[Ho[t]] += sqrt(M2);
          binChern[Ho[t]] += Chern;
          binE2[Ho[t]] += E2;
          binM2[Ho[t]] += M2;
          binE4[Ho[t]] += E2 * E2;
          binM4[Ho[t]] += M2 * M2;
          //binHel[Ho[t]] += (Helx * Helx + Hely * Hely + Helz * Helz) / 3;
        }
        //Parallel Tempering
        /*
        cnt += PTF;
        for(int p = 0; p < int(cnt); p++){
          tempering(Ms, accept);
          for(int t = 0; t < Hnum; t++) HHs[t] = Hls[Ho[t]];
          hipMemcpy(DHs, HHs, H_mem_size, hipMemcpyHostToDevice);
        }
        if(int(cnt))
          cnt = 0;
        */
      }
      for(int t = 0; t < Hnum; t++){
        Eout[t] = (double)binE[t] / BIN_SZ / N;
        Mout[t] = (double)binM[t] / BIN_SZ / N;
        Chernout[t] = (double)binChern[t] / BIN_SZ;
        E2out[t] = (double)binE2[t] / BIN_SZ / N / N;
        E4out[t] = (double)binE4[t] / BIN_SZ / N / N / N / N;
        M2out[t] = (double)binM2[t] / BIN_SZ / N / N;
        M4out[t] = (double)binM4[t] / BIN_SZ / N / N / N / N;
        //Helout[t] = (double)binHel[t] / BIN_SZ / N;
      }
      write(Efd, Eout, data_mem_size);
      write(Mfd, Mout, data_mem_size);
      write(Chernfd, Chernout, data_mem_size);
      write(E2fd, E2out, data_mem_size);
      write(E4fd, E4out, data_mem_size);
      write(M2fd, M2out, data_mem_size);
      write(M4fd, M4out, data_mem_size);
      //write(Helfd, Helout, data_mem_size);
      //hipMemcpy(Hcorr, Dcorr, Corr_mem_size, hipMemcpyDeviceToHost);
      //write(corrfd, Hcorr, Corr_mem_size);
      //close(corrfd);
      free(binE);
      free(binM);
      free(binChern);
      free(binE2);
      free(binE4);
      free(binM2);
      free(binM4);
      //free(binHel);
    }
    hipMemcpy(Hconfx, Dconfx, Spin_mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(Hconfy, Dconfy, Spin_mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(Hconfz, Dconfz, Spin_mem_size, hipMemcpyDeviceToHost);
    for (int iii = 0 ; iii < Hnum; iii ++){
    	ivHo[Ho[iii]] = Ho[iii];
    }
    write(Confxfd, Hconfx, Spin_mem_size);
    write(Confyfd, Hconfy, Spin_mem_size);
    write(Confzfd, Hconfz, Spin_mem_size);
#ifdef GET_CORR
    avgcorr<<<grid, block>>>(DSum_corr, double(corrcount));
    hipMemcpy(HSum_corr, DSum_corr, Spin_mem_size_d, hipMemcpyDeviceToHost);
    write(Corrfd, HSum_corr, Spin_mem_size_d);
    hipMemset((void*)DSum_corr, 0, Spin_mem_size_d);
#endif
  }
  free(Ms);
	sdkStopTimer(&timer);
    double time = 1.0e-3 * sdkGetTimerValue(&timer);
#ifdef GET_CORR
	close(Corrfd);
#endif
	close(Efd);
	close(Mfd);
	close(Chernfd);
	close(E2fd);
	close(E4fd);
	close(M2fd);
	close(M4fd);
	//close(Helfd);
	close(Confxfd);
	close(Confyfd);
	close(Confzfd);
	//close(corrfd);

	char detailFn[128];
	sprintf(detailFn, "%s/details", dir);
	FILE *detailFp = fopen(detailFn, "w");
	fprintf(detailFp, "elapsed time = %f (sec)\n", time);
	double speed = 0;
	speed = (N / time / 1000000000) * (BIN_SZ * BIN_NUM + EQUI_N) * Tnum;
	fprintf(detailFp, "speed = %f (GHz)\n", speed);
	fprintf(detailFp, "RNG: WarpStandard\n", SpinSize);
	fprintf(detailFp, "SpinSize = %d\n", SpinSize);
	fprintf(detailFp, "A = %4.3f\n", A);
	fprintf(detailFp, "D_Rashba = %4.3f\n", DR);
	fprintf(detailFp, "D_Dresselhaus = %4.3f\n", DD);
	fprintf(detailFp, "BlockSize_x = %d\n", BlockSize_x);
	fprintf(detailFp, "BlockSize_y = %d\n", BlockSize_y);
	fprintf(detailFp, "GridSize_x = %d\n", GridSize_x);
	fprintf(detailFp, "GridSize_y = %d\n", GridSize_y);
	//fprintf(detailFp, "CORR_L = %d\n", CORR_L);
	fprintf(detailFp, "Bin Size = %d\n", BIN_SZ);
	fprintf(detailFp, "Bin Number = %d\n", BIN_NUM);
	fprintf(detailFp, "Equilibration N = %d\n", EQUI_N);
	fprintf(detailFp, "Equilibration Ni = %d\n", EQUI_Ni);
	fprintf(detailFp, "f_CORR = %d\n", f_CORR);
	fprintf(detailFp, "PT frequency = %3.2f\n", PTF);
	fprintf(detailFp, "Relaxation N = %d\n", relax_N);
	fprintf(detailFp, "Tnum = %d\n", Tnum);
	fprintf(detailFp, "Temperature Set: ");
	for(int i = 0; i < Tnum; i++){
		fprintf(detailFp, "%.5f  ", Tls[i]);
  }
	fprintf(detailFp, "\nHnum = %d\n", Hnum);
	fprintf(detailFp, "field Set: ");
	for(int i = 0; i < Hnum; i++){
		fprintf(detailFp, "%.5f  ", Hls[i]/(DR*DR + DD*DD));
  }
	for(int i = 0; i < Hnum; i++){
    fprintf(detailFp, "\n");
    fprintf(detailFp, "Ho[%d]=%d",i,Ho[i]);
  }
	fprintf(detailFp, "\n");
	fprintf(detailFp, "Acceptance rates: ");
	if (PTF != 0 ){
	for(int i = 0; i < Tnum - 1; i++)
		fprintf(detailFp, "%4.3f  ", float(accept[i]) / (BIN_SZ * BIN_NUM * PTF));
	}
	fprintf(detailFp, "\n");
	fprintf(detailFp, "At all temperatures, configuration starts from ground state.\n");
	fprintf(detailFp, "Done by Po-Kuan Wu ^_^\n", EQUI_N);
	fclose(detailFp);

	//Set free memory
	free(Hconfx);
	free(Hconfy);
	free(Hconfz);
	free(Hout);
	//free(Hcorr);
	free(seedHost);
	free(Eout);
	free(Mout);
	free(Chernout);
	free(E2out);
	free(E4out);
	free(M2out);
	free(M4out);
	//free(Helout);
	hipFree(DHs);
	hipFree(Dconfx);
	hipFree(Dconfy);
	hipFree(Dconfz);
	hipFree(Dout);
	//hipFree(Dcorr);
	hipFree(seedDevice);
#ifdef GET_CORR
	free(HSum_corr);
  hipFree(Dcorr);
  hipFree(DTo);
  hipFree(DSum_corr);
#endif
	return 0;
}

void tempering(double *Ms, int *accept){
	int map[Hls.size()];	//map[t] the configuration of t'th temperature
	for(int i = 0; i < Hnum; i++){
		map[Ho[i]] = i;
  }
	double delta;
	int flag = 0;
	for(int i = 0; i < Hnum - 1; i++){
		delta = (Ms[i + 1] - Ms[i]) * ( Hls[i] - Hls[i + 1]) / Tcurrent;
		if(delta > 0)
			flag = 1;
		else if(uni01_sampler() < exp(delta))
			flag = 1;
		if(flag){
			int tmp = Ho[map[i]];
			Ho[map[i]] = Ho[map[i + 1]];
			Ho[map[i + 1]] = tmp;
			tmp = map[i];
			map[i] = map[i + 1];
			map[i + 1] = tmp;
			double tmpE = Ms[i];
			Ms[i] = Ms[i + 1];
			Ms[i + 1] = tmpE;
			accept[i] += 1;
			flag = 0;
		}
	}
}
