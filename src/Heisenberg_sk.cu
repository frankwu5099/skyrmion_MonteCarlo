#include "hip/hip_runtime.h"
//Ms PT function writedata  HHs
//Po -> Cpara
//why exchange conf and measurement works
using namespace std;


#include "params.cuh"
#include "updates.cuh"
#include "configuration.cuh"
#include "measurements_sk.cuh"
#include "extend.cu"
#define GET_CORR


unsigned seed = 73;
mt19937 rng(seed);
uniform_01<mt19937> uni01_sampler(rng);
void tempering_simple(double*, double*, int*);
void tempering(double*, double*, int*, int*, int*);
vector< vector<float> > Tls;
vector< vector<float> > Hls;
vector<int> Po;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
vector<int> ivPo;		//order of Temperature, Tls[To[t]] is the temperature of t'th configuration.
unsigned int Pnum;
unsigned int Cnum;
unsigned int Tnum;
unsigned int Hnum;
unsigned int f_CORR;
unsigned int CORR_N;
float Cparameter = 0.8;
int C_i = 0;
void var_examine();



int main(int argc, char *argv[]){
  //call GPU
  read_params(argv[1]);
  int deviceNum, gpu_i;
  hipGetDeviceCount(&deviceNum);
  device_0 = setDev();
  StreamN = deviceNum -device_0;

  if (device_0 == -1){
    return 1;
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    hipStreamCreate(&stream[gpu_i]);
  }
  hipGetLastError();
  CudaCheckError();
  //note: gpu part

  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    move_params_device_flip();
    move_params_device_cals();
    move_params_device_corr();
  }

  //examine variables
  var_examine();

  // ========================== initialize ===========================

  //begin (read in temperatures)
  unsigned int params_mem_size;

  if(argc > 2){
    float tmpT, tmpH;
    FILE *paramsfp = fopen(argv[2], "r");
    vector<float> tmpTls;
    vector<float> tmpHls;
    fscanf(paramsfp, "%d %d", &Tnum, &Hnum);
    while(fscanf(paramsfp, "%f %f", &tmpT, &tmpH) != EOF){
      if (tmpT < 0){
        Tls.push_back(tmpTls);
        Hls.push_back(tmpHls);
        tmpTls.clear();
        tmpHls.clear();
      }
      else {
        tmpTls.push_back(tmpT);
        tmpHls.push_back(tmpH*(DR*DR + DD*DD));
      }
    }
    fclose(paramsfp);
    Pnum = Tls[0].size();
    Cnum = Tls.size();
    if (Tnum * Hnum != Pnum){
      fprintf(stderr, "wrong temperatures and fields!!!\n");
      exit(0);
    }
    params_mem_size = Pnum * sizeof(float);

  }
  else{
    fprintf(stderr, "Give me a temperature set!!!\n");
    fprintf(stderr, "Give me a field set!!!\n");
    exit(0);
  }
  //end (read in temperatures)

  if (Pnum%StreamN != 0){
    printf("Fatal error: The number of replicas is not consistent with the number of streams.");
    return 1;
  }
  int Pnum_s = Pnum/StreamN;
  int params_mem_size_s = Pnum_s * sizeof(float);
	grid = Pnum_s * H_BN;


  //invTs is the inverse temperature in order of configurations on GPU.
  for (int i = 0; i < Pnum ; i++){
    Po.push_back(i);
    ivPo.push_back(i);
  }
  float *HHs;
  float **DHs;
  DHs = (float**)calloc(StreamN, sizeof(float*));
  float *invTs;
  float **DinvTs;
  DinvTs = (float**)calloc(StreamN, sizeof(float*));
  HHs = (float*)malloc(params_mem_size);
  invTs = (float*)malloc(params_mem_size);

  //note: gpu part
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void**)&DHs[gpu_i], params_mem_size/StreamN));
    CudaSafeCall(hipMalloc((void**)&DinvTs[gpu_i], params_mem_size/StreamN));
  }

  //begin (initialize random seeds)
  //Declare sizes
  unsigned int totalRngs = Pnum * H_BN * H_TN / WarpStandard_K;
  unsigned seedBytes = totalRngs * sizeof(unsigned int) * WarpStandard_STATE_WORDS;
  unsigned int **seedDevice;
  seedDevice = (unsigned int**)calloc(StreamN, sizeof(unsigned int*));
  //note: gpu part
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMalloc((void **)&seedDevice[gpu_i], seedBytes/StreamN));
  }
  unsigned int* seedHost = (unsigned int*)malloc(seedBytes);
  srand(seed);
  for(int i = 0; i < seedBytes / sizeof(unsigned int); i++)
    seedHost[i] = uni01_sampler() * UINT_MAX;
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(seedDevice[gpu_i], seedHost + (seedBytes/sizeof(unsigned int)/StreamN)*gpu_i, (seedBytes/StreamN), hipMemcpyHostToDevice, stream[gpu_i]));
  }
  //end (initialize random seeds)


  //Set up output data path
  char dir[128];
  sprintf(dir, "Data/L_%d-%s", H_SpinSize, Output);
  mkdir(dir, 0755);
  char Seedfn[128];
  sprintf(Seedfn, "Data/L_%d-%s/seed", H_SpinSize, Output);
  int seedfd = open(Seedfn, O_CREAT | O_WRONLY, 0644);
  write(seedfd, seedHost, seedBytes);
  close(seedfd);


  //MEASUREMENT initialize

  configuration CONF(Pnum, dir);
  measurements MEASURE(dir, Pnum, BIN_SZ); //Tnum for parallel tempering for T
#ifdef GET_CORR
  char Corrfn[128];
  sprintf(Corrfn, "%s/Corrpt", dir);
  correlation CORR(Pnum, Corrfn);
#endif

  StopWatchInterface *timer=NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  //Give initial configuration and settle the systems down to equilibrium states
  CONF.initialize(ORDER);
  int Eqii = 0;//150;
  for(int i = 0; i < Pnum; i++){
    HHs[i] = Hls[0][i];
    invTs[i] = 1.0/Tls[0][i];
  }
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
  }
  double *Ms = (double*)malloc(Pnum * sizeof(double));
  double *Es = (double*)malloc(Pnum * sizeof(double));
  int *accept1 = (int*)calloc((Tnum - 1)*Hnum + Tnum*(Hnum - 1), sizeof(int));
  int *stay = (int*)calloc(Tnum * Hnum, sizeof(int));
  int *staylargest = (int*)calloc(Tnum * Hnum, sizeof(int));
  int *staytmp = (int*)calloc(Tnum * Hnum, sizeof(int));
  float cnt = 0;

  for(int i = 0; i < EQUI_N; i++){
    if (i % 10 ==0) printf("%d\n",i);
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
    }
    //Parallel Tempering
    cnt += PTF;
    for(int p = 0; p < int(cnt); p++){
      MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
      tempering_simple(Ms, Es, accept1);
      for(int t = 0; t < Pnum; t++){
        HHs[t] = Hls[0][Po[t]];
        invTs[t] = 1.0/Tls[0][Po[t]];
      }
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
        CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
        CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
      }
    }
    if(int(cnt))
    cnt = 0;
  }

  //Do measurements (annealing)

  //int *accept = (int*)calloc(Pnum - 1, sizeof(int));
  int *accept = (int*)calloc((Tnum - 1)*Hnum + Tnum*(Hnum - 1), sizeof(int));
  for(C_i = 0 ; C_i < Cnum ; C_i ++){

    for(int t = 0; t < Pnum; t++){
      HHs[t] = Hls[C_i][Po[t]];
      invTs[t] = 1.0/Tls[C_i][Po[t]];
    }
    for (int i = 0; i< (Tnum - 1)*Hnum + Tnum*(Hnum - 1); i++) accept[i] = 0;
    for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
      hipSetDevice(device_0 + gpu_i);
      CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
      CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
    }

    for(int i = 0; i < EQUI_Ni; i++){
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
      }
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
        MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
        tempering_simple(Ms, Es, accept1);
        for(int t = 0; t < Pnum; t++){
          HHs[t] = Hls[C_i][Po[t]];
          invTs[t] = 1.0/Tls[C_i][Po[t]];
        }
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
          CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
        }
      }
      if(int(cnt))
        cnt = 0;
    }
    cnt = 0;
    for(int b = 0; b < BIN_NUM; b++){
      //Take the ensemble average
      for(int i = 0; i < BIN_SZ; i++){
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
					for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
						hipSetDevice(device_0 + gpu_i);
						SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
					}
        }
        MEASURE.measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
/*
#ifdef GET_CORR
        if ( i % f_CORR==0){
          CORR.extract(Po, CONF);//==
        }
#endif
*/
        cnt += PTF;
        for(int p = 0; p < int(cnt); p++){
          tempering(Ms, Es, accept, staytmp, stay);
          for(int t = 0; t < Pnum; t++){
            HHs[t] = Hls[C_i][Po[t]];
            invTs[t] = 1.0/Tls[C_i][Po[t]];
            if (stay[Po[t]] > staylargest[Po[t]]){
              CONF.Dominatestateback(Po[t],t);
              staylargest[Po[t]] = stay[Po[t]];
            }
          }
          for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
            hipSetDevice(device_0 + gpu_i);
            CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
            CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
          }
        }
        if(int(cnt))
          cnt = 0;
      }
      MEASURE.normalize_and_save_and_reset();
    }
    for (int iii = 0 ; iii < Pnum; iii ++){
      ivPo[Po[iii]] = iii;
    }
    //CONF.backtoHost(); //watch out! it must be compatible with the
    CONF.writedata();
#ifdef GET_CORR
    sprintf(Corrfn, "%s/Corr_%d", dir, b);
    CORR.changefile(Corrfn);
    for(int i = 0; i < CORR_N * f_CORR; i++){
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
      }
      if ( i % f_CORR==0){
        CORR.extract(Po, CONF);//==
      }
    }
    CORR.avg_write_reset(Po);
    for(int i = 0; i < 1000; i++){
      for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
        hipSetDevice(device_0 + gpu_i);
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF1(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF2(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
				for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
					hipSetDevice(device_0 + gpu_i);
					SSF3(CONF.Dx[gpu_i], CONF.Dy[gpu_i], CONF.Dz[gpu_i], seedDevice[gpu_i], DHs[gpu_i], DinvTs[gpu_i], stream[gpu_i]);
				}
      }
      //Parallel Tempering
      cnt += PTF;
      for(int p = 0; p < int(cnt); p++){
        MEASURE.virtual_measure(CONF.Dx, CONF.Dy, CONF.Dz, Po, Ms, Es, HHs);
        tempering_simple(Ms, Es, accept);
        for(int t = 0; t < Pnum; t++){
          HHs[t] = Hls[0][Po[t]];
          invTs[t] = 1.0/Tls[0][Po[t]];
        }
        for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
          hipSetDevice(device_0 + gpu_i);
          CudaSafeCall(hipMemcpyAsync(DinvTs[gpu_i], invTs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
          CudaSafeCall(hipMemcpyAsync(DHs[gpu_i], HHs+gpu_i*Pnum_s, params_mem_size_s, hipMemcpyHostToDevice, stream[gpu_i]));
        }
      }
      if(int(cnt))
      cnt = 0;
    }
#endif
  }
  char Histfn[128];
  sprintf(Histfn, "%s/%s", dir, "EHistogram");
  FILE *f_hist = fopen(Histfn, "w");
  fwrite(MEASURE.EHistogram, sizeof(unsigned int),Slice_NUM * Pnum, f_hist);
  fclose(f_hist);
  sprintf(Histfn, "%s/%s", dir, "ChernHistogram");
  FILE *f_chist = fopen(Histfn, "w");
  fwrite(MEASURE.ChernHistogram, sizeof(unsigned int), Slice_CNUM * Pnum, f_chist);
  fclose(f_chist);
  free(Ms);
  sdkStopTimer(&timer);
  double time = 1.0e-3 * sdkGetTimerValue(&timer);
    printf("G\n");
    fflush(stdout);


//======================= print details ==========================
  char detailFn[128];
  sprintf(detailFn, "%s/details", dir);
  FILE *detailFp = fopen(detailFn, "w");
  fprintf(detailFp, "elapsed time = %f (sec)\n", time);
  double speed = 0;
  speed = (H_N / time / 1000000000) * ((EQUI_Ni + BIN_SZ * BIN_NUM) * Cnum + EQUI_N) * Pnum;
  fprintf(detailFp, "speed = %f (GHz)\n", speed);
  fprintf(detailFp, "number of gpu = %d\n", StreamN);
  fprintf(detailFp, "RNG: WarpStandard\n", H_SpinSize);
  fprintf(detailFp, "SpinSize = %d\n", H_SpinSize);
  fprintf(detailFp, "A = %4.3f\n", H_A);
  fprintf(detailFp, "D_Rashba = %4.3f\n", DR);
  fprintf(detailFp, "D_Dresselhaus = %4.3f\n", DD);
  fprintf(detailFp, "BlockSize_x = %d\n", H_BlockSize_x);
  fprintf(detailFp, "BlockSize_y = %d\n", H_BlockSize_y);
  fprintf(detailFp, "GridSize_x = %d\n", H_GridSize_x);
  fprintf(detailFp, "GridSize_y = %d\n", H_GridSize_y);
  fprintf(detailFp, "Bin Size = %d\n", BIN_SZ);
  fprintf(detailFp, "Bin Number = %d\n", BIN_NUM);
  fprintf(detailFp, "Equilibration N = %d\n", EQUI_N);
  fprintf(detailFp, "Equilibration Ni = %d\n", EQUI_Ni);
  fprintf(detailFp, "f_CORR = %d\n", f_CORR);
  fprintf(detailFp, "CORR_N = %d\n", CORR_N);
  fprintf(detailFp, "PT frequency = %3.2f\n", PTF);
  fprintf(detailFp, "Pnum = %d\n", Pnum);
  fprintf(detailFp, "Temperature Set: ");
  for(int i = 0; i < Cnum; i++){
    for(int j = 0; j < Pnum; j++){
      fprintf(detailFp, "%.5f  ", Tls[i][j]);
    }
  }
  fprintf(detailFp, "\n");
  fprintf(detailFp, "field Set: ");
  for(int i = 0; i < Cnum; i++){
    for(int j = 0; j < Pnum; j++){
      fprintf(detailFp, "%.5f  ", Hls[i][j]/(DR*DR + DD*DD));
    }
  }
  for(int i = 0; i < Pnum; i++){
    fprintf(detailFp, "\n");
    fprintf(detailFp, "Po[%d]=%d",i,Po[i]);
  }
  fprintf(detailFp, "\n");
  fprintf(detailFp, "Acceptance rates: ");
  if (PTF != 0 ){
    for(int i = 0; i < (Tnum - 1)*Hnum + Tnum*(Hnum - 1); i++)
      fprintf(detailFp, "%4.3f  ", float(accept[i]) / (BIN_SZ * BIN_NUM * PTF));
  }
  fprintf(detailFp, "\n");
  if (ORDER){
    fprintf(detailFp, "Configurations start from ordered state.\n");
  }
  else {
    fprintf(detailFp, "Configurations start from random state.\n");
  }
  fprintf(detailFp, "N_histE = %d\n", Slice_NUM);
  fprintf(detailFp, "E_lowest = %4.3f\n", E_lowest);
  fprintf(detailFp, "E_highest = %4.3f\n", E_highest);
  fprintf(detailFp, "N_histChern = %d\n", Slice_CNUM);
  fprintf(detailFp, "Chern_lowest = %4.3f\n", Chern_lowest);
  fprintf(detailFp, "Chern_highest = %4.3f\n", Chern_highest);
  fprintf(detailFp, "Done by Po-Kuan Wu ^_^\n", EQUI_N);
  fclose(detailFp);
//===================== print details end =========================

  //Set free memory
  free(seedHost);
  for (gpu_i = 0; gpu_i < StreamN; gpu_i++){
    hipSetDevice(device_0 + gpu_i);
    CudaSafeCall(hipFree(DinvTs[gpu_i]));
    CudaSafeCall(hipFree(DHs[gpu_i]));
    CudaSafeCall(hipFree(seedDevice[gpu_i]));
  }
  //CORR.~correlation();
  //MEASURE.~measurements();
  //CONF.~configuration();
  return 0;
}


//=============================== functions ==================================
void tempering_simple(double *Ms, double *Es, int *accept){
  int map[Pnum];	//map[t] the configuration of t'th temperature
  int i, j, tmp, partT_num = (Tnum - 1) * Hnum;
  double tmpEM;

  for(i = 0; i < Pnum; i++)
    map[Po[i]] = i;

  double delta;
  int flag = 0;
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //T excnange
      if (i < Tnum -1){
	delta = (Es[j * Tnum + i] - Es[j * Tnum + i + 1]) * ((1.0 / Tls[C_i][j*Tnum + i]) - (1.0 / Tls[C_i][j*Tnum +i + 1]));
	if(delta > 0)
	  flag = 1;
	else if(uni01_sampler() < exp(delta))
	  flag = 1;
	if(flag){
	  tmp = Po[map[j * Tnum + i]];
	  Po[map[j * Tnum + i]] = Po[map[j * Tnum + i + 1]];
	  Po[map[j * Tnum + i + 1]] = tmp;
	  tmp = map[j * Tnum + i];
	  map[j * Tnum + i] = map[j * Tnum + i + 1];
	  map[j * Tnum + i + 1] = tmp;
	  tmpEM = Es[j * Tnum + i];
	  Es[j * Tnum + i] = Es[j * Tnum + i + 1];
	  Es[j * Tnum + i + 1] = tmpEM;
	  tmpEM = Ms[j * Tnum + i];
	  Ms[j * Tnum + i] = Ms[j * Tnum + i + 1];
	  Ms[j * Tnum + i + 1] = tmpEM;
	  accept[j * (Tnum - 1) + i] += 1;
	  flag = 0;
	}
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //H excnange
      if (j < Hnum -1){
        delta = (Ms[(j + 1) * Tnum + i] - Ms[j * Tnum + i]) * ( Hls[C_i][j * Tnum + i] - Hls[C_i][(j + 1) * Tnum + i]) / Tls[C_i][j * Tnum + i];
        if(delta > 0)
          flag = 1;
        else if(uni01_sampler() < exp(delta))
          flag = 1;
        if(flag){
          tmp = Po[map[j * Tnum + i]];
          Po[map[j * Tnum + i]] = Po[map[(j + 1) * Tnum + i]];
          Po[map[(j + 1) * Tnum + i]] = tmp;
          tmp = map[j * Tnum + i];
          map[j * Tnum + i] = map[(j + 1) * Tnum + i];
          map[(j + 1) * Tnum + i] = tmp;
          tmpEM = Es[j * Tnum + i];
          Es[j * Tnum + i] = Es[(j + 1) * Tnum + i];
          Es[(j + 1) * Tnum + i] = tmpEM;
          tmpEM = Ms[j * Tnum + i];
          Ms[j * Tnum + i] = Ms[(j + 1) * Tnum + i];
          Ms[(j + 1) * Tnum + i] = tmpEM;
          accept[partT_num + j * Tnum + i] += 1;
          flag = 0;
        }
      }
    }
  }
}

//=============================== functions ==================================
void tempering(double *Ms, double *Es, int *accept, int *staytmp, int *stay){
  int map[Pnum];	//map[t] the configuration of t'th temperature
  int i, j, tmp, partT_num = (Tnum - 1) * Hnum;
  double tmpEM;

  for(i = 0; i < Pnum; i++)
    map[Po[i]] = i;

  double delta;
  int flag = 0;
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
	staytmp[j * Tnum + i] = 1;
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //T excnange
      if (i < Tnum -1){
	delta = (Es[j * Tnum + i] - Es[j * Tnum + i + 1]) * ((1.0 / Tls[C_i][j*Tnum + i]) - (1.0 / Tls[C_i][j*Tnum +i + 1]));
	if(delta > 0)
	  flag = 1;
	else if(uni01_sampler() < exp(delta))
	  flag = 1;
	if(flag){
	  tmp = Po[map[j * Tnum + i]];
	  Po[map[j * Tnum + i]] = Po[map[j * Tnum + i + 1]];
	  Po[map[j * Tnum + i + 1]] = tmp;
	  tmp = map[j * Tnum + i];
	  map[j * Tnum + i] = map[j * Tnum + i + 1];
	  map[j * Tnum + i + 1] = tmp;
	  tmpEM = Es[j * Tnum + i];
	  Es[j * Tnum + i] = Es[j * Tnum + i + 1];
	  Es[j * Tnum + i + 1] = tmpEM;
	  tmpEM = Ms[j * Tnum + i];
	  Ms[j * Tnum + i] = Ms[j * Tnum + i + 1];
	  Ms[j * Tnum + i + 1] = tmpEM;
	  accept[j * (Tnum - 1) + i] += 1;
	  flag = 0;
	  staytmp[j * Tnum + i] *= 0;
	  staytmp[j * Tnum + i + 1] *= 0;
	}
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      //H excnange
      if (j < Hnum -1){
        delta = (Ms[(j + 1) * Tnum + i] - Ms[j * Tnum + i]) * ( Hls[C_i][j * Tnum + i] - Hls[C_i][(j + 1) * Tnum + i]) / Tls[C_i][j * Tnum + i];
        if(delta > 0)
          flag = 1;
        else if(uni01_sampler() < exp(delta))
          flag = 1;
        if(flag){
          tmp = Po[map[j * Tnum + i]];
          Po[map[j * Tnum + i]] = Po[map[(j + 1) * Tnum + i]];
          Po[map[(j + 1) * Tnum + i]] = tmp;
          tmp = map[j * Tnum + i];
          map[j * Tnum + i] = map[(j + 1) * Tnum + i];
          map[(j + 1) * Tnum + i] = tmp;
          tmpEM = Es[j * Tnum + i];
          Es[j * Tnum + i] = Es[(j + 1) * Tnum + i];
          Es[(j + 1) * Tnum + i] = tmpEM;
          tmpEM = Ms[j * Tnum + i];
          Ms[j * Tnum + i] = Ms[(j + 1) * Tnum + i];
          Ms[(j + 1) * Tnum + i] = tmpEM;
          accept[partT_num + j * Tnum + i] += 1;
          flag = 0;
	  staytmp[j * Tnum + i] *= 0;
	  staytmp[(j + 1) * Tnum + i] *= 0;
        }
      }
    }
  }
  for(i = 0; i < Tnum; i++){
    for (j = 0; j < Hnum; j++){
      stay[j * Tnum + i] = staytmp[j * Tnum + i]?(stay[j * Tnum + i]+1):0;
    }
  }
}
void var_examine(){
#ifndef TRI
  if(H_SpinSize % (H_BlockSize_x * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 2) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
#ifdef TRI
  if(H_SpinSize % (H_BlockSize_x * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_x * 2);
    exit(0);
  }
  if(H_SpinSize % (H_BlockSize_y * 3) != 0){
    fprintf(stderr, "SpinSize must be the multiple of %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
#ifndef THIN
  if (H_SpinSize_z != 1){
    fprintf(stderr, "SpinSize_z must be 1 %d\n", H_BlockSize_y * 2);
    exit(0);
  }
#endif
}
