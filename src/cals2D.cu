
#include <hip/hip_runtime.h>
#ifdef SQ

#include "measurements.cuh"
__global__ void cal2D(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	extern __shared__ double sD[];
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int txp = tx +1 ;
	const int typ = ty +1 ;
	//const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	const int dataoff = (blockIdx.x / BN) * MEASURE_NUM * BN;
	int bx, by;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + SpinSize - 1) % SpinSize;
	if((ty % SpinSize) == 0)	by = ty + SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.

	//Top-left corner
	sD[y][x] = -confx[coo2D(ty, tx)] * ( BXMxx * confx[coo2D(ty, bx)] + BYMxx * confx[coo2D(by, tx)])\
	           -confx[coo2D(ty, tx)] * ( BXMxy * confy[coo2D(ty, bx)] + BYMxy * confy[coo2D(by, tx)])\
	           -confx[coo2D(ty, tx)] * ( BXMxz * confz[coo2D(ty, bx)] + BYMxz * confz[coo2D(by, tx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyx * confx[coo2D(ty, bx)] + BYMyx * confx[coo2D(by, tx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyy * confy[coo2D(ty, bx)] + BYMyy * confy[coo2D(by, tx)])\
		         -confy[coo2D(ty, tx)] * ( BXMyz * confz[coo2D(ty, bx)] + BYMyz * confz[coo2D(by, tx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzx * confx[coo2D(ty, bx)] + BYMzx * confx[coo2D(by, tx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzy * confy[coo2D(ty, bx)] + BYMzy * confy[coo2D(by, tx)])\
		         -confz[coo2D(ty, tx)] * ( BXMzz * confz[coo2D(ty, bx)] + BYMzz * confz[coo2D(by, tx)] - A * confz[coo2D(ty, tx)]);
	//Bottom-left corner
	sD[y][x] -= confx[coo2D((ty+1), tx)] * ( BXMxx * confx[coo2D((ty+1), bx)] + BYMxx * confx[coo2D(ty, tx)])\
		         +confx[coo2D((ty+1), tx)] * ( BXMxy * confy[coo2D((ty+1), bx)] + BYMxy * confy[coo2D(ty, tx)])\
		         +confx[coo2D((ty+1), tx)] * ( BXMxz * confz[coo2D((ty+1), bx)] + BYMxz * confz[coo2D(ty, tx)])\
		         +confy[coo2D((ty+1), tx)] * ( BXMyx * confx[coo2D((ty+1), bx)] + BYMyx * confx[coo2D(ty, tx)])\
		         +confy[coo2D((ty+1), tx)] * ( BXMyy * confy[coo2D((ty+1), bx)] + BYMyy * confy[coo2D(ty, tx)])\
		         +confy[coo2D((ty+1), tx)] * ( BXMyz * confz[coo2D((ty+1), bx)] + BYMyz * confz[coo2D(ty, tx)])\
		         +confz[coo2D((ty+1), tx)] * ( BXMzx * confx[coo2D((ty+1), bx)] + BYMzx * confx[coo2D(ty, tx)])\
		         +confz[coo2D((ty+1), tx)] * ( BXMzy * confy[coo2D((ty+1), bx)] + BYMzy * confy[coo2D(ty, tx)])\
		         +confz[coo2D((ty+1), tx)] * ( BXMzz * confz[coo2D((ty+1), bx)] + BYMzz * confz[coo2D(ty, tx)] - A * confz[coo2D((ty+1), tx)]);
	//Top-right corner
	sD[y][x] -= confx[coo2D(ty, tx+1)] * ( BXMxx * confx[coo2D(ty, tx)] + BYMxx * confx[coo2D(by, tx+1)])\
		         +confx[coo2D(ty, tx+1)] * ( BXMxy * confy[coo2D(ty, tx)] + BYMxy * confy[coo2D(by, tx+1)])\
		         +confx[coo2D(ty, tx+1)] * ( BXMxz * confz[coo2D(ty, tx)] + BYMxz * confz[coo2D(by, tx+1)])\
		         +confy[coo2D(ty, tx+1)] * ( BXMyx * confx[coo2D(ty, tx)] + BYMyx * confx[coo2D(by, tx+1)])\
		         +confy[coo2D(ty, tx+1)] * ( BXMyy * confy[coo2D(ty, tx)] + BYMyy * confy[coo2D(by, tx+1)])\
		         +confy[coo2D(ty, tx+1)] * ( BXMyz * confz[coo2D(ty, tx)] + BYMyz * confz[coo2D(by, tx+1)])\
		         +confz[coo2D(ty, tx+1)] * ( BXMzx * confx[coo2D(ty, tx)] + BYMzx * confx[coo2D(by, tx+1)])\
		         +confz[coo2D(ty, tx+1)] * ( BXMzy * confy[coo2D(ty, tx)] + BYMzy * confy[coo2D(by, tx+1)])\
		         +confz[coo2D(ty, tx+1)] * ( BXMzz * confz[coo2D(ty, tx)] + BYMzz * confz[coo2D(by, tx+1)] - A * confz[coo2D(ty, tx+1)]);
	//Bottom-right corner
	sD[y][x] -= confx[coo2D((ty+1), tx+1)] * ( BXMxx * confx[coo2D((ty+1), tx)] + BYMxx * confx[coo2D(ty, tx+1)])\
		         +confx[coo2D((ty+1), tx+1)] * ( BXMxy * confy[coo2D((ty+1), tx)] + BYMxy * confy[coo2D(ty, tx+1)])\
		         +confx[coo2D((ty+1), tx+1)] * ( BXMxz * confz[coo2D((ty+1), tx)] + BYMxz * confz[coo2D(ty, tx+1)])\
		         +confy[coo2D((ty+1), tx+1)] * ( BXMyx * confx[coo2D((ty+1), tx)] + BYMyx * confx[coo2D(ty, tx+1)])\
		         +confy[coo2D((ty+1), tx+1)] * ( BXMyy * confy[coo2D((ty+1), tx)] + BYMyy * confy[coo2D(ty, tx+1)])\
		         +confy[coo2D((ty+1), tx+1)] * ( BXMyz * confz[coo2D((ty+1), tx)] + BYMyz * confz[coo2D(ty, tx+1)])\
		         +confz[coo2D((ty+1), tx+1)] * ( BXMzx * confx[coo2D((ty+1), tx)] + BYMzx * confx[coo2D(ty, tx+1)])\
		         +confz[coo2D((ty+1), tx+1)] * ( BXMzy * confy[coo2D((ty+1), tx)] + BYMzy * confy[coo2D(ty, tx+1)])\
		         +confz[coo2D((ty+1), tx+1)] * ( BXMzz * confz[coo2D((ty+1), tx)] + BYMzz * confz[coo2D(ty, tx+1)] - A * confz[coo2D((ty+1), tx+1)]);
	__syncthreads();


	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y<BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y<BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y<BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(y==0 && x==0)
		out[dataoff + (blockIdx.x % BN)] = sD[0][0];
	__syncthreads();
	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	sD[y][x]  = confx[coo2D(ty, tx)];
	sD[y][x] += confx[coo2D((ty+1), tx)];
	sD[y][x] += confx[coo2D(ty, tx+1)];
	sD[y][x] += confx[coo2D((ty+1), (tx+1))];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	//Top-left corner
	sD[y][x]  = confy[coo2D(ty, tx)];
	//Bottom-left corner
	sD[y][x] += confy[coo2D((ty+1), tx)];
	//Top-right corner
	sD[y][x] += confy[coo2D(ty, tx+1)];
	//Bottom-right corner
	sD[y][x] += confy[coo2D((ty+1), tx+1)];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 2 * BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	//Top-left corner
	sD[y][x]  = confz[coo2D(ty, tx)];
	//Bottom-left corner
	sD[y][x] += confz[coo2D((ty+1), tx)];
	//Top-right corner
	sD[y][x] += confz[coo2D(ty, tx+1)];
	//Bottom-right corner
	sD[y][x] += confz[coo2D((ty+1), tx+1)];
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 3 * BN] = sD[0][0];
	__syncthreads();
	//try to measure Chern number
	//Top-left corner
	sD[y][x]  = confx[coo2D(ty, tx)] * (
	(confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])*(confz[coo2D(ty, tx)]-confz[coo2D(by, tx)])
	-(confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])*(confy[coo2D(ty, tx)]-confy[coo2D(by, tx)])
	)+confy[coo2D(ty, tx)] * (
	(confz[coo2D(ty, tx)]-confz[coo2D(ty, bx)])*(confx[coo2D(ty, tx)]-confx[coo2D(by, tx)])
	-(confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])*(confz[coo2D(ty, tx)]-confz[coo2D(by, tx)])
	)+confz[coo2D(ty, tx)] * (
	(confx[coo2D(ty, tx)]-confx[coo2D(ty, bx)])*(confy[coo2D(ty, tx)]-confy[coo2D(by, tx)])
	-(confy[coo2D(ty, tx)]-confy[coo2D(ty, bx)])*(confx[coo2D(ty, tx)]-confx[coo2D(by, tx)])
	);
	//Bottom-left corner
	sD[y][x] += confx[coo2D(typ, tx)] * (
	(confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])*(confz[coo2D(typ, tx)]-confz[coo2D(ty, tx)])
	-(confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])*(confy[coo2D(typ, tx)]-confy[coo2D(ty, tx)])
	)+confy[coo2D(typ, tx)]*(
	(confz[coo2D(typ, tx)]-confz[coo2D(typ, bx)])*(confx[coo2D(typ, tx)]-confx[coo2D(ty, tx)])
	-(confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])*(confz[coo2D(typ, tx)]-confz[coo2D(ty, tx)])
	)+confz[coo2D(typ, tx)] * (
	(confx[coo2D(typ, tx)]-confx[coo2D(typ, bx)])*(confy[coo2D(typ, tx)]-confy[coo2D(ty, tx)])
	-(confy[coo2D(typ, tx)]-confy[coo2D(typ, bx)])*(confx[coo2D(typ, tx)]-confx[coo2D(ty, tx)])
	);
	//Top-right corner
	sD[y][x] += confx[coo2D(ty, txp)] * (
	(confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])*(confz[coo2D(ty, txp)]-confz[coo2D(by, txp)])
	-(confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])*(confy[coo2D(ty, txp)]-confy[coo2D(by, txp)])
	)+confy[coo2D(ty, txp)]*(
	(confz[coo2D(ty, txp)]-confz[coo2D(ty, tx)])*(confx[coo2D(ty, txp)]-confx[coo2D(by, txp)])
	-(confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])*(confz[coo2D(ty, txp)]-confz[coo2D(by, txp)])
	)+confz[coo2D(ty, txp)] * (
	(confx[coo2D(ty, txp)]-confx[coo2D(ty, tx)])*(confy[coo2D(ty, txp)]-confy[coo2D(by, txp)])
	-(confy[coo2D(ty, txp)]-confy[coo2D(ty, tx)])*(confx[coo2D(ty, txp)]-confx[coo2D(by, txp)])
	);
	//Bottom-right corner
	sD[y][x] += confx[coo2D(typ, txp)] * (
	(confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])*(confz[coo2D(typ, txp)]-confz[coo2D(ty, txp)])
	-(confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])*(confy[coo2D(typ, txp)]-confy[coo2D(ty, txp)])
	)+confy[coo2D(typ, txp)]*(
	(confz[coo2D(typ, txp)]-confz[coo2D(typ, tx)])*(confx[coo2D(typ, txp)]-confx[coo2D(ty, txp)])
	-(confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])*(confz[coo2D(typ, txp)]-confz[coo2D(ty, txp)])
	)+confz[coo2D(typ, txp)] * (
	(confx[coo2D(typ, txp)]-confx[coo2D(typ, tx)])*(confy[coo2D(typ, txp)]-confy[coo2D(ty, txp)])
	-(confy[coo2D(typ, txp)]-confy[coo2D(typ, tx)])*(confx[coo2D(typ, txp)]-confx[coo2D(ty, txp)])
	);
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 4 * BN] = sD[0][0];
	__syncthreads();
}
#endif
