#include "hip/hip_runtime.h"
//-----Spin-flip for top-right corner (0,0)                      //and bottom-left corner-----
__global__ void relaxTLBR(float *confx, float *confy, float *confz, unsigned int *rngState, float H){
	//Energy variables
	float phix, phiy;
	float sx, sy, sz;
	float s2x, s2y, s2z;
	float hx, hy, hz;
	float Az, Xxy;
	float norm;
	float dot;
	const int x = threadIdx.x % BlockSize_x;
	const int y = threadIdx.x / BlockSize_x;
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	int ib, jb, i, j;
	//----------Spin flip at the top and left corner of each thread sqare----------
	i = tx;
	j = ty;
	ib = (i + SpinSize - 1) % SpinSize;
	if((j % SpinSize) == 0)	jb = j + SpinSize - 1;
	else			jb = j - 1;
	//Spin flip!
	hx = BXPxx(bd) * confx[j * SpinSize + i+1] + BYPxx(bd) * confx[(j+1) * SpinSize + i] + BXMxx(bd) * confx[j * SpinSize + ib] + BYMxx(bd) * confx[jb * SpinSize + i]\
	   + BXPxy(bd) * confy[j * SpinSize + i+1] + BYPxy(bd) * confy[(j+1) * SpinSize + i] + BXMxy(bd) * confy[j * SpinSize + ib] + BYMxy(bd) * confy[jb * SpinSize + i]\
	   + BXPxz(bd) * confz[j * SpinSize + i+1] + BYPxz(bd) * confz[(j+1) * SpinSize + i] + BXMxz(bd) * confz[j * SpinSize + ib] + BYMxz(bd) * confz[jb * SpinSize + i];
	hy = BXPyx(bd) * confx[j * SpinSize + i+1] + BYPyx(bd) * confx[(j+1) * SpinSize + i] + BXMyx(bd) * confx[j * SpinSize + ib] + BYMyx(bd) * confx[jb * SpinSize + i]\
	   + BXPyy(bd) * confy[j * SpinSize + i+1] + BYPyy(bd) * confy[(j+1) * SpinSize + i] + BXMyy(bd) * confy[j * SpinSize + ib] + BYMyy(bd) * confy[jb * SpinSize + i]\
	   + BXPyz(bd) * confz[j * SpinSize + i+1] + BYPyz(bd) * confz[(j+1) * SpinSize + i] + BXMyz(bd) * confz[j * SpinSize + ib] + BYMyz(bd) * confz[jb * SpinSize + i];
	hz = BXPzx(bd) * confx[j * SpinSize + i+1] + BYPzx(bd) * confx[(j+1) * SpinSize + i] + BXMzx(bd) * confx[j * SpinSize + ib] + BYMzx(bd) * confx[jb * SpinSize + i]\
	   + BXPzy(bd) * confy[j * SpinSize + i+1] + BYPzy(bd) * confy[(j+1) * SpinSize + i] + BXMzy(bd) * confy[j * SpinSize + ib] + BYMzy(bd) * confy[jb * SpinSize + i]\
	   + BXPzz(bd) * confz[j * SpinSize + i+1] + BYPzz(bd) * confz[(j+1) * SpinSize + i] + BXMzz(bd) * confz[j * SpinSize + ib] + BYMzz(bd) * confz[jb * SpinSize + i] + H;
	if(hx * hx + hy * hy + hz * hz>0.001){
	norm = 1 / sqrt(hx * hx + hy * hy + hz * hz);
	hx *= norm;
	hy *= norm;
	hz *= norm;
	sx = confx[j*SpinSize + i];
	sy = confy[j*SpinSize + i];
	sz = confz[j*SpinSize + i];
	dot = hx * sx + hy * sy + hz * sz;
	Xxy = sx * hy - sy * hx;
	Az = sz - dot * hz;
	phix = (Az*Az + Xxy*Xxy >0.00001 && (Xxy*Xxy>0.0000001))?((Az*Az - Xxy*Xxy)/(Az*Az + Xxy*Xxy)):1.0;
	phiy = (Az*Az + Xxy*Xxy >0.00001 && (Xxy*Xxy>0.0000001))?((1-phix)*Az/Xxy):0.0;
	dot = dot * (1 - phix);
	s2x = sx * phix + hx * dot + (sy * hz - sz * hy) * phiy;
	s2y = sy * phix + hy * dot + (sz * hx - sx * hz) * phiy;
	s2z = sz * phix + hz * dot + Xxy * phiy;
	norm = 1 / sqrt(s2x * s2x + s2y * s2y + s2z * s2z);
	confx[j*SpinSize + i] = s2x * norm;
	confy[j*SpinSize + i] = s2y * norm;
	confz[j*SpinSize + i] = s2z * norm;
	}//exception
	__syncthreads();
	//----------Spin flip at the bottom and right corner of each thread sqare----------
	i = tx + 1;
	j = ty + 1;
	ib = (i + 1) % SpinSize;
	if((j % SpinSize) == SpinSize - 1)	jb = j - SpinSize + 1;
	else					jb = j + 1;
	//Spin flip!
	hx = BXPxx(bd) * confx[j * SpinSize + ib] + BYPxx(bd) * confx[jb * SpinSize + i] + BXMxx(bd) * confx[j * SpinSize + i-1] + BYMxx(bd) * confx[(j-1) * SpinSize + i]\
	   + BXPxy(bd) * confy[j * SpinSize + ib] + BYPxy(bd) * confy[jb * SpinSize + i] + BXMxy(bd) * confy[j * SpinSize + i-1] + BYMxy(bd) * confy[(j-1) * SpinSize + i]\
	   + BXPxz(bd) * confz[j * SpinSize + ib] + BYPxz(bd) * confz[jb * SpinSize + i] + BXMxz(bd) * confz[j * SpinSize + i-1] + BYMxz(bd) * confz[(j-1) * SpinSize + i];
	hy = BXPyx(bd) * confx[j * SpinSize + ib] + BYPyx(bd) * confx[jb * SpinSize + i] + BXMyx(bd) * confx[j * SpinSize + i-1] + BYMyx(bd) * confx[(j-1) * SpinSize + i]\
	   + BXPyy(bd) * confy[j * SpinSize + ib] + BYPyy(bd) * confy[jb * SpinSize + i] + BXMyy(bd) * confy[j * SpinSize + i-1] + BYMyy(bd) * confy[(j-1) * SpinSize + i]\
	   + BXPyz(bd) * confz[j * SpinSize + ib] + BYPyz(bd) * confz[jb * SpinSize + i] + BXMyz(bd) * confz[j * SpinSize + i-1] + BYMyz(bd) * confz[(j-1) * SpinSize + i];
	hz = BXPzx(bd) * confx[j * SpinSize + ib] + BYPzx(bd) * confx[jb * SpinSize + i] + BXMzx(bd) * confx[j * SpinSize + i-1] + BYMzx(bd) * confx[(j-1) * SpinSize + i]\
	   + BXPzy(bd) * confy[j * SpinSize + ib] + BYPzy(bd) * confy[jb * SpinSize + i] + BXMzy(bd) * confy[j * SpinSize + i-1] + BYMzy(bd) * confy[(j-1) * SpinSize + i]\
	   + BXPzz(bd) * confz[j * SpinSize + ib] + BYPzz(bd) * confz[jb * SpinSize + i] + BXMzz(bd) * confz[j * SpinSize + i-1] + BYMzz(bd) * confz[(j-1) * SpinSize + i] + H;
	if(hx * hx + hy * hy + hz * hz>0.001){
	norm = 1 / sqrt(hx * hx + hy * hy + hz * hz);
	hx *= norm;
	hy *= norm;
	hz *= norm;
	sx = confx[j*SpinSize + i];
	sy = confy[j*SpinSize + i];
	sz = confz[j*SpinSize + i];
	dot = hx * sx + hy * sy + hz * sz;
	Xxy = sx * hy - sy * hx;
	Az = sz - dot * hz;
	phix = (Az*Az + Xxy*Xxy >0.00001 && (Xxy*Xxy>0.0000001))?((Az*Az - Xxy*Xxy)/(Az*Az + Xxy*Xxy)):1.0;
	phiy = (Az*Az + Xxy*Xxy >0.00001 && (Xxy*Xxy>0.0000001))?((1-phix)*Az/Xxy):0.0;
	dot = dot * (1 - phix);
	s2x = sx * phix + hx * dot + (sy * hz - sz * hy) * phiy;
	s2y = sy * phix + hy * dot + (sz * hx - sx * hz) * phiy;
	s2z = sz * phix + hz * dot + Xxy * phiy;
	norm = 1 / sqrt(s2x * s2x + s2y * s2y + s2z * s2z);
	confx[j*SpinSize + i] = s2x * norm;
	confy[j*SpinSize + i] = s2y * norm;
	confz[j*SpinSize + i] = s2z * norm;
	}//exception
	__syncthreads();
	//Load random number back to global memory
}
