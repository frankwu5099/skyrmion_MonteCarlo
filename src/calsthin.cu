
#include <hip/hip_runtime.h>
#ifdef THIN
#include "measurements.cuh"
__global__ void calthin(float *confx, float *confy, float *confz, double *out){
	//Energy variables
	extern __shared__ double sD[];
	const int x = threadIdx.x % (BlockSize_x);
	const int y = (threadIdx.x / BlockSize_x);
	const int tx = 2 * (((blockIdx.x % BN) % GridSize_x) * BlockSize_x + x);
	const int ty =(blockIdx.x / BN) * SpinSize * SpinSize_z +  2 * ((((blockIdx.x % BN) / GridSize_x) % GridSize_y) * BlockSize_y + y);
	const int txp = tx + 1;
	const int typ = ty + 1;
	//const int ty = 2 * ((blockIdx.x / BN) * SpinSize + ((blockIdx.x % BN) / GridSize_x) * BlockSize_y + y);
	const int dataoff = (blockIdx.x / BN) * MEASURE_NUM * BN;
	int bx, by, z;
	//-----Calculate the energy of each spin pairs in the system-----
	//To avoid double counting, for each spin, choose the neighbor spin on the left hand side of each spin and also one above each spin as pairs. Each spin has two pairs.

	bx = (tx + SpinSize - 1) % SpinSize;
	if((ty % SpinSize) == 0)	by = ty + SpinSize - 1;
	else				by = ty - 1;
	//Calculate the two pair-energy of each spin on the thread square step by step and store the summing energy of each thread square in sD.
	z = 0;
  //Top-left corner
  sD[y][x] = -confx[coo(z, ty, tx)] * (BXMxx * confx[coo(z, ty, bx)] + BYMxx * confx[coo(z, by, tx)])\
             +confx[coo(z, ty, tx)] * (BXMxy * confy[coo(z, ty, bx)] + BYMxy * confy[coo(z, by, tx)])\
             +confx[coo(z, ty, tx)] * (BXMxz * confz[coo(z, ty, bx)] + BYMxz * confz[coo(z, by, tx)])\
             +confy[coo(z, ty, tx)] * (BXMyx * confx[coo(z, ty, bx)] + BYMyx * confx[coo(z, by, tx)])\
             +confy[coo(z, ty, tx)] * (BXMyy * confy[coo(z, ty, bx)] + BYMyy * confy[coo(z, by, tx)])\
             +confy[coo(z, ty, tx)] * (BXMyz * confz[coo(z, ty, bx)] + BYMyz * confz[coo(z, by, tx)])\
             +confz[coo(z, ty, tx)] * (BXMzx * confx[coo(z, ty, bx)] + BYMzx * confx[coo(z, by, tx)])\
             +confz[coo(z, ty, tx)] * (BXMzy * confy[coo(z, ty, bx)] + BYMzy * confy[coo(z, by, tx)])\
             +confz[coo(z, ty, tx)] * (BXMzz * confz[coo(z, ty, bx)] + BYMzz * confz[coo(z, by, tx)] - A * confz[coo(z, ty, tx)]);
  //Bottom-left corner
  sD[y][x] -= confx[coo(z, ty+1, tx)] * (BXMxx * confx[coo(z, ty+1, bx)] + BYMxx * confx[coo(z, ty, tx)])\
             +confx[coo(z, ty+1, tx)] * (BXMxy * confy[coo(z, ty+1, bx)] + BYMxy * confy[coo(z, ty, tx)])\
             +confx[coo(z, ty+1, tx)] * (BXMxz * confz[coo(z, ty+1, bx)] + BYMxz * confz[coo(z, ty, tx)])\
             +confy[coo(z, ty+1, tx)] * (BXMyx * confx[coo(z, ty+1, bx)] + BYMyx * confx[coo(z, ty, tx)])\
             +confy[coo(z, ty+1, tx)] * (BXMyy * confy[coo(z, ty+1, bx)] + BYMyy * confy[coo(z, ty, tx)])\
             +confy[coo(z, ty+1, tx)] * (BXMyz * confz[coo(z, ty+1, bx)] + BYMyz * confz[coo(z, ty, tx)])\
             +confz[coo(z, ty+1, tx)] * (BXMzx * confx[coo(z, ty+1, bx)] + BYMzx * confx[coo(z, ty, tx)])\
             +confz[coo(z, ty+1, tx)] * (BXMzy * confy[coo(z, ty+1, bx)] + BYMzy * confy[coo(z, ty, tx)])\
             +confz[coo(z, ty+1, tx)] * (BXMzz * confz[coo(z, ty+1, bx)] + BYMzz * confz[coo(z, ty, tx)] - A * confz[coo(z, ty+1, tx)]);
  //Top-right corner
  sD[y][x] -= confx[coo(z, ty, tx+1)] * (BXMxx * confx[coo(z, ty, tx)] + BYMxx * confx[coo(z, by, tx+1)])\
             +confx[coo(z, ty, tx+1)] * (BXMxy * confy[coo(z, ty, tx)] + BYMxy * confy[coo(z, by, tx+1)])\
             +confx[coo(z, ty, tx+1)] * (BXMxz * confz[coo(z, ty, tx)] + BYMxz * confz[coo(z, by, tx+1)])\
             +confy[coo(z, ty, tx+1)] * (BXMyx * confx[coo(z, ty, tx)] + BYMyx * confx[coo(z, by, tx+1)])\
             +confy[coo(z, ty, tx+1)] * (BXMyy * confy[coo(z, ty, tx)] + BYMyy * confy[coo(z, by, tx+1)])\
             +confy[coo(z, ty, tx+1)] * (BXMyz * confz[coo(z, ty, tx)] + BYMyz * confz[coo(z, by, tx+1)])\
             +confz[coo(z, ty, tx+1)] * (BXMzx * confx[coo(z, ty, tx)] + BYMzx * confx[coo(z, by, tx+1)])\
             +confz[coo(z, ty, tx+1)] * (BXMzy * confy[coo(z, ty, tx)] + BYMzy * confy[coo(z, by, tx+1)])\
             +confz[coo(z, ty, tx+1)] * (BXMzz * confz[coo(z, ty, tx)] + BYMzz * confz[coo(z, by, tx+1)] - A * confz[coo(z, ty, tx+1)]);
  //Bottom-right corner
  sD[y][x] -= confx[coo(z, ty+1, tx+1)] * (BXMxx * confx[coo(z, ty+1, tx)] + BYMxx * confx[coo(z, ty, tx+1)])\
             +confx[coo(z, ty+1, tx+1)] * (BXMxy * confy[coo(z, ty+1, tx)] + BYMxy * confy[coo(z, ty, tx+1)])\
             +confx[coo(z, ty+1, tx+1)] * (BXMxz * confz[coo(z, ty+1, tx)] + BYMxz * confz[coo(z, ty, tx+1)])\
             +confy[coo(z, ty+1, tx+1)] * (BXMyx * confx[coo(z, ty+1, tx)] + BYMyx * confx[coo(z, ty, tx+1)])\
             +confy[coo(z, ty+1, tx+1)] * (BXMyy * confy[coo(z, ty+1, tx)] + BYMyy * confy[coo(z, ty, tx+1)])\
             +confy[coo(z, ty+1, tx+1)] * (BXMyz * confz[coo(z, ty+1, tx)] + BYMyz * confz[coo(z, ty, tx+1)])\
             +confz[coo(z, ty+1, tx+1)] * (BXMzx * confx[coo(z, ty+1, tx)] + BYMzx * confx[coo(z, ty, tx+1)])\
             +confz[coo(z, ty+1, tx+1)] * (BXMzy * confy[coo(z, ty+1, tx)] + BYMzy * confy[coo(z, ty, tx+1)])\
             +confz[coo(z, ty+1, tx+1)] * (BXMzz * confz[coo(z, ty+1, tx)] + BYMzz * confz[coo(z, ty, tx+1)] - A * confz[coo(z, ty+1, tx+1)]);

  for (z = 1; z < SpinSize_z; z++){
    //Top-left corner
    sD[y][x] -= confx[coo(z, ty, tx)] * (BXMxx * confx[coo(z, ty, bx)] + BYMxx * confx[coo(z, by, tx)] + BZMxx * confx[coo(z-1, ty, tx)])\
               +confx[coo(z, ty, tx)] * (BXMxy * confy[coo(z, ty, bx)] + BYMxy * confy[coo(z, by, tx)] + BZMxy * confy[coo(z-1, ty, tx)])\
               +confx[coo(z, ty, tx)] * (BXMxz * confz[coo(z, ty, bx)] + BYMxz * confz[coo(z, by, tx)])\
               +confy[coo(z, ty, tx)] * (BXMyx * confx[coo(z, ty, bx)] + BYMyx * confx[coo(z, by, tx)] + BZMyx * confx[coo(z-1, ty, tx)])\
               +confy[coo(z, ty, tx)] * (BXMyy * confy[coo(z, ty, bx)] + BYMyy * confy[coo(z, by, tx)] + BZMyy * confy[coo(z-1, ty, tx)])\
               +confy[coo(z, ty, tx)] * (BXMyz * confz[coo(z, ty, bx)] + BYMyz * confz[coo(z, by, tx)])\
               +confz[coo(z, ty, tx)] * (BXMzx * confx[coo(z, ty, bx)] + BYMzx * confx[coo(z, by, tx)])\
               +confz[coo(z, ty, tx)] * (BXMzy * confy[coo(z, ty, bx)] + BYMzy * confy[coo(z, by, tx)])\
               +confz[coo(z, ty, tx)] * (BXMzz * confz[coo(z, ty, bx)] + BYMzz * confz[coo(z, by, tx)] + BZMzz * confz[coo(z-1, ty, tx)] - A * confz[coo(z, ty, tx)]);
    //Bottom-left corner
    sD[y][x] -= confx[coo(z, ty+1, tx)] * (BXMxx * confx[coo(z, ty+1, bx)] + BYMxx * confx[coo(z, ty, tx)] + BZMxx * confx[coo(z-1, ty+1, tx)])\
               +confx[coo(z, ty+1, tx)] * (BXMxy * confy[coo(z, ty+1, bx)] + BYMxy * confy[coo(z, ty, tx)] + BZMxy * confy[coo(z-1, ty+1, tx)])\
               +confx[coo(z, ty+1, tx)] * (BXMxz * confz[coo(z, ty+1, bx)] + BYMxz * confz[coo(z, ty, tx)])\
               +confy[coo(z, ty+1, tx)] * (BXMyx * confx[coo(z, ty+1, bx)] + BYMyx * confx[coo(z, ty, tx)] + BZMyx * confx[coo(z-1, ty+1, tx)])\
               +confy[coo(z, ty+1, tx)] * (BXMyy * confy[coo(z, ty+1, bx)] + BYMyy * confy[coo(z, ty, tx)] + BZMyy * confy[coo(z-1, ty+1, tx)])\
               +confy[coo(z, ty+1, tx)] * (BXMyz * confz[coo(z, ty+1, bx)] + BYMyz * confz[coo(z, ty, tx)])\
               +confz[coo(z, ty+1, tx)] * (BXMzx * confx[coo(z, ty+1, bx)] + BYMzx * confx[coo(z, ty, tx)])\
               +confz[coo(z, ty+1, tx)] * (BXMzy * confy[coo(z, ty+1, bx)] + BYMzy * confy[coo(z, ty, tx)])\
               +confz[coo(z, ty+1, tx)] * (BXMzz * confz[coo(z, ty+1, bx)] + BYMzz * confz[coo(z, ty, tx)] + BZMzz * confz[coo(z-1, ty+1, tx)] - A * confz[coo(z, ty+1, tx)]);
    //Top-right corner
    sD[y][x] -= confx[coo(z, ty, tx+1)] * (BXMxx * confx[coo(z, ty, tx)] + BYMxx * confx[coo(z, by, tx+1)] + BZMxx * confx[coo(z-1, ty, tx+1)])\
               +confx[coo(z, ty, tx+1)] * (BXMxy * confy[coo(z, ty, tx)] + BYMxy * confy[coo(z, by, tx+1)] + BZMxy * confy[coo(z-1, ty, tx+1)])\
               +confx[coo(z, ty, tx+1)] * (BXMxz * confz[coo(z, ty, tx)] + BYMxz * confz[coo(z, by, tx+1)])\
               +confy[coo(z, ty, tx+1)] * (BXMyx * confx[coo(z, ty, tx)] + BYMyx * confx[coo(z, by, tx+1)] + BZMyx * confx[coo(z-1, ty, tx+1)])\
               +confy[coo(z, ty, tx+1)] * (BXMyy * confy[coo(z, ty, tx)] + BYMyy * confy[coo(z, by, tx+1)] + BZMyy * confy[coo(z-1, ty, tx+1)])\
               +confy[coo(z, ty, tx+1)] * (BXMyz * confz[coo(z, ty, tx)] + BYMyz * confz[coo(z, by, tx+1)])\
               +confz[coo(z, ty, tx+1)] * (BXMzx * confx[coo(z, ty, tx)] + BYMzx * confx[coo(z, by, tx+1)])\
               +confz[coo(z, ty, tx+1)] * (BXMzy * confy[coo(z, ty, tx)] + BYMzy * confy[coo(z, by, tx+1)])\
               +confz[coo(z, ty, tx+1)] * (BXMzz * confz[coo(z, ty, tx)] + BYMzz * confz[coo(z, by, tx+1)] + BZMzz * confz[coo(z-1, ty, tx+1)] - A * confz[coo(z, ty, tx+1)]);
    //Bottom-right corner
    sD[y][x] -= confx[coo(z, ty+1, tx+1)] * (BXMxx * confx[coo(z, ty+1, tx)] + BYMxx * confx[coo(z, ty, tx+1)] + BZMxx * confx[coo(z-1, ty+1, tx+1)])\
               +confx[coo(z, ty+1, tx+1)] * (BXMxy * confy[coo(z, ty+1, tx)] + BYMxy * confy[coo(z, ty, tx+1)] + BZMxy * confy[coo(z-1, ty+1, tx+1)])\
               +confx[coo(z, ty+1, tx+1)] * (BXMxz * confz[coo(z, ty+1, tx)] + BYMxz * confz[coo(z, ty, tx+1)])\
               +confy[coo(z, ty+1, tx+1)] * (BXMyx * confx[coo(z, ty+1, tx)] + BYMyx * confx[coo(z, ty, tx+1)] + BZMyx * confx[coo(z-1, ty+1, tx+1)])\
               +confy[coo(z, ty+1, tx+1)] * (BXMyy * confy[coo(z, ty+1, tx)] + BYMyy * confy[coo(z, ty, tx+1)] + BZMyy * confy[coo(z-1, ty+1, tx+1)])\
               +confy[coo(z, ty+1, tx+1)] * (BXMyz * confz[coo(z, ty+1, tx)] + BYMyz * confz[coo(z, ty, tx+1)])\
               +confz[coo(z, ty+1, tx+1)] * (BXMzx * confx[coo(z, ty+1, tx)] + BYMzx * confx[coo(z, ty, tx+1)])\
               +confz[coo(z, ty+1, tx+1)] * (BXMzy * confy[coo(z, ty+1, tx)] + BYMzy * confy[coo(z, ty, tx+1)])\
               +confz[coo(z, ty+1, tx+1)] * (BXMzz * confz[coo(z, ty+1, tx)] + BYMzz * confz[coo(z, ty, tx+1)] + BZMzz * confz[coo(z-1, ty+1, tx+1)] - A * confz[coo(z, ty+1, tx+1)]);
  }
	__syncthreads();


	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y<BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y<BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y<BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(y==0 && x==0)
		out[dataoff + (blockIdx.x % BN)] = sD[0][0];
	__syncthreads();
	//Sum over the magnetic moments in x direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
  sD[y][x]  = 0;
  for (z = 0; z < SpinSize_z; z++){
    sD[y][x] += confx[coo(z, ty, tx)];
    sD[y][x] += confx[coo(z, ty+1, tx)];
    sD[y][x] += confx[coo(z, ty ,tx+1)];
    sD[y][x] += confx[coo(z, ty+1, tx+1)];
  }
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in y direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
	//Top-left corner
  sD[y][x]  = 0;
  for (z = 0; z < SpinSize_z; z++){
    sD[y][x] += confy[coo(z, ty, tx)];
    sD[y][x] += confy[coo(z, ty+1, tx)];
    sD[y][x] += confy[coo(z, ty ,tx+1)];
    sD[y][x] += confy[coo(z, ty+1, tx+1)];
  }
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 2 * BN] = sD[0][0];
	__syncthreads();

	//Sum over the magnetic moments in z direction of the eight spins on each thread cubic and store the result of each thread cubic in sD.
  sD[y][x]  = 0;
  for (z = 0; z < SpinSize_z; z++){
    sD[y][x] += confz[coo(z, ty, tx)];
    sD[y][x] += confz[coo(z, ty+1, tx)];
    sD[y][x] += confz[coo(z, ty ,tx+1)];
    sD[y][x] += confz[coo(z, ty+1, tx+1)];
  }
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 3 * BN] = sD[0][0];
	__syncthreads();
	//try to measure Chern number
	//Top-left corner
  sD[y][x]  = 0;
  for (z = 0; z < SpinSize_z; z++){
    sD[y][x]  = confx[coo(z, ty, tx)] * (
    (confy[coo(z, ty, tx)]-confy[coo(z, ty, bx)])*(confz[coo(z, ty, tx)]-confz[coo(z, by, tx)])
    -(confz[coo(z, ty, tx)]-confz[coo(z, ty, bx)])*(confy[coo(z, ty, tx)]-confy[coo(z, by, tx)])
    )+confy[coo(z, ty, tx)] * (
    (confz[coo(z, ty, tx)]-confz[coo(z, ty, bx)])*(confx[coo(z, ty, tx)]-confx[coo(z, by, tx)])
    -(confx[coo(z, ty, tx)]-confx[coo(z, ty, bx)])*(confz[coo(z, ty, tx)]-confz[coo(z, by, tx)])
    )+confz[coo(z, ty, tx)] * (
    (confx[coo(z, ty, tx)]-confx[coo(z, ty, bx)])*(confy[coo(z, ty, tx)]-confy[coo(z, by, tx)])
    -(confy[coo(z, ty, tx)]-confy[coo(z, ty, bx)])*(confx[coo(z, ty, tx)]-confx[coo(z, by, tx)])
    );
    //Bottom-left corner
    sD[y][x] += confx[coo(z, typ, tx)] * (
    (confy[coo(z, typ, tx)]-confy[coo(z, typ, bx)])*(confz[coo(z, typ, tx)]-confz[coo(z, ty, tx)])
    -(confz[coo(z, typ, tx)]-confz[coo(z, typ, bx)])*(confy[coo(z, typ, tx)]-confy[coo(z, ty, tx)])
    )+confy[coo(z, typ, tx)]*(
    (confz[coo(z, typ, tx)]-confz[coo(z, typ, bx)])*(confx[coo(z, typ, tx)]-confx[coo(z, ty, tx)])
    -(confx[coo(z, typ, tx)]-confx[coo(z, typ, bx)])*(confz[coo(z, typ, tx)]-confz[coo(z, ty, tx)])
    )+confz[coo(z, typ, tx)] * (
    (confx[coo(z, typ, tx)]-confx[coo(z, typ, bx)])*(confy[coo(z, typ, tx)]-confy[coo(z, ty, tx)])
    -(confy[coo(z, typ, tx)]-confy[coo(z, typ, bx)])*(confx[coo(z, typ, tx)]-confx[coo(z, ty, tx)])
    );
    //Top-right corner
    sD[y][x] += confx[coo(z, ty, txp)] * (
    (confy[coo(z, ty, txp)]-confy[coo(z, ty, tx)])*(confz[coo(z, ty, txp)]-confz[coo(z, by, txp)])
    -(confz[coo(z, ty, txp)]-confz[coo(z, ty, tx)])*(confy[coo(z, ty, txp)]-confy[coo(z, by, txp)])
    )+confy[coo(z, ty, txp)]*(
    (confz[coo(z, ty, txp)]-confz[coo(z, ty, tx)])*(confx[coo(z, ty, txp)]-confx[coo(z, by, txp)])
    -(confx[coo(z, ty, txp)]-confx[coo(z, ty, tx)])*(confz[coo(z, ty, txp)]-confz[coo(z, by, txp)])
    )+confz[coo(z, ty, txp)] * (
    (confx[coo(z, ty, txp)]-confx[coo(z, ty, tx)])*(confy[coo(z, ty, txp)]-confy[coo(z, by, txp)])
    -(confy[coo(z, ty, txp)]-confy[coo(z, ty, tx)])*(confx[coo(z, ty, txp)]-confx[coo(z, by, txp)])
    );
    //Bottom-right corner
    sD[y][x] += confx[coo(z, typ, txp)] * (
    (confy[coo(z, typ, txp)]-confy[coo(z, typ, tx)])*(confz[coo(z, typ, txp)]-confz[coo(z, ty, txp)])
    -(confz[coo(z, typ, txp)]-confz[coo(z, typ, tx)])*(confy[coo(z, typ, txp)]-confy[coo(z, ty, txp)])
    )+confy[coo(z, typ, txp)]*(
    (confz[coo(z, typ, txp)]-confz[coo(z, typ, tx)])*(confx[coo(z, typ, txp)]-confx[coo(z, ty, txp)])
    -(confx[coo(z, typ, txp)]-confx[coo(z, typ, tx)])*(confz[coo(z, typ, txp)]-confz[coo(z, ty, txp)])
    )+confz[coo(z, typ, txp)] * (
    (confx[coo(z, typ, txp)]-confx[coo(z, typ, tx)])*(confy[coo(z, typ, txp)]-confy[coo(z, ty, txp)])
    -(confy[coo(z, typ, txp)]-confy[coo(z, typ, tx)])*(confx[coo(z, typ, txp)]-confx[coo(z, ty, txp)])
    );
  }
	__syncthreads();

	//Sum over all elements in each sD
	if(y < BlockSize_y/2)
		sD[y][x] += sD[y+BlockSize_y/2] [x];
	__syncthreads();
	if(y < BlockSize_y/4)
		sD[y][x] += sD[y+BlockSize_y/4] [x];
	__syncthreads();
	if(y < BlockSize_y/8)
		sD[y][x] += sD[y+BlockSize_y/8] [x];
	__syncthreads();
	if(y < BlockSize_y/16)
		sD[y][x] += sD[y+BlockSize_y/16] [x];
	__syncthreads();
	if(y==0 && x<BlockSize_x/2)
		sD[y][x] += sD[y][x+BlockSize_x/2] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/4)
		sD[y][x] += sD[y][x+BlockSize_x/4] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/8)
		sD[y][x] += sD[y][x+BlockSize_x/8] ;
	__syncthreads();
	if(y==0 && x<BlockSize_x/16)
		sD[y][x] += sD[y][x+BlockSize_x/16] ;
	__syncthreads();

	if(x==0 && y==0)
		out[dataoff + (blockIdx.x % BN) + 4 * BN] = sD[0][0];
	__syncthreads();
}

#endif
