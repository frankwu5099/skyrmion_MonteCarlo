#include "hip/hip_runtime.h"
/* * This program uses the device CURAND API to calculate what * proportion of pseudo-random ints have low bit set. */ 
#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h> 
#include <iostream>
#include "port_mtgp32_host.h"  ///host part include this header 

 
//////////////////////////////////////////////////////////
//Device part : 
#include "port_mtgp32_kernel.h" ///device part include this header 
__global__ void generate_kernel(hiprandStateMtgp32 *state) { 
	
	///Generate uint32 
	///all thread in a block should not divergent around this. 
	unsigned int r = hiprand(&state[blockIdx.x]);
	r = hiprand(&state[blockIdx.x]);
	r = hiprand(&state[blockIdx.x]);
	r = hiprand(&state[blockIdx.x]);

}
/////////////////////////////////////////////////////////


using namespace std;
int main(int argc, char * argv[]){

	//Define state:
	hiprandStateMtgp32 *DStates;
	mtgp32_kernel_params *DParams;
	
	//Env:
	unsigned int NBlock = 64;
	unsigned int ThreadperBlock = 256; // Max limit at 256	
	unsigned int seed = 99;
	
	//Allocate Stats:
	hipMalloc((void**)&DStates,sizeof(hiprandStateMtgp32)*NBlock);
	
	//Set parameters:
	hipMalloc((void**)&DParams, sizeof(mtgp32_kernel_params));
	

	///Make constant:
	hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, DParams);
	//return 0;
        hiprandMakeMTGP32KernelState(DStates, mtgp32dc_params_fast_11213, DParams, NBlock, seed);


	generate_kernel<<<NBlock,ThreadperBlock>>>(DStates);

	///cleanup

	hipFree(DStates);
	hipFree(DParams);
	
	
	
		




}



